#include "hip/hip_runtime.h"
#include "constants.hpp"
#include "gpu_utils.cuh"
#include "kernel_utils.cuh"
#include "kernels/k_flat_bottom_bond.cuh"
#include "kernels/k_indices.cuh"
#include "kernels/k_local_md.cuh"
#include "local_md_potentials.hpp"
#include "math_utils.cuh"
#include <hipcub/hipcub.hpp>
#include <random>
#include <vector>

namespace timemachine {

// Struct representing the CUB < operation
struct LessThan {
    int compare;
    HIPCUB_RUNTIME_FUNCTION __device__ __forceinline__ explicit LessThan(int compare) : compare(compare) {}
    HIPCUB_RUNTIME_FUNCTION __device__ __forceinline__ bool operator()(const int &a) const { return (a < compare); }
};

LocalMDPotentials::LocalMDPotentials(const int N, const std::vector<std::shared_ptr<BoundPotential>> &bps)
    : N_(N), temp_storage_bytes_(0), all_potentials_(bps), d_restraint_pairs_(N_ * 2), d_bond_params_(N_ * 3),
      d_probability_buffer_(round_up_even(N_)), d_free_idxs_(N_), d_row_idxs_(N_), d_col_idxs_(N_), p_num_selected_(1),
      d_num_selected_buffer_(1) {

    std::vector<std::shared_ptr<BoundPotential>> nonbonded_pots;
    get_nonbonded_all_pair_potentials(bps, nonbonded_pots);

    if (nonbonded_pots.size() > 1) {
        throw std::runtime_error("found multiple NonbondedAllPairs potentials");
    }
    if (nonbonded_pots.size() != 1) {
        throw std::runtime_error("unable to find a NonbondedAllPairs potential");
    }

    // Only used to reference shared_ptr to potential and for Nonbonded parameters
    // modifications to the BoundPotential has no impact
    nonbonded_bp_ = nonbonded_pots[0];

    // Ensure that we allocate enough space for all potential bonds
    // default_bonds[i * 2 + 0] != default_bonds[i * 2 + 1], so set first value to 0, second to i + 1
    std::vector<int> default_bonds(N_ * 2);
    for (int i = 0; i < N_; i++) {
        default_bonds[i * 2 + 0] = 0;
        default_bonds[i * 2 + 1] = i + 1;
    }
    restraint_ = std::shared_ptr<FlatBottomBond<double>>(new FlatBottomBond<double>(default_bonds));
    // Construct a bound potential with 0 params
    bound_restraint_ = std::shared_ptr<BoundPotential>(new BoundPotential(restraint_, std::vector<int>({0}), nullptr));

    ixn_group_ =
        construct_ixn_group_potential(N_, nonbonded_bp_->potential, nonbonded_bp_->size(), nonbonded_bp_->d_p->data);

    // Add the restraint potential and ixn group potential
    all_potentials_.push_back(bound_restraint_);
    all_potentials_.push_back(ixn_group_);

    hipcub::DevicePartition::If(
        nullptr,
        temp_storage_bytes_,
        d_free_idxs_.data,
        d_row_idxs_.data,
        d_num_selected_buffer_.data,
        N_,
        LessThan(N_));
    // Allocate char as temp_storage_bytes_ is in raw bytes and the type doesn't matter in practice.
    // Equivalent to DeviceBuffer<int> buf(temp_storage_bytes_ / sizeof(int))
    d_temp_storage_buffer_.reset(new DeviceBuffer<char>(temp_storage_bytes_));

    curandErrchk(hiprandCreateGenerator(&cr_rng_, HIPRAND_RNG_PSEUDO_DEFAULT));
};

LocalMDPotentials::~LocalMDPotentials() { curandErrchk(hiprandDestroyGenerator(cr_rng_)); }

// setup_from_idxs takes a set of idxs, a temperature and a seed to determine the free particles. Fix the local_idxs to length
// one to ensure the same reference everytime, though the seed also handles the probabilities of selecting particles, and it is suggested
// to provide a new seed at each step.
void LocalMDPotentials::setup_from_idxs(
    double *d_x_t,
    double *d_box_t,
    const std::vector<int> &local_idxs,
    const double temperature,
    const int seed,
    const double radius,
    const double k,
    hipStream_t stream) {
    curandErrchk(hiprandSetStream(cr_rng_, stream));
    curandErrchk(hiprandSetPseudoRandomGeneratorSeed(cr_rng_, seed));
    // Reset the generator offset to ensure same values for the same seed are produced
    // Simply reseeding does NOT produce identical results
    curandErrchk(hiprandSetGeneratorOffset(cr_rng_, 0));

    // Set the array to all N, which indicates to ignore that idx
    k_initialize_array<unsigned int><<<ceil_divide(N_, warp_size), warp_size, 0, stream>>>(N_, d_free_idxs_.data, N_);
    gpuErrchk(hipPeekAtLastError());

    // Generate values between (0, 1.0]
    curandErrchk(hiprandGenerateUniform(cr_rng_, d_probability_buffer_.data, round_up_even(N_)));

    std::mt19937 rng;
    rng.seed(seed);
    std::uniform_int_distribution<unsigned int> random_dist(0, local_idxs.size() - 1);

    unsigned int reference_idx = local_idxs[random_dist(rng)];

    const double kBT = BOLTZ * temperature;
    // Select all of the particles that will be free
    k_log_probability_selection<float><<<ceil_divide(N_, warp_size), warp_size, 0, stream>>>(
        N_, kBT, radius, k, reference_idx, d_x_t, d_box_t, d_probability_buffer_.data, d_free_idxs_.data);
    gpuErrchk(hipPeekAtLastError());

    this->_setup_free_idxs_given_reference_idx(reference_idx, radius, k, stream);
}

// setup_from_idxs takes a set of idxs, flat-bottom restraint parameters (radius, k)
// assumes selection_idxs are sampled based on exp(-beta U_flat_bottom(distance_to_reference, radius, k))
// (or that the user is otherwise accounting for selection probabilities)
void LocalMDPotentials::setup_from_idxs(
    const int reference_idx,
    const std::vector<int> &selection_idxs,
    const double radius,
    const double k,
    const hipStream_t stream) {

    // Set the array to all N, which indicates to ignore that idx
    k_initialize_array<unsigned int><<<ceil_divide(N_, warp_size), warp_size, 0, stream>>>(N_, d_free_idxs_.data, N_);
    gpuErrchk(hipPeekAtLastError());

    k_initialize_array<unsigned int><<<ceil_divide(N_, warp_size), warp_size, 0, stream>>>(N_, d_row_idxs_.data, N_);
    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipMemcpyAsync(
        d_row_idxs_.data,
        &selection_idxs[0],
        selection_idxs.size() * sizeof(*d_row_idxs_.data),
        hipMemcpyHostToDevice,
        stream));

    // Split out the values from the selection idxs into the indices of the free
    k_unique_indices<<<ceil_divide(N_, warp_size), warp_size, 0, stream>>>(N_, N_, d_row_idxs_.data, d_free_idxs_.data);
    gpuErrchk(hipPeekAtLastError());

    this->_setup_free_idxs_given_reference_idx((unsigned int)reference_idx, radius, k, stream);
}

void LocalMDPotentials::_setup_free_idxs_given_reference_idx(
    const unsigned int reference_idx, const double radius, const double k, hipStream_t stream) {
    const int tpb = warp_size;

    LessThan select_op(N_);

    // Partition the free idxs into the row idxs
    gpuErrchk(hipcub::DevicePartition::If(
        d_temp_storage_buffer_->data,
        temp_storage_bytes_,
        d_free_idxs_.data,
        d_row_idxs_.data,
        d_num_selected_buffer_.data,
        N_,
        select_op,
        stream));

    gpuErrchk(hipMemcpyAsync(
        p_num_selected_.data,
        d_num_selected_buffer_.data,
        1 * sizeof(*p_num_selected_.data),
        hipMemcpyDeviceToHost,
        stream));
    gpuErrchk(hipStreamSynchronize(stream));

    // The row indices is all of the free indices, which excludes the reference
    const int num_row_idxs = p_num_selected_.data[0];
    // The col indices is all indices, except for the free, including the reference which is frozen.
    const int num_col_idxs = N_ - num_row_idxs;

    if (num_row_idxs == 0) {
        throw std::runtime_error("LocalMDPotentials setup has no free particles selected");
    }

    // The reference particle will always be in the column idxs
    if (num_row_idxs == N_ - 1) {
        fprintf(stderr, "LocalMDPotentials setup has entire system selected\n");
    }

    k_construct_bonded_params<<<ceil_divide(num_row_idxs, tpb), tpb, 0, stream>>>(
        num_row_idxs,
        N_,
        reference_idx,
        k,
        0.0,
        radius,
        d_row_idxs_.data,
        d_restraint_pairs_.data,
        d_bond_params_.data);
    gpuErrchk(hipPeekAtLastError());

    // Setup the flat bottom restraints
    bound_restraint_->set_params_device(std::vector<int>({num_row_idxs, 3}), d_bond_params_.data, stream);
    restraint_->set_bonds_device(num_row_idxs, d_restraint_pairs_.data, stream);

    // Set the nonbonded potential to compute forces of free particles
    set_nonbonded_potential_idxs(nonbonded_bp_->potential, num_row_idxs, d_row_idxs_.data, stream);

    // Invert to get column idxs
    k_invert_indices<<<ceil_divide(N_, tpb), tpb, 0, stream>>>(N_, d_free_idxs_.data);
    gpuErrchk(hipPeekAtLastError());

    // Partition the column idxs to the column buffer to setup the interaction group
    gpuErrchk(hipcub::DevicePartition::If(
        d_temp_storage_buffer_->data,
        temp_storage_bytes_,
        d_free_idxs_.data,
        d_col_idxs_.data,
        d_num_selected_buffer_.data,
        N_,
        select_op,
        stream));

    // Free particles should be in the row idxs
    set_nonbonded_ixn_potential_idxs(
        ixn_group_->potential, num_col_idxs, num_row_idxs, d_col_idxs_.data, d_row_idxs_.data, stream);
}

std::vector<std::shared_ptr<BoundPotential>> LocalMDPotentials::get_potentials() { return all_potentials_; }

unsigned int *LocalMDPotentials::get_free_idxs() { return d_row_idxs_.data; }

// reset_potentials resets the potentials passed in to the constructor to be in the original state. This is because
// they are passed by reference and so changes made to the potentials will persist otherwise beyond the scope of the local md.
void LocalMDPotentials::reset_potentials(hipStream_t stream) {
    // Set the row idxs back to the identity.
    k_arange<<<ceil_divide(N_, warp_size), warp_size, 0, stream>>>(N_, d_row_idxs_.data);
    gpuErrchk(hipPeekAtLastError());
    // Set back to the full system
    set_nonbonded_potential_idxs(nonbonded_bp_->potential, N_, d_row_idxs_.data, stream);
}

} // namespace timemachine
