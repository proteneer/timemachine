#include "device_buffer.hpp"
#include "exchange.hpp"
#include "gpu_utils.cuh"
#include "kernels/k_exchange.cuh"
#include "kernels/kernel_utils.cuh"
#include "math_utils.cuh"
#include "mol_utils.hpp"

namespace timemachine {

template <typename RealType>
std::array<std::vector<int>, 2> get_inner_and_outer_mols(
    const std::vector<int> &center_atoms,
    const std::vector<double> &coords,
    const std::vector<double> &box,
    const std::vector<std::vector<int>> &group_idxs,
    const RealType radius) {

    const int num_mols = group_idxs.size();
    DeviceBuffer<int> d_center_atoms(center_atoms);
    DeviceBuffer<double> d_coords(coords);
    DeviceBuffer<double> d_box(box);

    DeviceBuffer<int> d_flags(num_mols);

    DeviceBuffer<int> d_mol_offsets(get_mol_offsets(group_idxs));
    DeviceBuffer<int> d_atom_indices(get_atom_indices(group_idxs));

    DeviceBuffer<RealType> d_centroid(3);
    hipStream_t stream = static_cast<hipStream_t>(0);

    const int tpb = DEFAULT_THREADS_PER_BLOCK;

    k_compute_centroid_of_atoms<<<1, tpb, 0, stream>>>(
        static_cast<int>(d_center_atoms.length), d_center_atoms.data, d_coords.data, d_centroid.data);
    gpuErrchk(hipPeekAtLastError());

    const RealType square_radius = radius * radius;

    k_flag_mols_inner_outer<<<ceil_divide(num_mols, tpb), tpb, 0, stream>>>(
        num_mols,
        d_atom_indices.data,
        d_mol_offsets.data,
        d_centroid.data,
        square_radius,
        d_coords.data,
        d_box.data,
        d_flags.data);
    gpuErrchk(hipStreamSynchronize(stream));

    std::vector<int> h_inner_idxs(0);
    std::vector<int> h_outer_idxs(0);
    std::vector<int> h_flags(d_flags.length);
    d_flags.copy_to(&h_flags[0]);
    for (int i = 0; i < h_flags.size(); i++) {
        if (h_flags[i] == 1) {
            h_inner_idxs.push_back(i);
        } else {
            h_outer_idxs.push_back(i);
        }
    }

    return std::array<std::vector<int>, 2>({h_inner_idxs, h_outer_idxs});
}

template std::array<std::vector<int>, 2> get_inner_and_outer_mols<float>(
    const std::vector<int> &center_atoms,
    const std::vector<double> &coords,
    const std::vector<double> &box,
    const std::vector<std::vector<int>> &group_idxs,
    const float radius);
template std::array<std::vector<int>, 2> get_inner_and_outer_mols<double>(
    const std::vector<int> &center_atoms,
    const std::vector<double> &coords,
    const std::vector<double> &box,
    const std::vector<std::vector<int>> &group_idxs,
    const double radius);

} // namespace timemachine
