#include "hip/hip_runtime.h"
#include "bd_exchange_move.hpp"

#include "constants.hpp"
#include "gpu_utils.cuh"
#include "kernels/k_exchange.cuh"
#include "kernels/k_indices.cuh"
#include "kernels/k_nonbonded.cuh"
#include "kernels/k_probability.cuh"
#include "kernels/k_rotations.cuh"
#include "math_utils.cuh"
#include "mol_utils.hpp"

// The number of threads per block for the setting of the final weight of the moved mol is low
// if using the same number as in the rest of the kernels of DEFAULT_THREADS_PER_BLOCK
#define WEIGHT_THREADS_PER_BLOCK 512

namespace timemachine {

template <typename RealType>
BDExchangeMove<RealType>::BDExchangeMove(
    const int N,
    const std::vector<std::vector<int>> &target_mols,
    const std::vector<double> &params,
    const double temperature,
    const double nb_beta,
    const double cutoff,
    const int seed,
    const int proposals_per_move,
    const int interval)
    : Mover(interval), N_(N), mol_size_(target_mols[0].size()), proposals_per_move_(proposals_per_move),
      num_target_mols_(target_mols.size()), nb_beta_(static_cast<RealType>(nb_beta)),
      beta_(static_cast<RealType>(1.0 / (BOLTZ * temperature))),
      cutoff_squared_(static_cast<RealType>(cutoff * cutoff)), noise_offset_(0), num_attempted_(0),
      mol_potential_(N, target_mols, nb_beta, cutoff), sampler_(num_target_mols_, seed), logsumexp_(num_target_mols_),
      d_intermediate_coords_(N * 3), d_params_(params), d_mol_energy_buffer_(num_target_mols_),
      d_sample_per_atom_energy_buffer_(mol_size_ * N), d_atom_idxs_(get_atom_indices(target_mols)),
      d_mol_offsets_(get_mol_offsets(target_mols)), d_log_weights_before_(num_target_mols_),
      d_log_weights_after_(num_target_mols_), d_log_sum_exp_before_(2), d_log_sum_exp_after_(2), d_samples_(1),
      d_quaternions_(round_up_even(QUATERNIONS_PER_STEP * this->RANDOM_BATCH_SIZE)), d_translations_(round_up_even(4)),
      d_num_accepted_(1), d_target_mol_atoms_(mol_size_), d_target_mol_offsets_(num_target_mols_ + 1),
      d_intermediate_sample_weights_(ceil_divide(N_, WEIGHT_THREADS_PER_BLOCK)) {

    if (proposals_per_move_ <= 0) {
        throw std::runtime_error("proposals per move must be greater than 0");
    }
    if (mol_size_ == 0) {
        throw std::runtime_error("must provide non-empty molecule indices");
    }
    verify_mols_contiguous(target_mols);
    for (int i = 0; i < target_mols.size(); i++) {
        if (target_mols[i].size() != mol_size_) {
            throw std::runtime_error("only support running with mols with constant size, got mixed sizes");
        }
    }
    // Clear out the logsumexp values so the log probability starts off as zero
    gpuErrchk(hipMemset(d_log_sum_exp_before_.data, 0, d_log_sum_exp_before_.size()));
    gpuErrchk(hipMemset(d_log_sum_exp_after_.data, 0, d_log_sum_exp_after_.size()));
    curandErrchk(hiprandCreateGenerator(&cr_rng_, HIPRAND_RNG_PSEUDO_DEFAULT));
    curandErrchk(hiprandSetPseudoRandomGeneratorSeed(cr_rng_, seed));

    // Set the offset to the length of the random vector to ensure noise is triggered on first step
    noise_offset_ = d_quaternions_.length;
}

template <typename RealType> BDExchangeMove<RealType>::~BDExchangeMove() {
    curandErrchk(hiprandDestroyGenerator(cr_rng_));
}

template <typename RealType>
void BDExchangeMove<RealType>::move(
    const int N,
    double *d_coords, // [N, 3]
    double *d_box,    // [3, 3]
    hipStream_t stream) {

    if (N != N_) {
        throw std::runtime_error("N != N_");
    }
    this->step_++;
    if (this->step_ % this->interval_ != 0) {
        return;
    }

    // Set the stream for the generator
    curandErrchk(hiprandSetStream(cr_rng_, stream));

    const int tpb = DEFAULT_THREADS_PER_BLOCK;

    this->compute_initial_weights(N, d_coords, d_box, stream);

    for (int move = 0; move < proposals_per_move_; move++) {
        if (noise_offset_ >= this->d_quaternions_.length) {
            // reset the noise to zero and generate more noise
            noise_offset_ = 0;
            curandErrchk(templateCurandNormal(cr_rng_, d_quaternions_.data, d_quaternions_.length, 0.0, 1.0));
        }
        // Run only after the first pass, to maintain meaningful `log_probability_host` values
        if (move > 0) {
            // Run a separate kernel to replace the before log probs and weights with the after if accepted a move
            // Need the weights to sample a value and the log probs are just because they aren't expensive to copy
            k_store_accepted_log_probability<RealType><<<1, tpb, 0>>>(
                num_target_mols_,
                d_translations_.data + 3, // Offset to get the last value for the acceptance criteria
                d_log_sum_exp_before_.data,
                d_log_sum_exp_after_.data,
                d_log_weights_before_.data,
                d_log_weights_after_.data);
            gpuErrchk(hipPeekAtLastError());
        }

        gpuErrchk(hipMemcpyAsync(
            d_log_weights_after_.data,
            d_log_weights_before_.data,
            d_log_weights_after_.size(),
            hipMemcpyDeviceToDevice,
            stream));

        // The d_translation_ buffer is [x,y,z,w] where [x,y,z] are a random translation and w is used for acceptance
        curandErrchk(templateCurandUniform(cr_rng_, d_translations_.data, d_translations_.length));

        // We only ever sample a single molecule
        sampler_.sample_device(num_target_mols_, 1, d_log_weights_before_.data, d_samples_.data, stream);

        // Don't move translations into computation of the incremental, as different translations can be used
        // by different bias deletion movers (such as targeted insertion)
        // scale the translations as they are between [0, 1]
        this->compute_incremental_weights(N, true, d_coords, d_box, this->d_quaternions_.data + noise_offset_, stream);

        logsumexp_.sum_device(num_target_mols_, d_log_weights_after_.data, d_log_sum_exp_after_.data, stream);

        k_attempt_exchange_move<RealType><<<ceil_divide(N_, tpb), tpb, 0, stream>>>(
            N,
            d_translations_.data + 3, // Offset to get the last value for the acceptance criteria
            d_log_sum_exp_before_.data,
            d_log_sum_exp_after_.data,
            d_intermediate_coords_.data,
            d_coords,
            d_num_accepted_.data);
        gpuErrchk(hipPeekAtLastError());
        num_attempted_++;
        noise_offset_ += QUATERNIONS_PER_STEP;
    }
}

template <typename RealType>
void BDExchangeMove<RealType>::compute_initial_weights(
    const int N, double *d_coords, double *d_box, hipStream_t stream) {
    const int tpb = DEFAULT_THREADS_PER_BLOCK;
    const int mol_blocks = ceil_divide(num_target_mols_, tpb);
    mol_potential_.mol_energies_device(
        N,
        num_target_mols_,
        d_coords,
        d_params_.data,
        d_box,
        d_mol_energy_buffer_.data, // Don't need to zero, will be overridden
        stream);

    // Don't need to normalize to sample
    k_compute_log_weights_from_energies<RealType><<<mol_blocks, tpb, 0, stream>>>(
        num_target_mols_, beta_, d_mol_energy_buffer_.data, d_log_weights_before_.data);
    gpuErrchk(hipPeekAtLastError());

    // Compute logsumexp of energies once upfront to get log probabilities
    logsumexp_.sum_device(num_target_mols_, d_log_weights_before_.data, d_log_sum_exp_before_.data, stream);
}

template <typename RealType>
void BDExchangeMove<RealType>::compute_incremental_weights(
    const int N,
    const bool scale,
    double *d_coords,        // [N, 3]
    double *d_box,           // [3, 3]
    RealType *d_quaternions, // [4]
    hipStream_t stream) {
    const int tpb = DEFAULT_THREADS_PER_BLOCK;
    dim3 atom_by_atom_grid(ceil_divide(N, tpb), mol_size_, 1);

    // Make a copy of the coordinates
    gpuErrchk(hipMemcpyAsync(
        d_intermediate_coords_.data, d_coords, d_intermediate_coords_.size(), hipMemcpyDeviceToDevice, stream));

    // Only support sampling a single mol at this time, so only one block
    k_setup_sample_atoms<<<1, tpb, 0, stream>>>(
        mol_size_,
        d_samples_.data,
        d_atom_idxs_.data,
        d_mol_offsets_.data,
        d_target_mol_atoms_.data,
        d_target_mol_offsets_.data);
    gpuErrchk(hipPeekAtLastError());

    if (scale) {
        k_rotate_and_translate_mols<RealType, true><<<1, tpb, 0, stream>>>(
            1,
            d_coords,
            d_box,
            d_samples_.data,
            d_target_mol_offsets_.data,
            d_quaternions,
            d_translations_.data,
            d_intermediate_coords_.data);
        gpuErrchk(hipPeekAtLastError());
    } else {
        k_rotate_and_translate_mols<RealType, false><<<1, tpb, 0, stream>>>(
            1,
            d_coords,
            d_box,
            d_samples_.data,
            d_target_mol_offsets_.data,
            d_quaternions,
            d_translations_.data,
            d_intermediate_coords_.data);
        gpuErrchk(hipPeekAtLastError());
    }

    k_atom_by_atom_energies<<<atom_by_atom_grid, tpb, 0, stream>>>(
        N,
        mol_size_,
        d_target_mol_atoms_.data,
        d_coords,
        d_params_.data,
        d_box,
        nb_beta_,
        cutoff_squared_,
        d_sample_per_atom_energy_buffer_.data);
    gpuErrchk(hipPeekAtLastError());

    // Subtract off the weights for the individual waters from the sampled water.
    // It modifies the sampled mol energy value, leaving it in an invalid state, which is why
    // we later call k_set_sampled_weight to set the weight of the sampled mol
    k_adjust_weights<RealType, true><<<ceil_divide(num_target_mols_, tpb), tpb, 0, stream>>>(
        N,
        num_target_mols_,
        mol_size_,
        d_atom_idxs_.data,
        d_mol_offsets_.data,
        d_sample_per_atom_energy_buffer_.data,
        beta_, // 1 / kT
        d_log_weights_after_.data);
    gpuErrchk(hipPeekAtLastError());

    k_atom_by_atom_energies<<<atom_by_atom_grid, tpb, 0, stream>>>(
        N,
        mol_size_,
        d_target_mol_atoms_.data,
        d_intermediate_coords_.data,
        d_params_.data,
        d_box,
        nb_beta_,
        cutoff_squared_,
        d_sample_per_atom_energy_buffer_.data);
    gpuErrchk(hipPeekAtLastError());

    // Add in the new weights from the individual waters
    // the sampled weight continues to be garbage
    k_adjust_weights<RealType, false><<<ceil_divide(num_target_mols_, tpb), tpb, 0, stream>>>(
        N,
        num_target_mols_,
        mol_size_,
        d_atom_idxs_.data,
        d_mol_offsets_.data,
        d_sample_per_atom_energy_buffer_.data,
        beta_, // 1 / kT
        d_log_weights_after_.data);
    gpuErrchk(hipPeekAtLastError());

    // Set the sampled weight to be the correct value
    k_set_sampled_weight_block<RealType, WEIGHT_THREADS_PER_BLOCK>
        <<<static_cast<int>(d_intermediate_sample_weights_.length), WEIGHT_THREADS_PER_BLOCK, 0, stream>>>(
            N,
            mol_size_,
            d_target_mol_atoms_.data,
            d_sample_per_atom_energy_buffer_.data,
            beta_, // 1 / kT
            d_intermediate_sample_weights_.data);
    gpuErrchk(hipPeekAtLastError());

    k_set_sampled_weight_reduce<RealType, WEIGHT_THREADS_PER_BLOCK><<<1, WEIGHT_THREADS_PER_BLOCK, 0, stream>>>(
        static_cast<int>(d_intermediate_sample_weights_.length), // Number of intermediates
        d_samples_.data,                                         // where to set the value
        d_intermediate_sample_weights_.data,                     // intermediate fixed point weights
        d_log_weights_after_.data);
    gpuErrchk(hipPeekAtLastError());
}

template <typename RealType> double BDExchangeMove<RealType>::raw_log_probability_host() {
    std::vector<RealType> h_log_exp_before(2);
    std::vector<RealType> h_log_exp_after(2);
    d_log_sum_exp_before_.copy_to(&h_log_exp_before[0]);
    d_log_sum_exp_after_.copy_to(&h_log_exp_after[0]);

    RealType before_log_prob = convert_nan_to_inf(compute_logsumexp_final(&h_log_exp_before[0]));
    RealType after_log_prob = convert_nan_to_inf(compute_logsumexp_final(&h_log_exp_after[0]));

    return static_cast<double>(before_log_prob - after_log_prob);
}

template <typename RealType> double BDExchangeMove<RealType>::log_probability_host() {
    return min(raw_log_probability_host(), 0.0);
}

template <typename RealType> size_t BDExchangeMove<RealType>::n_accepted() const {
    size_t h_accepted;
    d_num_accepted_.copy_to(&h_accepted);
    return h_accepted;
}

template <typename RealType> std::vector<double> BDExchangeMove<RealType>::get_params() {
    std::vector<double> h_params(d_params_.length);
    d_params_.copy_to(&h_params[0]);
    return h_params;
};

template <typename RealType> void BDExchangeMove<RealType>::set_params(const std::vector<double> &params) {
    hipStream_t stream = static_cast<hipStream_t>(0);
    DeviceBuffer<double> d_params(params.size());
    d_params.copy_from(&params[0]);
    this->set_params_device(params.size(), d_params.data, stream);
    gpuErrchk(hipStreamSynchronize(stream));
};

template <typename RealType>
void BDExchangeMove<RealType>::set_params_device(const int size, const double *d_p, const hipStream_t stream) {
    if (d_params_.length != size) {
        throw std::runtime_error("number of params don't match");
    }
    gpuErrchk(hipMemcpyAsync(d_params_.data, d_p, d_params_.size(), hipMemcpyDeviceToDevice, stream));
};

template class BDExchangeMove<float>;
template class BDExchangeMove<double>;

} // namespace timemachine
