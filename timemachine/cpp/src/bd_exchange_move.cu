#include "hip/hip_runtime.h"
#include "bd_exchange_move.hpp"

#include "constants.hpp"
#include "gpu_utils.cuh"
#include "kernels/k_exchange.cuh"
#include "kernels/k_indices.cuh"
#include "kernels/k_nonbonded.cuh"
#include "kernels/k_probability.cuh"
#include "kernels/k_rotations.cuh"
#include "math_utils.cuh"
#include "mol_utils.hpp"

namespace timemachine {
// The number of threads per block for the setting of the final weight of the moved mol is low
// if using the same number as in the rest of the kernels of DEFAULT_THREADS_PER_BLOCK
static const int WEIGHT_THREADS_PER_BLOCK = 512;
// The number of translations to generate each step. The first three values are a unit vector translation and the fourth
// value is used for the metropolis hasting check
static const int BD_TRANSLATIONS_PER_STEP_XYZW = 4;

template <typename RealType>
BDExchangeMove<RealType>::BDExchangeMove(
    const int N,
    const std::vector<std::vector<int>> &target_mols,
    const std::vector<double> &params,
    const double temperature,
    const double nb_beta,
    const double cutoff,
    const int seed,
    const int num_proposals_per_move,
    const int interval,
    const int batch_size)
    : BDExchangeMove<RealType>(
          N,
          target_mols,
          params,
          temperature,
          nb_beta,
          cutoff,
          seed,
          num_proposals_per_move,
          interval,
          batch_size,
          round_up_even(BD_TRANSLATIONS_PER_STEP_XYZW * num_proposals_per_move)) {}

template <typename RealType>
BDExchangeMove<RealType>::BDExchangeMove(
    const int N,
    const std::vector<std::vector<int>> &target_mols,
    const std::vector<double> &params,
    const double temperature,
    const double nb_beta,
    const double cutoff,
    const int seed,
    const int num_proposals_per_move,
    const int interval,
    const int batch_size,
    const int translation_buffer_size)
    : Mover(interval), N_(N), mol_size_(target_mols[0].size()), num_proposals_per_move_(num_proposals_per_move),
      steps_per_move_(num_proposals_per_move_ / batch_size), num_target_mols_(target_mols.size()),
      nb_beta_(static_cast<RealType>(nb_beta)), beta_(static_cast<RealType>(1.0 / (BOLTZ * temperature))),
      cutoff_squared_(static_cast<RealType>(cutoff * cutoff)), batch_size_(batch_size), num_attempted_(0),
      mol_potential_(N, target_mols, nb_beta, cutoff), sampler_(num_target_mols_, batch_size_, seed),
      logsumexp_(num_target_mols_, batch_size_), d_intermediate_coords_(batch_size_ * mol_size_ * 3), d_params_(params),
      d_mol_energy_buffer_(batch_size_ * num_target_mols_),
      d_sample_per_atom_energy_buffer_(batch_size_ * mol_size_ * N), d_atom_idxs_(get_atom_indices(target_mols)),
      d_mol_offsets_(get_mol_offsets(target_mols)), d_log_weights_before_(num_target_mols_),
      d_log_weights_after_(num_target_mols_), d_lse_max_before_(1), d_lse_exp_sum_before_(1),
      d_lse_max_after_(batch_size_), d_lse_exp_sum_after_(batch_size_), d_samples_(batch_size_),
      d_quaternions_(round_up_even(QUATERNIONS_PER_STEP * num_proposals_per_move_ * batch_size_)), d_num_accepted_(1),
      d_target_mol_atoms_(batch_size_ * mol_size_), d_target_mol_offsets_(num_target_mols_ + 1),
      d_intermediate_sample_weights_(batch_size_ * ceil_divide(N_, WEIGHT_THREADS_PER_BLOCK)),
      d_sample_noise_(round_up_even(num_target_mols_ * num_proposals_per_move_)),
      d_sampling_intermediate_(num_target_mols_ * batch_size_), d_translations_(translation_buffer_size),
      d_sample_segments_offsets_(batch_size_ + 1) {

    if (num_proposals_per_move_ <= 0) {
        throw std::runtime_error("proposals per move must be greater than 0");
    }
    if (mol_size_ == 0) {
        throw std::runtime_error("must provide non-empty molecule indices");
    }
    if (num_proposals_per_move_ % batch_size_ != 0) {
        throw std::runtime_error("num_proposals_per_move must be a multiple of batch size");
    }
    verify_mols_contiguous(target_mols);
    for (int i = 0; i < target_mols.size(); i++) {
        if (target_mols[i].size() != mol_size_) {
            throw std::runtime_error("only support running with mols with constant size, got mixed sizes");
        }
    }
    // Clear out the logsumexp values so the log probability starts off as zero
    gpuErrchk(hipMemset(d_lse_exp_sum_before_.data, 0, d_lse_exp_sum_before_.size()));
    gpuErrchk(hipMemset(d_lse_max_before_.data, 0, d_lse_max_before_.size()));
    gpuErrchk(hipMemset(d_lse_exp_sum_after_.data, 0, d_lse_exp_sum_after_.size()));
    gpuErrchk(hipMemset(d_lse_max_after_.data, 0, d_lse_max_after_.size()));
    gpuErrchk(hipMemset(d_num_accepted_.data, 0, d_num_accepted_.size()));

    // Initialize several different RNGs to allow for determinism between numbers of steps per move
    curandErrchk(hiprandCreateGenerator(&cr_rng_quat_, HIPRAND_RNG_PSEUDO_DEFAULT));
    curandErrchk(hiprandSetPseudoRandomGeneratorSeed(cr_rng_quat_, seed));

    curandErrchk(hiprandCreateGenerator(&cr_rng_translations_, HIPRAND_RNG_PSEUDO_DEFAULT));
    curandErrchk(hiprandSetPseudoRandomGeneratorSeed(cr_rng_translations_, seed + 1));

    curandErrchk(hiprandCreateGenerator(&cr_rng_samples_, HIPRAND_RNG_PSEUDO_DEFAULT));
    curandErrchk(hiprandSetPseudoRandomGeneratorSeed(cr_rng_samples_, seed + 2));

    // Setup the sample segments
    // constant for BDExchangeMove since the sample size is always batches of num_target_mols_ weights
    std::vector<int> h_sample_segments(d_sample_segments_offsets_.length);
    int offset = 0;
    for (unsigned int i = 0; i < h_sample_segments.size(); i++) {
        h_sample_segments[i] = offset;
        offset += num_target_mols_;
    }
    d_sample_segments_offsets_.copy_from(&h_sample_segments[0]);
}

template <typename RealType> BDExchangeMove<RealType>::~BDExchangeMove() {
    curandErrchk(hiprandDestroyGenerator(cr_rng_quat_));
    curandErrchk(hiprandDestroyGenerator(cr_rng_translations_));
    curandErrchk(hiprandDestroyGenerator(cr_rng_samples_));
}

template <typename RealType>
void BDExchangeMove<RealType>::move(
    const int N,
    double *d_coords, // [N, 3]
    double *d_box,    // [3, 3]
    hipStream_t stream) {

    if (N != N_) {
        throw std::runtime_error("N != N_");
    }
    this->step_++;
    if (this->step_ % this->interval_ != 0) {
        return;
    }
    if (d_translations_.length / BD_TRANSLATIONS_PER_STEP_XYZW !=
        this->d_quaternions_.length / this->QUATERNIONS_PER_STEP) {
        throw std::runtime_error("bug in the code: buffers with random values don't match in batch size");
    }

    // Set the stream for the generators
    curandErrchk(hiprandSetStream(cr_rng_quat_, stream));
    curandErrchk(hiprandSetStream(cr_rng_translations_, stream));
    curandErrchk(hiprandSetStream(cr_rng_samples_, stream));

    const int tpb = DEFAULT_THREADS_PER_BLOCK;

    this->compute_initial_weights(N, d_coords, d_box, stream);

    // All of the noise is generated upfront
    curandErrchk(templateCurandNormal(cr_rng_quat_, d_quaternions_.data, d_quaternions_.length, 0.0, 1.0));
    // The d_translation_ buffer contains uniform noise over [0, 1] containing [x,y,z,w] where [x,y,z] are a random
    // translation and w is used in the metropolis-hastings check
    curandErrchk(templateCurandUniform(cr_rng_translations_, d_translations_.data, d_translations_.length));
    curandErrchk(templateCurandUniform(cr_rng_samples_, d_sample_noise_.data, d_sample_noise_.length));
    for (int step = 0; step < steps_per_move_; step++) {
        // Run only after the first pass, to maintain meaningful `log_probability_host` values
        if (step > 0) {
            // Run a separate kernel to replace the before log probs and weights with the after if accepted a move
            // Need the weights to sample a value and the log probs are just because they aren't expensive to copy
            k_store_accepted_log_probability<RealType><<<1, tpb, 0>>>(
                num_target_mols_,
                d_translations_.data + (step * BD_TRANSLATIONS_PER_STEP_XYZW * batch_size_) +
                    (BD_TRANSLATIONS_PER_STEP_XYZW - 1), // Offset to get the last value for the acceptance criteria
                d_lse_max_before_.data,
                d_lse_exp_sum_before_.data,
                d_lse_max_after_.data,
                d_lse_exp_sum_after_.data,
                d_log_weights_before_.data,
                d_log_weights_after_.data);
            gpuErrchk(hipPeekAtLastError());
        }

        gpuErrchk(hipMemcpyAsync(
            d_log_weights_after_.data,
            d_log_weights_before_.data,
            d_log_weights_after_.size(),
            hipMemcpyDeviceToDevice,
            stream));

        // We only ever sample a single molecule
        sampler_.sample_given_noise_device(
            num_target_mols_ * batch_size_,
            batch_size_,
            d_sample_segments_offsets_.data,
            d_log_weights_before_.data,
            d_sample_noise_.data + (step * num_target_mols_ * batch_size_),
            d_sampling_intermediate_.data,
            d_samples_.data,
            stream);

        // Don't move translations into computation of the incremental, as different translations can be used
        // by different bias deletion movers (such as targeted insertion)
        // scale the translations as they are between [0, 1]
        this->compute_incremental_weights(
            N,
            true,
            d_box,
            d_coords,
            this->d_quaternions_.data + (step * QUATERNIONS_PER_STEP * batch_size_),
            this->d_translations_.data + (step * BD_TRANSLATIONS_PER_STEP_XYZW * batch_size_),
            stream);

        logsumexp_.sum_device(
            num_target_mols_ * batch_size_,
            batch_size_,
            d_sample_segments_offsets_.data,
            d_log_weights_after_.data,
            d_lse_max_after_.data,
            d_lse_exp_sum_after_.data,
            stream);

        k_attempt_exchange_move<RealType><<<1, 1, 0, stream>>>(
            N,
            d_translations_.data + (step * BD_TRANSLATIONS_PER_STEP_XYZW * batch_size_) +
                (BD_TRANSLATIONS_PER_STEP_XYZW - 1), // Offset to get the last value for the acceptance criteria
            d_lse_max_before_.data,
            d_lse_exp_sum_before_.data,
            d_lse_max_after_.data,
            d_lse_exp_sum_after_.data,
            d_target_mol_offsets_.data,
            d_samples_.data,
            d_intermediate_coords_.data,
            d_coords,
            d_num_accepted_.data);
        gpuErrchk(hipPeekAtLastError());
        num_attempted_++;
    }
}

template <typename RealType>
void BDExchangeMove<RealType>::compute_initial_weights(
    const int N, double *d_coords, double *d_box, hipStream_t stream) {
    const int tpb = DEFAULT_THREADS_PER_BLOCK;
    const int mol_blocks = ceil_divide(num_target_mols_, tpb);
    mol_potential_.mol_energies_device(
        N,
        num_target_mols_,
        d_coords,
        d_params_.data,
        d_box,
        d_mol_energy_buffer_.data, // Don't need to zero, will be overridden
        stream);

    // Don't need to normalize to sample
    k_compute_log_weights_from_energies<RealType><<<mol_blocks, tpb, 0, stream>>>(
        num_target_mols_, beta_, d_mol_energy_buffer_.data, d_log_weights_before_.data);
    gpuErrchk(hipPeekAtLastError());

    // Compute logsumexp of energies once upfront to get log probabilities
    logsumexp_.sum_device(
        num_target_mols_,
        1,
        d_sample_segments_offsets_.data,
        d_log_weights_before_.data,
        d_lse_max_before_.data,
        d_lse_exp_sum_before_.data,
        stream);
}

template <typename RealType>
void BDExchangeMove<RealType>::compute_incremental_weights(
    const int N,
    const bool scale,
    const double *d_box,            // [3, 3]
    const double *d_coords,         // [N, 3]
    const RealType *d_quaternions,  // [batch_size_, 4]
    const RealType *d_translations, // [batch_size_, 3]
    hipStream_t stream) {
    const int tpb = DEFAULT_THREADS_PER_BLOCK;
    dim3 atom_by_atom_grid(ceil_divide(N, tpb), mol_size_ * batch_size_, 1);

    k_setup_proposals<<<ceil_divide(batch_size_, tpb), tpb, 0, stream>>>(
        batch_size_,
        mol_size_,
        d_samples_.data,
        d_atom_idxs_.data,
        d_mol_offsets_.data,
        d_target_mol_atoms_.data,
        d_target_mol_offsets_.data);
    gpuErrchk(hipPeekAtLastError());

    if (scale) {
        k_rotate_and_translate_mols<RealType, true><<<ceil_divide(batch_size_, tpb), tpb, 0, stream>>>(
            batch_size_,
            d_coords,
            d_box,
            d_samples_.data,
            d_target_mol_offsets_.data,
            d_quaternions,
            d_translations,
            d_intermediate_coords_.data);
        gpuErrchk(hipPeekAtLastError());
    } else {
        k_rotate_and_translate_mols<RealType, false><<<ceil_divide(batch_size_, tpb), tpb, 0, stream>>>(
            batch_size_,
            d_coords,
            d_box,
            d_samples_.data,
            d_target_mol_offsets_.data,
            d_quaternions,
            d_translations,
            d_intermediate_coords_.data);
        gpuErrchk(hipPeekAtLastError());
    }

    k_atom_by_atom_energies<<<atom_by_atom_grid, tpb, 0, stream>>>(
        N,
        mol_size_,
        d_target_mol_atoms_.data,
        nullptr,
        d_coords,
        d_params_.data,
        d_box,
        nb_beta_,
        cutoff_squared_,
        d_sample_per_atom_energy_buffer_.data);
    gpuErrchk(hipPeekAtLastError());

    // Subtract off the weights for the individual waters from the sampled water.
    // It modifies the sampled mol energy value, leaving it in an invalid state, which is why
    // we later call k_set_sampled_weight to set the weight of the sampled mol
    k_adjust_weights<RealType, true><<<ceil_divide(num_target_mols_, tpb), tpb, 0, stream>>>(
        N,
        num_target_mols_,
        mol_size_,
        d_atom_idxs_.data,
        d_mol_offsets_.data,
        d_sample_per_atom_energy_buffer_.data,
        beta_, // 1 / kT
        d_log_weights_after_.data);
    gpuErrchk(hipPeekAtLastError());

    k_atom_by_atom_energies<<<atom_by_atom_grid, tpb, 0, stream>>>(
        N,
        mol_size_,
        d_target_mol_atoms_.data,
        d_intermediate_coords_.data,
        d_coords,
        d_params_.data,
        d_box,
        nb_beta_,
        cutoff_squared_,
        d_sample_per_atom_energy_buffer_.data);
    gpuErrchk(hipPeekAtLastError());

    // Add in the new weights from the individual waters
    // the sampled weight continues to be garbage
    k_adjust_weights<RealType, false><<<ceil_divide(num_target_mols_, tpb), tpb, 0, stream>>>(
        N,
        num_target_mols_,
        mol_size_,
        d_atom_idxs_.data,
        d_mol_offsets_.data,
        d_sample_per_atom_energy_buffer_.data,
        beta_, // 1 / kT
        d_log_weights_after_.data);
    gpuErrchk(hipPeekAtLastError());

    // Set the sampled weight to be the correct value
    k_set_sampled_weight_block<RealType, WEIGHT_THREADS_PER_BLOCK>
        <<<static_cast<int>(d_intermediate_sample_weights_.length), WEIGHT_THREADS_PER_BLOCK, 0, stream>>>(
            N,
            mol_size_,
            d_target_mol_atoms_.data,
            d_sample_per_atom_energy_buffer_.data,
            beta_, // 1 / kT
            d_intermediate_sample_weights_.data);
    gpuErrchk(hipPeekAtLastError());

    k_set_sampled_weight_reduce<RealType, WEIGHT_THREADS_PER_BLOCK><<<1, WEIGHT_THREADS_PER_BLOCK, 0, stream>>>(
        static_cast<int>(d_intermediate_sample_weights_.length), // Number of intermediates
        d_samples_.data,                                         // where to set the value
        d_intermediate_sample_weights_.data,                     // intermediate fixed point weights
        d_log_weights_after_.data);
    gpuErrchk(hipPeekAtLastError());
}

template <typename RealType> double BDExchangeMove<RealType>::raw_log_probability_host() {
    std::vector<RealType> h_log_exp_before(2);
    std::vector<RealType> h_log_exp_after(2);
    d_lse_max_before_.copy_to(&h_log_exp_before[0]);
    d_lse_exp_sum_before_.copy_to(&h_log_exp_before[1]);
    d_lse_max_after_.copy_to(&h_log_exp_after[0]);
    d_lse_exp_sum_after_.copy_to(&h_log_exp_after[1]);

    RealType before_log_prob = convert_nan_to_inf(compute_logsumexp_final(h_log_exp_before[0], h_log_exp_before[1]));
    RealType after_log_prob = convert_nan_to_inf(compute_logsumexp_final(h_log_exp_after[0], h_log_exp_after[1]));

    return static_cast<double>(before_log_prob - after_log_prob);
}

template <typename RealType> double BDExchangeMove<RealType>::log_probability_host() {
    return min(raw_log_probability_host(), 0.0);
}

template <typename RealType> size_t BDExchangeMove<RealType>::n_accepted() const {
    size_t h_accepted;
    d_num_accepted_.copy_to(&h_accepted);
    return h_accepted;
}

template <typename RealType> std::vector<double> BDExchangeMove<RealType>::get_params() {
    std::vector<double> h_params(d_params_.length);
    d_params_.copy_to(&h_params[0]);
    return h_params;
};

template <typename RealType> void BDExchangeMove<RealType>::set_params(const std::vector<double> &params) {
    hipStream_t stream = static_cast<hipStream_t>(0);
    DeviceBuffer<double> d_params(params.size());
    d_params.copy_from(&params[0]);
    this->set_params_device(params.size(), d_params.data, stream);
    gpuErrchk(hipStreamSynchronize(stream));
};

template <typename RealType>
void BDExchangeMove<RealType>::set_params_device(const int size, const double *d_p, const hipStream_t stream) {
    if (d_params_.length != size) {
        throw std::runtime_error("number of params don't match");
    }
    gpuErrchk(hipMemcpyAsync(d_params_.data, d_p, d_params_.size(), hipMemcpyDeviceToDevice, stream));
};

template class BDExchangeMove<float>;
template class BDExchangeMove<double>;

} // namespace timemachine
