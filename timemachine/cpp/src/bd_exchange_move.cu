#include "hip/hip_runtime.h"
#include "bd_exchange_move.hpp"

#include "constants.hpp"
#include "gpu_utils.cuh"
#include "kernels/k_exchange.cuh"
#include "kernels/k_probability.cuh"
#include "kernels/k_rotations.cuh"
#include "math_utils.cuh"
#include "mol_utils.hpp"

namespace timemachine {

template <typename RealType>
BDExchangeMove<RealType>::BDExchangeMove(
    const int N,
    const std::vector<std::vector<int>> &target_mols,
    const std::vector<double> &params,
    const double temperature,
    const double nb_beta,
    const double cutoff,
    const int seed,
    const int proposals_per_move)
    : N_(N), proposals_per_move_(proposals_per_move), num_target_mols_(target_mols.size()),
      beta_(static_cast<RealType>(1.0 / (BOLTZ * temperature))), mol_potential_(N, target_mols, nb_beta, cutoff),
      sampler_(num_target_mols_, seed), logsumexp_(N), d_intermediate_coords_(N * 3), d_params_(params.size()),
      d_mol_energy_buffer_(num_target_mols_), d_mol_offsets_(get_mol_offsets(target_mols).size()),
      d_log_weights_before_(num_target_mols_), d_log_weights_after_(num_target_mols_),
      d_log_probabilities_before_(num_target_mols_), d_log_probabilities_after_(num_target_mols_),
      d_log_sum_exp_before_(2), d_log_sum_exp_after_(2), d_samples_(1), d_quaternions_(round_up_even(4)),
      d_translations_(round_up_even(4)), d_num_accepted_(1), num_attempted_(0) {
    d_params_.copy_from(&params[0]);
    d_mol_offsets_.copy_from(&get_mol_offsets(target_mols)[0]);

    // Clear out the logsumexp values so the log probability starts off as zero
    gpuErrchk(hipMemset(d_log_sum_exp_before_.data, 0, d_log_sum_exp_before_.size()));
    gpuErrchk(hipMemset(d_log_sum_exp_after_.data, 0, d_log_sum_exp_after_.size()));
    curandErrchk(hiprandCreateGenerator(&cr_rng_, HIPRAND_RNG_PSEUDO_DEFAULT));
    curandErrchk(hiprandSetPseudoRandomGeneratorSeed(cr_rng_, seed));
}

template <typename RealType> BDExchangeMove<RealType>::~BDExchangeMove() {
    curandErrchk(hiprandDestroyGenerator(cr_rng_));
}

template <typename RealType>
void BDExchangeMove<RealType>::move_device(
    const int N,
    double *d_coords, // [N, 3]
    double *d_box,    // [3, 3]
    hipStream_t stream) {

    if (N != N_) {
        throw std::runtime_error("N != N_");
    }

    // Set the stream for the generator
    curandErrchk(hiprandSetStream(cr_rng_, stream));

    const int tpb = DEFAULT_THREADS_PER_BLOCK;
    const int mol_blocks = ceil_divide(num_target_mols_, tpb);
    // Compute logsumexp of energies once upfront to get log probabilities
    mol_potential_.mol_energies_device(
        N,
        num_target_mols_,
        d_coords,
        d_params_.data,
        d_box,
        d_mol_energy_buffer_.data, // Don't need to zero, will be overridden
        stream);

    // Don't need to normalize to sample
    k_compute_log_weights_from_energies<RealType><<<mol_blocks, tpb, 0, stream>>>(
        num_target_mols_, beta_, d_mol_energy_buffer_.data, d_log_weights_before_.data);
    gpuErrchk(hipPeekAtLastError());

    logsumexp_.sum_device(num_target_mols_, d_log_weights_before_.data, d_log_sum_exp_before_.data, stream);

    const int num_samples = 1;
    for (int move = 0; move < proposals_per_move_; move++) {
        // Run only after the first pass, to maintain meaningful `log_probability_host` values
        if (move > 0) {
            // Run a separate kernel to replace the before log probs and weights with the after if accepted a move
            // Need the weights to sample a value and the log probs are just because they aren't expensive to copy
            k_store_accepted_log_probability<RealType><<<1, tpb, 0>>>(
                num_target_mols_,
                d_translations_.data + 3, // Offset to get the last value for the acceptance criteria
                d_log_sum_exp_before_.data,
                d_log_sum_exp_after_.data,
                d_log_weights_before_.data,
                d_log_weights_after_.data);
            gpuErrchk(hipPeekAtLastError());
        }
        // Make a copy of the coordinates
        gpuErrchk(hipMemcpyAsync(
            d_intermediate_coords_.data, d_coords, d_intermediate_coords_.size(), hipMemcpyDeviceToDevice, stream));

        // Quaternions generated from normal noise generate uniform rotations
        curandErrchk(templateCurandNormal(cr_rng_, d_quaternions_.data, d_quaternions_.length, 0.0, 1.0));
        // The d_translation_ buffer is [x,y,z,w] where [x,y,z] are a random translation and w is used for acceptance
        curandErrchk(templateCurandUniform(cr_rng_, d_translations_.data, d_translations_.length));

        sampler_.sample_device(num_target_mols_, num_samples, d_log_weights_before_.data, d_samples_.data, stream);
        k_rotate_and_translate_mols<RealType><<<ceil_divide(num_samples, tpb), tpb, 0, stream>>>(
            num_samples,
            d_coords,
            d_box,
            d_samples_.data,
            d_mol_offsets_.data,
            d_quaternions_.data,
            d_translations_.data,
            d_intermediate_coords_.data);
        gpuErrchk(hipPeekAtLastError());

        mol_potential_.mol_energies_device(
            N,
            num_target_mols_,
            d_intermediate_coords_.data, // Use the moved coords
            d_params_.data,
            d_box,
            d_mol_energy_buffer_.data, // Don't need to zero, will be overridden
            stream);

        k_compute_log_weights_from_energies<RealType><<<mol_blocks, tpb, 0, stream>>>(
            num_target_mols_, beta_, d_mol_energy_buffer_.data, d_log_weights_after_.data);
        gpuErrchk(hipPeekAtLastError());

        logsumexp_.sum_device(num_target_mols_, d_log_weights_after_.data, d_log_sum_exp_after_.data, stream);

        k_attempt_exchange_move<RealType><<<ceil_divide(N_, tpb), tpb, 0, stream>>>(
            N,
            d_translations_.data + 3, // Offset to get the last value for the acceptance criteria
            d_log_sum_exp_before_.data,
            d_log_sum_exp_after_.data,
            d_intermediate_coords_.data,
            d_coords,
            d_num_accepted_.data);
        gpuErrchk(hipPeekAtLastError());
        num_attempted_++;
    }
}

template <typename RealType>
std::array<std::vector<double>, 2>
BDExchangeMove<RealType>::move_host(const int N, const double *h_coords, const double *h_box) {

    DeviceBuffer<double> d_coords(N * 3);
    d_coords.copy_from(h_coords);

    DeviceBuffer<double> d_box(3 * 3);
    d_box.copy_from(h_box);

    hipStream_t stream = static_cast<hipStream_t>(0);

    this->move_device(N, d_coords.data, d_box.data, stream);
    gpuErrchk(hipStreamSynchronize(stream));

    std::vector<double> out_coords(d_coords.length);
    d_coords.copy_to(&out_coords[0]);

    std::vector<double> out_box(d_box.length);
    d_box.copy_to(&out_box[0]);

    return std::array<std::vector<double>, 2>({out_coords, out_box});
}

template <typename RealType> double BDExchangeMove<RealType>::log_probability_host() {
    std::vector<RealType> h_log_exp_before(2);
    std::vector<RealType> h_log_exp_after(2);
    d_log_sum_exp_before_.copy_to(&h_log_exp_before[0]);
    d_log_sum_exp_after_.copy_to(&h_log_exp_after[0]);

    RealType before_log_prob = convert_nan_to_inf(compute_logsumexp_final(&h_log_exp_before[0]));
    RealType after_log_prob = convert_nan_to_inf(compute_logsumexp_final(&h_log_exp_after[0]));

    return min(static_cast<double>(before_log_prob - after_log_prob), 0.0);
}

template <typename RealType> size_t BDExchangeMove<RealType>::n_accepted() const {
    size_t h_accepted;
    d_num_accepted_.copy_to(&h_accepted);
    return h_accepted;
}

template class BDExchangeMove<float>;
template class BDExchangeMove<double>;

} // namespace timemachine
