#include "hip/hip_runtime.h"
#include "bd_exchange_move.hpp"

#include "constants.hpp"
#include "gpu_utils.cuh"
#include "kernels/k_exchange.cuh"
#include "kernels/k_indices.cuh"
#include "kernels/k_nonbonded.cuh"
#include "kernels/k_probability.cuh"
#include "kernels/k_rotations.cuh"
#include "math_utils.cuh"
#include "mol_utils.hpp"

namespace timemachine {
// The number of threads per block for the setting of the final weight of the moved mol is low
// if using the same number as in the rest of the kernels of DEFAULT_THREADS_PER_BLOCK
static const int WEIGHT_THREADS_PER_BLOCK = 512;
// The number of translations to generate each step. The first three values are a unit vector translation and the fourth
// value is used for the metropolis hasting check
static const int BD_TRANSLATIONS_PER_STEP_XYZW = 4;

template <typename RealType>
BDExchangeMove<RealType>::BDExchangeMove(
    const int N,
    const std::vector<std::vector<int>> &target_mols,
    const std::vector<double> &params,
    const double temperature,
    const double nb_beta,
    const double cutoff,
    const int seed,
    const int proposals_per_move,
    const int interval)
    : BDExchangeMove<RealType>(
          N,
          target_mols,
          params,
          temperature,
          nb_beta,
          cutoff,
          seed,
          proposals_per_move,
          interval,
          round_up_even(BD_TRANSLATIONS_PER_STEP_XYZW * proposals_per_move)) {}

template <typename RealType>
BDExchangeMove<RealType>::BDExchangeMove(
    const int N,
    const std::vector<std::vector<int>> &target_mols,
    const std::vector<double> &params,
    const double temperature,
    const double nb_beta,
    const double cutoff,
    const int seed,
    const int proposals_per_move,
    const int interval,
    const int translation_buffer_size)
    : Mover(interval), N_(N), mol_size_(target_mols[0].size()), proposals_per_move_(proposals_per_move),
      num_target_mols_(target_mols.size()), nb_beta_(static_cast<RealType>(nb_beta)),
      beta_(static_cast<RealType>(1.0 / (BOLTZ * temperature))),
      cutoff_squared_(static_cast<RealType>(cutoff * cutoff)),
      samples_per_proposal_(1), // Hardcoded to 1 currently, will be exposed in the constructor eventually
      num_attempted_(0), mol_potential_(N, target_mols, nb_beta, cutoff),
      sampler_(num_target_mols_, samples_per_proposal_, seed), logsumexp_(num_target_mols_),
      d_intermediate_coords_(N * 3), d_params_(params), d_mol_energy_buffer_(num_target_mols_),
      d_sample_per_atom_energy_buffer_(mol_size_ * N), d_atom_idxs_(get_atom_indices(target_mols)),
      d_mol_offsets_(get_mol_offsets(target_mols)), d_log_weights_before_(num_target_mols_),
      d_log_weights_after_(num_target_mols_), d_log_sum_exp_before_(2), d_log_sum_exp_after_(2),
      d_samples_(samples_per_proposal_), d_quaternions_(round_up_even(QUATERNIONS_PER_STEP * proposals_per_move_)),
      d_num_accepted_(1), d_target_mol_atoms_(mol_size_), d_target_mol_offsets_(num_target_mols_ + 1),
      d_intermediate_sample_weights_(ceil_divide(N_, WEIGHT_THREADS_PER_BLOCK)),
      d_sample_noise_(round_up_even(num_target_mols_ * proposals_per_move_)),
      d_sampling_intermediate_(num_target_mols_), d_translations_(translation_buffer_size),
      d_sample_segments_offsets_(samples_per_proposal_ + 1) {

    if (proposals_per_move_ <= 0) {
        throw std::runtime_error("proposals per move must be greater than 0");
    }
    if (mol_size_ == 0) {
        throw std::runtime_error("must provide non-empty molecule indices");
    }
    verify_mols_contiguous(target_mols);
    for (int i = 0; i < target_mols.size(); i++) {
        if (target_mols[i].size() != mol_size_) {
            throw std::runtime_error("only support running with mols with constant size, got mixed sizes");
        }
    }
    // Clear out the logsumexp values so the log probability starts off as zero
    gpuErrchk(hipMemset(d_log_sum_exp_before_.data, 0, d_log_sum_exp_before_.size()));
    gpuErrchk(hipMemset(d_log_sum_exp_after_.data, 0, d_log_sum_exp_after_.size()));
    gpuErrchk(hipMemset(d_num_accepted_.data, 0, d_num_accepted_.size()));

    // Initialize several different RNGs to allow for determinism between numbers of proposals per move
    curandErrchk(hiprandCreateGenerator(&cr_rng_quat_, HIPRAND_RNG_PSEUDO_DEFAULT));
    curandErrchk(hiprandSetPseudoRandomGeneratorSeed(cr_rng_quat_, seed));

    curandErrchk(hiprandCreateGenerator(&cr_rng_translations_, HIPRAND_RNG_PSEUDO_DEFAULT));
    curandErrchk(hiprandSetPseudoRandomGeneratorSeed(cr_rng_translations_, seed + 1));

    curandErrchk(hiprandCreateGenerator(&cr_rng_samples_, HIPRAND_RNG_PSEUDO_DEFAULT));
    curandErrchk(hiprandSetPseudoRandomGeneratorSeed(cr_rng_samples_, seed + 2));

    // Setup the sample segments
    // constant for BDExchangeMove since the sample size is always batches of num_target_mols_ weights
    std::vector<int> h_sample_segments(d_sample_segments_offsets_.length);
    int offset = 0;
    for (unsigned int i = 0; i < h_sample_segments.size(); i++) {
        h_sample_segments[i] = offset;
        offset += num_target_mols_;
    }
    d_sample_segments_offsets_.copy_from(&h_sample_segments[0]);
}

template <typename RealType> BDExchangeMove<RealType>::~BDExchangeMove() {
    curandErrchk(hiprandDestroyGenerator(cr_rng_quat_));
    curandErrchk(hiprandDestroyGenerator(cr_rng_translations_));
    curandErrchk(hiprandDestroyGenerator(cr_rng_samples_));
}

template <typename RealType>
void BDExchangeMove<RealType>::move(
    const int N,
    double *d_coords, // [N, 3]
    double *d_box,    // [3, 3]
    hipStream_t stream) {

    if (N != N_) {
        throw std::runtime_error("N != N_");
    }
    this->step_++;
    if (this->step_ % this->interval_ != 0) {
        return;
    }
    if (d_translations_.length / BD_TRANSLATIONS_PER_STEP_XYZW !=
        this->d_quaternions_.length / this->QUATERNIONS_PER_STEP) {
        throw std::runtime_error("bug in the code: buffers with random values don't match in batch size");
    }

    // Set the stream for the generators
    curandErrchk(hiprandSetStream(cr_rng_quat_, stream));
    curandErrchk(hiprandSetStream(cr_rng_translations_, stream));
    curandErrchk(hiprandSetStream(cr_rng_samples_, stream));

    const int tpb = DEFAULT_THREADS_PER_BLOCK;

    this->compute_initial_weights(N, d_coords, d_box, stream);

    // All of the noise is generated upfront
    curandErrchk(templateCurandNormal(cr_rng_quat_, d_quaternions_.data, d_quaternions_.length, 0.0, 1.0));
    // The d_translation_ buffer contains uniform noise over [0, 1] containing [x,y,z,w] where [x,y,z] are a random
    // translation and w is used in the metropolis-hastings check
    curandErrchk(templateCurandUniform(cr_rng_translations_, d_translations_.data, d_translations_.length));
    curandErrchk(templateCurandUniform(cr_rng_samples_, d_sample_noise_.data, d_sample_noise_.length));
    for (int step = 0; step < proposals_per_move_; step++) {
        // Run only after the first pass, to maintain meaningful `log_probability_host` values
        if (step > 0) {
            // Run a separate kernel to replace the before log probs and weights with the after if accepted a move
            // Need the weights to sample a value and the log probs are just because they aren't expensive to copy
            k_store_accepted_log_probability<RealType><<<1, tpb, 0>>>(
                num_target_mols_,
                d_translations_.data + (step * BD_TRANSLATIONS_PER_STEP_XYZW * samples_per_proposal_) +
                    (BD_TRANSLATIONS_PER_STEP_XYZW - 1), // Offset to get the last value for the acceptance criteria
                d_log_sum_exp_before_.data,
                d_log_sum_exp_after_.data,
                d_log_weights_before_.data,
                d_log_weights_after_.data);
            gpuErrchk(hipPeekAtLastError());
        }

        gpuErrchk(hipMemcpyAsync(
            d_log_weights_after_.data,
            d_log_weights_before_.data,
            d_log_weights_after_.size(),
            hipMemcpyDeviceToDevice,
            stream));

        // We only ever sample a single molecule
        sampler_.sample_given_noise_device(
            num_target_mols_ * samples_per_proposal_,
            samples_per_proposal_,
            d_sample_segments_offsets_.data,
            d_log_weights_before_.data,
            d_sample_noise_.data + (step * num_target_mols_ * samples_per_proposal_),
            d_sampling_intermediate_.data,
            d_samples_.data,
            stream);

        // Don't move translations into computation of the incremental, as different translations can be used
        // by different bias deletion movers (such as targeted insertion)
        // scale the translations as they are between [0, 1]
        this->compute_incremental_weights(
            N,
            true,
            d_box,
            d_coords,
            this->d_quaternions_.data + (step * QUATERNIONS_PER_STEP * samples_per_proposal_),
            this->d_translations_.data + (step * BD_TRANSLATIONS_PER_STEP_XYZW * samples_per_proposal_),
            stream);

        logsumexp_.sum_device(num_target_mols_, d_log_weights_after_.data, d_log_sum_exp_after_.data, stream);

        k_attempt_exchange_move<RealType><<<ceil_divide(N_, tpb), tpb, 0, stream>>>(
            N,
            d_translations_.data + (step * BD_TRANSLATIONS_PER_STEP_XYZW * samples_per_proposal_) +
                (BD_TRANSLATIONS_PER_STEP_XYZW - 1), // Offset to get the last value for the acceptance criteria
            d_log_sum_exp_before_.data,
            d_log_sum_exp_after_.data,
            d_intermediate_coords_.data,
            d_coords,
            d_num_accepted_.data);
        gpuErrchk(hipPeekAtLastError());
        num_attempted_++;
    }
}

template <typename RealType>
void BDExchangeMove<RealType>::compute_initial_weights(
    const int N, double *d_coords, double *d_box, hipStream_t stream) {
    const int tpb = DEFAULT_THREADS_PER_BLOCK;
    const int mol_blocks = ceil_divide(num_target_mols_, tpb);
    mol_potential_.mol_energies_device(
        N,
        num_target_mols_,
        d_coords,
        d_params_.data,
        d_box,
        d_mol_energy_buffer_.data, // Don't need to zero, will be overridden
        stream);

    // Don't need to normalize to sample
    k_compute_log_weights_from_energies<RealType><<<mol_blocks, tpb, 0, stream>>>(
        num_target_mols_, beta_, d_mol_energy_buffer_.data, d_log_weights_before_.data);
    gpuErrchk(hipPeekAtLastError());

    // Compute logsumexp of energies once upfront to get log probabilities
    logsumexp_.sum_device(num_target_mols_, d_log_weights_before_.data, d_log_sum_exp_before_.data, stream);
}

template <typename RealType>
void BDExchangeMove<RealType>::compute_incremental_weights(
    const int N,
    const bool scale,
    const double *d_box,            // [3, 3]
    const double *d_coords,         // [N, 3]
    const RealType *d_quaternions,  // [4]
    const RealType *d_translations, // [3]
    hipStream_t stream) {
    const int tpb = DEFAULT_THREADS_PER_BLOCK;
    dim3 atom_by_atom_grid(ceil_divide(N, tpb), mol_size_, 1);

    // Make a copy of the coordinates
    gpuErrchk(hipMemcpyAsync(
        d_intermediate_coords_.data, d_coords, d_intermediate_coords_.size(), hipMemcpyDeviceToDevice, stream));

    // Only support sampling a single mol at this time, so only one block
    k_setup_sample_atoms<<<1, tpb, 0, stream>>>(
        mol_size_,
        d_samples_.data,
        d_atom_idxs_.data,
        d_mol_offsets_.data,
        d_target_mol_atoms_.data,
        d_target_mol_offsets_.data);
    gpuErrchk(hipPeekAtLastError());

    if (scale) {
        k_rotate_and_translate_mols<RealType, true><<<1, tpb, 0, stream>>>(
            1,
            d_coords,
            d_box,
            d_samples_.data,
            d_target_mol_offsets_.data,
            d_quaternions,
            d_translations,
            d_intermediate_coords_.data);
        gpuErrchk(hipPeekAtLastError());
    } else {
        k_rotate_and_translate_mols<RealType, false><<<1, tpb, 0, stream>>>(
            1,
            d_coords,
            d_box,
            d_samples_.data,
            d_target_mol_offsets_.data,
            d_quaternions,
            d_translations,
            d_intermediate_coords_.data);
        gpuErrchk(hipPeekAtLastError());
    }

    k_atom_by_atom_energies<<<atom_by_atom_grid, tpb, 0, stream>>>(
        N,
        mol_size_,
        d_target_mol_atoms_.data,
        d_coords,
        d_params_.data,
        d_box,
        nb_beta_,
        cutoff_squared_,
        d_sample_per_atom_energy_buffer_.data);
    gpuErrchk(hipPeekAtLastError());

    // Subtract off the weights for the individual waters from the sampled water.
    // It modifies the sampled mol energy value, leaving it in an invalid state, which is why
    // we later call k_set_sampled_weight to set the weight of the sampled mol
    k_adjust_weights<RealType, true><<<ceil_divide(num_target_mols_, tpb), tpb, 0, stream>>>(
        N,
        num_target_mols_,
        mol_size_,
        d_atom_idxs_.data,
        d_mol_offsets_.data,
        d_sample_per_atom_energy_buffer_.data,
        beta_, // 1 / kT
        d_log_weights_after_.data);
    gpuErrchk(hipPeekAtLastError());

    k_atom_by_atom_energies<<<atom_by_atom_grid, tpb, 0, stream>>>(
        N,
        mol_size_,
        d_target_mol_atoms_.data,
        d_intermediate_coords_.data,
        d_params_.data,
        d_box,
        nb_beta_,
        cutoff_squared_,
        d_sample_per_atom_energy_buffer_.data);
    gpuErrchk(hipPeekAtLastError());

    // Add in the new weights from the individual waters
    // the sampled weight continues to be garbage
    k_adjust_weights<RealType, false><<<ceil_divide(num_target_mols_, tpb), tpb, 0, stream>>>(
        N,
        num_target_mols_,
        mol_size_,
        d_atom_idxs_.data,
        d_mol_offsets_.data,
        d_sample_per_atom_energy_buffer_.data,
        beta_, // 1 / kT
        d_log_weights_after_.data);
    gpuErrchk(hipPeekAtLastError());

    // Set the sampled weight to be the correct value
    k_set_sampled_weight_block<RealType, WEIGHT_THREADS_PER_BLOCK>
        <<<static_cast<int>(d_intermediate_sample_weights_.length), WEIGHT_THREADS_PER_BLOCK, 0, stream>>>(
            N,
            mol_size_,
            d_target_mol_atoms_.data,
            d_sample_per_atom_energy_buffer_.data,
            beta_, // 1 / kT
            d_intermediate_sample_weights_.data);
    gpuErrchk(hipPeekAtLastError());

    k_set_sampled_weight_reduce<RealType, WEIGHT_THREADS_PER_BLOCK><<<1, WEIGHT_THREADS_PER_BLOCK, 0, stream>>>(
        static_cast<int>(d_intermediate_sample_weights_.length), // Number of intermediates
        d_samples_.data,                                         // where to set the value
        d_intermediate_sample_weights_.data,                     // intermediate fixed point weights
        d_log_weights_after_.data);
    gpuErrchk(hipPeekAtLastError());
}

template <typename RealType> double BDExchangeMove<RealType>::raw_log_probability_host() {
    std::vector<RealType> h_log_exp_before(2);
    std::vector<RealType> h_log_exp_after(2);
    d_log_sum_exp_before_.copy_to(&h_log_exp_before[0]);
    d_log_sum_exp_after_.copy_to(&h_log_exp_after[0]);

    RealType before_log_prob = convert_nan_to_inf(compute_logsumexp_final(&h_log_exp_before[0]));
    RealType after_log_prob = convert_nan_to_inf(compute_logsumexp_final(&h_log_exp_after[0]));

    return static_cast<double>(before_log_prob - after_log_prob);
}

template <typename RealType> double BDExchangeMove<RealType>::log_probability_host() {
    return min(raw_log_probability_host(), 0.0);
}

template <typename RealType> size_t BDExchangeMove<RealType>::n_accepted() const {
    size_t h_accepted;
    d_num_accepted_.copy_to(&h_accepted);
    return h_accepted;
}

template <typename RealType> std::vector<double> BDExchangeMove<RealType>::get_params() {
    std::vector<double> h_params(d_params_.length);
    d_params_.copy_to(&h_params[0]);
    return h_params;
};

template <typename RealType> void BDExchangeMove<RealType>::set_params(const std::vector<double> &params) {
    hipStream_t stream = static_cast<hipStream_t>(0);
    DeviceBuffer<double> d_params(params.size());
    d_params.copy_from(&params[0]);
    this->set_params_device(params.size(), d_params.data, stream);
    gpuErrchk(hipStreamSynchronize(stream));
};

template <typename RealType>
void BDExchangeMove<RealType>::set_params_device(const int size, const double *d_p, const hipStream_t stream) {
    if (d_params_.length != size) {
        throw std::runtime_error("number of params don't match");
    }
    gpuErrchk(hipMemcpyAsync(d_params_.data, d_p, d_params_.size(), hipMemcpyDeviceToDevice, stream));
};

template class BDExchangeMove<float>;
template class BDExchangeMove<double>;

} // namespace timemachine
