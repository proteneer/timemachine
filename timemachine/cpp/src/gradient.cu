
#include <iostream>

#include "gpu_utils.cuh"
#include "gradient.hpp"
#include "surreal.cuh"

namespace timemachine {

void Potential::execute_host(
    const int N,
    const double *h_x,           // [N,3]
    const double *h_params,      // [P,]
    const double *h_box,         // [3, 3]
    const double lambda,         // [1]
    unsigned long long *h_du_dx, // [N,3]
    double *h_du_dp,             // [P]
    double *h_du_dl,             //
    double *h_u) {

    double *d_x;
    double *d_box;

    const int D = 3;

    gpuErrchk(hipMalloc(&d_x, N * D * sizeof(double)));
    gpuErrchk(hipMemcpy(d_x, h_x, N * D * sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_box, D * D * sizeof(double)));
    gpuErrchk(hipMemcpy(d_box, h_box, D * D * sizeof(double), hipMemcpyHostToDevice));

    unsigned long long *d_du_dx; // du/dx

    double *d_du_dl;       // du/dl
    double *d_u = nullptr; // u

    // very important that these are initialized to zero since the kernels themselves just accumulate
    gpuErrchk(hipMalloc(&d_du_dx, N * D * sizeof(unsigned long long)));
    gpuErrchk(hipMemset(d_du_dx, 0, N * D * sizeof(unsigned long long)));
    gpuErrchk(hipMalloc(&d_du_dl, sizeof(double)));
    gpuErrchk(hipMemset(d_du_dl, 0, sizeof(double)));
    gpuErrchk(hipMalloc(&d_u, sizeof(double)));
    gpuErrchk(hipMemset(d_u, 0, sizeof(double)));

    this->execute_device(N, d_x, lambda, d_du_dx, d_du_dl, d_u, static_cast<hipStream_t>(0));

    gpuErrchk(hipMemcpy(h_du_dx, d_du_dx, N * D * sizeof(*h_du_dx), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(d_du_dx));
    gpuErrchk(hipMemcpy(h_du_dl, d_du_dl, sizeof(*h_du_dl), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(d_du_dl));
    gpuErrchk(hipMemcpy(h_u, d_u, sizeof(*h_u), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(d_u));
    gpuErrchk(hipFree(d_coords));
    gpuErrchk(hipFree(d_box));
};

// void Gradient::execute_lambda_jvp_host(
//     const int N,
//     const double *h_in_coords_primals,
//     const double *h_in_coords_tangents,
//     const double lambda_primal,
//     const double lambda_tangent,
//     double *h_out_coords_primals,
//     double *h_out_coords_tangents) {

//     double *d_in_coords_primals;
//     double *d_in_params_primals;

//     const int D = 3;

//     gpuErrchk(hipMalloc(&d_in_coords_primals, N*D*sizeof(double)));
//     gpuErrchk(hipMemcpy(d_in_coords_primals, h_in_coords_primals, N*D*sizeof(double), hipMemcpyHostToDevice));

//     double *d_in_coords_tangents; // x tangent
//     double *d_out_coords_primals; // du/dx
//     double *d_out_coords_tangents; // du/dx

//     gpuErrchk(hipMalloc(&d_in_coords_tangents, N*D*sizeof(double)));
//     gpuErrchk(hipMemcpy(d_in_coords_tangents, h_in_coords_tangents, N*D*sizeof(double), hipMemcpyHostToDevice));

//     gpuErrchk(hipMalloc(&d_out_coords_tangents, N*D*sizeof(double)));
//     gpuErrchk(hipMemset(d_out_coords_tangents, 0, N*D*sizeof(double)));

//     gpuErrchk(hipMalloc(&d_out_coords_primals, N*D*sizeof(double)));
//     gpuErrchk(hipMemset(d_out_coords_primals, 0, N*D*sizeof(double)));

//     this->execute_lambda_jvp_device(
//         N,
//         d_in_coords_primals,
//         d_in_coords_tangents,
//         lambda_primal,
//         lambda_tangent,
//         d_out_coords_primals,
//         d_out_coords_tangents,
//         static_cast<hipStream_t>(0)
//     );

//     gpuErrchk(hipMemcpy(h_out_coords_tangents, d_out_coords_tangents, N*D*sizeof(double), hipMemcpyDeviceToHost));
//     gpuErrchk(hipFree(d_out_coords_tangents));

//     gpuErrchk(hipMemcpy(h_out_coords_primals, d_out_coords_primals, N*D*sizeof(double), hipMemcpyDeviceToHost));
//     gpuErrchk(hipFree(d_out_coords_primals));

//     gpuErrchk(hipFree(d_in_coords_primals));
//     gpuErrchk(hipFree(d_in_coords_tangents));

// };

} // namespace timemachine
