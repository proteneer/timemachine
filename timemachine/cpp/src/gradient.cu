
#include "gradient.hpp"
#include "kernel_utils.cuh"
#include "surreal.cuh"

namespace timemachine {

template<int D>
void Gradient<D>::execute_host(
    const int N,
    const int P,
    const double *h_in_coords,
    const double *h_in_coords_tangents,
    const double *h_in_params,
    unsigned long long *h_out_coords,
    double *h_out_coords_tangents,
    double *h_out_params_tangents) {

    double *d_in_coords;
    double *d_in_params;

    gpuErrchk(hipMalloc(&d_in_coords, N*D*sizeof(double)));
    gpuErrchk(hipMalloc(&d_in_params, P*sizeof(double)));
    gpuErrchk(hipMemcpy(d_in_coords, h_in_coords, N*D*sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_in_params, h_in_params, P*sizeof(double), hipMemcpyHostToDevice));

    unsigned long long *d_out_coords;

    // very important that we initialize these
    double *d_in_coords_tangents = nullptr;
    double *d_out_coords_tangents = nullptr;
    double *d_out_params_tangents = nullptr;
    if(h_in_coords_tangents == nullptr) {
        gpuErrchk(hipMalloc(&d_out_coords, N*D*sizeof(unsigned long long)));
        gpuErrchk(hipMemset(d_out_coords, 0, N*D*sizeof(unsigned long long)));
    } else {

        gpuErrchk(hipMalloc(&d_in_coords_tangents, N*D*sizeof(double)));
        gpuErrchk(hipMemcpy(d_in_coords_tangents, h_in_coords_tangents, N*D*sizeof(double), hipMemcpyHostToDevice));

        gpuErrchk(hipMalloc(&d_out_coords_tangents, N*D*sizeof(double)));
        gpuErrchk(hipMalloc(&d_out_params_tangents, P*sizeof(double)));

        gpuErrchk(hipMemset(d_out_coords_tangents, 0, N*D*sizeof(double)));
        gpuErrchk(hipMemset(d_out_params_tangents, 0, P*sizeof(double)));
    }

    this->execute_device(
        N,
        P,
        d_in_coords, 
        d_in_coords_tangents,
        d_in_params,
        d_out_coords,
        d_out_coords_tangents,
        d_out_params_tangents
    );

    if(h_in_coords_tangents == nullptr) {
        gpuErrchk(hipMemcpy(h_out_coords, d_out_coords, N*D*sizeof(*h_out_coords), hipMemcpyDeviceToHost));
        gpuErrchk(hipFree(d_out_coords));
    } else {
        gpuErrchk(hipMemcpy(h_out_coords_tangents, d_out_coords_tangents, N*D*sizeof(double), hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(h_out_params_tangents, d_out_params_tangents, P*sizeof(double), hipMemcpyDeviceToHost));
        gpuErrchk(hipFree(d_out_coords_tangents));
        gpuErrchk(hipFree(d_out_params_tangents));
    }

    gpuErrchk(hipFree(d_in_coords));
    gpuErrchk(hipFree(d_in_params));

};

template class Gradient<4>; 
template class Gradient<3>;

}

