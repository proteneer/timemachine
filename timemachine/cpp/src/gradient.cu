
#include <iostream>

#include "gradient.hpp"
#include "gpu_utils.cuh"
#include "surreal.cuh"

namespace timemachine {

void Gradient::execute_lambda_inference_host(
    const int N,
    // const int P,
    const double *h_in_coords_primals,
    // const double *h_in_params_primals,
    const double lambda_primal,
    unsigned long long *h_out_coords_primals,
    double *h_out_lambda_primals,
    double *h_out_energy_primal) {

    double *d_in_coords_primals;
    // double *d_in_params_primals;

    const int D = 3;

    gpuErrchk(hipMalloc(&d_in_coords_primals, N*D*sizeof(double)));
    // gpuErrchk(hipMalloc(&d_in_params_primals, P*sizeof(double)));
    gpuErrchk(hipMemcpy(d_in_coords_primals, h_in_coords_primals, N*D*sizeof(double), hipMemcpyHostToDevice));
    // gpuErrchk(hipMemcpy(d_in_params_primals, h_in_params_primals, P*sizeof(double), hipMemcpyHostToDevice));

    unsigned long long *d_out_coords_primals; // du/dx
    double *d_out_lambda_primals; // du/dl
    double *d_out_energy = nullptr; // u

    // very important that these are initialized to zero since the kernels themselves just accumulate
    gpuErrchk(hipMalloc(&d_out_coords_primals, N*D*sizeof(unsigned long long)));
    gpuErrchk(hipMemset(d_out_coords_primals, 0, N*D*sizeof(unsigned long long)));
    gpuErrchk(hipMalloc(&d_out_lambda_primals, sizeof(double)));
    gpuErrchk(hipMemset(d_out_lambda_primals, 0, sizeof(double)));
    gpuErrchk(hipMalloc(&d_out_energy, sizeof(double)));
    gpuErrchk(hipMemset(d_out_energy, 0, sizeof(double)));


    this->execute_lambda_inference_device(
        N,
        // P,
        d_in_coords_primals, 
        // d_in_params_primals,
        lambda_primal,
        d_out_coords_primals,
        d_out_lambda_primals,
        d_out_energy,
        static_cast<hipStream_t>(0)
    );

    gpuErrchk(hipMemcpy(h_out_coords_primals, d_out_coords_primals, N*D*sizeof(*h_out_coords_primals), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(d_out_coords_primals));
    gpuErrchk(hipMemcpy(h_out_lambda_primals, d_out_lambda_primals, sizeof(*h_out_lambda_primals), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(d_out_lambda_primals));
    gpuErrchk(hipMemcpy(h_out_energy_primal, d_out_energy, sizeof(*h_out_energy_primal), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(d_out_energy));

    gpuErrchk(hipFree(d_in_coords_primals));
    // gpuErrchk(hipFree(d_in_params_primals));

};

void Gradient::execute_lambda_jvp_host(
    const int N,
    // const int P,
    const double *h_in_coords_primals,
    const double *h_in_coords_tangents,
    // const double *h_in_params_primals,
    const double lambda_primal,
    const double lambda_tangent,
    double *h_out_coords_primals,
    double *h_out_coords_tangents
    // double *h_out_params_primals,
    // double *h_out_params_tangents
    ) {

    double *d_in_coords_primals;
    double *d_in_params_primals;

    const int D = 3;

    gpuErrchk(hipMalloc(&d_in_coords_primals, N*D*sizeof(double)));
    // gpuErrchk(hipMalloc(&d_in_params_primals, P*sizeof(double)));
    gpuErrchk(hipMemcpy(d_in_coords_primals, h_in_coords_primals, N*D*sizeof(double), hipMemcpyHostToDevice));
    // gpuErrchk(hipMemcpy(d_in_params_primals, h_in_params_primals, P*sizeof(double), hipMemcpyHostToDevice));

    double *d_in_coords_tangents; // x tangent
    double *d_out_coords_primals; // du/dx
    double *d_out_coords_tangents; // du/dx
    // double *d_out_params_primals; // du/dp
    // double *d_out_params_tangents; // du/dp

    gpuErrchk(hipMalloc(&d_in_coords_tangents, N*D*sizeof(double)));
    gpuErrchk(hipMemcpy(d_in_coords_tangents, h_in_coords_tangents, N*D*sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_out_coords_tangents, N*D*sizeof(double)));
    // gpuErrchk(hipMalloc(&d_out_params_tangents, P*sizeof(double)));
    gpuErrchk(hipMemset(d_out_coords_tangents, 0, N*D*sizeof(double)));
    // gpuErrchk(hipMemset(d_out_params_tangents, 0, P*sizeof(double)));

    gpuErrchk(hipMalloc(&d_out_coords_primals, N*D*sizeof(double)));
    // gpuErrchk(hipMalloc(&d_out_params_primals, P*sizeof(double)));
    gpuErrchk(hipMemset(d_out_coords_primals, 0, N*D*sizeof(double)));
    // gpuErrchk(hipMemset(d_out_params_primals, 0, P*sizeof(double)));

    this->execute_lambda_jvp_device(
        N,
        // P,
        d_in_coords_primals, 
        d_in_coords_tangents,
        // d_in_params_primals,
        lambda_primal,
        lambda_tangent,
        d_out_coords_primals,
        d_out_coords_tangents,
        // d_out_params_primals,
        // d_out_params_tangents,
        static_cast<hipStream_t>(0)
    );

    gpuErrchk(hipMemcpy(h_out_coords_tangents, d_out_coords_tangents, N*D*sizeof(double), hipMemcpyDeviceToHost));
    // gpuErrchk(hipMemcpy(h_out_params_tangents, d_out_params_tangents, P*sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(d_out_coords_tangents));
    // gpuErrchk(hipFree(d_out_params_tangents));

    gpuErrchk(hipMemcpy(h_out_coords_primals, d_out_coords_primals, N*D*sizeof(double), hipMemcpyDeviceToHost));
    // gpuErrchk(hipMemcpy(h_out_params_primals, d_out_params_primals, P*sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(d_out_coords_primals));
    // gpuErrchk(hipFree(d_out_params_primals));

    gpuErrchk(hipFree(d_in_coords_primals));
    gpuErrchk(hipFree(d_in_coords_tangents));
    // gpuErrchk(hipFree(d_in_params_primals));

};

}

