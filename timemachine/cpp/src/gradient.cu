
#include "gradient.hpp"
#include "gpu_utils.cuh"
#include "surreal.cuh"

namespace timemachine {

template<int D>
void Gradient<D>::execute_lambda_host(
    const int N,
    const int P,
    const double *h_in_coords_primals,
    const double *h_in_coords_tangents,
    const double *h_in_params_primals,
    const double in_lambda_primal,
    const double in_lambda_tangent,
    unsigned long long *h_out_coords_primals,
    double *h_out_lambda_primals,
    double *h_out_coords_tangents,
    double *h_out_params_tangents) {

    double *d_in_coords_primals;
    double *d_in_params_primals;

    gpuErrchk(hipMalloc(&d_in_coords_primals, N*D*sizeof(double)));
    gpuErrchk(hipMalloc(&d_in_params_primals, P*sizeof(double)));
    gpuErrchk(hipMemcpy(d_in_coords_primals, h_in_coords_primals, N*D*sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_in_params_primals, h_in_params_primals, P*sizeof(double), hipMemcpyHostToDevice));

    unsigned long long *d_out_coords_primals;
    double *d_out_lambda_primals;

    // very important that we initialize these
    double *d_in_coords_tangents = nullptr;
    double *d_out_coords_tangents = nullptr;
    double *d_out_params_tangents = nullptr;
    if(h_in_coords_tangents == nullptr) {
        gpuErrchk(hipMalloc(&d_out_coords_primals, N*D*sizeof(unsigned long long)));
        gpuErrchk(hipMemset(d_out_coords_primals, 0, N*D*sizeof(unsigned long long)));
        gpuErrchk(hipMalloc(&d_out_lambda_primals, sizeof(double)));
        gpuErrchk(hipMemset(d_out_lambda_primals, 0, sizeof(double)));
    } else {

        gpuErrchk(hipMalloc(&d_in_coords_tangents, N*D*sizeof(double)));
        gpuErrchk(hipMemcpy(d_in_coords_tangents, h_in_coords_tangents, N*D*sizeof(double), hipMemcpyHostToDevice));

        gpuErrchk(hipMalloc(&d_out_coords_tangents, N*D*sizeof(double)));
        gpuErrchk(hipMalloc(&d_out_params_tangents, P*sizeof(double)));

        gpuErrchk(hipMemset(d_out_coords_tangents, 0, N*D*sizeof(double)));
        gpuErrchk(hipMemset(d_out_params_tangents, 0, P*sizeof(double)));
    }

    this->execute_lambda_device(
        N,
        P,
        d_in_coords_primals, 
        d_in_coords_tangents,
        d_in_params_primals,
        in_lambda_primal,
        in_lambda_tangent,
        d_out_coords_primals,
        d_out_lambda_primals,
        d_out_coords_tangents,
        d_out_params_tangents,
        static_cast<hipStream_t>(0)
    );

    if(h_in_coords_tangents == nullptr) {
        gpuErrchk(hipMemcpy(h_out_coords_primals, d_out_coords_primals, N*D*sizeof(*h_out_coords_primals), hipMemcpyDeviceToHost));
        gpuErrchk(hipFree(d_out_coords_primals));
        gpuErrchk(hipMemcpy(h_out_lambda_primals, d_out_lambda_primals, sizeof(*h_out_lambda_primals), hipMemcpyDeviceToHost));
        gpuErrchk(hipFree(d_out_lambda_primals));

    } else {
        gpuErrchk(hipMemcpy(h_out_coords_tangents, d_out_coords_tangents, N*D*sizeof(double), hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(h_out_params_tangents, d_out_params_tangents, P*sizeof(double), hipMemcpyDeviceToHost));
        gpuErrchk(hipFree(d_out_coords_tangents));
        gpuErrchk(hipFree(d_out_params_tangents));
    }


    gpuErrchk(hipFree(d_in_coords_primals));
    gpuErrchk(hipFree(d_in_coords_tangents));
    gpuErrchk(hipFree(d_in_params_primals));

};

template class Gradient<4>; 
template class Gradient<3>;

}

