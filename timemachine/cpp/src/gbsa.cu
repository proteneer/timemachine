#include <stdexcept>
#include <iostream> 
#include <chrono>
#include "fixed_point.hpp"
#include "gbsa.hpp"
#include "gbsa_jvp.cuh"
#include "gpu_utils.cuh"
#include "math_utils.cuh"
#include "k_gbsa.cuh"
#include "k_gbsa_jvp.cuh"

namespace timemachine {

template <typename RealType>
GBSA<RealType>::GBSA(
    const std::vector<int> &charge_param_idxs, // [N]
    const std::vector<int> &atomic_radii_idxs, // [N]
    const std::vector<int> &scale_factor_idxs, // [E,2]
    const std::vector<int> &lambda_plane_idxs, // [N]
    const std::vector<int> &lambda_offset_idxs, // [N]
    double alpha,
    double beta,
    double gamma,
    double dielectric_offset,
    double surface_tension,
    double solute_dielectric,
    double solvent_dielectric,
    double probe_radius,
    double cutoff_radii,
    double cutoff_force
) : N_(charge_param_idxs.size()),
    alpha_(alpha),
    beta_(beta),
    gamma_(gamma),
    dielectric_offset_(dielectric_offset),
    surface_tension_(surface_tension),
    solute_dielectric_(solute_dielectric),
    solvent_dielectric_(solvent_dielectric),
    probe_radius_(probe_radius),
    cutoff_radii_(cutoff_radii),
    cutoff_force_(cutoff_force),
    nblist_(charge_param_idxs.size(), 3) {

    if(cutoff_radii != cutoff_force) {
      throw std::runtime_error("GB currently requires that cutoff_radii be equal to cutoff_force!");
    }

    gpuErrchk(hipMalloc(&d_lambda_plane_idxs_, N_*sizeof(*d_lambda_plane_idxs_)));
    gpuErrchk(hipMemcpy(d_lambda_plane_idxs_, &lambda_plane_idxs[0], N_*sizeof(*d_lambda_plane_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_lambda_offset_idxs_, N_*sizeof(*d_lambda_offset_idxs_)));
    gpuErrchk(hipMemcpy(d_lambda_offset_idxs_, &lambda_offset_idxs[0], N_*sizeof(*d_lambda_offset_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_charge_param_idxs_, N_*sizeof(*d_charge_param_idxs_)));
    gpuErrchk(hipMalloc(&d_scale_factor_idxs_, N_*sizeof(*d_scale_factor_idxs_)));
    gpuErrchk(hipMalloc(&d_atomic_radii_idxs_, N_*sizeof(*d_atomic_radii_idxs_)));
    gpuErrchk(hipMemcpy(d_charge_param_idxs_, &charge_param_idxs[0], N_*sizeof(*d_charge_param_idxs_), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_scale_factor_idxs_, &scale_factor_idxs[0], N_*sizeof(*d_scale_factor_idxs_), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_atomic_radii_idxs_, &atomic_radii_idxs[0], N_*sizeof(*d_atomic_radii_idxs_), hipMemcpyHostToDevice));

    // we probaly don't need *all* these buffers if we do just one pass, but they take up only
    // O(N) ram so we don't really care and just pre-allocate everything to keep things simple.
    // it also ensures that we can RAII properly.

    const int N = charge_param_idxs.size();

    gpuErrchk(hipMalloc(&d_born_psi_buffer_, N*sizeof(*d_born_psi_buffer_)));
    gpuErrchk(hipMalloc(&d_born_radii_buffer_, N*sizeof(*d_born_radii_buffer_)));
    gpuErrchk(hipMalloc(&d_obc_buffer_, N*sizeof(*d_obc_buffer_)));
    gpuErrchk(hipMalloc(&d_born_forces_buffer_, N*sizeof(*d_born_forces_buffer_)));

    gpuErrchk(hipMalloc(&d_born_radii_buffer_jvp_, N*sizeof(*d_born_radii_buffer_jvp_)));
    gpuErrchk(hipMalloc(&d_obc_buffer_jvp_, N*sizeof(*d_obc_buffer_jvp_)));
    gpuErrchk(hipMalloc(&d_obc_ri_buffer_jvp_, N*sizeof(*d_obc_ri_buffer_jvp_)));
    gpuErrchk(hipMalloc(&d_born_forces_buffer_jvp_, N*sizeof(*d_born_forces_buffer_jvp_)));


}

template <typename RealType>
GBSA<RealType>::~GBSA() {

  gpuErrchk(hipFree(d_charge_param_idxs_));
  gpuErrchk(hipFree(d_atomic_radii_idxs_));
  gpuErrchk(hipFree(d_scale_factor_idxs_));
  gpuErrchk(hipFree(d_lambda_plane_idxs_));
  gpuErrchk(hipFree(d_lambda_offset_idxs_));

  gpuErrchk(hipFree(d_born_psi_buffer_));
  gpuErrchk(hipFree(d_born_radii_buffer_));
  gpuErrchk(hipFree(d_obc_buffer_));
  gpuErrchk(hipFree(d_born_forces_buffer_));

  gpuErrchk(hipFree(d_born_radii_buffer_jvp_));
  gpuErrchk(hipFree(d_obc_buffer_jvp_));
  gpuErrchk(hipFree(d_obc_ri_buffer_jvp_));
  gpuErrchk(hipFree(d_born_forces_buffer_jvp_));

};

template <typename RealType>
void GBSA<RealType>::execute_lambda_inference_device(
    const int N,
    const int P,
    const double *d_coords,
    const double *d_params,
    const double lambda,
    unsigned long long *d_out_coords,
    double *d_out_lambda,
    double *d_out_energy,
    hipStream_t stream) {

    // std::cout << "exec GB" << std::endl;

    int tpb = 32;
    int B = (N_+tpb-1)/tpb;
    const int D = 3;

    dim3 dimGrid(B, B, 1); // x, y, z dims

    double prefactor;
    if (solute_dielectric_ != 0.0 && solvent_dielectric_ != 0.0) {
        // prefactor = -screening_*((1.0/solute_dielectric_) - (1.0/solvent_dielectric_));
        prefactor = -((1.0/solute_dielectric_) - (1.0/solvent_dielectric_));
    } else {
        prefactor = 0.0;
    }
    // std::cout << "cutoff 12: " << cutoff_radii_ << " " << cutoff_force_ << std::endl;

    // hipDeviceSynchronize();
    nblist_.compute_block_bounds(N_, D, d_coords, stream);

    auto start = std::chrono::high_resolution_clock::now();

    gpuErrchk(hipMemsetAsync(d_born_psi_buffer_, 0, N*sizeof(*d_born_psi_buffer_), stream));
    gpuErrchk(hipMemsetAsync(d_born_radii_buffer_, 0, N*sizeof(*d_born_radii_buffer_), stream));
    gpuErrchk(hipMemsetAsync(d_obc_buffer_, 0, N*sizeof(*d_obc_buffer_), stream));
    gpuErrchk(hipMemsetAsync(d_born_forces_buffer_, 0, N*sizeof(*d_born_forces_buffer_), stream));

    k_compute_born_radii<RealType><<<dimGrid, tpb, 0, stream>>>(
      N_,
      d_coords,
      d_params,
      lambda,
      d_lambda_plane_idxs_,
      d_lambda_offset_idxs_,
      d_atomic_radii_idxs_,
      d_scale_factor_idxs_,
      dielectric_offset_,
      cutoff_radii_,
      nblist_.get_block_bounds_ctr(),
      nblist_.get_block_bounds_ext(),
      d_born_psi_buffer_
    );

    // hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    k_reduce_born_radii<<<B, tpb, 0, stream>>>(
      N_,
      d_params,
      d_atomic_radii_idxs_,
      dielectric_offset_,
      alpha_,
      beta_,
      gamma_,
      d_born_psi_buffer_,
      d_born_radii_buffer_,
      d_obc_buffer_
    );

    // hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    k_compute_born_first_loop_gpu<RealType><<<dimGrid, tpb, 0, stream>>>(
      N_,
      d_coords,
      d_params,
      lambda,
      d_lambda_plane_idxs_,
      d_lambda_offset_idxs_,
      d_charge_param_idxs_,
      d_born_radii_buffer_,
      prefactor,
      cutoff_force_,
      nblist_.get_block_bounds_ctr(),
      nblist_.get_block_bounds_ext(),
      d_born_forces_buffer_, // output
      d_out_coords,
      d_out_lambda, // output
      d_out_energy
    );

    // hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    k_reduce_born_forces<<<B, tpb, 0, stream>>>(
      N_,
      d_params,
      d_atomic_radii_idxs_,
      d_born_radii_buffer_,
      d_obc_buffer_,
      surface_tension_,
      probe_radius_,
      d_born_forces_buffer_,
      d_out_energy
    );

    // // hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    k_compute_born_energy_and_forces<RealType><<<dimGrid, tpb, 0, stream>>>(
      N_,
      d_coords,
      d_params,
      lambda,
      d_lambda_plane_idxs_,
      d_lambda_offset_idxs_,
      d_atomic_radii_idxs_,
      d_scale_factor_idxs_,
      d_born_radii_buffer_,
      d_obc_buffer_,
      dielectric_offset_,
      cutoff_force_,
      nblist_.get_block_bounds_ctr(),
      nblist_.get_block_bounds_ext(),
      d_born_forces_buffer_,
      d_out_coords,
      d_out_lambda
    );

    // hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

}

template <typename RealType>
void GBSA<RealType>::execute_lambda_jvp_device(
    const int N,
    const int P,
    const double *d_coords_primals,
    const double *d_coords_tangents,
    const double *d_params_primals,
    const double lambda_primal,
    const double lambda_tangent,
    double *d_out_coords_primals,
    double *d_out_coords_tangents,
    double *d_out_params_primals,
    double *d_out_params_tangents,
    hipStream_t stream) {

    // std::cout << "exec GB" << std::endl;

    int tpb = 32;
    int B = (N_+tpb-1)/tpb;
    const int D = 3;

    dim3 dimGrid(B, B, 1); // x, y, z dims

    double prefactor;
    if (solute_dielectric_ != 0.0 && solvent_dielectric_ != 0.0) {
        prefactor = -((1.0/solute_dielectric_) - (1.0/solvent_dielectric_));
    } else {
        prefactor = 0.0;
    }

    nblist_.compute_block_bounds(N_, D, d_coords_primals, stream);

    auto start = std::chrono::high_resolution_clock::now();

    gpuErrchk(hipMemsetAsync(d_born_radii_buffer_jvp_, 0, N*sizeof(*d_born_radii_buffer_jvp_), stream));
    gpuErrchk(hipMemsetAsync(d_obc_buffer_jvp_, 0, N*sizeof(*d_obc_buffer_jvp_), stream));
    gpuErrchk(hipMemsetAsync(d_obc_ri_buffer_jvp_, 0, N*sizeof(*d_obc_ri_buffer_jvp_), stream));
    gpuErrchk(hipMemsetAsync(d_born_forces_buffer_jvp_, 0, N*sizeof(*d_born_forces_buffer_jvp_), stream));

    k_compute_born_radii_gpu_jvp<RealType><<<dimGrid, tpb, 0, stream>>>(
        N_,
        d_coords_primals,
        d_coords_tangents,
        d_params_primals,
        lambda_primal,
        lambda_tangent,
        d_lambda_plane_idxs_,
        d_lambda_offset_idxs_,
        d_atomic_radii_idxs_,
        d_scale_factor_idxs_,
        dielectric_offset_,
        cutoff_radii_,
        nblist_.get_block_bounds_ctr(),
        nblist_.get_block_bounds_ext(),
        d_born_radii_buffer_jvp_
    );

    // hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    k_reduce_born_radii_jvp<<<B, tpb, 0, stream>>>(
      N_,
      d_params_primals,
      d_atomic_radii_idxs_,
      dielectric_offset_,
      alpha_,
      beta_,
      gamma_,
      d_born_radii_buffer_jvp_,
      d_obc_buffer_jvp_,
      d_obc_ri_buffer_jvp_
    );

    // hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    k_compute_born_first_loop_gpu_jvp<RealType><<<dimGrid, tpb, 0, stream>>>(
        N_,
        d_coords_primals,
        d_coords_tangents,
        d_params_primals,
        lambda_primal,
        lambda_tangent,
        d_lambda_plane_idxs_,
        d_lambda_offset_idxs_,
        d_charge_param_idxs_,
        d_born_radii_buffer_jvp_,
        prefactor,
        cutoff_force_,
        nblist_.get_block_bounds_ctr(),
        nblist_.get_block_bounds_ext(),
        d_born_forces_buffer_jvp_, // output
        d_out_coords_primals, // output
        d_out_coords_tangents, // output
        d_out_params_primals, // output
        d_out_params_tangents // output
    );

    // hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    k_reduce_born_forces_jvp<<<B, tpb, 0, stream>>>(
        N_,
        d_params_primals,
        d_atomic_radii_idxs_,
        d_born_radii_buffer_jvp_,
        d_obc_buffer_jvp_,
        d_obc_ri_buffer_jvp_,
        surface_tension_,
        probe_radius_,
        d_born_forces_buffer_jvp_,
        d_out_params_primals,
        d_out_params_tangents
    );

    // hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());


    // auto start = std::chrono::high_resolution_clock::now();
    k_compute_born_energy_and_forces_jvp<RealType, D><<<dimGrid, tpb, 0, stream>>>(
        N_,
        d_coords_primals,
        d_coords_tangents,
        d_params_primals,
        lambda_primal,
        lambda_tangent,
        d_lambda_plane_idxs_,
        d_lambda_offset_idxs_,
        d_atomic_radii_idxs_,
        d_scale_factor_idxs_,
        d_born_radii_buffer_jvp_,
        d_obc_buffer_jvp_,
        d_obc_ri_buffer_jvp_,
        dielectric_offset_,
        cutoff_force_,
        nblist_.get_block_bounds_ctr(),
        nblist_.get_block_bounds_ext(),
        d_born_forces_buffer_jvp_,
        d_out_coords_primals, // output
        d_out_coords_tangents, // output
        d_out_params_primals, // output
        d_out_params_tangents // output
    );

    // hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

}

template class GBSA<double>;
template class GBSA<float>;


}

