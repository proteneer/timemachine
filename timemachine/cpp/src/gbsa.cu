#include <stdexcept>
#include <iostream> 
#include <chrono>
#include "fixed_point.hpp"
#include "gbsa.hpp"
#include "gbsa_jvp.cuh"
#include "gpu_utils.cuh"
#include "math_utils.cuh"
#include "k_gbsa.cuh"
#include "k_gbsa_jvp.cuh"

namespace timemachine {

template <typename RealType, int D>
GBSA<RealType, D>::GBSA(
    const std::vector<int> &charge_param_idxs, // [N]
    const std::vector<int> &atomic_radii_idxs, // [N]
    const std::vector<int> &scale_factor_idxs, // [E,2]
    const std::vector<int> &lambda_idxs, // [N]
    double alpha,
    double beta,
    double gamma,
    double dielectric_offset,
    double surface_tension,
    double solute_dielectric,
    double solvent_dielectric,
    double probe_radius,
    double cutoff_radii,
    double cutoff_force
) : N_(charge_param_idxs.size()),
    alpha_(alpha),
    beta_(beta),
    gamma_(gamma),
    dielectric_offset_(dielectric_offset),
    surface_tension_(surface_tension),
    solute_dielectric_(solute_dielectric),
    solvent_dielectric_(solvent_dielectric),
    probe_radius_(probe_radius),
    cutoff_radii_(cutoff_radii),
    cutoff_force_(cutoff_force),
    nblist_(charge_param_idxs.size(), D) {

    if(cutoff_radii != cutoff_force) {
      throw std::runtime_error("GB currently requires that cutoff_radii be equal to cutoff_force!");
    }


    gpuErrchk(hipMalloc(&d_lambda_idxs_, N_*sizeof(*d_lambda_idxs_)));
    gpuErrchk(hipMemcpy(d_lambda_idxs_, &lambda_idxs[0], N_*sizeof(*d_lambda_idxs_), hipMemcpyHostToDevice));


    gpuErrchk(hipMalloc(&d_charge_param_idxs_, N_*sizeof(*d_charge_param_idxs_)));
    gpuErrchk(hipMalloc(&d_scale_factor_idxs_, N_*sizeof(*d_scale_factor_idxs_)));
    gpuErrchk(hipMalloc(&d_atomic_radii_idxs_, N_*sizeof(*d_atomic_radii_idxs_)));
    gpuErrchk(hipMemcpy(d_charge_param_idxs_, &charge_param_idxs[0], N_*sizeof(*d_charge_param_idxs_), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_scale_factor_idxs_, &scale_factor_idxs[0], N_*sizeof(*d_scale_factor_idxs_), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_atomic_radii_idxs_, &atomic_radii_idxs[0], N_*sizeof(*d_atomic_radii_idxs_), hipMemcpyHostToDevice));

    // we probaly don't need *all* these buffers if we do just one pass, but they take up only
    // O(N) ram so we don't really care and just pre-allocate everything to keep things simple.
    // it also ensures that we can RAII properly.

    const int N = charge_param_idxs.size();

    gpuErrchk(hipMalloc(&d_born_psi_buffer_, N*sizeof(*d_born_psi_buffer_)));
    gpuErrchk(hipMalloc(&d_born_radii_buffer_, N*sizeof(*d_born_radii_buffer_)));
    gpuErrchk(hipMalloc(&d_obc_buffer_, N*sizeof(*d_obc_buffer_)));
    gpuErrchk(hipMalloc(&d_born_forces_buffer_, N*sizeof(*d_born_forces_buffer_)));

    gpuErrchk(hipMalloc(&d_born_radii_buffer_jvp_, N*sizeof(*d_born_radii_buffer_jvp_)));
    gpuErrchk(hipMalloc(&d_obc_buffer_jvp_, N*sizeof(*d_obc_buffer_jvp_)));
    gpuErrchk(hipMalloc(&d_obc_ri_buffer_jvp_, N*sizeof(*d_obc_ri_buffer_jvp_)));
    gpuErrchk(hipMalloc(&d_born_forces_buffer_jvp_, N*sizeof(*d_born_forces_buffer_jvp_)));


}

template <typename RealType, int D>
GBSA<RealType, D>::~GBSA() {

  gpuErrchk(hipFree(d_charge_param_idxs_));
  gpuErrchk(hipFree(d_atomic_radii_idxs_));
  gpuErrchk(hipFree(d_scale_factor_idxs_));
  gpuErrchk(hipFree(d_lambda_idxs_));

  gpuErrchk(hipFree(d_born_psi_buffer_));
  gpuErrchk(hipFree(d_born_radii_buffer_));
  gpuErrchk(hipFree(d_obc_buffer_));
  gpuErrchk(hipFree(d_born_forces_buffer_));

  gpuErrchk(hipFree(d_born_radii_buffer_jvp_));
  gpuErrchk(hipFree(d_obc_buffer_jvp_));
  gpuErrchk(hipFree(d_obc_ri_buffer_jvp_));
  gpuErrchk(hipFree(d_born_forces_buffer_jvp_));


};




template <typename RealType, int D>
void GBSA<RealType, D>::execute_lambda_device(
    const int N,
    const int P,
    const double *d_coords,
    const double *d_coords_tangents,
    const double *d_params,
    const double lambda,
    const double lambda_tangent,
    unsigned long long *d_out_coords,
    double *d_out_lambda,
    double *d_out_coords_tangents,
    double *d_out_params_tangents,
    hipStream_t stream
) {

    // std::cout << "exec GB" << std::endl;

    int tpb = 32;
    int B = (N_+tpb-1)/tpb;

    dim3 dimGrid(B, B, 1); // x, y, z dims

    double prefactor;
    if (solute_dielectric_ != 0.0 && solvent_dielectric_ != 0.0) {
        // prefactor = -screening_*((1.0/solute_dielectric_) - (1.0/solvent_dielectric_));
        prefactor = -((1.0/solute_dielectric_) - (1.0/solvent_dielectric_));
    } else {
        prefactor = 0.0;
    }


    // std::cout << "cutoff 12: " << cutoff_radii_ << " " << cutoff_force_ << std::endl;

    // hipDeviceSynchronize();
    nblist_.compute_block_bounds(N_, D, d_coords, stream);

    auto start = std::chrono::high_resolution_clock::now();
    if(d_coords_tangents == nullptr) {

        // inference mode  
        gpuErrchk(hipMemsetAsync(d_born_psi_buffer_, 0, N*sizeof(*d_born_psi_buffer_), stream));
        gpuErrchk(hipMemsetAsync(d_born_radii_buffer_, 0, N*sizeof(*d_born_radii_buffer_), stream));
        gpuErrchk(hipMemsetAsync(d_obc_buffer_, 0, N*sizeof(*d_obc_buffer_), stream));
        gpuErrchk(hipMemsetAsync(d_born_forces_buffer_, 0, N*sizeof(*d_born_forces_buffer_), stream));

        k_compute_born_radii<RealType><<<dimGrid, tpb, 0, stream>>>(
          N_,
          d_coords,
          d_params,
          lambda,
          d_lambda_idxs_,
          d_atomic_radii_idxs_,
          d_scale_factor_idxs_,
          dielectric_offset_,
          cutoff_radii_,
          nblist_.get_block_bounds_ctr(),
          nblist_.get_block_bounds_ext(),
          d_born_psi_buffer_
        );

        // hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        k_reduce_born_radii<<<B, tpb, 0, stream>>>(
          N_,
          d_params,
          d_atomic_radii_idxs_,
          dielectric_offset_,
          alpha_,
          beta_,
          gamma_,
          d_born_psi_buffer_,
          d_born_radii_buffer_,
          d_obc_buffer_
        );

        // hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());


        k_compute_born_first_loop_gpu<RealType><<<dimGrid, tpb, 0, stream>>>(
          N_,
          d_coords,
          d_params,
          lambda,
          d_lambda_idxs_,
          d_charge_param_idxs_,
          d_born_radii_buffer_,
          prefactor,
          cutoff_force_,
          nblist_.get_block_bounds_ctr(),
          nblist_.get_block_bounds_ext(),
          d_born_forces_buffer_, // output
          d_out_coords,
          d_out_lambda // ouput
        );

        // hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        k_reduce_born_forces<<<B, tpb, 0, stream>>>(
          N_,
          d_params,
          d_atomic_radii_idxs_,
          d_born_radii_buffer_,
          d_obc_buffer_,
          surface_tension_,
          probe_radius_,
          d_born_forces_buffer_
        );

        // // hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        k_compute_born_energy_and_forces<RealType><<<dimGrid, tpb, 0, stream>>>(
          N_,
          d_coords,
          d_params,
          lambda,
          d_lambda_idxs_,
          d_atomic_radii_idxs_,
          d_scale_factor_idxs_,
          d_born_radii_buffer_,
          d_obc_buffer_,
          dielectric_offset_,
          cutoff_force_,
          nblist_.get_block_bounds_ctr(),
          nblist_.get_block_bounds_ext(),
          d_born_forces_buffer_,
          d_out_coords,
          d_out_lambda
        );

        // hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        // // auto finish = std::chrono::high_resolution_clock::now();
        // // std::chrono::duration<double> elapsed = finish - start;
        // // std::cout << "Nonbonded Elapsed time: " << elapsed.count() << " s\n";

    } else {

        // std::cout << "exec GB JVP" << std::endl;

        gpuErrchk(hipMemsetAsync(d_born_radii_buffer_jvp_, 0, N*sizeof(*d_born_radii_buffer_jvp_), stream));
        gpuErrchk(hipMemsetAsync(d_obc_buffer_jvp_, 0, N*sizeof(*d_obc_buffer_jvp_), stream));
        gpuErrchk(hipMemsetAsync(d_obc_ri_buffer_jvp_, 0, N*sizeof(*d_obc_ri_buffer_jvp_), stream));
        gpuErrchk(hipMemsetAsync(d_born_forces_buffer_jvp_, 0, N*sizeof(*d_born_forces_buffer_jvp_), stream));

        k_compute_born_radii_gpu_jvp<RealType><<<dimGrid, tpb, 0, stream>>>(
            N_,
            d_coords,
            d_coords_tangents,
            d_params,
            lambda,
            lambda_tangent,
            d_lambda_idxs_,
            d_atomic_radii_idxs_,
            d_scale_factor_idxs_,
            dielectric_offset_,
            cutoff_radii_,
            nblist_.get_block_bounds_ctr(),
            nblist_.get_block_bounds_ext(),
            d_born_radii_buffer_jvp_
        );

        // hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        k_reduce_born_radii_jvp<<<B, tpb, 0, stream>>>(
          N_,
          d_params,
          d_atomic_radii_idxs_,
          dielectric_offset_,
          alpha_,
          beta_,
          gamma_,
          d_born_radii_buffer_jvp_,
          d_obc_buffer_jvp_,
          d_obc_ri_buffer_jvp_
        );

        // hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        k_compute_born_first_loop_gpu_jvp<RealType><<<dimGrid, tpb, 0, stream>>>(
            N_,
            d_coords,
            d_coords_tangents,
            d_params,
            lambda,
            lambda_tangent,
            d_lambda_idxs_,
            d_charge_param_idxs_,
            d_born_radii_buffer_jvp_,
            prefactor,
            cutoff_force_,
            nblist_.get_block_bounds_ctr(),
            nblist_.get_block_bounds_ext(),
            d_born_forces_buffer_jvp_, // output
            d_out_coords_tangents, // ouput
            d_out_params_tangents // ouput
        );

        // hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        k_reduce_born_forces_jvp<<<B, tpb, 0, stream>>>(
            N_,
            d_params,
            d_atomic_radii_idxs_,
            d_born_radii_buffer_jvp_,
            d_obc_buffer_jvp_,
            d_obc_ri_buffer_jvp_,
            surface_tension_,
            probe_radius_,
            d_born_forces_buffer_jvp_,
            d_out_params_tangents
        );

        // hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());


        // auto start = std::chrono::high_resolution_clock::now();
        k_compute_born_energy_and_forces_jvp<RealType, D><<<dimGrid, tpb, 0, stream>>>(
            N_,
            d_coords,
            d_coords_tangents,
            d_params,
            lambda,
            lambda_tangent,
            d_lambda_idxs_,
            d_atomic_radii_idxs_,
            d_scale_factor_idxs_,
            d_born_radii_buffer_jvp_,
            d_obc_buffer_jvp_,
            d_obc_ri_buffer_jvp_,
            dielectric_offset_,
            cutoff_force_,
            nblist_.get_block_bounds_ctr(),
            nblist_.get_block_bounds_ext(),
            d_born_forces_buffer_jvp_,
            d_out_coords_tangents,
            d_out_params_tangents
        );

        // hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        // auto finish = std::chrono::high_resolution_clock::now();
        // std::chrono::duration<double> elapsed = finish - start;
        // std::cout << "Nonbonded JVP Elapsed time: " << elapsed.count() << " s\n";


    }

    // hipDeviceSynchronize();
    // auto finish = std::chrono::high_resolution_clock::now();
    // std::chrono::duration<double> elapsed = finish - start;
    // std::cout << "GBSA Elapsed time: " << elapsed.count() << " s\n";


}

template class GBSA<double, 4>;
template class GBSA<double, 3>;

template class GBSA<float, 4>;
template class GBSA<float, 3>;


}

