#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "device_buffer.hpp"
#include "gpu_utils.cuh"
#include "k_neighborlist.cuh"
#include "neighborlist.hpp"
#include "nonbonded_common.cuh"

namespace timemachine {

template <typename RealType> Neighborlist<RealType>::Neighborlist(const int N) : max_size_(N), N_(N), NC_(N), NR_(N) {
    const int tpb = warp_size;
    const int column_blocks = this->num_column_blocks();
    const int row_blocks = this->num_row_blocks();
    const int Y = this->Y();

    const unsigned long long MAX_TILE_BUFFER = row_blocks * column_blocks;
    const unsigned long long MAX_ATOM_BUFFER = MAX_TILE_BUFFER * tpb;

    // interaction buffers
    gpuErrchk(hipMalloc(&d_ixn_count_, 1 * sizeof(*d_ixn_count_)));
    gpuErrchk(hipMalloc(&d_ixn_tiles_, MAX_TILE_BUFFER * sizeof(*d_ixn_tiles_)));
    gpuErrchk(hipMalloc(&d_ixn_atoms_, MAX_ATOM_BUFFER * sizeof(*d_ixn_atoms_)));
    gpuErrchk(hipMalloc(&d_trim_atoms_, column_blocks * Y * tpb * sizeof(*d_trim_atoms_)));

    // bounding box buffers
    gpuErrchk(hipMalloc(&d_row_block_bounds_ctr_, column_blocks * 3 * sizeof(*d_row_block_bounds_ctr_)));
    gpuErrchk(hipMalloc(&d_row_block_bounds_ext_, column_blocks * 3 * sizeof(*d_row_block_bounds_ext_)));
    gpuErrchk(hipMalloc(&d_column_block_bounds_ctr_, column_blocks * 3 * sizeof(*d_column_block_bounds_ctr_)));
    gpuErrchk(hipMalloc(&d_column_block_bounds_ext_, column_blocks * 3 * sizeof(*d_column_block_bounds_ext_)));

    // Row and column indices arrays
    gpuErrchk(hipMalloc(&d_column_idxs_, max_size_ * sizeof(*d_column_idxs_)));
    gpuErrchk(hipMalloc(&d_row_idxs_, max_size_ * sizeof(*d_row_idxs_)));

    this->reset_row_idxs();
}

template <typename RealType> Neighborlist<RealType>::~Neighborlist() {
    gpuErrchk(hipFree(d_column_idxs_));
    gpuErrchk(hipFree(d_row_idxs_));

    gpuErrchk(hipFree(d_ixn_count_));
    gpuErrchk(hipFree(d_ixn_tiles_));
    gpuErrchk(hipFree(d_ixn_atoms_));
    gpuErrchk(hipFree(d_trim_atoms_));

    gpuErrchk(hipFree(d_row_block_bounds_ctr_));
    gpuErrchk(hipFree(d_row_block_bounds_ext_));
    gpuErrchk(hipFree(d_column_block_bounds_ctr_));
    gpuErrchk(hipFree(d_column_block_bounds_ext_));
}

template <typename RealType>
void Neighborlist<RealType>::compute_block_bounds_host(
    const int N, const double *h_coords, const double *h_box, double *h_bb_ctrs, double *h_bb_exts) {

    const int D = 3;
    DeviceBuffer<double> d_coords(N * D);
    DeviceBuffer<double> d_box(D * D);

    d_coords.copy_from(h_coords);
    d_box.copy_from(h_box);

    this->compute_block_bounds_device(N, D, d_coords.data, d_box.data, static_cast<hipStream_t>(0));
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(
        h_bb_ctrs,
        d_column_block_bounds_ctr_,
        this->num_column_blocks() * 3 * sizeof(*d_column_block_bounds_ctr_),
        hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(
        h_bb_exts,
        d_column_block_bounds_ext_,
        this->num_column_blocks() * 3 * sizeof(*d_column_block_bounds_ext_),
        hipMemcpyDeviceToHost));
}

template <typename RealType>
std::vector<std::vector<int>>
Neighborlist<RealType>::get_nblist_host(int N, const double *h_coords, const double *h_box, const double cutoff) {

    if (N != N_) {
        throw std::runtime_error("N != N_");
    }

    DeviceBuffer<double> d_coords(N * 3);
    DeviceBuffer<double> d_box(3 * 3);
    d_coords.copy_from(h_coords);
    d_box.copy_from(h_box);

    this->build_nblist_device(N, d_coords.data, d_box.data, cutoff, static_cast<hipStream_t>(0));

    gpuErrchk(hipDeviceSynchronize());
    const int tpb = warp_size;
    const int column_blocks = this->num_column_blocks();
    const int row_blocks = this->num_row_blocks();

    unsigned long long MAX_TILE_BUFFER = row_blocks * column_blocks;
    unsigned long long MAX_ATOM_BUFFER = MAX_TILE_BUFFER * tpb;

    unsigned int h_ixn_count;
    gpuErrchk(hipMemcpy(&h_ixn_count, d_ixn_count_, 1 * sizeof(*d_ixn_count_), hipMemcpyDeviceToHost));
    std::vector<int> h_ixn_tiles(MAX_TILE_BUFFER);
    std::vector<unsigned int> h_ixn_atoms(MAX_ATOM_BUFFER);
    gpuErrchk(hipMemcpy(&h_ixn_tiles[0], d_ixn_tiles_, MAX_TILE_BUFFER * sizeof(int), hipMemcpyDeviceToHost));
    gpuErrchk(
        hipMemcpy(&h_ixn_atoms[0], d_ixn_atoms_, MAX_ATOM_BUFFER * sizeof(unsigned int), hipMemcpyDeviceToHost));

    std::vector<std::vector<int>> ixn_list(row_blocks, std::vector<int>());
    for (int i = 0; i < h_ixn_count; i++) {
        int tile_idx = h_ixn_tiles[i];
        for (int j = 0; j < tpb; j++) {
            int atom_j_idx = h_ixn_atoms[i * tpb + j];
            if (atom_j_idx < N) {
                ixn_list[tile_idx].push_back(atom_j_idx);
            }
        }
    }

    return ixn_list;
}

template <typename RealType>
void Neighborlist<RealType>::build_nblist_device(
    const int N, const double *d_coords, const double *d_box, const double cutoff, const hipStream_t stream) {
    gpuErrchk(hipMemsetAsync(d_ixn_count_, 0, 1 * sizeof(*d_ixn_count_), stream));

    const int D = 3;
    this->compute_block_bounds_device(N, D, d_coords, d_box, stream);
    const int tpb = warp_size;
    const int row_blocks = this->num_row_blocks();
    const int Y = this->Y();

    dim3 dimGrid(row_blocks, Y, 1); // block x, y, z dims

    // (ytz): TBD shared memory, stream
    if (this->compute_upper_triangular()) {
        // Compute only the upper triangle as rows and cols are the same
        // pass duplicates of column coords and the bounding boxes
        k_find_blocks_with_ixns<RealType, true><<<dimGrid, tpb, 0, stream>>>(
            N_,
            NC_,
            NR_,
            d_column_idxs_,
            d_row_idxs_,
            d_column_block_bounds_ctr_,
            d_column_block_bounds_ext_,
            d_column_block_bounds_ctr_,
            d_column_block_bounds_ext_,
            d_coords,
            d_box,
            d_ixn_count_,
            d_ixn_tiles_,
            d_ixn_atoms_,
            d_trim_atoms_,
            cutoff);
    } else {
        k_find_blocks_with_ixns<RealType, false><<<dimGrid, tpb, 0, stream>>>(
            N_,
            NC_,
            NR_,
            d_column_idxs_,
            d_row_idxs_,
            d_column_block_bounds_ctr_,
            d_column_block_bounds_ext_,
            d_row_block_bounds_ctr_,
            d_row_block_bounds_ext_,
            d_coords,
            d_box,
            d_ixn_count_,
            d_ixn_tiles_,
            d_ixn_atoms_,
            d_trim_atoms_,
            cutoff);
    }

    gpuErrchk(hipPeekAtLastError());
    k_compact_trim_atoms<<<row_blocks, tpb, 0, stream>>>(
        N_, Y, d_trim_atoms_, d_ixn_count_, d_ixn_tiles_, d_ixn_atoms_);

    gpuErrchk(hipPeekAtLastError());
}

template <typename RealType>
void Neighborlist<RealType>::compute_block_bounds_device(
    const int N,            // Number of atoms
    const int D,            // Box dimensions
    const double *d_coords, // [N*3]
    const double *d_box,    // [D*3]
    const hipStream_t stream) {

    if (D != 3) {
        throw std::runtime_error("D != 3");
    }

    const int tpb = warp_size;
    const int column_blocks = this->num_column_blocks(); // total number of blocks we need to process

    k_find_block_bounds<RealType><<<column_blocks, tpb, 0, stream>>>(
        N, column_blocks, NC_, d_column_idxs_, d_coords, d_box, d_column_block_bounds_ctr_, d_column_block_bounds_ext_);
    gpuErrchk(hipPeekAtLastError());
    // In the case of upper triangle of the matrix, the column and row indices are the same, so only compute block ixns for both
    // when they are different
    if (!this->compute_upper_triangular()) {
        const int row_blocks = this->num_row_blocks();
        k_find_block_bounds<RealType><<<row_blocks, tpb, 0, stream>>>(
            N, row_blocks, NR_, d_row_idxs_, d_coords, d_box, d_row_block_bounds_ctr_, d_row_block_bounds_ext_);
        gpuErrchk(hipPeekAtLastError());
    }
};

template <typename RealType> void Neighborlist<RealType>::set_row_idxs(std::vector<unsigned int> row_idxs) {
    if (row_idxs.size() == 0) {
        throw std::runtime_error("idxs can't be empty");
    }
    std::set<unsigned int> unique_idxs(row_idxs.begin(), row_idxs.end());
    if (unique_idxs.size() != row_idxs.size()) {
        throw std::runtime_error("atom indices must be unique");
    }
    if (row_idxs.size() >= N_) {
        throw std::runtime_error("number of idxs must be less than N");
    }
    if (*std::max_element(row_idxs.begin(), row_idxs.end()) >= N_) {
        throw std::runtime_error("indices values must be less than N");
    }
    std::vector<unsigned int> column_indices = get_indices_difference<unsigned int>(N_, unique_idxs);
    const size_t row_count = row_idxs.size();
    const size_t col_count = column_indices.size();

    DeviceBuffer<unsigned int> row_idx_buffer(row_count);
    DeviceBuffer<unsigned int> column_idx_buffer(col_count);

    row_idx_buffer.copy_from(&row_idxs[0]);

    column_idx_buffer.copy_from(&column_indices[0]);

    this->set_idxs_device(
        col_count, row_count, column_idx_buffer.data, row_idx_buffer.data, static_cast<hipStream_t>(0));
    gpuErrchk(hipDeviceSynchronize());
}

template <typename RealType> void Neighborlist<RealType>::reset_row_idxs() {
    const hipStream_t stream = static_cast<hipStream_t>(0);
    this->reset_row_idxs_device(stream);
    gpuErrchk(hipStreamSynchronize(stream));
}

template <typename RealType> void Neighborlist<RealType>::reset_row_idxs_device(const hipStream_t stream) {
    const int tpb = warp_size;
    const int blocks = ceil_divide(N_, tpb);
    // Fill the indices with the 0 to N-1 indices, indicating 'normal' neighborlist operation
    k_arange<<<blocks, tpb, 0, stream>>>(N_, d_column_idxs_);
    gpuErrchk(hipPeekAtLastError());
    k_arange<<<blocks, tpb, 0, stream>>>(N_, d_row_idxs_);
    gpuErrchk(hipPeekAtLastError());
    this->NR_ = N_;
    this->NC_ = N_;
}

template <typename RealType> void Neighborlist<RealType>::resize(const int size) {
    const hipStream_t stream = static_cast<hipStream_t>(0);
    this->resize_device(size, stream);
    gpuErrchk(hipStreamSynchronize(stream));
}

// Resize the Neighborlist to function on a different size. Note that this only allows
// finding interactions on a smaller set of the system, will not increase the size of the underlying buffers.
template <typename RealType> void Neighborlist<RealType>::resize_device(const int size, const hipStream_t stream) {
    if (size <= 0) {
        throw std::runtime_error("size is must be at least 1");
    }
    if (size > max_size_) {
        throw std::runtime_error("size is greater than max size");
    }
    this->N_ = size;
    this->reset_row_idxs_device(stream);
}

// set_idxs_device is for use when idxs exist on the GPU already and are used as the new idxs to compute the neighborlist on.
template <typename RealType>
void Neighborlist<RealType>::set_idxs_device(
    const int NC,
    const int NR,
    unsigned int *d_in_column_idxs,
    unsigned int *d_in_row_idxs,
    const hipStream_t stream) {
    if (NC + NR != N_) {
        throw std::runtime_error("Total of indices must equal N");
    }
    if (NC == 0 || NR == 0) {
        throw std::runtime_error("Number of column and row indices must be non-zero");
    }
    const size_t tpb = warp_size;

    // The indices must already be on the GPU and are copied into the neighborlist buffers.
    gpuErrchk(hipMemcpyAsync(
        d_column_idxs_, d_in_column_idxs, NC * sizeof(*d_column_idxs_), hipMemcpyDeviceToDevice, stream));
    gpuErrchk(hipMemcpyAsync(d_row_idxs_, d_in_row_idxs, NR * sizeof(*d_row_idxs_), hipMemcpyDeviceToDevice, stream));

    // Fill in the rest of values with N, potentially redundant
    k_initialize_array<unsigned int><<<ceil_divide(NR, tpb), tpb, 0, stream>>>(NR, d_column_idxs_ + NC, N_);
    gpuErrchk(hipPeekAtLastError());
    k_initialize_array<unsigned int><<<ceil_divide(NC, tpb), tpb, 0, stream>>>(NC, d_row_idxs_ + NR, N_);
    gpuErrchk(hipPeekAtLastError());

    // Update the row and column counts
    this->NR_ = NR;
    this->NC_ = NC;

    // Clear the atom ixns, to avoid reuse
    const int row_blocks = this->num_row_blocks();
    const int column_blocks = this->num_column_blocks();
    unsigned long long MAX_ATOM_BUFFER = row_blocks * column_blocks * tpb;
    // Set to max value, ie greater than N. Note that Memset is on bytes, which is why it is UCHAR_MAX
    gpuErrchk(hipMemsetAsync(d_ixn_atoms_, UCHAR_MAX, MAX_ATOM_BUFFER * sizeof(*d_ixn_atoms_), stream));
}

template <typename RealType> bool Neighborlist<RealType>::compute_upper_triangular() const {
    return NR_ == N_ && NC_ == N_;
};

template <typename RealType> int Neighborlist<RealType>::num_column_blocks() const {
    return ceil_divide(NC_, tile_size);
};

template <typename RealType> int Neighborlist<RealType>::Y() const {
    return ceil_divide(this->num_column_blocks(), warp_size);
};

template <typename RealType> int Neighborlist<RealType>::num_row_blocks() const { return ceil_divide(NR_, tile_size); }

template class Neighborlist<double>;
template class Neighborlist<float>;

} // namespace timemachine
