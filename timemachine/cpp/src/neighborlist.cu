#include <cassert>
#include <vector>
#include <iostream>

#include "neighborlist.hpp"
#include "k_neighborlist.cuh"
#include "gpu_utils.cuh"

namespace timemachine {

template<typename RealType>
Neighborlist<RealType>::Neighborlist(
	int N) : N_(N) {

    const int B = this->B(); //(N+32-1)/32;
    const int Y = this->Y(); //(B+32-1)/32;

    unsigned long long MAX_TILE_BUFFER = B*B;
    unsigned long long MAX_ATOM_BUFFER = B*B*32;

    // interaction buffers
    gpuErrchk(hipMalloc(&d_ixn_count_, 1*sizeof(*d_ixn_count_)));
    gpuErrchk(hipMalloc(&d_ixn_tiles_, MAX_TILE_BUFFER*sizeof(*d_ixn_tiles_)));
    gpuErrchk(hipMalloc(&d_ixn_atoms_, MAX_ATOM_BUFFER*sizeof(*d_ixn_atoms_)));
    gpuErrchk(hipMalloc(&d_trim_atoms_, B*Y*32*sizeof(*d_trim_atoms_)));


    // bounding box buffers
    gpuErrchk(hipMalloc(&d_block_bounds_ctr_, B*3*sizeof(*d_block_bounds_ctr_)));
    gpuErrchk(hipMalloc(&d_block_bounds_ext_, B*3*sizeof(*d_block_bounds_ext_)));

}

template<typename RealType>
Neighborlist<RealType>::~Neighborlist() {

    gpuErrchk(hipFree(d_ixn_count_));
    gpuErrchk(hipFree(d_ixn_tiles_));
    gpuErrchk(hipFree(d_ixn_atoms_));
    gpuErrchk(hipFree(d_trim_atoms_));

    gpuErrchk(hipFree(d_block_bounds_ctr_));
    gpuErrchk(hipFree(d_block_bounds_ext_));

}


bool is_pow_2(int x) {
    return (x & (x - 1)) == 0;
}

int log2_int(int v) {
    int bits = 0;
    while (v >>= 1) ++bits;
    return bits;
}

int pow_int(int x, int p) {
  if (p == 0) return 1;
  if (p == 1) return x;

  int tmp = pow_int(x, p/2);
  if (p%2 == 0) return tmp * tmp;
  else return x * tmp * tmp;
}

template<typename RealType>
void Neighborlist<RealType>::compute_block_bounds_host(
    const int N,
    const int D,
    const int block_size,
    const double *h_coords,
    const double *h_box,
    double *h_bb_ctrs,
    double *h_bb_exts) {

    assert(N == N_);
    assert(D == 3);

    double *d_coords = gpuErrchkCudaMallocAndCopy(h_coords, N*3*sizeof(double));
    double *d_box = gpuErrchkCudaMallocAndCopy(h_box, 3*3*sizeof(double));  
   
    this->compute_block_bounds_device(
        N,
        D,
        d_coords,
        d_box,
        static_cast<hipStream_t>(0)
    );
    // Does this need to peek at the last error?
    hipDeviceSynchronize();

    gpuErrchk(hipMemcpy(h_bb_ctrs, d_block_bounds_ctr_, this->B()*3*sizeof(*d_block_bounds_ctr_), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_bb_exts, d_block_bounds_ext_, this->B()*3*sizeof(*d_block_bounds_ext_), hipMemcpyDeviceToHost));

}

template<typename RealType>
std::vector<std::vector<int> > Neighborlist<RealType>::get_nblist_host(
    int N,
    const double *h_coords,
    const double *h_box,
    const double cutoff) {

    // assert(N==N_);

    double *d_coords = gpuErrchkCudaMallocAndCopy(h_coords, N*3*sizeof(double));
    double *d_box = gpuErrchkCudaMallocAndCopy(h_box, 3*3*sizeof(double));    

    this->build_nblist_device(
        N,
        d_coords,
        d_box,
        cutoff,
        static_cast<hipStream_t>(0)
    );

    hipDeviceSynchronize();
    const int B = this->B(); //(N+32-1)/32;

    unsigned long long MAX_TILE_BUFFER = B*B;
    unsigned long long MAX_ATOM_BUFFER = B*B*32;

    unsigned int h_ixn_count;
    gpuErrchk(hipMemcpy(&h_ixn_count, d_ixn_count_, 1*sizeof(*d_ixn_count_), hipMemcpyDeviceToHost));
    std::vector<int> h_ixn_tiles(MAX_TILE_BUFFER);
    std::vector<unsigned int> h_ixn_atoms(MAX_ATOM_BUFFER);
    gpuErrchk(hipMemcpy(&h_ixn_tiles[0], d_ixn_tiles_, MAX_TILE_BUFFER*sizeof(int), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(&h_ixn_atoms[0], d_ixn_atoms_, MAX_ATOM_BUFFER*sizeof(unsigned int), hipMemcpyDeviceToHost));

    std::vector<std::vector<int> > ixn_list(B, std::vector<int>());

    for(int i=0; i < h_ixn_count; i++) {
        int tile_idx = h_ixn_tiles[i];
        for(int j=0; j < 32; j++) {
            int atom_j_idx = h_ixn_atoms[i*32+j];
            if(atom_j_idx < N) {
                ixn_list[tile_idx].push_back(atom_j_idx);
            }
        }
    }

    gpuErrchk(hipFree(d_coords));
    gpuErrchk(hipFree(d_box));

    return ixn_list;

}

template<typename RealType>
void Neighborlist<RealType>::build_nblist_device(
    const int N,
    const double *d_coords,
    const double *d_box,
    const double cutoff,
    hipStream_t stream) {

    // assert(N == N_);

    // reset the interaction count
    gpuErrchk(hipMemsetAsync(d_ixn_count_, 0, 1*sizeof(*d_ixn_count_), stream));

    const int D = 3;
    this->compute_block_bounds_device(
        N,
        D,
        d_coords,
        d_box,
        stream
    );

    int tpb = 32;
    const int B = this->B(); // (N+32-1)/32;
    const int Y = this->Y(); // (B+32-1)/32;

    dim3 dimGrid(B, Y, 1); // block x, y, z dims

    // (ytz): TBD shared memory, stream
    k_find_blocks_with_ixns<RealType><<<dimGrid, tpb, 0, stream>>>(
        N,
        d_block_bounds_ctr_,
        d_block_bounds_ext_,
        d_coords,
        d_box,
        d_ixn_count_,
        d_ixn_tiles_,
        d_ixn_atoms_,
        d_trim_atoms_,
        cutoff
    );

    gpuErrchk(hipPeekAtLastError());

    k_compact_trim_atoms<<<B, tpb, 0, stream>>>(
        N,
        Y,
        d_trim_atoms_,
        d_ixn_count_,
        d_ixn_tiles_,
        d_ixn_atoms_
    );

    gpuErrchk(hipPeekAtLastError());

}

template <typename RealType>
void Neighborlist<RealType>::compute_block_bounds_device(
	int N, // Number of atoms
	int D, // Box dimensions
	const double *d_coords, // [N*3]
    const double *d_box, // [D*3]
	hipStream_t stream) {

    assert(N == N_);
    assert(D == 3);

    const int tpb = 32;
    const int B = (N+tpb-1)/tpb; // total number of blocks we need to process

    gpuErrchk(hipMemsetAsync(d_block_bounds_ctr_, 0, B*D*sizeof(*d_block_bounds_ctr_), stream));
    gpuErrchk(hipMemsetAsync(d_block_bounds_ext_, 0, B*D*sizeof(*d_block_bounds_ext_), stream));

    k_find_block_bounds<<<B, tpb, 0, stream>>>(
        N,
        D,
        B,
        d_coords,
        d_box,
        d_block_bounds_ctr_,
        d_block_bounds_ext_
    );	

    gpuErrchk(hipPeekAtLastError());

};


template class Neighborlist<double>;
template class Neighborlist<float>;


}