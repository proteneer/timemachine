#include <chrono>
#include <iostream>
#include <vector>
#include <complex>
#include "harmonic_angle.hpp"
#include "gpu_utils.cuh"
#include "k_bonded_deterministic.cuh"

namespace timemachine {

template <typename RealType>
HarmonicAngle<RealType>::HarmonicAngle(
    const std::vector<int> &angle_idxs, // [A, 3]
    const std::vector<int> &param_idxs // [A, 2]
) : A_(angle_idxs.size()/3) {

    if(angle_idxs.size() % 3 != 0) {
        throw std::runtime_error("angle_idxs.size() must be exactly 3*k");
    }

    for(int a=0; a < A_; a++) {
        auto i = angle_idxs[a*3+0];
        auto j = angle_idxs[a*3+1];
        auto k = angle_idxs[a*3+2];
        if(i == j || j == k || i == k) {
            throw std::runtime_error("angle triplets must be unique");
        }
    }

    gpuErrchk(hipMalloc(&d_angle_idxs_, A_*3*sizeof(*d_angle_idxs_)));
    gpuErrchk(hipMemcpy(d_angle_idxs_, &angle_idxs[0], A_*3*sizeof(*d_angle_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_param_idxs_, A_*3*sizeof(*d_param_idxs_)));
    gpuErrchk(hipMemcpy(d_param_idxs_, &param_idxs[0], A_*3*sizeof(*d_param_idxs_), hipMemcpyHostToDevice));

};

template <typename RealType>
HarmonicAngle<RealType>::~HarmonicAngle() {
    gpuErrchk(hipFree(d_angle_idxs_));
    gpuErrchk(hipFree(d_param_idxs_));
};

template <typename RealType>
void HarmonicAngle<RealType>::execute_lambda_inference_device(
    const int N,
    const int P,
    const double *d_coords_primals,
    const double *d_params_primals,
    const double lambda_primal,
    unsigned long long *d_out_coords_primals, // du/dx
    double *d_out_lambda_primal, // du/dl
    double *d_out_energy_primal, // U
    hipStream_t stream) {

    int tpb = 32;
    int blocks = (A_+tpb-1)/tpb;

    auto start = std::chrono::high_resolution_clock::now();
    // if(d_coords_tangents == nullptr) {

    k_harmonic_angle_inference<RealType, 3><<<blocks, tpb, 0, stream>>>(
        A_,
        d_coords_primals,
        d_params_primals,
        d_angle_idxs_,
        d_param_idxs_,
        d_out_coords_primals,
        d_out_energy_primal
    );

    gpuErrchk(hipPeekAtLastError());

}



template <typename RealType>
void HarmonicAngle<RealType>::execute_lambda_jvp_device(
    const int N,
    const int P,
    const double *d_coords_primals,
    const double *d_coords_tangents,
    const double *d_params_primals,
    const double lambda_primal, // unused
    const double lambda_tangent, // unused
    double *d_out_coords_primals,
    double *d_out_coords_tangents,
    double *d_out_params_primals,
    double *d_out_params_tangents,
    hipStream_t stream) {

    int tpb = 32;
    int blocks = (A_+tpb-1)/tpb;

    k_harmonic_angle_jvp<RealType, 3><<<blocks, tpb,  0, stream>>>(
        A_,
        d_coords_primals,
        d_coords_tangents,
        d_params_primals,
        d_angle_idxs_,
        d_param_idxs_,
        d_out_coords_primals,
        d_out_coords_tangents,
        d_out_params_primals,
        d_out_params_tangents
    );

    gpuErrchk(hipPeekAtLastError());

    //     gpuErrchk(hipPeekAtLastError());

    //     // hipDeviceSynchronize();
    //     // auto finish = std::chrono::high_resolution_clock::now();
    //     // std::chrono::duration<double> elapsed = finish - start;
    //     // std::cout << "HarmonicAngle JVP Elapsed time: " << elapsed.count() << " s\n";


    // }


};

template class HarmonicAngle<double>;
template class HarmonicAngle<float>;

} // namespace timemachine