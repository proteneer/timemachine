#include "summed_potential.hpp"
#include <memory>
#include <numeric>
#include <stdexcept>

namespace timemachine {

SummedPotential::SummedPotential(
    const std::vector<std::shared_ptr<Potential>> potentials, const std::vector<int> params_sizes)
    : potentials_(potentials), params_sizes_(params_sizes),
      P_(std::accumulate(params_sizes.begin(), params_sizes.end(), 0)) {
    if (potentials_.size() != params_sizes_.size()) {
        throw std::runtime_error("number of potentials != number of parameter sizes");
    }
};

const std::vector<std::shared_ptr<Potential>> &SummedPotential::get_potentials() { return potentials_; }

void SummedPotential::execute_device(
    const int N,
    const int P,
    const double *d_x,
    const double *d_p,
    const double *d_box,
    const double lambda,
    unsigned long long *d_du_dx,
    unsigned long long *d_du_dp,
    unsigned long long *d_du_dl,
    unsigned long long *d_u,
    hipStream_t stream) {

    if (P != P_) {
        throw std::runtime_error(
            "SummedPotential::execute_device(): expected " + std::to_string(P_) + " parameters, got " +
            std::to_string(P));
    }

    int offset = 0;

    for (auto i = 0; i < potentials_.size(); i++) {

        potentials_[i]->execute_device(
            N,
            params_sizes_[i],
            d_x,
            d_p + offset,
            d_box,
            lambda,
            d_du_dx,
            d_du_dp == nullptr ? nullptr : d_du_dp + offset,
            d_du_dl,
            d_u,
            stream);

        offset += params_sizes_[i];
    }
};

void SummedPotential::du_dp_fixed_to_float(
    const int N, const int P, const unsigned long long *du_dp, double *du_dp_float) {

    int offset = 0;

    for (auto i = 0; i < potentials_.size(); i++) {
        potentials_[i]->du_dp_fixed_to_float(N, params_sizes_[i], du_dp + offset, du_dp_float + offset);
        offset += params_sizes_[i];
    }
}

} // namespace timemachine
