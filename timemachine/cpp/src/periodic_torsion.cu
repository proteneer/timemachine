#include <chrono>
#include <iostream>
#include <vector>
#include <complex>
#include "periodic_torsion.hpp"
#include "gpu_utils.cuh"
#include "k_bonded.cuh"

namespace timemachine {

template <typename RealType>
PeriodicTorsion<RealType>::PeriodicTorsion(
    const std::vector<int> &torsion_idxs, // [A, 4]
    const std::vector<double> &params // [A, 3]
) : T_(torsion_idxs.size()/4) {

    if(torsion_idxs.size() % 4 != 0) {
        throw std::runtime_error("torsion_idxs.size() must be exactly 4*k");
    }

    for(int a=0; a < T_; a++) {
        auto i = torsion_idxs[a*4+0];
        auto j = torsion_idxs[a*4+1];
        auto k = torsion_idxs[a*4+2];
        auto l = torsion_idxs[a*4+3];
        if(i == j || i == k || i == l || j == k || j == l || k == l) {
            throw std::runtime_error("torsion quads must be unique");
        }
    }

    gpuErrchk(hipMalloc(&d_torsion_idxs_, T_*4*sizeof(*d_torsion_idxs_)));
    gpuErrchk(hipMemcpy(d_torsion_idxs_, &torsion_idxs[0], T_*4*sizeof(*d_torsion_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_params_, T_*3*sizeof(*d_params_)));
    gpuErrchk(hipMemcpy(d_params_, &params[0], T_*3*sizeof(*d_params_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_du_dp_primals_, T_*3*sizeof(*d_du_dp_primals_)));
    gpuErrchk(hipMemset(d_du_dp_primals_, 0, T_*3*sizeof(*d_du_dp_primals_)));

    gpuErrchk(hipMalloc(&d_du_dp_tangents_, T_*3*sizeof(*d_du_dp_tangents_)));
    gpuErrchk(hipMemset(d_du_dp_tangents_, 0, T_*3*sizeof(*d_du_dp_tangents_)));

};

template <typename RealType>
PeriodicTorsion<RealType>::~PeriodicTorsion() {
    gpuErrchk(hipFree(d_torsion_idxs_));

    gpuErrchk(hipFree(d_params_));
    gpuErrchk(hipFree(d_du_dp_primals_));
    gpuErrchk(hipFree(d_du_dp_tangents_));

};

template <typename RealType>
void PeriodicTorsion<RealType>::get_du_dp_primals(double *buf) {
    gpuErrchk(hipMemcpy(buf, d_du_dp_primals_, T_*3*sizeof(*d_params_), hipMemcpyDeviceToHost));
}

template <typename RealType>
void PeriodicTorsion<RealType>::get_du_dp_tangents(double *buf) {
    gpuErrchk(hipMemcpy(buf, d_du_dp_tangents_, T_*3*sizeof(*d_params_), hipMemcpyDeviceToHost));
}

template <typename RealType>
void PeriodicTorsion<RealType>::execute_lambda_inference_device(
    const int N,
    const double *d_coords_primals,
    const double lambda_primal,
    unsigned long long *d_out_coords_primals, // du/dx
    double *d_out_lambda_primal, // du/dl, unused
    double *d_out_energy_primal, // U
    hipStream_t stream) {

    int tpb = 32;
    int blocks = (T_+tpb-1)/tpb;

    const int D = 3;

    k_periodic_torsion_inference<RealType, D><<<blocks, tpb, 0, stream>>>(
        T_,
        d_coords_primals,
        d_params_,
        d_torsion_idxs_,
        d_out_coords_primals,
        d_out_energy_primal
    );

    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

};

template <typename RealType>
void PeriodicTorsion<RealType>::execute_lambda_jvp_device(
    const int N,
    const double *d_coords_primals,
    const double *d_coords_tangents,
    const double lambda_primal, // unused
    const double lambda_tangent, // unused
    double *d_out_coords_primals,
    double *d_out_coords_tangents,
    hipStream_t stream) {

    int tpb = 32;
    int blocks = (T_+tpb-1)/tpb;
    const int D = 3;
    k_periodic_torsion_jvp<RealType, D><<<blocks, tpb, 0, stream>>>(
        T_,
        d_coords_primals,
        d_coords_tangents,
        d_params_,
        d_torsion_idxs_,
        d_out_coords_primals,
        d_out_coords_tangents,
        d_du_dp_primals_,
        d_du_dp_tangents_
    );

    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

};

template class PeriodicTorsion<double>;
template class PeriodicTorsion<float>;

} // namespace timemachine