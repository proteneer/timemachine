#include <chrono>
#include <iostream>
#include <vector>
#include <complex>
#include "periodic_torsion.hpp"
#include "kernel_utils.cuh"
#include "k_bonded_deterministic.cuh"

namespace timemachine {

template <typename RealType, int D>
PeriodicTorsion<RealType, D>::PeriodicTorsion(
    const std::vector<int> &torsion_idxs, // [A, 4]
    const std::vector<int> &param_idxs // [A, 3]
) : T_(torsion_idxs.size()/4) {

    if(torsion_idxs.size() % 4 != 0) {
        throw std::runtime_error("torsion_idxs.size() must be exactly 4*k");
    }

    for(int a=0; a < T_; a++) {
        auto i = torsion_idxs[a*4+0];
        auto j = torsion_idxs[a*4+1];
        auto k = torsion_idxs[a*4+2];
        auto l = torsion_idxs[a*4+3];
        if(i == j || i == k || i == l || j == k || j == l || k == l) {
            throw std::runtime_error("torsion quads must be unique");
        }
    }

    gpuErrchk(hipMalloc(&d_torsion_idxs_, T_*4*sizeof(*d_torsion_idxs_)));
    gpuErrchk(hipMemcpy(d_torsion_idxs_, &torsion_idxs[0], T_*4*sizeof(*d_torsion_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_param_idxs_, T_*3*sizeof(*d_param_idxs_)));
    gpuErrchk(hipMemcpy(d_param_idxs_, &param_idxs[0], T_*3*sizeof(*d_param_idxs_), hipMemcpyHostToDevice));

};

template <typename RealType, int D>
PeriodicTorsion<RealType, D>::~PeriodicTorsion() {
    gpuErrchk(hipFree(d_torsion_idxs_));
    gpuErrchk(hipFree(d_param_idxs_));
};

template <typename RealType, int D>
void PeriodicTorsion<RealType, D>::execute_device(
    const int N,
    const int P,
    const double *d_coords,
    const double *d_coords_tangents,
    const double *d_params,
    unsigned long long *d_out_coords,
    double *d_out_coords_tangents,
    double *d_out_params_tangents
) {

    int tpb = 32;
    int blocks = (T_+tpb-1)/tpb;

    auto start = std::chrono::high_resolution_clock::now();
    if(d_coords_tangents == nullptr) {

        k_periodic_torsion_inference<RealType, D><<<blocks, tpb>>>(
            T_,
            d_coords,
            d_params,
            d_torsion_idxs_,
            d_param_idxs_,
            d_out_coords
        );

        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        // auto finish = std::chrono::high_resolution_clock::now();
        // std::chrono::duration<double> elapsed = finish - start;
        // std::cout << "PeriodicTorsion Elapsed time: " << elapsed.count() << " s\n";

    } else {


        k_periodic_torsion_jvp<RealType, D><<<blocks, tpb>>>(
            T_,
            d_coords,
            d_coords_tangents,
            d_params,
            d_torsion_idxs_,
            d_param_idxs_,
            d_out_coords_tangents,
            d_out_params_tangents
        );

        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        // auto finish = std::chrono::high_resolution_clock::now();
        // std::chrono::duration<double> elapsed = finish - start;
        // std::cout << "PeriodicTorsion JVP Elapsed time: " << elapsed.count() << " s\n";


    }


};

template class PeriodicTorsion<double, 4>;
template class PeriodicTorsion<double, 3>;

template class PeriodicTorsion<float, 4>;
template class PeriodicTorsion<float, 3>;

} // namespace timemachine