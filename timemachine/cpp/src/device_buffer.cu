#include "hiprand/hiprand_kernel.h"
#include "device_buffer.hpp"
#include "gpu_utils.cuh"
#include <cstddef>
#include <cub/util_type.cuh>

namespace timemachine {

template <typename T> static T *allocate_gpu_memory(const std::size_t length) {
    T *buffer;
    cudaSafeMalloc(&buffer, length * sizeof(T));
    return buffer;
}

template <typename T> DeviceBuffer<T>::DeviceBuffer(const std::vector<T> &vec) : DeviceBuffer(vec.size()) {
    this->copy_from(&vec[0]);
}

template <typename T> DeviceBuffer<T>::DeviceBuffer() : DeviceBuffer(0) {}

template <typename T>
DeviceBuffer<T>::DeviceBuffer(const std::size_t length) : length(length), data(allocate_gpu_memory<T>(length)) {}

template <typename T> void DeviceBuffer<T>::realloc(const size_t new_length) {
    // Print a warning if buffers were non-zero when resized, this can have real performance impacts
    if (this->length > 0) {
        std::cout << "warning:: resizing device buffer that is non-zero" << std::endl;
    }
    // Free the existing data
    gpuErrchk(hipFree(data));
    this->length = new_length;
    this->data = allocate_gpu_memory<T>(new_length);
}

template <typename T> DeviceBuffer<T>::~DeviceBuffer() {
    // TODO: the file/line context reported by gpuErrchk on failure is
    // not very useful when it's called from here. Is there a way to
    // report a stack trace?
    gpuErrchk(hipFree(data));
}

template <typename T> size_t DeviceBuffer<T>::size() const { return this->length * sizeof(T); }

template <typename T> void DeviceBuffer<T>::copy_from(const T *host_buffer) const {
    gpuErrchk(hipMemcpy(data, host_buffer, this->size(), hipMemcpyHostToDevice));
}

template <typename T> void DeviceBuffer<T>::copy_to(T *host_buffer) const {
    gpuErrchk(hipMemcpy(host_buffer, data, this->size(), hipMemcpyDeviceToHost));
}

template class DeviceBuffer<double>;
template class DeviceBuffer<float>;
template class DeviceBuffer<int>;
template class DeviceBuffer<size_t>;
template class DeviceBuffer<char>;
template class DeviceBuffer<unsigned int>;
template class DeviceBuffer<unsigned long long>;
template class DeviceBuffer<__int128>;
template class DeviceBuffer<hipcub::KeyValuePair<int, double>>;
template class DeviceBuffer<hipcub::KeyValuePair<int, float>>;
template class DeviceBuffer<hiprandState_t>;
} // namespace timemachine
