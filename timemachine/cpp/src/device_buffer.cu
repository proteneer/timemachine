#include "device_buffer.hpp"
#include "gpu_utils.cuh"

namespace timemachine {

void *allocate(const std::size_t size) {
    void *buffer;
    gpuErrchk(hipMalloc(&buffer, size));
    return buffer;
}

template <typename T>
DeviceBuffer<T>::DeviceBuffer(const std::size_t length)
    : size(length * sizeof(T)), data(static_cast<T *>(allocate(size))) {}

template <typename T> DeviceBuffer<T>::~DeviceBuffer() { gpuErrchk(hipFree(data)); }

template <typename T> void DeviceBuffer<T>::copy_from_host(const T *host_buffer) const {
    gpuErrchk(hipMemcpy(data, host_buffer, size, hipMemcpyHostToDevice));
}

template <typename T> void DeviceBuffer<T>::copy_to_host(T *host_buffer) const {
    gpuErrchk(hipMemcpy(host_buffer, data, size, hipMemcpyDeviceToHost));
}

template <typename T> void DeviceBuffer<T>::copy_from_device(const T *device_buffer) const {
    gpuErrchk(hipMemcpy(data, device_buffer, size, hipMemcpyDeviceToDevice));
}

template <typename T> void DeviceBuffer<T>::copy_to_device(T *device_buffer) const {
    gpuErrchk(hipMemcpy(device_buffer, data, size, hipMemcpyDeviceToDevice));
}

template <typename T> void DeviceBuffer<T>::copy_from_host_async(const T *host_buffer, hipStream_t stream) const {
    gpuErrchk(hipMemcpyAsync(data, host_buffer, size, hipMemcpyHostToDevice, stream));
}

template <typename T> void DeviceBuffer<T>::copy_to_host_async(T *host_buffer, hipStream_t stream) const {
    gpuErrchk(hipMemcpyAsync(host_buffer, data, size, hipMemcpyDeviceToHost, stream));
}

template <typename T> void DeviceBuffer<T>::copy_from_device_async(const T *device_buffer, hipStream_t stream) const {
    gpuErrchk(hipMemcpyAsync(data, device_buffer, size, hipMemcpyDeviceToDevice, stream));
}

template <typename T> void DeviceBuffer<T>::copy_to_device_async(T *device_buffer, hipStream_t stream) const {
    gpuErrchk(hipMemcpyAsync(device_buffer, data, size, hipMemcpyDeviceToDevice, stream));
}

template class DeviceBuffer<char>;
template class DeviceBuffer<int>;
template class DeviceBuffer<unsigned int>;
template class DeviceBuffer<unsigned long long>;
template class DeviceBuffer<double>;
} // namespace timemachine
