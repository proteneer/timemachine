#include "device_buffer.hpp"
#include "gpu_utils.cuh"
#include <cstddef>

namespace timemachine {

template <typename T> T *allocate(const std::size_t size) {
    T *buffer;
    gpuErrchk(hipMalloc(&buffer, size));
    return buffer;
}

template <typename T>
DeviceBuffer<T>::DeviceBuffer(const std::size_t length) : size(length * sizeof(T)), data(allocate<T>(size)) {}

template <typename T> DeviceBuffer<T>::~DeviceBuffer() { gpuErrchk(hipFree(data)); }

template class DeviceBuffer<double>;
template class DeviceBuffer<unsigned long long>;
} // namespace timemachine
