#include "device_buffer.hpp"
#include "gpu_utils.cuh"
#include <cstddef>

namespace timemachine {

template <typename T> T *allocate(const std::size_t length) {
    T *buffer;
    gpuErrchk(hipMalloc(&buffer, length * sizeof(T)));
    return buffer;
}

template <typename T>
DeviceBuffer<T>::DeviceBuffer(const std::size_t length) : size(length * sizeof(T)), data(allocate<T>(length)) {}

template <typename T> DeviceBuffer<T>::~DeviceBuffer() {
    // TODO: the file/line context reported by gpuErrchk on failure is
    // not very useful when it's called from here. Is there a way to
    // report a stack trace?
    gpuErrchk(hipFree(data));
}

template <typename T> void DeviceBuffer<T>::copy_from(const T *host_buffer) const {
    gpuErrchk(hipMemcpy(data, host_buffer, size, hipMemcpyHostToDevice));
}

template <typename T> void DeviceBuffer<T>::copy_to(T *host_buffer) const {
    gpuErrchk(hipMemcpy(host_buffer, data, size, hipMemcpyDeviceToHost));
}

template class DeviceBuffer<double>;
template class DeviceBuffer<unsigned long long>;
} // namespace timemachine
