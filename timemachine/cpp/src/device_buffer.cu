#include "device_buffer.hpp"
#include "gpu_utils.cuh"
#include <cstddef>

namespace timemachine {

template <typename T>
DeviceBuffer<T>::DeviceBuffer(const std::size_t length) : size(length * sizeof(T)), data(allocate_(size)) {}

template <typename T> DeviceBuffer<T>::~DeviceBuffer() { gpuErrchk(hipFree(data)); }

template <typename T> T *DeviceBuffer<T>::allocate_(const std::size_t size) {
    T *buffer;
    gpuErrchk(hipMalloc(&buffer, size));
    return buffer;
}

template class DeviceBuffer<double>;
template class DeviceBuffer<unsigned long long>;
} // namespace timemachine
