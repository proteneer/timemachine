#include "context.hpp"
#include "gpu_utils.cuh"
#include "fixed_point.hpp"
#include <iostream>
#include <chrono>
#include <hipcub/hipcub.hpp>

namespace timemachine {



Context::Context(
    int N,
    const double *x_0,
    const double *v_0,
    const double *box_0,
    Integrator* intg,
    std::vector<BoundPotential *> bps,
    MonteCarloBarostat* barostat) :
    N_(N),
    intg_(intg),
    bps_(bps),
    step_(0),
    d_sum_storage_(nullptr),
    d_sum_storage_bytes_(0),
    barostat_(barostat) {

    d_x_t_ = gpuErrchkCudaMallocAndCopy(x_0, N*3);
    d_v_t_ = gpuErrchkCudaMallocAndCopy(v_0, N*3);
    d_box_t_ = gpuErrchkCudaMallocAndCopy(box_0, 3*3);

    gpuErrchk(hipMalloc(&d_du_dx_t_, N*3*sizeof(*d_du_dx_t_)));
    gpuErrchk(hipMalloc(&d_du_dl_buffer_, N*sizeof(*d_du_dl_buffer_)));

    unsigned long long *d_in_tmp_ = nullptr; // dummy
    unsigned long long *d_out_tmp_ = nullptr; // dummy

    hipcub::DeviceReduce::Sum(d_sum_storage_, d_sum_storage_bytes_, d_in_tmp_, d_out_tmp_, N_);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMalloc(&d_sum_storage_, d_sum_storage_bytes_));

    // for(int i=0; i < bps.size(); i++) {
        // hipStream_t stream;
        // gpuErrchk(hipStreamCreate(&stream));
        // streams_.push_back(stream);
    // }


};

Context::~Context() {
    gpuErrchk(hipFree(d_x_t_));
    gpuErrchk(hipFree(d_v_t_));
    gpuErrchk(hipFree(d_box_t_));
    gpuErrchk(hipFree(d_du_dx_t_));
    gpuErrchk(hipFree(d_du_dl_buffer_));
    gpuErrchk(hipFree(d_sum_storage_));

    // for(int i=0; i < streams_.size(); i++) {
        // gpuErrchk(hipStreamDestroy(streams_[i]));
    // }
};

void Context::add_observable(Observable *obs) {
    this->observables_.push_back(obs);
}

std::array<std::vector<double>, 2> Context::multiple_steps(
    const std::vector<double> &lambda_schedule,
    int store_du_dl_interval,
    int store_x_interval) {
    unsigned long long *d_du_dl_buffer = nullptr;
    // try catch block is to deal with leaks in d_du_dl_buffer
    if(store_du_dl_interval <= 0) {
        throw std::runtime_error("store_du_dl_interval <= 0");
    }
    if(store_x_interval <= 0) {
        throw std::runtime_error("store_x_interval <= 0");
    }
    int du_dl_buffer_size = (lambda_schedule.size() + store_du_dl_interval - 1) / store_du_dl_interval;
    int x_buffer_size = (lambda_schedule.size() + store_x_interval - 1) / store_x_interval;

    std::vector<double> h_x_buffer(x_buffer_size*N_*3);

    try {
        // indicator so we can set it to a default arg.
        gpuErrchk(hipMalloc(&d_du_dl_buffer, du_dl_buffer_size*sizeof(*d_du_dl_buffer)));
        gpuErrchk(hipMemset(d_du_dl_buffer, 0, du_dl_buffer_size*sizeof(*d_du_dl_buffer)));

        for(int i=0; i < lambda_schedule.size(); i++) {
            // decide if we need to store the du_dl for this step
            unsigned long long *du_dl_ptr = nullptr;
            if(i % store_du_dl_interval == 0) {
                // pemdas but just to make it clear we're doing pointer arithmetic
                du_dl_ptr = d_du_dl_buffer + (i / store_du_dl_interval);
            }

            if(i % store_x_interval == 0) {
                gpuErrchk(hipMemcpy(
                    &h_x_buffer[0] + (i / store_x_interval)*N_*3,
                    d_x_t_,
                    N_*3*sizeof(double),
                    hipMemcpyDeviceToHost)
                );
            }

            double lambda = lambda_schedule[i];
            this->_step(lambda, du_dl_ptr);
        }

        hipDeviceSynchronize();

        std::vector<unsigned long long> h_du_dl_buffer_ull(du_dl_buffer_size);
        gpuErrchk(hipMemcpy(
            &h_du_dl_buffer_ull[0],
            d_du_dl_buffer,
            du_dl_buffer_size*sizeof(*d_du_dl_buffer),
            hipMemcpyDeviceToHost)
        );

        std::vector<double> h_du_dl_buffer_double(du_dl_buffer_size);
        for(int i=0; i < h_du_dl_buffer_ull.size(); i++) {
            h_du_dl_buffer_double[i] = FIXED_TO_FLOAT<double>(h_du_dl_buffer_ull[i]);
        }

        gpuErrchk(hipFree(d_du_dl_buffer));
        return std::array<std::vector<double>, 2>({h_du_dl_buffer_double, h_x_buffer});

    } catch(...) {
        gpuErrchk(hipFree(d_du_dl_buffer));
        throw;
    }

}

void Context::step(double lambda) {
    this->_step(lambda, nullptr);
    hipDeviceSynchronize();
}

void Context::_step(double lambda, unsigned long long *du_dl_out) {

    // the observables decide on whether or not to act on given
    // data (cheap pointers in any case)

    for(int i=0; i < observables_.size(); i++) {
        observables_[i]->observe(
            step_,
            N_,
            d_x_t_,
            d_box_t_,
            lambda
        );
    }

    gpuErrchk(hipMemset(d_du_dx_t_, 0, N_*3*sizeof(*d_du_dx_t_)));

    if(du_dl_out) {
        gpuErrchk(hipMemset(d_du_dl_buffer_, 0, N_*sizeof(*d_du_dl_buffer_)));
    }

    auto start = std::chrono::high_resolution_clock::now();

    for(int i=0; i < bps_.size(); i++) {

        bps_[i]->execute_device(
            N_,
            d_x_t_,
            d_box_t_,
            lambda,
            d_du_dx_t_, // we only need the forces
            nullptr,
            du_dl_out ? d_du_dl_buffer_ : nullptr,
            nullptr,
            static_cast<hipStream_t>(0) // TBD: parallelize me!
            // streams_[i]
        );
    }

    // compute du_dl
    if(du_dl_out) {
        hipcub::DeviceReduce::Sum(
            d_sum_storage_,
            d_sum_storage_bytes_,
            d_du_dl_buffer_,
            du_dl_out,
            N_,
            static_cast<hipStream_t>(0)
        );
        gpuErrchk(hipPeekAtLastError());
    }


    // for(int i=0; i < streams_.size(); i++) {
        // gpuErrchk(hipStreamSynchronize(streams_[i]));
    // }

    intg_->step_fwd(
        d_x_t_,
        d_v_t_,
        d_du_dx_t_,
        d_box_t_
    );

    if(barostat_) {
        // May modify coords and box size
        barostat_->inplace_move(d_x_t_, d_box_t_, lambda);
    }


    step_ += 1;

};


int Context::num_atoms() const {
    return N_;
}

void Context::get_du_dx_t_minus_1(unsigned long long *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_du_dx_t_, N_*3*sizeof(*out_buffer), hipMemcpyDeviceToHost));
}

void Context::get_x_t(double *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_x_t_, N_*3*sizeof(*out_buffer), hipMemcpyDeviceToHost));
}

void Context::get_v_t(double *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_v_t_, N_*3*sizeof(*out_buffer), hipMemcpyDeviceToHost));
}

void Context::get_box(double *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_box_t_, 3*3*sizeof(*out_buffer), hipMemcpyDeviceToHost));
}

}
