#include "context.hpp"
#include "gpu_utils.cuh"
#include <iostream>
#include <chrono>

namespace timemachine {

Context::Context(
    int N,
    const double *x_0,
    const double *v_0,
    const double *box_0,
    // double lambda,
    Integrator* intg,
    std::vector<BoundPotential *> bps) :
    // std::vector<Observable *> obs) : 
    N_(N),
    intg_(intg),
    bps_(bps),
    // observables_(obs),
    // lambda_(lambda),
    step_(0) {

    d_x_t_ = gpuErrchkCudaMallocAndCopy(x_0, N*3);
    d_v_t_ = gpuErrchkCudaMallocAndCopy(v_0, N*3);
    d_box_t_ = gpuErrchkCudaMallocAndCopy(box_0, 3*3);

    gpuErrchk(hipMalloc(&d_du_dx_t_, N*3*sizeof(*d_du_dx_t_)));
    gpuErrchk(hipMalloc(&d_u_t_, 1*sizeof(*d_u_t_)));

    // for(int i=0; i < bps.size(); i++) {
    //     hipStream_t stream;
    //     gpuErrchk(hipStreamCreate(&stream));
    //     streams_.push_back(stream);
    // }


};

Context::~Context() {
    gpuErrchk(hipFree(d_x_t_));
    gpuErrchk(hipFree(d_v_t_));
    gpuErrchk(hipFree(d_box_t_));
    gpuErrchk(hipFree(d_u_t_));
    gpuErrchk(hipFree(d_du_dx_t_));

    // for(int i=0; i < streams_.size(); i++) {
    //     gpuErrchk(hipStreamDestroy(streams_[i]));
    // }
};

void Context::add_observable(Observable *obs) {
    this->observables_.push_back(obs);
}

void Context::step(double lambda) {

    // the observables decide on whether or not to act on given
    // data (cheap pointers in any case)

    for(int i=0; i < observables_.size(); i++) {
        observables_[i]->observe(
            step_,
            N_,
            d_x_t_,
            d_box_t_,
            lambda
        );
    }

    gpuErrchk(hipMemset(d_du_dx_t_, 0, N_*3*sizeof(*d_du_dx_t_)));
    gpuErrchk(hipMemset(d_u_t_, 0, 1*sizeof(*d_du_dx_t_)));

    auto start = std::chrono::high_resolution_clock::now();

    for(int i=0; i < bps_.size(); i++) {
        // std::cout << i << std::endl;
        bps_[i]->execute_device(
            N_,
            d_x_t_,
            d_box_t_,
            lambda,
            d_du_dx_t_,
            nullptr,
            nullptr,
            d_u_t_, // energies aren't really needed tbh
            static_cast<hipStream_t>(0) // TBD: parallelize me!
            // streams_[i]
        );
    }

    hipDeviceSynchronize();

    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> elapsed = end - start;
    // std::cout << "Forces time: " << elapsed.count() << "ms\n";

    // for(int i=0; i < streams_.size(); i++) {
        // gpuErrchk(hipStreamSynchronize(streams_[i]));
    // }

    intg_->step_fwd(
        d_x_t_,
        d_v_t_,
        d_du_dx_t_,
        d_box_t_
    );

    hipDeviceSynchronize();

    step_ += 1;

};


int Context::num_atoms() const {
    return N_;
}

double Context::get_u_t() const {
    double u;
    gpuErrchk(hipMemcpy(&u, d_u_t_, 1*sizeof(*d_u_t_), hipMemcpyDeviceToHost));
    return u;
}

void Context::get_du_dx_t(unsigned long long *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_du_dx_t_, N_*3*sizeof(*out_buffer), hipMemcpyDeviceToHost));
}

void Context::get_x_t(double *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_x_t_, N_*3*sizeof(*out_buffer), hipMemcpyDeviceToHost));
}

void Context::get_v_t(double *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_v_t_, N_*3*sizeof(*out_buffer), hipMemcpyDeviceToHost));
}


// void Context::compute(unsigned int flags) {

//     double *u = (flags & ComputeFlags::u) ? d_u_t_ : nullptr;
//     unsigned long long *du_dx = (flags & ComputeFlags::du_dx) ? d_du_dx_t_ : nullptr;
//     double *du_dl = (flags & ComputeFlags::du_dl) ? d_du_dl_t_ : nullptr;

//     for(int i=0; i < potentials_.size(); i++) {

//         DualParams *dp = dual_params_[i];

//         // note that dp->d_du_dp itself may be null if the end-user
//         // does not care about du_dp.
//         double *du_dp = (flags & ComputeFlags::du_dp) ? dp->d_du_dp : nullptr;

//         potentials_[i]->execute_device(
//             N_,
//             dp->size(),
//             d_x_t_,
//             dp->d_p,
//             d_box_t_,
//             lambda_,
//             du_dx,
//             du_dp,
//             du_dl,
//             u,
//             static_cast<hipStream_t>(0)
//         );

//     }

// };


}