#include "constants.hpp"
#include "context.hpp"
#include "fanout_summed_potential.hpp"
#include "fixed_point.hpp"
#include "flat_bottom_bond.hpp"
#include "gpu_utils.cuh"
#include "kernels/k_flat_bottom_bond.cuh"
#include "kernels/k_indices.cuh"
#include "kernels/k_local_md.cuh"
#include "kernels/kernel_utils.cuh"
#include "langevin_integrator.hpp"
#include "neighborlist.hpp"
#include "nonbonded_all_pairs.hpp"
#include "pinned_host_buffer.hpp"
#include "set_utils.hpp"
#include "summed_potential.hpp"
#include <hipcub/hipcub.hpp>
#include <memory>
#include <random>
#include <typeinfo>

namespace timemachine {

Context::Context(
    int N,
    const double *x_0,
    const double *v_0,
    const double *box_0,
    Integrator *intg,
    std::vector<BoundPotential *> bps,
    MonteCarloBarostat *barostat)
    : N_(N), barostat_(barostat), step_(0), d_sum_storage_(nullptr), d_sum_storage_bytes_(0), intg_(intg), bps_(bps) {
    d_x_t_ = gpuErrchkCudaMallocAndCopy(x_0, N * 3);
    d_v_t_ = gpuErrchkCudaMallocAndCopy(v_0, N * 3);
    d_box_t_ = gpuErrchkCudaMallocAndCopy(box_0, 3 * 3);
    cudaSafeMalloc(&d_u_buffer_, N * sizeof(*d_u_buffer_));

    unsigned long long *d_in_tmp = nullptr;  // dummy
    unsigned long long *d_out_tmp = nullptr; // dummy

    // Compute the storage size necessary to reduce energies
    hipcub::DeviceReduce::Sum(d_sum_storage_, d_sum_storage_bytes_, d_in_tmp, d_out_tmp, N_);
    gpuErrchk(hipPeekAtLastError());
    cudaSafeMalloc(&d_sum_storage_, d_sum_storage_bytes_);
};

Context::~Context() {
    gpuErrchk(hipFree(d_x_t_));
    gpuErrchk(hipFree(d_v_t_));
    gpuErrchk(hipFree(d_box_t_));
    gpuErrchk(hipFree(d_u_buffer_));
    gpuErrchk(hipFree(d_sum_storage_));
};

double get_nonbonded_potential_cutoff(std::shared_ptr<Potential> pot) {
    if (std::shared_ptr<NonbondedAllPairs<float>> nb_pot = std::dynamic_pointer_cast<NonbondedAllPairs<float>>(pot);
        nb_pot) {
        return nb_pot->get_cutoff();
    } else if (std::shared_ptr<NonbondedAllPairs<double>> nb_pot =
                   std::dynamic_pointer_cast<NonbondedAllPairs<double>>(pot);
               nb_pot) {
        return nb_pot->get_cutoff();
    } else {
        throw std::runtime_error("Unable to cast potential to NonbondedAllPairs");
    }
}

bool is_nonbonded_potential(std::shared_ptr<Potential> pot) {
    if (std::shared_ptr<NonbondedAllPairs<float>> nb_pot = std::dynamic_pointer_cast<NonbondedAllPairs<float>>(pot);
        nb_pot) {
        return true;
    } else if (std::shared_ptr<NonbondedAllPairs<double>> nb_pot =
                   std::dynamic_pointer_cast<NonbondedAllPairs<double>>(pot);
               nb_pot) {
        return true;
    }
    return false;
}

void set_nonbonded_potential_idxs(
    std::shared_ptr<Potential> pot, const int num_idxs, const unsigned int *d_idxs, const hipStream_t stream) {
    if (std::shared_ptr<NonbondedAllPairs<float>> nb_pot = std::dynamic_pointer_cast<NonbondedAllPairs<float>>(pot);
        nb_pot) {
        nb_pot->set_atom_idxs_device(num_idxs, d_idxs, stream);
    } else if (std::shared_ptr<NonbondedAllPairs<double>> nb_pot =
                   std::dynamic_pointer_cast<NonbondedAllPairs<double>>(pot);
               nb_pot) {
        nb_pot->set_atom_idxs_device(num_idxs, d_idxs, stream);
    } else {
        throw std::runtime_error("unable to cast potential to NonbondedAllPairs");
    }
}

// Recursively flatten the potentials. Important to find specific NonbondedAllPairs potentials for multiple_steps_local which
// can be wrapped in FanoutSummedPotential or SummedPotential objects.
void flatten_potentials(
    std::vector<std::shared_ptr<Potential>> input, std::vector<std::shared_ptr<Potential>> &flattened) {
    for (std::shared_ptr<Potential> pot : input) {
        std::shared_ptr<FanoutSummedPotential> fanned_potential = std::dynamic_pointer_cast<FanoutSummedPotential>(pot);
        if (fanned_potential != nullptr) {
            flatten_potentials(fanned_potential->get_potentials(), flattened);
            continue;
        }
        std::shared_ptr<SummedPotential> summed_potential = std::dynamic_pointer_cast<SummedPotential>(pot);
        if (summed_potential != nullptr) {
            flatten_potentials(summed_potential->get_potentials(), flattened);
            continue;
        }
        flattened.push_back(pot);
    }
}

double Context::_get_temperature() {
    if (LangevinIntegrator *langevin = dynamic_cast<LangevinIntegrator *>(intg_); langevin != nullptr) {
        return langevin->get_temperature();
    } else {
        throw std::runtime_error("integrator must be LangevinIntegrator.");
    }
}

std::array<std::vector<double>, 2> Context::multiple_steps_local(
    const int n_steps,
    const std::vector<int> &local_idxs,
    const int burn_in,
    const int store_x_interval,
    const double radius,
    const double k,
    const int seed) {
    if (store_x_interval <= 0) {
        throw std::runtime_error("store_x_interval <= 0");
    }
    const double temperature = this->_get_temperature();

    const int x_buffer_size = n_steps / store_x_interval;

    const int box_buffer_size = x_buffer_size * 3 * 3;

    std::vector<std::shared_ptr<Potential>> initial_potentials(bps_.size());

    for (BoundPotential *pot : bps_) {
        initial_potentials.push_back(pot->potential);
    }
    std::vector<std::shared_ptr<Potential>> flattened_potentials;
    flatten_potentials(initial_potentials, flattened_potentials);

    std::shared_ptr<Potential> nonbonded_potential;

    // Find the nonbonded potential
    for (std::shared_ptr<Potential> pot : flattened_potentials) {
        if (is_nonbonded_potential(pot)) {
            if (nonbonded_potential) {
                throw std::runtime_error("found multiple NonbondedAllPairs potentials");
            }
            nonbonded_potential = pot;
        }
    }
    if (!nonbonded_potential) {
        throw std::runtime_error("unable to find a NonbondedAllPairs potential");
    }

    std::mt19937 rng;
    rng.seed(seed);
    std::uniform_int_distribution<unsigned int> random_dist(0, local_idxs.size() - 1);

    // Construct neighborlist to find the inner and outer shell
    Neighborlist<float> nblist(N_);

    // Store coordinates in host memory as it can be very large
    std::vector<double> h_x_buffer(x_buffer_size * N_ * 3);
    // Store boxes on GPU as boxes are a constant size and relatively small
    std::unique_ptr<DeviceBuffer<double>> d_box_traj(nullptr);
    if (box_buffer_size > 0) {
        d_box_traj.reset(new DeviceBuffer<double>(box_buffer_size));
    }

    const size_t tpb = warp_size;

    DeviceBuffer<unsigned int> d_shell_idxs_inner(N_);

    DeviceBuffer<unsigned int> d_row_idxs(N_);
    // d_col indices used both for column indices for neighborlist as well as outer shell.
    DeviceBuffer<unsigned int> d_col_idxs(N_);

    // Pinned memory for getting lengths of indice arrays
    PinnedHostBuffer<int> p_num_selected(1);
    DeviceBuffer<int> num_selected_buffer(1);
    LessThan select_op(N_);

    std::size_t temp_storage_bytes = 0;
    hipcub::DevicePartition::If(
        nullptr, temp_storage_bytes, d_shell_idxs_inner.data, d_row_idxs.data, num_selected_buffer.data, N_, select_op);
    // Allocate char as temp_storage_bytes is in raw bytes and the type doesn't matter in practice.
    // Equivalent to DeviceBuffer<int> buf(temp_storage_bytes / sizeof(int))
    DeviceBuffer<char> d_temp_storage_buffer(temp_storage_bytes);

    const double outer_cutoff = get_nonbonded_potential_cutoff(nonbonded_potential);

    DeviceBuffer<int> restraints(N_ * 2);
    DeviceBuffer<double> bond_params(N_ * 3);
    // Ensure that we allocate enough space for all potential bonds
    std::vector<int> default_bonds(2 * N_);
    for (int i = 0; i < N_; i++) {
        default_bonds[i * 2 + 0] = 0;
        default_bonds[i * 2 + 1] = i + 1;
    }
    std::shared_ptr<FlatBottomBond<double>> restraint_ptr(new FlatBottomBond<double>(default_bonds));
    // Construct a bound potential with 0 params
    BoundPotential bound_shell_restraint(restraint_ptr, std::vector<int>({0}), nullptr);

    // Copy constructor to get new set of bound potentials
    std::vector<BoundPotential *> local_bps = bps_;
    local_bps.push_back(&bound_shell_restraint);

    const double kBT = BOLTZ * temperature;

    hipStream_t stream;
    // Create stream that doesn't sync with the default stream
    gpuErrchk(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    hiprandGenerator_t cr_rng;
    DeviceBuffer<float> probability_buffer(round_up_even(N_));
    curandErrchk(hiprandCreateGenerator(&cr_rng, HIPRAND_RNG_PSEUDO_DEFAULT));
    curandErrchk(hiprandSetPseudoRandomGeneratorSeed(cr_rng, seed));
    curandErrchk(hiprandSetStream(cr_rng, stream));

    try {

        // Set the array to all N, which means it will be ignored as an idx
        k_initialize_array<unsigned int><<<ceil_divide(N_, tpb), tpb, 0, stream>>>(N_, d_shell_idxs_inner.data, N_);
        gpuErrchk(hipPeekAtLastError());
        // Generate values between (0, 1.0]
        curandErrchk(hiprandGenerateUniform(cr_rng, probability_buffer.data, round_up_even(N_)));

        unsigned int reference_idx = local_idxs[random_dist(rng)];

        k_log_probability_selection<double><<<ceil_divide(N_, tpb), tpb, 0, stream>>>(
            N_, kBT, radius, k, reference_idx, d_x_t_, d_box_t_, probability_buffer.data, d_shell_idxs_inner.data);
        gpuErrchk(hipPeekAtLastError());

        // Partition the valid row indices to the front of the array, defines the complete inner shell
        hipcub::DevicePartition::If(
            d_temp_storage_buffer.data,
            temp_storage_bytes,
            d_shell_idxs_inner.data,
            d_row_idxs.data,
            num_selected_buffer.data,
            N_,
            select_op,
            stream);
        gpuErrchk(hipPeekAtLastError());

        // Copy the num out, that is the new num_row_indices, num_col_indices == N_ - num_row_indices
        gpuErrchk(hipMemcpyAsync(
            p_num_selected.data,
            num_selected_buffer.data,
            1 * sizeof(*p_num_selected.data),
            hipMemcpyDeviceToHost,
            stream));
        gpuErrchk(hipStreamSynchronize(stream));

        int num_row_indices = p_num_selected.data[0];
        int num_col_indices = N_ - num_row_indices;
        if (num_row_indices == 0 || num_col_indices == 0) {
            throw std::runtime_error("local md no longer stable, check system");
        }

        k_construct_bonded_params<<<ceil_divide(num_row_indices, tpb), tpb, 0, stream>>>(
            num_row_indices, N_, reference_idx, k, 0.0, radius, d_row_idxs.data, restraints.data, bond_params.data);
        gpuErrchk(hipPeekAtLastError());
        // Setup the flat bottom restraints
        bound_shell_restraint.set_params_device(std::vector<int>({num_row_indices, 3}), bond_params.data, stream);
        restraint_ptr->set_bonds_device(num_row_indices, restraints.data, stream);
        // Invert to get the column indices
        k_invert_indices<<<ceil_divide(N_, tpb), tpb, 0, stream>>>(N_, d_shell_idxs_inner.data);
        gpuErrchk(hipPeekAtLastError());

        // Partition the col idxs to the front
        hipcub::DevicePartition::If(
            d_temp_storage_buffer.data,
            temp_storage_bytes,
            d_shell_idxs_inner.data,
            d_col_idxs.data,
            num_selected_buffer.data,
            N_,
            select_op,
            stream);
        gpuErrchk(hipPeekAtLastError());

        // Invert to get back to the inner shell idxs
        k_invert_indices<<<ceil_divide(N_, tpb), tpb, 0, stream>>>(N_, d_shell_idxs_inner.data);
        gpuErrchk(hipPeekAtLastError());

        nblist.set_idxs_device(num_col_indices, num_row_indices, d_col_idxs.data, d_row_idxs.data, stream);
        int max_interactions = nblist.max_ixn_count();
        // Build the neighborlist around the inner idxs to get the outer shell. Use the nonbonded potential's cutoff
        // to ensure correctness and to avoid wasted computation (ie radius >> outer_cutoff).
        nblist.build_nblist_device(N_, d_x_t_, d_box_t_, outer_cutoff, stream);

        // Now reuse the d_col_idxs for the outer idxs, to reduce memory consumption
        // Set the array to all N, which means it will be ignored as an idx
        k_initialize_array<unsigned int><<<ceil_divide(N_, tpb), tpb, 0, stream>>>(N_, d_col_idxs.data, N_);
        gpuErrchk(hipPeekAtLastError());

        k_unique_indices<<<ceil_divide(max_interactions, tpb), tpb, 0, stream>>>(
            max_interactions, N_, nblist.get_ixn_atoms(), d_col_idxs.data);
        gpuErrchk(hipPeekAtLastError());

        // Add the inner indices to the outer indices
        k_unique_indices<<<ceil_divide(num_row_indices, tpb), tpb, 0, stream>>>(
            num_row_indices, N_, d_row_idxs.data, d_col_idxs.data);
        gpuErrchk(hipPeekAtLastError());

        // Partition all the indices that make up the the inner and outer shell, reuse the d_row_idxs
        hipcub::DevicePartition::If(
            d_temp_storage_buffer.data,
            temp_storage_bytes,
            d_col_idxs.data,
            d_row_idxs.data,
            num_selected_buffer.data,
            N_,
            select_op,
            stream);
        gpuErrchk(hipPeekAtLastError());

        // Copy out the number of indices in the outer indices to the row indices
        gpuErrchk(hipMemcpyAsync(
            p_num_selected.data,
            num_selected_buffer.data,
            1 * sizeof(*p_num_selected.data),
            hipMemcpyDeviceToHost,
            stream));
        gpuErrchk(hipStreamSynchronize(stream));

        // Set the nonbonded potential to compute forces of inner+outer shell.
        set_nonbonded_potential_idxs(nonbonded_potential, p_num_selected.data[0], d_row_idxs.data, stream);
        for (int i = 0; i < burn_in; i++) {
            this->_step(bps_, d_shell_idxs_inner.data, stream);
        }
        for (int i = 1; i <= n_steps; i++) {
            this->_step(bps_, d_shell_idxs_inner.data, stream);
            if (i % store_x_interval == 0) {
                gpuErrchk(hipMemcpyAsync(
                    &h_x_buffer[0] + ((i / store_x_interval) - 1) * N_ * 3,
                    d_x_t_,
                    N_ * 3 * sizeof(double),
                    hipMemcpyDeviceToHost,
                    stream));
                gpuErrchk(hipMemcpyAsync(
                    &d_box_traj->data[0] + ((i / store_x_interval) - 1) * 3 * 3,
                    d_box_t_,
                    3 * 3 * sizeof(*d_box_traj->data),
                    hipMemcpyDeviceToDevice,
                    stream));
            }
        }
        // Set the row indices back to the identity.
        k_arange<<<ceil_divide(N_, tpb), tpb, 0, stream>>>(N_, d_row_idxs.data);
        gpuErrchk(hipPeekAtLastError());
        // Set back to the full system, for when the loop ends
        set_nonbonded_potential_idxs(nonbonded_potential, N_, d_row_idxs.data, stream);
    } catch (...) {
        gpuErrchk(hipStreamSynchronize(stream));
        gpuErrchk(hipStreamDestroy(stream));
        curandErrchk(hiprandDestroyGenerator(cr_rng));
        throw;
    }
    gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk(hipStreamDestroy(stream));
    curandErrchk(hiprandDestroyGenerator(cr_rng));

    std::vector<double> h_box_buffer(box_buffer_size);

    if (box_buffer_size > 0) {
        d_box_traj->copy_to(&h_box_buffer[0]);
    }
    return std::array<std::vector<double>, 2>({h_x_buffer, h_box_buffer});
}

std::array<std::vector<double>, 2> Context::multiple_steps(const int n_steps, int store_x_interval) {
    if (store_x_interval <= 0) {
        throw std::runtime_error("store_x_interval <= 0");
    }
    if (n_steps % store_x_interval != 0) {
        std::cout << "warning:: n_steps modulo store_x_interval does not equal zero" << std::endl;
    }

    int x_buffer_size = n_steps / store_x_interval;
    int box_buffer_size = x_buffer_size * 3 * 3;

    std::vector<double> h_x_buffer(x_buffer_size * N_ * 3);

    hipStream_t stream = static_cast<hipStream_t>(0);

    std::unique_ptr<DeviceBuffer<double>> d_box_buffer(nullptr);
    if (box_buffer_size > 0) {
        d_box_buffer.reset(new DeviceBuffer<double>(box_buffer_size));
    }

    intg_->initialize(bps_, d_x_t_, d_v_t_, d_box_t_, nullptr, stream);
    for (int i = 1; i <= n_steps; i++) {
        this->_step(bps_, nullptr, stream);

        if (i % store_x_interval == 0) {
            gpuErrchk(hipMemcpyAsync(
                &h_x_buffer[0] + ((i / store_x_interval) - 1) * N_ * 3,
                d_x_t_,
                N_ * 3 * sizeof(double),
                hipMemcpyDeviceToHost,
                stream));
            gpuErrchk(hipMemcpyAsync(
                &d_box_buffer->data[0] + ((i / store_x_interval) - 1) * 3 * 3,
                d_box_t_,
                3 * 3 * sizeof(*d_box_buffer->data),
                hipMemcpyDeviceToDevice,
                stream));
        }
    }
    intg_->finalize(bps_, d_x_t_, d_v_t_, d_box_t_, nullptr, stream);

    gpuErrchk(hipStreamSynchronize(stream));

    std::vector<double> h_box_buffer(box_buffer_size);
    if (box_buffer_size > 0) {
        d_box_buffer->copy_to(&h_box_buffer[0]);
    }

    return std::array<std::vector<double>, 2>({h_x_buffer, h_box_buffer});
}

std::array<std::vector<double>, 3>
Context::multiple_steps_U(const int n_steps, int store_u_interval, int store_x_interval) {

    if (store_u_interval <= 0) {
        throw std::runtime_error("store_u_interval <= 0");
    }

    if (store_x_interval <= 0) {
        throw std::runtime_error("store_x_interval <= 0");
    }

    if (n_steps % store_x_interval != 0) {
        std::cout << "warning:: n_steps modulo store_x_interval does not equal zero" << std::endl;
    }

    if (n_steps % store_u_interval != 0) {
        std::cout << "warning:: n_steps modulo store_u_interval does not equal zero" << std::endl;
    }

    int u_traj_size = n_steps / store_u_interval;
    int x_traj_size = n_steps / store_x_interval;

    std::vector<double> h_x_traj(x_traj_size * N_ * 3);

    hipStream_t stream = static_cast<hipStream_t>(0);

    std::unique_ptr<DeviceBuffer<double>> d_box_traj(nullptr);
    if (x_traj_size > 0) {
        d_box_traj.reset(new DeviceBuffer<double>(x_traj_size * 3 * 3));
    }
    std::unique_ptr<DeviceBuffer<unsigned long long>> d_u_traj(nullptr);
    if (u_traj_size > 0) {
        d_u_traj.reset(new DeviceBuffer<unsigned long long>(u_traj_size));
        gpuErrchk(hipMemsetAsync(d_u_traj->data, 0, d_u_traj->size, stream));
    }

    intg_->initialize(bps_, d_x_t_, d_v_t_, d_box_t_, nullptr, stream);
    for (int step = 1; step <= n_steps; step++) {

        this->_step(bps_, nullptr, stream);

        if (step % store_x_interval == 0) {
            gpuErrchk(hipMemcpyAsync(
                &h_x_traj[0] + ((step / store_x_interval) - 1) * N_ * 3,
                d_x_t_,
                N_ * 3 * sizeof(double),
                hipMemcpyDeviceToHost,
                stream));
            gpuErrchk(hipMemcpyAsync(
                &d_box_traj->data[0] + ((step / store_x_interval) - 1) * 3 * 3,
                d_box_t_,
                3 * 3 * sizeof(*d_box_traj->data),
                hipMemcpyDeviceToDevice,
                stream));
        }

        // we need to compute aggregate energies
        if (u_traj_size > 0 && step % store_u_interval == 0) {
            // reset buffers on each pass.
            gpuErrchk(hipMemsetAsync(d_u_buffer_, 0, N_ * sizeof(*d_u_buffer_), stream));
            unsigned long long *u_ptr = d_u_traj->data + (step / store_u_interval) - 1;
            for (int i = 0; i < bps_.size(); i++) {
                bps_[i]->execute_device(N_, d_x_t_, d_box_t_, nullptr, nullptr, d_u_buffer_, stream);
            }
            hipcub::DeviceReduce::Sum(d_sum_storage_, d_sum_storage_bytes_, d_u_buffer_, u_ptr, N_, stream);
            gpuErrchk(hipPeekAtLastError());
        }
    }
    intg_->finalize(bps_, d_x_t_, d_v_t_, d_box_t_, nullptr, stream);

    gpuErrchk(hipStreamSynchronize(stream));

    std::vector<unsigned long long> h_u_traj_ull(u_traj_size);
    if (u_traj_size > 0) {
        d_u_traj->copy_to(&h_u_traj_ull[0]);
    }

    std::vector<double> h_u_traj_double(u_traj_size);
    for (int i = 0; i < h_u_traj_ull.size(); i++) {
        h_u_traj_double[i] = FIXED_TO_FLOAT<double>(h_u_traj_ull[i]);
    }
    std::vector<double> h_box_traj(x_traj_size * 3 * 3);
    if (x_traj_size > 0) {
        d_box_traj->copy_to(&h_box_traj[0]);
    }

    return std::array<std::vector<double>, 3>({h_u_traj_double, h_x_traj, h_box_traj});
}

void Context::step() {
    hipStream_t stream = static_cast<hipStream_t>(0);
    this->_step(bps_, nullptr, stream);
    gpuErrchk(hipStreamSynchronize(stream));
}

void Context::finalize() {
    hipStream_t stream = static_cast<hipStream_t>(0);
    intg_->finalize(bps_, d_x_t_, d_v_t_, d_box_t_, nullptr, stream);
    gpuErrchk(hipStreamSynchronize(stream));
}

void Context::initialize() {
    hipStream_t stream = static_cast<hipStream_t>(0);
    intg_->initialize(bps_, d_x_t_, d_v_t_, d_box_t_, nullptr, stream);
    gpuErrchk(hipStreamSynchronize(stream));
}

void Context::_step(std::vector<BoundPotential *> &bps, unsigned int *d_atom_idxs, const hipStream_t stream) {
    intg_->step_fwd(bps, d_x_t_, d_v_t_, d_box_t_, d_atom_idxs, stream);

    // If atom idxs are passed, indicates that only a subset of the system should move. Don't
    // run the barostat in this situation.
    if (d_atom_idxs == nullptr && barostat_) {
        // May modify coords, du_dx and box size
        barostat_->inplace_move(d_x_t_, d_box_t_, stream);
    }

    step_ += 1;
};

int Context::num_atoms() const { return N_; }

void Context::set_x_t(const double *in_buffer) {
    gpuErrchk(hipMemcpy(d_x_t_, in_buffer, N_ * 3 * sizeof(*in_buffer), hipMemcpyHostToDevice));
}

void Context::set_v_t(const double *in_buffer) {
    gpuErrchk(hipMemcpy(d_v_t_, in_buffer, N_ * 3 * sizeof(*in_buffer), hipMemcpyHostToDevice));
}

void Context::get_x_t(double *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_x_t_, N_ * 3 * sizeof(*out_buffer), hipMemcpyDeviceToHost));
}

void Context::get_v_t(double *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_v_t_, N_ * 3 * sizeof(*out_buffer), hipMemcpyDeviceToHost));
}

void Context::get_box(double *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_box_t_, 3 * 3 * sizeof(*out_buffer), hipMemcpyDeviceToHost));
}

} // namespace timemachine
