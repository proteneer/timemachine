#include <iostream>
#include "context.hpp"
#include "gpu_utils.cuh"

namespace timemachine {

template<typename RealType>
Context<RealType>::Context(
    const std::vector<Potential<RealType>* > system,
    const Optimizer<RealType> *optimizer,
    const RealType *h_params,
    const RealType *h_x0,
    const RealType *h_v0,
    const int N,
    const int D,
    const int P,
    const int *h_gather_param_idxs,
    const int DP) : system_(system),
    optimizer_(optimizer),
    step_(0),
    N_(N),
    P_(P),
    D_(D),
    DP_(DP) {
    // if DP == 0 then this is null

    // 1. allocate
    gpuErrchk(hipMalloc((void**)&d_params_, P*sizeof(RealType)));
    gpuErrchk(hipMalloc((void**)&d_gather_param_idxs_, P*sizeof(int)));
    gpuErrchk(hipMalloc((void**)&d_x_t_, N*D*sizeof(RealType)));
    gpuErrchk(hipMalloc((void**)&d_v_t_, N*D*sizeof(RealType)));

    gpuErrchk(hipMalloc((void**)&d_E_, sizeof(RealType)));
    gpuErrchk(hipMalloc((void**)&d_dE_dx_, N*D*sizeof(RealType)));
    gpuErrchk(hipMalloc((void**)&d_dE_dp_, DP*sizeof(RealType)));
    gpuErrchk(hipMalloc((void**)&d_d2E_dx2_, N*N*D*D*sizeof(RealType)));
    gpuErrchk(hipMalloc((void**)&d_d2E_dxdp_, DP*N*D*sizeof(RealType)));
    gpuErrchk(hipMalloc((void**)&d_dx_dp_t_, DP*N*D*sizeof(RealType)));
    gpuErrchk(hipMalloc((void**)&d_dv_dp_t_, DP*N*D*sizeof(RealType)));

    // 2. memcpy and memset to initialize
    gpuErrchk(hipMemcpy(d_params_, h_params, P*sizeof(RealType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_gather_param_idxs_, h_gather_param_idxs, P*sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_x_t_, h_x0, N*D*sizeof(RealType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_v_t_, h_v0, N*D*sizeof(RealType), hipMemcpyHostToDevice));

    gpuErrchk(hipMemset(d_dx_dp_t_, 0, DP*N*D*sizeof(RealType)));
    gpuErrchk(hipMemset(d_dv_dp_t_, 0, DP*N*D*sizeof(RealType)));

}

template<typename RealType>
Context<RealType>::~Context() {
    gpuErrchk(hipFree(d_params_));
    gpuErrchk(hipFree(d_gather_param_idxs_));
    gpuErrchk(hipFree(d_x_t_));
    gpuErrchk(hipFree(d_v_t_));

    gpuErrchk(hipFree(d_E_));
    gpuErrchk(hipFree(d_dE_dx_));
    gpuErrchk(hipFree(d_dE_dp_));
    gpuErrchk(hipFree(d_d2E_dx2_));
    gpuErrchk(hipFree(d_d2E_dxdp_));
    gpuErrchk(hipFree(d_dx_dp_t_));
    gpuErrchk(hipFree(d_dv_dp_t_));
}

template<typename RealType>
void Context<RealType>::step() {

    // reset force buffers
    gpuErrchk(hipMemset(d_E_, 0, sizeof(RealType)));
    gpuErrchk(hipMemset(d_dE_dx_, 0, N_*D_*sizeof(RealType)));
    gpuErrchk(hipMemset(d_dE_dp_, 0, DP_*sizeof(RealType)));
    gpuErrchk(hipMemset(d_d2E_dx2_, 0, N_*N_*D_*D_*sizeof(RealType)));
    gpuErrchk(hipMemset(d_d2E_dxdp_, 0, DP_*N_*D_*sizeof(RealType)));

    for(auto nrg : system_) {
        nrg->derivatives_device(
            1, // one conformer when doing dynamics
            N_,
            D_, // FIX ME
            d_x_t_,
            d_params_,
            d_E_, // this is likely optional as well
            d_dE_dx_,
            d_d2E_dx2_,
            DP_,
            d_gather_param_idxs_,
            d_dE_dp_, // (ytz) we don't actually need to compute dE_dp, so can probably speed this up later
            d_d2E_dxdp_
        );
    }

    optimizer_->step(
        N_,
        D_,
        DP_,
        d_dE_dx_,
        d_d2E_dx2_,
        d_d2E_dxdp_,
        d_x_t_,
        d_v_t_,
        d_dx_dp_t_,
        d_dv_dp_t_
    );
    step_++;

}

template<typename RealType>
void Context<RealType>::debug_compute_dE_dx(
        const RealType *h_new_x,
        RealType *h_E,
        RealType *h_dE_dx) {

    gpuErrchk(hipMemset(d_E_, 0, sizeof(RealType)));
    gpuErrchk(hipMemset(d_dE_dx_, 0, N_*3*sizeof(RealType)));

    gpuErrchk(hipMemcpy(d_x_t_, h_new_x, N_*3*sizeof(RealType), hipMemcpyHostToDevice));
    
    for(auto nrg : system_) {
        nrg->derivatives_device(
            1, // one conformer when doing dynamics
            N_,
            d_x_t_,
            d_params_,
            d_E_, // this is likely optional as well
            d_dE_dx_,
            d_d2E_dx2_,
            DP_,
            d_gather_param_idxs_,
            d_dE_dp_, // (ytz) we don't actually need to compute dE_dp, so can probably speed this up later
            d_d2E_dxdp_
        );
    }

    gpuErrchk(hipMemcpy(h_dE_dx, d_dE_dx_, N_*3*sizeof(RealType), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_E, d_E_, sizeof(RealType), hipMemcpyDeviceToHost));


}

template<typename RealType>
void Context<RealType>::get_x(RealType *buffer) const {
    gpuErrchk(hipMemcpy(buffer, d_x_t_, N_*D_*sizeof(RealType), hipMemcpyDeviceToHost));
}

template<typename RealType>
void Context<RealType>::get_v(RealType *buffer) const {
    gpuErrchk(hipMemcpy(buffer, d_v_t_, N_*D_*sizeof(RealType), hipMemcpyDeviceToHost));
}

template<typename RealType>
void Context<RealType>::get_E(RealType *buffer) const {
    gpuErrchk(hipMemcpy(buffer, d_E_, sizeof(RealType), hipMemcpyDeviceToHost));
}

template<typename RealType>
void Context<RealType>::get_dE_dx(RealType *buffer) const {
    gpuErrchk(hipMemcpy(buffer, d_dE_dx_, N_*D_*sizeof(RealType), hipMemcpyDeviceToHost));
}

template<typename RealType>
void Context<RealType>::get_dE_dp(RealType *buffer) const {
    gpuErrchk(hipMemcpy(buffer, d_dE_dp_, DP_*sizeof(RealType), hipMemcpyDeviceToHost));
}

template<typename RealType>
void Context<RealType>::get_dx_dp(RealType *buffer) const {
    gpuErrchk(hipMemcpy(buffer, d_dx_dp_t_, DP_*N_*D_*sizeof(RealType), hipMemcpyDeviceToHost));
}

template<typename RealType>
void Context<RealType>::get_dv_dp(RealType *buffer) const {
    gpuErrchk(hipMemcpy(buffer, d_dv_dp_t_, DP_*N_*D_*sizeof(RealType), hipMemcpyDeviceToHost));
}

template class Context<float>;
template class Context<double>;

}