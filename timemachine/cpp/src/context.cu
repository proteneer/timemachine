#include "context.hpp"
#include "gpu_utils.cuh"
#include "fixed_point.hpp"
#include <iostream>
#include <chrono>
#include <hipcub/hipcub.hpp>

namespace timemachine {

Context::Context(
    int N,
    const double *x_0,
    const double *v_0,
    const double *box_0,
    Integrator* intg,
    std::vector<BoundPotential *> bps) :
    N_(N),
    intg_(intg),
    bps_(bps),
    step_(0),
    d_sum_storage_(nullptr),
    d_sum_storage_bytes_(0) {

    d_x_t_ = gpuErrchkCudaMallocAndCopy(x_0, N*3);
    d_v_t_ = gpuErrchkCudaMallocAndCopy(v_0, N*3);
    d_box_t_ = gpuErrchkCudaMallocAndCopy(box_0, 3*3);

    gpuErrchk(hipMalloc(&d_du_dx_t_, N*3*sizeof(*d_du_dx_t_)));
    gpuErrchk(hipMalloc(&d_du_dl_buffer_, N*sizeof(*d_du_dl_buffer_)));

    unsigned long long *d_in_tmp_ = nullptr; // dummy
    unsigned long long *d_out_tmp_ = nullptr; // dummy

    hipcub::DeviceReduce::Sum(d_sum_storage_, d_sum_storage_bytes_, d_in_tmp_, d_out_tmp_, N_);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMalloc(&d_sum_storage_, d_sum_storage_bytes_));

    // for(int i=0; i < bps.size(); i++) {
        // hipStream_t stream;
        // gpuErrchk(hipStreamCreate(&stream));
        // streams_.push_back(stream);
    // }


};

Context::~Context() {
    gpuErrchk(hipFree(d_x_t_));
    gpuErrchk(hipFree(d_v_t_));
    gpuErrchk(hipFree(d_box_t_));
    gpuErrchk(hipFree(d_du_dx_t_));
    gpuErrchk(hipFree(d_du_dl_buffer_));
    gpuErrchk(hipFree(d_sum_storage_));

    // for(int i=0; i < streams_.size(); i++) {
        // gpuErrchk(hipStreamDestroy(streams_[i]));
    // }
};

void Context::add_observable(Observable *obs) {
    this->observables_.push_back(obs);
}

std::vector<double> Context::multiple_steps(
    const std::vector<double> &lambda_schedule,
    int store_du_dl_freq) {
    unsigned long long *d_du_dl_buffer = nullptr;
    // try catch block is to deal with leaks in d_du_dl_buffer
    if(store_du_dl_freq==0) {
        store_du_dl_freq = lambda_schedule.size();
    }
    int buffer_size = (lambda_schedule.size() + store_du_dl_freq - 1) / store_du_dl_freq;
    try {
        // indicator so we can set it to a default arg.
        gpuErrchk(hipMalloc(&d_du_dl_buffer, buffer_size*sizeof(*d_du_dl_buffer)));
        gpuErrchk(hipMemset(d_du_dl_buffer, 0, buffer_size*sizeof(*d_du_dl_buffer)));
        for(int i=0; i < lambda_schedule.size(); i++) {
            // decide if we need to store the du_dl for this step
            unsigned long long *du_dl_ptr = nullptr;
            if(i % store_du_dl_freq == 0) {
                // pemdas but just to make it clear we're doing pointer arithmetic
                du_dl_ptr = d_du_dl_buffer + (i / store_du_dl_freq);
            }
            double lambda = lambda_schedule[i];
            this->_step(lambda, du_dl_ptr);
        }
        hipDeviceSynchronize();

        std::vector<unsigned long long> h_du_dl_buffer_ull(buffer_size);
        gpuErrchk(hipMemcpy(
            &h_du_dl_buffer_ull[0],
            d_du_dl_buffer,
            buffer_size*sizeof(*d_du_dl_buffer),
            hipMemcpyDeviceToHost)
        );

        std::vector<double> h_du_dl_buffer_double(buffer_size);
        for(int i=0; i < h_du_dl_buffer_ull.size(); i++) {
            h_du_dl_buffer_double[i] = FIXED_TO_FLOAT<double>(h_du_dl_buffer_ull[i]);
        }

        gpuErrchk(hipFree(d_du_dl_buffer));
        return h_du_dl_buffer_double;

    } catch(...) {
        gpuErrchk(hipFree(d_du_dl_buffer));
        throw;
    }

}

void Context::step(double lambda) {
    this->_step(lambda, nullptr);
    hipDeviceSynchronize();
}

void Context::_step(double lambda, unsigned long long *du_dl_out) {

    // the observables decide on whether or not to act on given
    // data (cheap pointers in any case)

    for(int i=0; i < observables_.size(); i++) {
        observables_[i]->observe(
            step_,
            N_,
            d_x_t_,
            d_box_t_,
            lambda
        );
    }

    gpuErrchk(hipMemset(d_du_dx_t_, 0, N_*3*sizeof(*d_du_dx_t_)));

    if(du_dl_out) {
        gpuErrchk(hipMemset(d_du_dl_buffer_, 0, N_*sizeof(*d_du_dl_buffer_)));
    }

    auto start = std::chrono::high_resolution_clock::now();

    for(int i=0; i < bps_.size(); i++) {

        bps_[i]->execute_device(
            N_,
            d_x_t_,
            d_box_t_,
            lambda,
            d_du_dx_t_, // we only need the forces
            nullptr,
            du_dl_out ? d_du_dl_buffer_ : nullptr,
            nullptr,
            static_cast<hipStream_t>(0) // TBD: parallelize me!
            // streams_[i]
        );
    }

    // compute du_dl
    if(du_dl_out) {
        hipcub::DeviceReduce::Sum(
            d_sum_storage_,
            d_sum_storage_bytes_,
            d_du_dl_buffer_,
            du_dl_out,
            N_,
            static_cast<hipStream_t>(0)
        );
        gpuErrchk(hipPeekAtLastError());
    }


    // for(int i=0; i < streams_.size(); i++) {
        // gpuErrchk(hipStreamSynchronize(streams_[i]));
    // }

    intg_->step_fwd(
        d_x_t_,
        d_v_t_,
        d_du_dx_t_,
        d_box_t_
    );

    step_ += 1;

};


int Context::num_atoms() const {
    return N_;
}

void Context::get_du_dx_t_minus_1(unsigned long long *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_du_dx_t_, N_*3*sizeof(*out_buffer), hipMemcpyDeviceToHost));
}

void Context::get_x_t(double *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_x_t_, N_*3*sizeof(*out_buffer), hipMemcpyDeviceToHost));
}

void Context::get_v_t(double *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_v_t_, N_*3*sizeof(*out_buffer), hipMemcpyDeviceToHost));
}

}