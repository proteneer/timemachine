#include "context.hpp"
#include "fixed_point.hpp"
#include "gpu_utils.cuh"
#include <chrono>
#include <hipcub/hipcub.hpp>
#include <iostream>

namespace timemachine {

Context::Context(
    int N,
    const double *x_0,
    const double *v_0,
    const double *box_0,
    Integrator *intg,
    std::vector<BoundPotential *> bps,
    MonteCarloBarostat *barostat)
    : N_(N), barostat_(barostat), step_(0), d_sum_storage_(nullptr), d_sum_storage_bytes_(0), intg_(intg), bps_(bps) {
    d_x_t_ = gpuErrchkCudaMallocAndCopy(x_0, N * 3);
    d_v_t_ = gpuErrchkCudaMallocAndCopy(v_0, N * 3);
    d_box_t_ = gpuErrchkCudaMallocAndCopy(box_0, 3 * 3);
    gpuErrchk(hipMalloc(&d_u_buffer_, N * sizeof(*d_u_buffer_)));

    unsigned long long *d_in_tmp = nullptr;  // dummy
    unsigned long long *d_out_tmp = nullptr; // dummy

    // Compute the storage size necessary to reduce energies
    hipcub::DeviceReduce::Sum(d_sum_storage_, d_sum_storage_bytes_, d_in_tmp, d_out_tmp, N_);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMalloc(&d_sum_storage_, d_sum_storage_bytes_));
};

Context::~Context() {
    gpuErrchk(hipFree(d_x_t_));
    gpuErrchk(hipFree(d_v_t_));
    gpuErrchk(hipFree(d_box_t_));
    gpuErrchk(hipFree(d_u_buffer_));
    gpuErrchk(hipFree(d_sum_storage_));
};

std::array<std::vector<double>, 2> Context::multiple_steps(const int n_steps, int store_x_interval) {
    if (store_x_interval <= 0) {
        throw std::runtime_error("store_x_interval <= 0");
    }
    if (n_steps % store_x_interval != 0) {
        std::cout << "warning:: n_steps modulo store_x_interval does not equal zero" << std::endl;
    }

    int x_buffer_size = n_steps / store_x_interval;
    int box_buffer_size = x_buffer_size * 3 * 3;

    std::vector<double> h_x_buffer(x_buffer_size * N_ * 3);

    hipStream_t stream = static_cast<hipStream_t>(0);

    std::unique_ptr<DeviceBuffer<double>> d_box_buffer(nullptr);
    if (box_buffer_size > 0) {
        d_box_buffer.reset(new DeviceBuffer<double>(box_buffer_size));
    }

    intg_->initialize(bps_, d_x_t_, d_v_t_, d_box_t_, stream);
    for (int i = 1; i <= n_steps; i++) {
        this->_step(bps_, stream);

        if (i % store_x_interval == 0) {
            gpuErrchk(hipMemcpyAsync(
                &h_x_buffer[0] + ((i / store_x_interval) - 1) * N_ * 3,
                d_x_t_,
                N_ * 3 * sizeof(double),
                hipMemcpyDeviceToHost,
                stream));
            gpuErrchk(hipMemcpyAsync(
                &d_box_buffer->data[0] + ((i / store_x_interval) - 1) * 3 * 3,
                d_box_t_,
                3 * 3 * sizeof(*d_box_buffer->data),
                hipMemcpyDeviceToDevice,
                stream));
        }
    }
    intg_->finalize(bps_, d_x_t_, d_v_t_, d_box_t_, stream);

    gpuErrchk(hipStreamSynchronize(stream));

    std::vector<double> h_box_buffer(box_buffer_size);
    if (box_buffer_size > 0) {
        d_box_buffer->copy_to(&h_box_buffer[0]);
    }

    return std::array<std::vector<double>, 2>({h_x_buffer, h_box_buffer});
}

std::array<std::vector<double>, 3>
Context::multiple_steps_U(const int n_steps, int store_u_interval, int store_x_interval) {

    if (store_u_interval <= 0) {
        throw std::runtime_error("store_u_interval <= 0");
    }

    if (store_x_interval <= 0) {
        throw std::runtime_error("store_x_interval <= 0");
    }

    if (n_steps % store_x_interval != 0) {
        std::cout << "warning:: n_steps modulo store_x_interval does not equal zero" << std::endl;
    }

    if (n_steps % store_u_interval != 0) {
        std::cout << "warning:: n_steps modulo store_u_interval does not equal zero" << std::endl;
    }

    int u_traj_size = n_steps / store_u_interval;
    int x_traj_size = n_steps / store_x_interval;

    std::vector<double> h_x_traj(x_traj_size * N_ * 3);

    hipStream_t stream = static_cast<hipStream_t>(0);

    std::unique_ptr<DeviceBuffer<double>> d_box_traj(nullptr);
    if (x_traj_size > 0) {
        d_box_traj.reset(new DeviceBuffer<double>(x_traj_size * 3 * 3));
    }
    std::unique_ptr<DeviceBuffer<unsigned long long>> d_u_traj(nullptr);
    if (u_traj_size > 0) {
        d_u_traj.reset(new DeviceBuffer<unsigned long long>(u_traj_size));
        gpuErrchk(hipMemsetAsync(d_u_traj->data, 0, d_u_traj->size, stream));
    }

    intg_->initialize(bps_, d_x_t_, d_v_t_, d_box_t_, stream);
    for (int step = 1; step <= n_steps; step++) {

        this->_step(bps_, stream);

        if (step % store_x_interval == 0) {
            gpuErrchk(hipMemcpyAsync(
                &h_x_traj[0] + ((step / store_x_interval) - 1) * N_ * 3,
                d_x_t_,
                N_ * 3 * sizeof(double),
                hipMemcpyDeviceToHost,
                stream));
            gpuErrchk(hipMemcpyAsync(
                &d_box_traj->data[0] + ((step / store_x_interval) - 1) * 3 * 3,
                d_box_t_,
                3 * 3 * sizeof(*d_box_traj->data),
                hipMemcpyDeviceToDevice,
                stream));
        }

        // we need to compute aggregate energies
        if (u_traj_size > 0 && step % store_u_interval == 0) {
            // reset buffers on each pass.
            gpuErrchk(hipMemsetAsync(d_u_buffer_, 0, N_ * sizeof(*d_u_buffer_), stream));
            unsigned long long *u_ptr = d_u_traj->data + (step / store_u_interval) - 1;
            for (int i = 0; i < bps_.size(); i++) {
                bps_[i]->execute_device(N_, d_x_t_, d_box_t_, nullptr, nullptr, d_u_buffer_, stream);
            }
            hipcub::DeviceReduce::Sum(d_sum_storage_, d_sum_storage_bytes_, d_u_buffer_, u_ptr, N_, stream);
            gpuErrchk(hipPeekAtLastError());
        }
    }
    intg_->finalize(bps_, d_x_t_, d_v_t_, d_box_t_, stream);

    gpuErrchk(hipStreamSynchronize(stream));

    std::vector<unsigned long long> h_u_traj_ull(u_traj_size);
    if (u_traj_size > 0) {
        d_u_traj->copy_to(&h_u_traj_ull[0]);
    }

    std::vector<double> h_u_traj_double(u_traj_size);
    for (int i = 0; i < h_u_traj_ull.size(); i++) {
        h_u_traj_double[i] = FIXED_TO_FLOAT<double>(h_u_traj_ull[i]);
    }
    std::vector<double> h_box_traj(x_traj_size * 3 * 3);
    if (x_traj_size > 0) {
        d_box_traj->copy_to(&h_box_traj[0]);
    }

    return std::array<std::vector<double>, 3>({h_u_traj_double, h_x_traj, h_box_traj});
}

void Context::step() {
    hipStream_t stream = static_cast<hipStream_t>(0);
    this->_step(bps_, stream);
    gpuErrchk(hipDeviceSynchronize());
}

void Context::finalize() {
    hipStream_t stream = static_cast<hipStream_t>(0);
    intg_->finalize(bps_, d_x_t_, d_v_t_, d_box_t_, stream);
    gpuErrchk(hipStreamSynchronize(stream));
}

void Context::initialize() {
    hipStream_t stream = static_cast<hipStream_t>(0);
    intg_->initialize(bps_, d_x_t_, d_v_t_, d_box_t_, stream);
    gpuErrchk(hipStreamSynchronize(stream));
}

void Context::_step(std::vector<BoundPotential *> &bps, const hipStream_t stream) {

    intg_->step_fwd(bps, d_x_t_, d_v_t_, d_box_t_, stream);

    if (barostat_) {
        // May modify coords, du_dx and box size
        barostat_->inplace_move(d_x_t_, d_box_t_, stream);
    }

    step_ += 1;
};

int Context::num_atoms() const { return N_; }

void Context::set_x_t(const double *in_buffer) {
    gpuErrchk(hipMemcpy(d_x_t_, in_buffer, N_ * 3 * sizeof(*in_buffer), hipMemcpyHostToDevice));
}

void Context::set_v_t(const double *in_buffer) {
    gpuErrchk(hipMemcpy(d_v_t_, in_buffer, N_ * 3 * sizeof(*in_buffer), hipMemcpyHostToDevice));
}

void Context::get_x_t(double *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_x_t_, N_ * 3 * sizeof(*out_buffer), hipMemcpyDeviceToHost));
}

void Context::get_v_t(double *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_v_t_, N_ * 3 * sizeof(*out_buffer), hipMemcpyDeviceToHost));
}

void Context::get_box(double *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_box_t_, 3 * 3 * sizeof(*out_buffer), hipMemcpyDeviceToHost));
}

} // namespace timemachine
