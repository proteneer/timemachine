#include "context.hpp"
#include "fixed_point.hpp"
#include "gpu_utils.cuh"
#include <chrono>
#include <hipcub/hipcub.hpp>
#include <iostream>

namespace timemachine {

Context::Context(
    int N,
    const double *x_0,
    const double *v_0,
    const double *box_0,
    Integrator *intg,
    std::vector<BoundPotential *> bps,
    MonteCarloBarostat *barostat)
    : N_(N), barostat_(barostat), step_(0), d_sum_storage_(nullptr), d_sum_storage_bytes_(0), intg_(intg), bps_(bps) {
    d_x_t_ = gpuErrchkCudaMallocAndCopy(x_0, N * 3);
    d_v_t_ = gpuErrchkCudaMallocAndCopy(v_0, N * 3);
    d_box_t_ = gpuErrchkCudaMallocAndCopy(box_0, 3 * 3);
    gpuErrchk(hipMalloc(&d_du_dl_buffer_, N * sizeof(*d_du_dl_buffer_)));
    gpuErrchk(hipMalloc(&d_u_buffer_, N * sizeof(*d_u_buffer_)));

    unsigned long long *d_in_tmp = nullptr;  // dummy
    unsigned long long *d_out_tmp = nullptr; // dummy

    // Compute the storage size necessary to reduce du_dl
    hipcub::DeviceReduce::Sum(d_sum_storage_, d_sum_storage_bytes_, d_in_tmp, d_out_tmp, N_);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMalloc(&d_sum_storage_, d_sum_storage_bytes_));
};

Context::~Context() {
    gpuErrchk(hipFree(d_x_t_));
    gpuErrchk(hipFree(d_v_t_));
    gpuErrchk(hipFree(d_box_t_));
    gpuErrchk(hipFree(d_du_dl_buffer_));
    gpuErrchk(hipFree(d_u_buffer_));
    gpuErrchk(hipFree(d_sum_storage_));
};

std::array<std::vector<double>, 3>
Context::multiple_steps(const std::vector<double> &lambda_schedule, int store_du_dl_interval, int store_x_interval) {
    if (store_du_dl_interval <= 0) {
        throw std::runtime_error("store_du_dl_interval <= 0");
    }
    if (store_x_interval <= 0) {
        throw std::runtime_error("store_x_interval <= 0");
    }
    if (lambda_schedule.size() % store_x_interval != 0) {
        std::cout << "warning:: length of lambda_schedule modulo store_x_interval does not equal zero" << std::endl;
    }

    if (lambda_schedule.size() % store_du_dl_interval != 0) {
        std::cout << "warning:: length of lambda_schedule modulo store_du_dl_interval does not equal zero" << std::endl;
    }

    int du_dl_buffer_size = lambda_schedule.size() / store_du_dl_interval;
    int x_buffer_size = lambda_schedule.size() / store_x_interval;
    int box_buffer_size = x_buffer_size * 3 * 3;

    std::vector<double> h_x_buffer(x_buffer_size * N_ * 3);

    hipStream_t stream = static_cast<hipStream_t>(0);

    std::unique_ptr<DeviceBuffer<double>> d_box_buffer(nullptr);
    if (box_buffer_size > 0) {
        d_box_buffer.reset(new DeviceBuffer<double>(box_buffer_size));
    }
    std::unique_ptr<DeviceBuffer<unsigned long long>> d_du_dl_buffer(nullptr);
    if (du_dl_buffer_size > 0) {
        d_du_dl_buffer.reset(new DeviceBuffer<unsigned long long>(du_dl_buffer_size));
        gpuErrchk(hipMemsetAsync(d_du_dl_buffer->data, 0, d_du_dl_buffer->size, stream));
    }

    intg_->initialize(bps_, lambda_schedule[0], d_x_t_, d_v_t_, d_box_t_, stream);
    for (int i = 1; i <= lambda_schedule.size(); i++) {
        // decide if we need to store the du_dl for this step
        unsigned long long *du_dl_ptr = nullptr;
        if (i % store_du_dl_interval == 0) {
            // pemdas but just to make it clear we're doing pointer arithmetic
            du_dl_ptr = d_du_dl_buffer->data + ((i / store_du_dl_interval) - 1);
        }

        double lambda = lambda_schedule[i - 1];
        this->_step(bps_, lambda, du_dl_ptr, stream);

        if (i % store_x_interval == 0) {
            gpuErrchk(hipMemcpyAsync(
                &h_x_buffer[0] + ((i / store_x_interval) - 1) * N_ * 3,
                d_x_t_,
                N_ * 3 * sizeof(double),
                hipMemcpyDeviceToHost,
                stream));
            gpuErrchk(hipMemcpyAsync(
                &d_box_buffer->data[0] + ((i / store_x_interval) - 1) * 3 * 3,
                d_box_t_,
                3 * 3 * sizeof(*d_box_buffer->data),
                hipMemcpyDeviceToDevice,
                stream));
        }
    }
    intg_->finalize(bps_, lambda_schedule[lambda_schedule.size() - 1], d_x_t_, d_v_t_, d_box_t_, stream);

    gpuErrchk(hipStreamSynchronize(stream));

    std::vector<unsigned long long> h_du_dl_buffer_ull(du_dl_buffer_size);
    if (du_dl_buffer_size > 0) {
        d_du_dl_buffer->copy_to(&h_du_dl_buffer_ull[0]);
    }

    std::vector<double> h_du_dl_buffer_double(du_dl_buffer_size);
    for (int i = 0; i < h_du_dl_buffer_ull.size(); i++) {
        h_du_dl_buffer_double[i] = FIXED_TO_FLOAT<double>(h_du_dl_buffer_ull[i]);
    }
    std::vector<double> h_box_buffer(box_buffer_size);
    if (box_buffer_size > 0) {
        d_box_buffer->copy_to(&h_box_buffer[0]);
    }

    return std::array<std::vector<double>, 3>({h_du_dl_buffer_double, h_x_buffer, h_box_buffer});
}

std::array<std::vector<double>, 3> Context::multiple_steps_U(
    const double lambda, // which lambda window we run the integrator over
    const int n_steps,
    const std::vector<double> &lambda_windows, // which lambda windows we wish to evaluate U at
    int store_u_interval,
    int store_x_interval) {

    if (store_u_interval <= 0) {
        throw std::runtime_error("store_u_interval <= 0");
    }

    if (store_x_interval <= 0) {
        throw std::runtime_error("store_x_interval <= 0");
    }

    if (n_steps % store_x_interval != 0) {
        std::cout << "warning:: n_steps modulo store_x_interval does not equal zero" << std::endl;
    }

    if (n_steps % store_u_interval != 0) {
        std::cout << "warning:: n_steps modulo store_u_interval does not equal zero" << std::endl;
    }

    int n_windows = lambda_windows.size();
    int u_traj_size = (n_steps / store_u_interval) * n_windows;
    int x_traj_size = n_steps / store_x_interval;
    int box_traj_size = x_traj_size * 3 * 3;

    std::vector<double> h_x_traj(x_traj_size * N_ * 3);

    hipStream_t stream = static_cast<hipStream_t>(0);

    std::unique_ptr<DeviceBuffer<double>> d_box_traj(nullptr);
    if (box_traj_size > 0) {
        d_box_traj.reset(new DeviceBuffer<double>(box_traj_size));
    }
    std::unique_ptr<DeviceBuffer<unsigned long long>> d_u_traj(nullptr);
    if (u_traj_size > 0) {
        d_u_traj.reset(new DeviceBuffer<unsigned long long>(u_traj_size));
        gpuErrchk(hipMemsetAsync(d_u_traj->data, 0, d_u_traj->size, stream));
    }

    intg_->initialize(bps_, lambda, d_x_t_, d_v_t_, d_box_t_, stream);
    for (int step = 1; step <= n_steps; step++) {

        this->_step(bps_, lambda, nullptr, stream);

        if (step % store_x_interval == 0) {
            gpuErrchk(hipMemcpyAsync(
                &h_x_traj[0] + ((step / store_x_interval) - 1) * N_ * 3,
                d_x_t_,
                N_ * 3 * sizeof(double),
                hipMemcpyDeviceToHost,
                stream));
            gpuErrchk(hipMemcpyAsync(
                &d_box_traj->data[0] + ((step / store_x_interval) - 1) * 3 * 3,
                d_box_t_,
                3 * 3 * sizeof(*d_box_traj->data),
                hipMemcpyDeviceToDevice,
                stream));
        }

        // we need to compute aggregate energies
        if (u_traj_size > 0 && step % store_u_interval == 0) {
            unsigned long long *u_ptr = d_u_traj->data + ((step / store_u_interval) - 1) * n_windows;
            for (int w = 0; w < n_windows; w++) {
                // reset buffers on each pass.
                gpuErrchk(hipMemsetAsync(d_u_buffer_, 0, N_ * sizeof(*d_u_buffer_), stream));
                for (int i = 0; i < bps_.size(); i++) {
                    bps_[i]->execute_device(
                        N_, d_x_t_, d_box_t_, lambda_windows[w], nullptr, nullptr, nullptr, d_u_buffer_, stream);
                }
                hipcub::DeviceReduce::Sum(d_sum_storage_, d_sum_storage_bytes_, d_u_buffer_, u_ptr + w, N_, stream);
                gpuErrchk(hipPeekAtLastError());
            }
        }
    }
    intg_->finalize(bps_, lambda, d_x_t_, d_v_t_, d_box_t_, stream);

    gpuErrchk(hipStreamSynchronize(stream));

    std::vector<unsigned long long> h_u_traj_ull(u_traj_size);
    if (u_traj_size > 0) {
        d_u_traj->copy_to(&h_u_traj_ull[0]);
    }

    std::vector<double> h_u_traj_double(u_traj_size);
    for (int i = 0; i < h_u_traj_ull.size(); i++) {
        h_u_traj_double[i] = FIXED_TO_FLOAT<double>(h_u_traj_ull[i]);
    }
    std::vector<double> h_box_traj(box_traj_size);
    if (box_traj_size > 0) {
        d_box_traj->copy_to(&h_box_traj[0]);
    }

    return std::array<std::vector<double>, 3>({h_u_traj_double, h_x_traj, h_box_traj});
}

void Context::step(double lambda) {
    hipStream_t stream = static_cast<hipStream_t>(0);
    this->_step(bps_, lambda, nullptr, stream);
    gpuErrchk(hipDeviceSynchronize());
}

void Context::finalize(double lambda) {
    hipStream_t stream = static_cast<hipStream_t>(0);
    intg_->finalize(bps_, lambda, d_x_t_, d_v_t_, d_box_t_, stream);
    gpuErrchk(hipStreamSynchronize(stream));
}

void Context::initialize(double lambda) {
    hipStream_t stream = static_cast<hipStream_t>(0);
    intg_->initialize(bps_, lambda, d_x_t_, d_v_t_, d_box_t_, stream);
    gpuErrchk(hipStreamSynchronize(stream));
}

void Context::_step(
    std::vector<BoundPotential *> &bps, const double lambda, unsigned long long *du_dl_out, const hipStream_t stream) {

    if (du_dl_out) {
        gpuErrchk(hipMemsetAsync(d_du_dl_buffer_, 0, N_ * sizeof(*d_du_dl_buffer_), stream));
    }

    intg_->step_fwd(bps, lambda, d_x_t_, d_v_t_, d_box_t_, du_dl_out ? d_du_dl_buffer_ : nullptr, stream);

    // compute du_dl
    if (du_dl_out) {
        hipcub::DeviceReduce::Sum(d_sum_storage_, d_sum_storage_bytes_, d_du_dl_buffer_, du_dl_out, N_, stream);
        gpuErrchk(hipPeekAtLastError());
    }

    if (barostat_) {
        // May modify coords, du_dx and box size
        barostat_->inplace_move(d_x_t_, d_box_t_, lambda, stream);
    }

    step_ += 1;
};

int Context::num_atoms() const { return N_; }

void Context::set_x_t(const double *in_buffer) {
    gpuErrchk(hipMemcpy(d_x_t_, in_buffer, N_ * 3 * sizeof(*in_buffer), hipMemcpyHostToDevice));
}

void Context::set_v_t(const double *in_buffer) {
    gpuErrchk(hipMemcpy(d_v_t_, in_buffer, N_ * 3 * sizeof(*in_buffer), hipMemcpyHostToDevice));
}

void Context::get_x_t(double *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_x_t_, N_ * 3 * sizeof(*out_buffer), hipMemcpyDeviceToHost));
}

void Context::get_v_t(double *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_v_t_, N_ * 3 * sizeof(*out_buffer), hipMemcpyDeviceToHost));
}

void Context::get_box(double *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_box_t_, 3 * 3 * sizeof(*out_buffer), hipMemcpyDeviceToHost));
}

} // namespace timemachine
