#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#include "context.hpp"
#include "integrator.hpp"
#include "assert.h"
#include "kernel_utils.cuh"
#include "stepper.hpp"

namespace timemachine {

ReversibleContext::~ReversibleContext() {
    gpuErrchk(hipFree(d_params_));
    gpuErrchk(hipFree(d_params_grads_));

    gpuErrchk(hipFree(d_coeff_cbs_));

    gpuErrchk(hipFree(d_coords_));
    gpuErrchk(hipFree(d_du_dls_));

    gpuErrchk(hipFree(d_velocities_));
    gpuErrchk(hipFree(d_forces_));

    gpuErrchk(hipFree(d_x_t_tangent_));
    gpuErrchk(hipFree(d_param_adjoint_accum_));
    gpuErrchk(hipFree(d_x_t_adjoint_));
    gpuErrchk(hipFree(d_v_t_adjoint_));

    gpuErrchk(hipFree(d_dE_dx_jvp_));
    gpuErrchk(hipFree(d_dE_dp_jvp_));
};

ReversibleContext::ReversibleContext(
    Stepper *stepper,
    const int N,
    const std::vector<double> &x0,
    const std::vector<double> &v0,
    const std::vector<double> &coeff_cas,
    const std::vector<double> &coeff_cbs,
    const std::vector<double> &step_sizes,
    const std::vector<double> &params) :
        N_(N),
        P_(params.size()),
        stepper_(stepper),
        coeff_cas_(coeff_cas),
        step_sizes_(step_sizes) {

    size_t T = step_sizes.size();
    size_t P = P_;
    size_t F = T+1; // number of frames is number of steps + 1

    assert(x0.size() == N*D);
    assert(v0.size() == N*D);
    assert(coeff_cas.size() == T);
    assert(coeff_cbs.size() == N);
    assert(step_sizes.size() == T);
    assert(params.size() == P);

    gpuErrchk(hipMalloc(&d_coeff_cbs_, N*sizeof(double)));
    gpuErrchk(hipMemcpy(d_coeff_cbs_, &coeff_cbs[0], N*sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_coords_, F*N*D*sizeof(double)));
    gpuErrchk(hipMalloc(&d_du_dls_, F*sizeof(double)));
    hipMemset(d_du_dls_, 0, F*sizeof(double));
    gpuErrchk(hipMemcpy(d_coords_, &x0[0], N*D*sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_velocities_, N*D*sizeof(double)));
    gpuErrchk(hipMemcpy(d_velocities_, &v0[0], N*D*sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_forces_, N*D*sizeof(*d_forces_)));

    gpuErrchk(hipMalloc(&d_params_, P*sizeof(double)));
    gpuErrchk(hipMemcpy(d_params_, &params[0], P*sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_x_t_tangent_, N*D*sizeof(double))); // [NxD]
    gpuErrchk(hipMalloc(&d_param_adjoint_accum_, P*sizeof(double))); // [P]
    gpuErrchk(hipMalloc(&d_x_t_adjoint_, N*D*sizeof(double))); // [NxD]
    gpuErrchk(hipMalloc(&d_v_t_adjoint_, N*D*sizeof(double))); // [NxD]

    gpuErrchk(hipMalloc(&d_dE_dx_jvp_, N*D*sizeof(double))); // [NxD]
    gpuErrchk(hipMalloc(&d_dE_dp_jvp_, P*sizeof(double))); // [P]

    gpuErrchk(hipMalloc(&d_params_grads_, P*sizeof(double))); // [P]
};

void ReversibleContext::forward_mode() {

    for(int t=0; t < step_sizes_.size(); t++) {

        // compute gradients
        gpuErrchk(hipMemset(d_forces_, 0, N_*D*sizeof(*d_forces_)));
        // gpuErrchk(hipMemset(d_params_grads_, 0, P_*sizeof(double))); # not used

	auto start0 = std::chrono::high_resolution_clock::now();
        stepper_->forward_step(
            N_,
            P_,
            d_coords_ + t*N_*D,
            d_params_,
            d_forces_
        );
	auto finish0 = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed0 = finish0 - start0;
        // std::cout << "Stepper Elapsed time: " << elapsed0.count() << " s\n";
	auto start = std::chrono::high_resolution_clock::now();
        step_forward<double>(
            N_,
            D,
            coeff_cas_[t],
            d_coeff_cbs_,
            d_coords_ + t*N_*D,
            d_velocities_, 
            d_forces_,
            step_sizes_[t],
            d_coords_ + (t+1)*N_*D,
            d_velocities_
        );

	auto finish = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed = finish - start;
	// std::cout << "Context Elapsed time: " << elapsed.count() << " s\n";

    }

}

__global__ void update_backward_1(
    const int N,
    const int D,
    const double *d_coeff_bs,
    const double *d_adjoint_x_t_new,
    double *d_adjoint_v_t_new,
    double *d_x_t_tangent,
    const double dt) {

    int atom_idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(atom_idx >= N) {
        return;
    }

    int d_idx = blockIdx.y;
    int local_idx = atom_idx*D + d_idx;
    d_adjoint_v_t_new[local_idx] += dt*d_adjoint_x_t_new[local_idx];
    d_x_t_tangent[local_idx] = d_coeff_bs[atom_idx]*d_adjoint_v_t_new[local_idx];

};

__global__ void update_backward_2(
    int P,
    const double *d_dE_dp_jvp,
    double *d_adjoint_params) {

    int p_idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(p_idx >= P) {
        return;
    }
    d_adjoint_params[p_idx] += d_dE_dp_jvp[p_idx];
}

__global__ void update_backward_3(
    const int N,
    const int D,
    const double ca,
    const double *d_adjoint_x_new,
    const double *d_adjoint_v_new,
    const double *d_x_t_jvp,
    double *d_adjoint_x_old,
    double *d_adjoint_v_old) {

    int atom_idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(atom_idx >= N) {
        return;
    }
    int d_idx = blockIdx.y;
    int local_idx = atom_idx*D + d_idx;

    d_adjoint_x_old[local_idx] += d_x_t_jvp[local_idx];
    d_adjoint_v_old[local_idx] = ca*d_adjoint_v_new[local_idx];

};

void ReversibleContext::set_x_t_adjoint(const double *buffer) {
    gpuErrchk(hipMemcpy(d_x_t_adjoint_, buffer, N_*D*sizeof(*buffer), hipMemcpyHostToDevice));
}

void ReversibleContext::backward_mode() {

    // initialized
    // d_x_t adjoint has been set via set_x_t_adjoint() but we should make it possible
    // to set the rest of the initial adjoints as well.
    gpuErrchk(hipMemset(d_v_t_adjoint_, 0, N_*D*sizeof(*d_v_t_adjoint_)));
    gpuErrchk(hipMemset(d_param_adjoint_accum_, 0, P_*sizeof(double)));

    // compute derivatives
    for(int t = step_sizes_.size()-1; t >= 0; t--) {
        size_t tpb = 32;
        size_t n_blocks = (N_*D + tpb - 1) / tpb;
        dim3 dimGrid_dx(n_blocks, D);
        gpuErrchk(hipMemset(d_x_t_tangent_, 0, N_*D*sizeof(*d_x_t_tangent_)));

        update_backward_1<<<dimGrid_dx, tpb>>>(
            N_,
            D,
            d_coeff_cbs_,
            d_x_t_adjoint_,
            d_v_t_adjoint_,
            d_x_t_tangent_,
            step_sizes_[t] // this is out of bounds
        );

        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        // important that we set the memory addresses to zero
        gpuErrchk(hipMemset(d_dE_dx_jvp_, 0, N_*D*sizeof(*d_dE_dx_jvp_)));
        gpuErrchk(hipMemset(d_dE_dp_jvp_, 0, P_*sizeof(*d_dE_dp_jvp_)));

        stepper_->backward_step(
            N_,
            P_,
            d_coords_ + t*N_*D,
            d_params_,
            d_x_t_tangent_,
            d_dE_dx_jvp_,
            d_dE_dp_jvp_
        );

        size_t n_block_params = (P_ + tpb - 1) / tpb;

        // we can probably *directly* atomic add into this (and into the d_adjoint_xol in the above func)
        // also would let us save a little bit more buffer room
        update_backward_2<<<n_block_params, tpb>>>(P_, d_dE_dp_jvp_, d_param_adjoint_accum_);

        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        update_backward_3<<<dimGrid_dx, tpb>>>(
            N_,
            D,
            coeff_cas_[t],
            d_x_t_adjoint_,
            d_v_t_adjoint_,
            d_dE_dx_jvp_,
            d_x_t_adjoint_,
            d_v_t_adjoint_  
        );

        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

    }

}

void ReversibleContext::get_all_coords(double *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_coords_, (step_sizes_.size()+1)*N_*D*sizeof(double), hipMemcpyDeviceToHost));
}

void ReversibleContext::get_param_adjoint_accum(double *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_param_adjoint_accum_, P_*sizeof(double), hipMemcpyDeviceToHost));
}

void ReversibleContext::get_x_t_adjoint(double *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_x_t_adjoint_, N_*D*sizeof(double), hipMemcpyDeviceToHost));
}

void ReversibleContext::get_v_t_adjoint(double *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_v_t_adjoint_, N_*D*sizeof(double), hipMemcpyDeviceToHost));
}

// template class ReversibleContext<double, 4>;
// template class ReversibleContext<double, 3>;

// template class ReversibleContext<float, 4>;
// template class ReversibleContext<float, 3>;

};
