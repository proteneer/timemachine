#include "hip/hip_runtime.h"
#include "vendored/jitify.hpp"
#include <algorithm>
#include <cassert>
#include <chrono>
#include <complex>
#include <cstdlib>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <numeric>
#include <set>
#include <vector>

#include "fixed_point.hpp"
#include "gpu_utils.cuh"
#include "nonbonded_interaction_group.hpp"
#include "vendored/hilbert.h"

#include "k_nonbonded.cuh"

#include <fstream>
#include <streambuf>
#include <string>

namespace timemachine {

std::vector<int> set_to_vector(const std::set<int> &s) {
    std::vector<int> v(s.begin(), s.end());
    return v;
}

template <typename RealType, bool Interpolated>
NonbondedInteractionGroup<RealType, Interpolated>::NonbondedInteractionGroup(
    const std::set<int> &row_atom_idxs,
    const std::vector<int> &lambda_plane_idxs,  // [N]
    const std::vector<int> &lambda_offset_idxs, // [N]
    const double beta,
    const double cutoff,
    const std::string &kernel_src)
    : N_(lambda_offset_idxs.size()), NR_(row_atom_idxs.size()), NC_(N_ - NR_), cutoff_(cutoff), nblist_(NC_, NR_),
      beta_(beta), d_sort_storage_(nullptr), d_sort_storage_bytes_(0), nblist_padding_(0.1), disable_hilbert_(false),
      kernel_ptrs_({// enumerate over every possible kernel combination
                    // U: Compute U
                    // X: Compute DU_DL
                    // L: Compute DU_DX
                    // P: Compute DU_DP
                    //                             U  X  L  P
                    &k_nonbonded_unified<RealType, 0, 0, 0, 0>,
                    &k_nonbonded_unified<RealType, 0, 0, 0, 1>,
                    &k_nonbonded_unified<RealType, 0, 0, 1, 0>,
                    &k_nonbonded_unified<RealType, 0, 0, 1, 1>,
                    &k_nonbonded_unified<RealType, 0, 1, 0, 0>,
                    &k_nonbonded_unified<RealType, 0, 1, 0, 1>,
                    &k_nonbonded_unified<RealType, 0, 1, 1, 0>,
                    &k_nonbonded_unified<RealType, 0, 1, 1, 1>,
                    &k_nonbonded_unified<RealType, 1, 0, 0, 0>,
                    &k_nonbonded_unified<RealType, 1, 0, 0, 1>,
                    &k_nonbonded_unified<RealType, 1, 0, 1, 0>,
                    &k_nonbonded_unified<RealType, 1, 0, 1, 1>,
                    &k_nonbonded_unified<RealType, 1, 1, 0, 0>,
                    &k_nonbonded_unified<RealType, 1, 1, 0, 1>,
                    &k_nonbonded_unified<RealType, 1, 1, 1, 0>,
                    &k_nonbonded_unified<RealType, 1, 1, 1, 1>}),
      compute_w_coords_instance_(kernel_cache_.program(kernel_src.c_str()).kernel("k_compute_w_coords").instantiate()),
      compute_gather_interpolated_(
          kernel_cache_.program(kernel_src.c_str()).kernel("k_gather_interpolated").instantiate()),
      compute_add_du_dp_interpolated_(
          kernel_cache_.program(kernel_src.c_str()).kernel("k_add_du_dp_interpolated").instantiate()) {

    if (NR_ == 0) {
        throw std::runtime_error("row_atom_idxs must be nonempty");
    }

    if (lambda_offset_idxs.size() != lambda_plane_idxs.size()) {
        throw std::runtime_error("lambda offset idxs and plane idxs need to be equivalent");
    }

    // compute set of column atoms as set difference
    std::vector<int> all_atom_idxs(N_);
    std::iota(all_atom_idxs.begin(), all_atom_idxs.end(), 0);
    std::set<int> col_atom_idxs;
    std::set_difference(
        all_atom_idxs.begin(),
        all_atom_idxs.end(),
        row_atom_idxs.begin(),
        row_atom_idxs.end(),
        std::inserter(col_atom_idxs, col_atom_idxs.end()));

    std::vector<int> col_atom_idxs_v(set_to_vector(col_atom_idxs));
    gpuErrchk(hipMalloc(&d_col_atom_idxs_, NC_ * sizeof(*d_col_atom_idxs_)));
    gpuErrchk(
        hipMemcpy(d_col_atom_idxs_, &col_atom_idxs_v[0], NC_ * sizeof(*d_col_atom_idxs_), hipMemcpyHostToDevice));

    std::vector<int> row_atom_idxs_v(set_to_vector(row_atom_idxs));
    gpuErrchk(hipMalloc(&d_row_atom_idxs_, NR_ * sizeof(*d_row_atom_idxs_)));
    gpuErrchk(
        hipMemcpy(d_row_atom_idxs_, &row_atom_idxs_v[0], NR_ * sizeof(*d_row_atom_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_lambda_plane_idxs_, N_ * sizeof(*d_lambda_plane_idxs_)));
    gpuErrchk(hipMemcpy(
        d_lambda_plane_idxs_, &lambda_plane_idxs[0], N_ * sizeof(*d_lambda_plane_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_lambda_offset_idxs_, N_ * sizeof(*d_lambda_offset_idxs_)));
    gpuErrchk(hipMemcpy(
        d_lambda_offset_idxs_, &lambda_offset_idxs[0], N_ * sizeof(*d_lambda_offset_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_perm_, N_ * sizeof(*d_perm_)));

    gpuErrchk(hipMalloc(&d_sorted_x_, N_ * 3 * sizeof(*d_sorted_x_)));

    gpuErrchk(hipMalloc(&d_w_, N_ * sizeof(*d_w_)));
    gpuErrchk(hipMalloc(&d_dw_dl_, N_ * sizeof(*d_dw_dl_)));
    gpuErrchk(hipMalloc(&d_sorted_w_, N_ * sizeof(*d_sorted_w_)));
    gpuErrchk(hipMalloc(&d_sorted_dw_dl_, N_ * sizeof(*d_sorted_dw_dl_)));

    gpuErrchk(hipMalloc(&d_sorted_p_, N_ * 3 * sizeof(*d_sorted_p_)));         // interpolated
    gpuErrchk(hipMalloc(&d_sorted_dp_dl_, N_ * 3 * sizeof(*d_sorted_dp_dl_))); // interpolated
    gpuErrchk(hipMalloc(&d_sorted_du_dx_, N_ * 3 * sizeof(*d_sorted_du_dx_)));
    gpuErrchk(hipMalloc(&d_sorted_du_dp_, N_ * 3 * sizeof(*d_sorted_du_dp_)));
    gpuErrchk(hipMalloc(&d_du_dp_buffer_, N_ * 3 * sizeof(*d_du_dp_buffer_)));

    gpuErrchk(hipHostMalloc(&p_ixn_count_, 1 * sizeof(*p_ixn_count_)));

    gpuErrchk(hipMalloc(&d_nblist_x_, N_ * 3 * sizeof(*d_nblist_x_)));
    gpuErrchk(hipMemset(d_nblist_x_, 0, N_ * 3 * sizeof(*d_nblist_x_))); // set non-sensical positions
    gpuErrchk(hipMalloc(&d_nblist_box_, 3 * 3 * sizeof(*d_nblist_x_)));
    gpuErrchk(hipMemset(d_nblist_box_, 0, 3 * 3 * sizeof(*d_nblist_x_)));
    gpuErrchk(hipMalloc(&d_rebuild_nblist_, 1 * sizeof(*d_rebuild_nblist_)));
    gpuErrchk(hipHostMalloc(&p_rebuild_nblist_, 1 * sizeof(*p_rebuild_nblist_)));

    gpuErrchk(hipMalloc(&d_sort_keys_in_, N_ * sizeof(d_sort_keys_in_)));
    gpuErrchk(hipMalloc(&d_sort_keys_out_, N_ * sizeof(d_sort_keys_out_)));
    gpuErrchk(hipMalloc(&d_sort_vals_in_, N_ * sizeof(d_sort_vals_in_)));

    // initialize hilbert curve
    std::vector<unsigned int> bin_to_idx(HILBERT_GRID_DIM * HILBERT_GRID_DIM * HILBERT_GRID_DIM);
    for (int i = 0; i < HILBERT_GRID_DIM; i++) {
        for (int j = 0; j < HILBERT_GRID_DIM; j++) {
            for (int k = 0; k < HILBERT_GRID_DIM; k++) {

                bitmask_t hilbert_coords[3];
                hilbert_coords[0] = i;
                hilbert_coords[1] = j;
                hilbert_coords[2] = k;

                unsigned int bin = static_cast<unsigned int>(hilbert_c2i(3, HILBERT_N_BITS, hilbert_coords));
                bin_to_idx[i * HILBERT_GRID_DIM * HILBERT_GRID_DIM + j * HILBERT_GRID_DIM + k] = bin;
            }
        }
    }

    gpuErrchk(
        hipMalloc(&d_bin_to_idx_, HILBERT_GRID_DIM * HILBERT_GRID_DIM * HILBERT_GRID_DIM * sizeof(*d_bin_to_idx_)));
    gpuErrchk(hipMemcpy(
        d_bin_to_idx_,
        &bin_to_idx[0],
        HILBERT_GRID_DIM * HILBERT_GRID_DIM * HILBERT_GRID_DIM * sizeof(*d_bin_to_idx_),
        hipMemcpyHostToDevice));

    // estimate size needed to do radix sorting, this can use uninitialized data.
    hipcub::DeviceRadixSort::SortPairs(
        d_sort_storage_,
        d_sort_storage_bytes_,
        d_sort_keys_in_,
        d_sort_keys_out_,
        d_sort_vals_in_,
        d_perm_,
        std::max(NC_, NR_));

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMalloc(&d_sort_storage_, d_sort_storage_bytes_));
};

template <typename RealType, bool Interpolated>
NonbondedInteractionGroup<RealType, Interpolated>::~NonbondedInteractionGroup() {
    gpuErrchk(hipFree(d_col_atom_idxs_));
    gpuErrchk(hipFree(d_row_atom_idxs_));

    gpuErrchk(hipFree(d_lambda_plane_idxs_));
    gpuErrchk(hipFree(d_lambda_offset_idxs_));
    gpuErrchk(hipFree(d_du_dp_buffer_));
    gpuErrchk(hipFree(d_perm_));

    gpuErrchk(hipFree(d_bin_to_idx_));
    gpuErrchk(hipFree(d_sorted_x_));

    gpuErrchk(hipFree(d_w_));
    gpuErrchk(hipFree(d_dw_dl_));
    gpuErrchk(hipFree(d_sorted_w_));
    gpuErrchk(hipFree(d_sorted_dw_dl_));
    gpuErrchk(hipFree(d_sorted_p_));
    gpuErrchk(hipFree(d_sorted_dp_dl_));
    gpuErrchk(hipFree(d_sorted_du_dx_));
    gpuErrchk(hipFree(d_sorted_du_dp_));

    gpuErrchk(hipFree(d_sort_keys_in_));
    gpuErrchk(hipFree(d_sort_keys_out_));
    gpuErrchk(hipFree(d_sort_vals_in_));
    gpuErrchk(hipFree(d_sort_storage_));

    gpuErrchk(hipHostFree(p_ixn_count_));

    gpuErrchk(hipFree(d_nblist_x_));
    gpuErrchk(hipFree(d_nblist_box_));
    gpuErrchk(hipFree(d_rebuild_nblist_));
    gpuErrchk(hipHostFree(p_rebuild_nblist_));
};

template <typename RealType, bool Interpolated>
void NonbondedInteractionGroup<RealType, Interpolated>::set_nblist_padding(double val) {
    nblist_padding_ = val;
}

template <typename RealType, bool Interpolated>
void NonbondedInteractionGroup<RealType, Interpolated>::disable_hilbert_sort() {
    disable_hilbert_ = true;
}

template <typename RealType, bool Interpolated>
void NonbondedInteractionGroup<RealType, Interpolated>::hilbert_sort(
    const int N,
    const unsigned int *d_atom_idxs,
    const double *d_coords,
    const double *d_box,
    unsigned int *d_perm,
    hipStream_t stream) {

    const int tpb = 32;
    const int B = ceil_divide(N, tpb);

    k_coords_to_kv_gather<<<B, tpb, 0, stream>>>(
        N, d_atom_idxs, d_coords, d_box, d_bin_to_idx_, d_sort_keys_in_, d_sort_vals_in_);

    gpuErrchk(hipPeekAtLastError());

    hipcub::DeviceRadixSort::SortPairs(
        d_sort_storage_,
        d_sort_storage_bytes_,
        d_sort_keys_in_,
        d_sort_keys_out_,
        d_sort_vals_in_,
        d_perm,
        N,
        0,                            // begin bit
        sizeof(*d_sort_keys_in_) * 8, // end bit
        stream                        // cudaStream
    );

    gpuErrchk(hipPeekAtLastError());
}

template <typename RealType, bool Interpolated>
void NonbondedInteractionGroup<RealType, Interpolated>::execute_device(
    const int N,
    const int P,
    const double *d_x,
    const double *d_p,   // 2 * N * 3
    const double *d_box, // 3 * 3
    const double lambda,
    unsigned long long *d_du_dx,
    unsigned long long *d_du_dp,
    unsigned long long *d_du_dl,
    unsigned long long *d_u,
    hipStream_t stream) {

    // (ytz) the nonbonded algorithm proceeds as follows:

    // (done in constructor), construct a hilbert curve mapping each of the HILBERT_GRID_DIM x HILBERT_GRID_DIM x HILBERT_GRID_DIM cells into an index.
    // a. decide if we need to rebuild the neighborlist, if so:
    //     - look up which cell each particle belongs to, and its linear index along the hilbert curve.
    //     - use radix pair sort keyed on the hilbert index with values equal to the atomic index
    //     - resulting sorted values is the permutation array.
    //     - permute lambda plane/offsets, coords
    // b. else:
    //     - permute new coords
    // c. permute parameters
    // d. compute the nonbonded interactions using the neighborlist
    // e. inverse permute the forces, du/dps into the original index.
    // f. u and du/dl is buffered into a per-particle array, and then reduced.
    // g. note that du/dl is not an exact per-particle du/dl - it is only used for reduction purposes.

    if (N != N_) {
        throw std::runtime_error(
            "NonbondedAllPairs::execute_device(): expected N == N_, got N=" + std::to_string(N) +
            ", N_=" + std::to_string(N_));
    }

    const int M = Interpolated ? 2 : 1;

    if (P != M * N_ * 3) {
        throw std::runtime_error(
            "NonbondedAllPairs::execute_device(): expected P == M*N_*3, got P=" + std::to_string(P) +
            ", M*N_*3=" + std::to_string(M * N_ * 3));
    }

    // identify which tiles contain interpolated parameters

    const int tpb = 32;
    const int B = ceil_divide(N_, tpb);

    // (ytz) see if we need to rebuild the neighborlist.
    k_check_rebuild_coords_and_box<RealType>
        <<<B, tpb, 0, stream>>>(N_, d_x, d_nblist_x_, d_box, d_nblist_box_, nblist_padding_, d_rebuild_nblist_);

    gpuErrchk(hipPeekAtLastError());

    // we can optimize this away by doing the check on the GPU directly.
    gpuErrchk(hipMemcpyAsync(
        p_rebuild_nblist_, d_rebuild_nblist_, 1 * sizeof(*p_rebuild_nblist_), hipMemcpyDeviceToHost, stream));
    gpuErrchk(hipStreamSynchronize(stream)); // slow!

    dim3 dimGrid(B, 3, 1);

    if (p_rebuild_nblist_[0] > 0) {

        // (ytz): update the permutation index before building neighborlist, as the neighborlist is tied
        // to a particular sort order
        if (!disable_hilbert_) {
            this->hilbert_sort(NC_, d_col_atom_idxs_, d_x, d_box, d_perm_, stream);
            this->hilbert_sort(NR_, d_row_atom_idxs_, d_x, d_box, d_perm_ + NC_, stream);
        } else {
            gpuErrchk(hipMemcpyAsync(
                d_perm_, d_col_atom_idxs_, NC_ * sizeof(*d_col_atom_idxs_), hipMemcpyDeviceToDevice, stream));
            gpuErrchk(hipMemcpyAsync(
                d_perm_ + NC_, d_row_atom_idxs_, NR_ * sizeof(*d_row_atom_idxs_), hipMemcpyDeviceToDevice, stream));
        }

        // compute new coordinates, new lambda_idxs, new_plane_idxs
        k_gather<<<dimGrid, tpb, 0, stream>>>(N_, d_perm_, d_x, d_sorted_x_);
        gpuErrchk(hipPeekAtLastError());

        nblist_.build_nblist_device(
            NC_, NR_, d_sorted_x_, d_sorted_x_ + 3 * NC_, d_box, cutoff_ + nblist_padding_, stream);
        gpuErrchk(hipMemcpyAsync(
            p_ixn_count_, nblist_.get_ixn_count(), 1 * sizeof(*p_ixn_count_), hipMemcpyDeviceToHost, stream));

        std::vector<double> h_box(9);
        gpuErrchk(hipMemcpyAsync(&h_box[0], d_box, 3 * 3 * sizeof(*d_box), hipMemcpyDeviceToHost, stream));

        // this stream needs to be synchronized so we can be sure that p_ixn_count_ is properly set.
        hipStreamSynchronize(stream);

        // Verify that the cutoff and box size are valid together. If cutoff is greater than half the box
        // then a particle can interact with multiple periodic copies.
        const double db_cutoff = (cutoff_ + nblist_padding_) * 2;

        // Verify that box is orthogonal and the width of the box in all dimensions is greater than twice the cutoff
        for (int i = 0; i < 9; i++) {
            if (i == 0 || i == 4 || i == 8) {
                if (h_box[i] < db_cutoff) {
                    throw std::runtime_error(
                        "Cutoff with padding is more than half of the box width, neighborlist is no longer reliable");
                }
            } else if (h_box[i] != 0.0) {
                throw std::runtime_error("Provided non-ortholinear box, unable to compute nonbonded energy");
            }
        }

        gpuErrchk(hipMemsetAsync(d_rebuild_nblist_, 0, sizeof(*d_rebuild_nblist_), stream));
        gpuErrchk(hipMemcpyAsync(d_nblist_x_, d_x, N * 3 * sizeof(*d_x), hipMemcpyDeviceToDevice, stream));
        gpuErrchk(hipMemcpyAsync(d_nblist_box_, d_box, 3 * 3 * sizeof(*d_box), hipMemcpyDeviceToDevice, stream));
    } else {
        k_gather<<<dimGrid, tpb, 0, stream>>>(N, d_perm_, d_x, d_sorted_x_);
        gpuErrchk(hipPeekAtLastError());
    }

    // if the neighborlist is empty, we can return early
    if (p_ixn_count_[0] == 0) {
        return;
    }

    // do parameter interpolation here
    if (Interpolated) {
        hipError_t result = compute_gather_interpolated_.configure(dimGrid, tpb, 0, stream)
                              .launch(lambda, N, d_perm_, d_p, d_p + N * 3, d_sorted_p_, d_sorted_dp_dl_);
        if (result != 0) {
            throw std::runtime_error("Driver call to k_gather_interpolated failed");
        }
    } else {
        k_gather<<<dimGrid, tpb, 0, stream>>>(N, d_perm_, d_p, d_sorted_p_);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipMemsetAsync(d_sorted_dp_dl_, 0, N * 3 * sizeof(*d_sorted_dp_dl_), stream))
    }

    // reset buffers and sorted accumulators
    if (d_du_dx) {
        gpuErrchk(hipMemsetAsync(d_sorted_du_dx_, 0, N * 3 * sizeof(*d_sorted_du_dx_), stream))
    }
    if (d_du_dp) {
        gpuErrchk(hipMemsetAsync(d_sorted_du_dp_, 0, N * 3 * sizeof(*d_sorted_du_dp_), stream))
    }

    // update new w coordinates
    // (tbd): cache lambda value for equilibrium calculations
    hipError_t result = compute_w_coords_instance_.configure(B, tpb, 0, stream)
                          .launch(N, lambda, cutoff_, d_lambda_plane_idxs_, d_lambda_offset_idxs_, d_w_, d_dw_dl_);
    if (result != 0) {
        throw std::runtime_error("Driver call to k_compute_w_coords");
    }

    gpuErrchk(hipPeekAtLastError());
    k_gather_2x<<<B, tpb, 0, stream>>>(N, d_perm_, d_w_, d_dw_dl_, d_sorted_w_, d_sorted_dw_dl_);
    gpuErrchk(hipPeekAtLastError());

    // look up which kernel we need for this computation
    int kernel_idx = 0;
    kernel_idx |= d_du_dp ? 1 << 0 : 0;
    kernel_idx |= d_du_dl ? 1 << 1 : 0;
    kernel_idx |= d_du_dx ? 1 << 2 : 0;
    kernel_idx |= d_u ? 1 << 3 : 0;

    kernel_ptrs_[kernel_idx]<<<p_ixn_count_[0], tpb, 0, stream>>>(
        NC_,
        NR_,
        d_sorted_x_,
        d_sorted_p_,
        d_box,
        d_sorted_dp_dl_,
        d_sorted_w_,
        d_sorted_dw_dl_,
        beta_,
        cutoff_,
        nblist_.get_ixn_tiles(),
        nblist_.get_ixn_atoms(),
        d_sorted_du_dx_,
        d_sorted_du_dp_,
        d_du_dl, // switch to nullptr if we don't request du_dl
        d_u      // switch to nullptr if we don't request energies
    );

    gpuErrchk(hipPeekAtLastError());

    // coords are N,3
    if (d_du_dx) {
        k_scatter_accum<<<dimGrid, tpb, 0, stream>>>(N, d_perm_, d_sorted_du_dx_, d_du_dx);
        gpuErrchk(hipPeekAtLastError());
    }

    // params are N,3
    // this needs to be an accumulated permute
    if (d_du_dp) {
        k_scatter_assign<<<dimGrid, tpb, 0, stream>>>(N, d_perm_, d_sorted_du_dp_, d_du_dp_buffer_);
        gpuErrchk(hipPeekAtLastError());
    }

    if (d_du_dp) {
        if (Interpolated) {
            hipError_t result = compute_add_du_dp_interpolated_.configure(dimGrid, tpb, 0, stream)
                                  .launch(lambda, N, d_du_dp_buffer_, d_du_dp);
            if (result != 0) {
                throw std::runtime_error("Driver call to k_add_du_dp_interpolated failed");
            }
        } else {
            k_add_ull_to_ull<<<dimGrid, tpb, 0, stream>>>(N, d_du_dp_buffer_, d_du_dp);
        }
        gpuErrchk(hipPeekAtLastError());
    }
}

template <typename RealType, bool Interpolated>
void NonbondedInteractionGroup<RealType, Interpolated>::du_dp_fixed_to_float(
    const int N, const int P, const unsigned long long *du_dp, double *du_dp_float) {

    // In the interpolated case we have derivatives for the initial and final parameters
    const int num_tuples = Interpolated ? N * 2 : N;

    for (int i = 0; i < num_tuples; i++) {
        const int idx_charge = i * 3 + 0;
        const int idx_sig = i * 3 + 1;
        const int idx_eps = i * 3 + 2;
        du_dp_float[idx_charge] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DCHARGE>(du_dp[idx_charge]);
        du_dp_float[idx_sig] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DSIG>(du_dp[idx_sig]);
        du_dp_float[idx_eps] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DEPS>(du_dp[idx_eps]);
    }
}

template class NonbondedInteractionGroup<double, true>;
template class NonbondedInteractionGroup<float, true>;
template class NonbondedInteractionGroup<double, false>;
template class NonbondedInteractionGroup<float, false>;

} // namespace timemachine
