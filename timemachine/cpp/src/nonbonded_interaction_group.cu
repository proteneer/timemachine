#include "hip/hip_runtime.h"
#include <complex>
#include <hipcub/hipcub.hpp>
#include <set>
#include <string>
#include <vector>

#include "fixed_point.hpp"
#include "gpu_utils.cuh"
#include "kernel_utils.cuh"
#include "nonbonded_common.cuh"
#include "nonbonded_interaction_group.hpp"
#include "vendored/hilbert.h"

#include "k_nonbonded.cuh"

namespace timemachine {

template <typename RealType>
NonbondedInteractionGroup<RealType>::NonbondedInteractionGroup(
    const int N, const std::set<int> &row_atom_idxs, const double beta, const double cutoff)
    : N_(N), NR_(row_atom_idxs.size()), NC_(N_ - NR_),

      kernel_ptrs_({// enumerate over every possible kernel combination
                    // U: Compute U
                    // X: Compute DU_DX
                    // P: Compute DU_DP
                    //                             U  X  P
                    &k_nonbonded_unified<RealType, 0, 0, 0>,
                    &k_nonbonded_unified<RealType, 0, 0, 1>,
                    &k_nonbonded_unified<RealType, 0, 1, 0>,
                    &k_nonbonded_unified<RealType, 0, 1, 1>,
                    &k_nonbonded_unified<RealType, 1, 0, 0>,
                    &k_nonbonded_unified<RealType, 1, 0, 1>,
                    &k_nonbonded_unified<RealType, 1, 1, 0>,
                    &k_nonbonded_unified<RealType, 1, 1, 1>}),

      beta_(beta), cutoff_(cutoff), nblist_(N_), nblist_padding_(0.1), d_sort_storage_(nullptr),
      d_sort_storage_bytes_(0), disable_hilbert_(false) {

    if (NR_ == 0) {
        throw std::runtime_error("row_atom_idxs must be nonempty");
    }

    // compute set of column atoms as set difference
    std::vector<int> col_atom_idxs_v = get_indices_difference(N_, row_atom_idxs);
    gpuErrchk(hipMalloc(&d_col_atom_idxs_, NC_ * sizeof(*d_col_atom_idxs_)));
    gpuErrchk(
        hipMemcpy(d_col_atom_idxs_, &col_atom_idxs_v[0], NC_ * sizeof(*d_col_atom_idxs_), hipMemcpyHostToDevice));

    std::vector<int> row_atom_idxs_v(set_to_vector(row_atom_idxs));
    gpuErrchk(hipMalloc(&d_row_atom_idxs_, NR_ * sizeof(*d_row_atom_idxs_)));
    gpuErrchk(
        hipMemcpy(d_row_atom_idxs_, &row_atom_idxs_v[0], NR_ * sizeof(*d_row_atom_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_perm_, N_ * sizeof(*d_perm_)));

    gpuErrchk(hipMalloc(&d_sorted_x_, N_ * 3 * sizeof(*d_sorted_x_)));

    gpuErrchk(hipMalloc(&d_sorted_p_, N_ * PARAMS_PER_ATOM * sizeof(*d_sorted_p_)));
    gpuErrchk(hipMalloc(&d_sorted_du_dx_, N_ * 3 * sizeof(*d_sorted_du_dx_)));
    gpuErrchk(hipMalloc(&d_sorted_du_dp_, N_ * PARAMS_PER_ATOM * sizeof(*d_sorted_du_dp_)));
    gpuErrchk(hipMalloc(&d_du_dp_buffer_, N_ * PARAMS_PER_ATOM * sizeof(*d_du_dp_buffer_)));

    gpuErrchk(hipHostMalloc(&p_ixn_count_, 1 * sizeof(*p_ixn_count_)));

    gpuErrchk(hipMalloc(&d_nblist_x_, N_ * 3 * sizeof(*d_nblist_x_)));
    gpuErrchk(hipMemset(d_nblist_x_, 0, N_ * 3 * sizeof(*d_nblist_x_))); // set non-sensical positions
    gpuErrchk(hipMalloc(&d_nblist_box_, 3 * 3 * sizeof(*d_nblist_x_)));
    gpuErrchk(hipMemset(d_nblist_box_, 0, 3 * 3 * sizeof(*d_nblist_x_)));
    gpuErrchk(hipMalloc(&d_rebuild_nblist_, 1 * sizeof(*d_rebuild_nblist_)));
    gpuErrchk(hipHostMalloc(&p_rebuild_nblist_, 1 * sizeof(*p_rebuild_nblist_)));

    gpuErrchk(hipMalloc(&d_sort_keys_in_, N_ * sizeof(d_sort_keys_in_)));
    gpuErrchk(hipMalloc(&d_sort_keys_out_, N_ * sizeof(d_sort_keys_out_)));
    gpuErrchk(hipMalloc(&d_sort_vals_in_, N_ * sizeof(d_sort_vals_in_)));

    // initialize hilbert curve
    std::vector<unsigned int> bin_to_idx(HILBERT_GRID_DIM * HILBERT_GRID_DIM * HILBERT_GRID_DIM);
    for (int i = 0; i < HILBERT_GRID_DIM; i++) {
        for (int j = 0; j < HILBERT_GRID_DIM; j++) {
            for (int k = 0; k < HILBERT_GRID_DIM; k++) {

                bitmask_t hilbert_coords[3];
                hilbert_coords[0] = i;
                hilbert_coords[1] = j;
                hilbert_coords[2] = k;

                unsigned int bin = static_cast<unsigned int>(hilbert_c2i(3, HILBERT_N_BITS, hilbert_coords));
                bin_to_idx[i * HILBERT_GRID_DIM * HILBERT_GRID_DIM + j * HILBERT_GRID_DIM + k] = bin;
            }
        }
    }

    gpuErrchk(
        hipMalloc(&d_bin_to_idx_, HILBERT_GRID_DIM * HILBERT_GRID_DIM * HILBERT_GRID_DIM * sizeof(*d_bin_to_idx_)));
    gpuErrchk(hipMemcpy(
        d_bin_to_idx_,
        &bin_to_idx[0],
        HILBERT_GRID_DIM * HILBERT_GRID_DIM * HILBERT_GRID_DIM * sizeof(*d_bin_to_idx_),
        hipMemcpyHostToDevice));

    // estimate size needed to do radix sorting, this can use uninitialized data.
    hipcub::DeviceRadixSort::SortPairs(
        d_sort_storage_,
        d_sort_storage_bytes_,
        d_sort_keys_in_,
        d_sort_keys_out_,
        d_sort_vals_in_,
        d_perm_,
        std::max(NC_, NR_));

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMalloc(&d_sort_storage_, d_sort_storage_bytes_));
    // We will sort so that the row atoms are always first for the nblist. Cheaper to set once than to
    // recompute the idxs from the permuation
    std::vector<unsigned int> row_atoms(NR_);
    std::iota(row_atoms.begin(), row_atoms.end(), 0);
    nblist_.set_row_idxs(row_atoms);
};

template <typename RealType> NonbondedInteractionGroup<RealType>::~NonbondedInteractionGroup() {
    gpuErrchk(hipFree(d_col_atom_idxs_));
    gpuErrchk(hipFree(d_row_atom_idxs_));

    gpuErrchk(hipFree(d_du_dp_buffer_));
    gpuErrchk(hipFree(d_perm_));

    gpuErrchk(hipFree(d_bin_to_idx_));
    gpuErrchk(hipFree(d_sorted_x_));

    gpuErrchk(hipFree(d_sorted_p_));
    gpuErrchk(hipFree(d_sorted_du_dx_));
    gpuErrchk(hipFree(d_sorted_du_dp_));

    gpuErrchk(hipFree(d_sort_keys_in_));
    gpuErrchk(hipFree(d_sort_keys_out_));
    gpuErrchk(hipFree(d_sort_vals_in_));
    gpuErrchk(hipFree(d_sort_storage_));

    gpuErrchk(hipHostFree(p_ixn_count_));

    gpuErrchk(hipFree(d_nblist_x_));
    gpuErrchk(hipFree(d_nblist_box_));
    gpuErrchk(hipFree(d_rebuild_nblist_));
    gpuErrchk(hipHostFree(p_rebuild_nblist_));
};

template <typename RealType> void NonbondedInteractionGroup<RealType>::set_nblist_padding(double val) {
    nblist_padding_ = val;
}

template <typename RealType> void NonbondedInteractionGroup<RealType>::disable_hilbert_sort() {
    disable_hilbert_ = true;
}

template <typename RealType>
void NonbondedInteractionGroup<RealType>::hilbert_sort(
    const int N,
    const unsigned int *d_atom_idxs,
    const double *d_coords,
    const double *d_box,
    unsigned int *d_perm,
    hipStream_t stream) {

    const int tpb = warp_size;
    const int B = ceil_divide(N, tpb);

    k_coords_to_kv_gather<<<B, tpb, 0, stream>>>(
        N, d_atom_idxs, d_coords, d_box, d_bin_to_idx_, d_sort_keys_in_, d_sort_vals_in_);

    gpuErrchk(hipPeekAtLastError());

    hipcub::DeviceRadixSort::SortPairs(
        d_sort_storage_,
        d_sort_storage_bytes_,
        d_sort_keys_in_,
        d_sort_keys_out_,
        d_sort_vals_in_,
        d_perm,
        N,
        0,                            // begin bit
        sizeof(*d_sort_keys_in_) * 8, // end bit
        stream                        // cudaStream
    );

    gpuErrchk(hipPeekAtLastError());
}

template <typename RealType>
void NonbondedInteractionGroup<RealType>::execute_device(
    const int N,
    const int P,
    const double *d_x,
    const double *d_p,   // N * PARAMS_PER_ATOM
    const double *d_box, // 3 * 3
    const double lambda,
    unsigned long long *d_du_dx,
    unsigned long long *d_du_dp,
    unsigned long long *d_du_dl,
    unsigned long long *d_u,
    hipStream_t stream) {

    // (ytz) the nonbonded algorithm proceeds as follows:

    // (done in constructor), construct a hilbert curve mapping each of the HILBERT_GRID_DIM x HILBERT_GRID_DIM x HILBERT_GRID_DIM cells into an index.
    // a. decide if we need to rebuild the neighborlist, if so:
    //     - look up which cell each particle belongs to, and its linear index along the hilbert curve.
    //     - use radix pair sort keyed on the hilbert index with values equal to the atomic index
    //     - resulting sorted values is the permutation array.
    //     - permute coords
    // b. else:
    //     - permute new coords
    // c. permute parameters
    // d. compute the nonbonded interactions using the neighborlist
    // e. inverse permute the forces, du/dps into the original index.
    // f. u and du/dl is buffered into a per-particle array, and then reduced.
    // g. note that du/dl is not an exact per-particle du/dl - it is only used for reduction purposes.

    if (N != N_) {
        throw std::runtime_error(
            "NonbondedInteractionGroup::execute_device(): expected N == N_, got N=" + std::to_string(N) +
            ", N_=" + std::to_string(N_));
    }

    if (P != N_ * PARAMS_PER_ATOM) {
        throw std::runtime_error(
            "NonbondedInteractionGroup::execute_device(): expected P == N_*" + std::to_string(PARAMS_PER_ATOM) +
            ", got P=" + std::to_string(P) + ", N_*" + std::to_string(PARAMS_PER_ATOM) + "=" +
            std::to_string(N_ * PARAMS_PER_ATOM));
    }

    const int tpb = warp_size;
    const int B = ceil_divide(N_, tpb);

    // (ytz) see if we need to rebuild the neighborlist.
    k_check_rebuild_coords_and_box<RealType>
        <<<B, tpb, 0, stream>>>(N_, d_x, d_nblist_x_, d_box, d_nblist_box_, nblist_padding_, d_rebuild_nblist_);

    gpuErrchk(hipPeekAtLastError());

    // we can optimize this away by doing the check on the GPU directly.
    gpuErrchk(hipMemcpyAsync(
        p_rebuild_nblist_, d_rebuild_nblist_, 1 * sizeof(*p_rebuild_nblist_), hipMemcpyDeviceToHost, stream));
    gpuErrchk(hipStreamSynchronize(stream)); // slow!

    if (p_rebuild_nblist_[0] > 0) {

        // (ytz): update the permutation index before building neighborlist, as the neighborlist is tied
        // to a particular sort order
        if (!disable_hilbert_) {
            this->hilbert_sort(NR_, d_row_atom_idxs_, d_x, d_box, d_perm_, stream);
            this->hilbert_sort(NC_, d_col_atom_idxs_, d_x, d_box, d_perm_ + NR_, stream);
        } else {
            gpuErrchk(hipMemcpyAsync(
                d_perm_, d_row_atom_idxs_, NR_ * sizeof(*d_row_atom_idxs_), hipMemcpyDeviceToDevice, stream));
            gpuErrchk(hipMemcpyAsync(
                d_perm_ + NR_, d_col_atom_idxs_, NC_ * sizeof(*d_col_atom_idxs_), hipMemcpyDeviceToDevice, stream));
        }

        // compute new coordinates
        k_gather<<<dim3(B, 3, 1), tpb, 0, stream>>>(N_, d_perm_, d_x, d_sorted_x_);
        gpuErrchk(hipPeekAtLastError());

        nblist_.build_nblist_device(N_, d_sorted_x_, d_box, cutoff_ + nblist_padding_, stream);
        gpuErrchk(hipMemcpyAsync(
            p_ixn_count_, nblist_.get_ixn_count(), 1 * sizeof(*p_ixn_count_), hipMemcpyDeviceToHost, stream));

        std::vector<double> h_box(9);
        gpuErrchk(hipMemcpyAsync(&h_box[0], d_box, 3 * 3 * sizeof(*d_box), hipMemcpyDeviceToHost, stream));

        // this stream needs to be synchronized so we can be sure that p_ixn_count_ is properly set.
        gpuErrchk(hipStreamSynchronize(stream));

        // Verify that the cutoff and box size are valid together. If cutoff is greater than half the box
        // then a particle can interact with multiple periodic copies.
        const double db_cutoff = (cutoff_ + nblist_padding_) * 2;

        // Verify that box is orthogonal and the width of the box in all dimensions is greater than twice the cutoff
        for (int i = 0; i < 9; i++) {
            if (i == 0 || i == 4 || i == 8) {
                if (h_box[i] < db_cutoff) {
                    throw std::runtime_error(
                        "Cutoff with padding is more than half of the box width, neighborlist is no longer reliable");
                }
            } else if (h_box[i] != 0.0) {
                throw std::runtime_error("Provided non-ortholinear box, unable to compute nonbonded energy");
            }
        }

        gpuErrchk(hipMemsetAsync(d_rebuild_nblist_, 0, sizeof(*d_rebuild_nblist_), stream));
        gpuErrchk(hipMemcpyAsync(d_nblist_x_, d_x, N * 3 * sizeof(*d_x), hipMemcpyDeviceToDevice, stream));
        gpuErrchk(hipMemcpyAsync(d_nblist_box_, d_box, 3 * 3 * sizeof(*d_box), hipMemcpyDeviceToDevice, stream));
    } else {
        k_gather<<<dim3(B, 3, 1), tpb, 0, stream>>>(N, d_perm_, d_x, d_sorted_x_);
        gpuErrchk(hipPeekAtLastError());
    }

    // if the neighborlist is empty, we can return early
    if (p_ixn_count_[0] == 0) {
        return;
    }

    k_gather<<<dim3(B, PARAMS_PER_ATOM, 1), tpb, 0, stream>>>(N, d_perm_, d_p, d_sorted_p_);
    gpuErrchk(hipPeekAtLastError());

    // reset buffers and sorted accumulators
    if (d_du_dx) {
        gpuErrchk(hipMemsetAsync(d_sorted_du_dx_, 0, N * 3 * sizeof(*d_sorted_du_dx_), stream))
    }
    if (d_du_dp) {
        gpuErrchk(hipMemsetAsync(d_sorted_du_dp_, 0, N * PARAMS_PER_ATOM * sizeof(*d_sorted_du_dp_), stream))
    }

    gpuErrchk(hipPeekAtLastError());

    // look up which kernel we need for this computation
    int kernel_idx = 0;
    kernel_idx |= d_du_dp ? 1 << 0 : 0;
    kernel_idx |= d_du_dx ? 1 << 1 : 0;
    kernel_idx |= d_u ? 1 << 2 : 0;

    kernel_ptrs_[kernel_idx]<<<p_ixn_count_[0], tpb, 0, stream>>>(
        N,
        nblist_.get_num_row_idxs(),
        d_sorted_x_,
        d_sorted_p_,
        d_box,
        beta_,
        cutoff_,
        nblist_.get_row_idxs(),
        nblist_.get_ixn_tiles(),
        nblist_.get_ixn_atoms(),
        d_sorted_du_dx_,
        d_sorted_du_dp_,
        d_u // switch to nullptr if we don't request energies
    );

    gpuErrchk(hipPeekAtLastError());

    // coords are N,3
    if (d_du_dx) {
        k_scatter_accum<<<dim3(B, 3, 1), tpb, 0, stream>>>(N, d_perm_, d_sorted_du_dx_, d_du_dx);
        gpuErrchk(hipPeekAtLastError());
    }

    // params are N, PARAMS_PER_ATOM
    // this needs to be an accumulated permute
    if (d_du_dp) {
        k_scatter_assign<<<dim3(B, PARAMS_PER_ATOM, 1), tpb, 0, stream>>>(N, d_perm_, d_sorted_du_dp_, d_du_dp_buffer_);
        gpuErrchk(hipPeekAtLastError());
    }

    if (d_du_dp) {
        k_add_ull_to_ull<<<dim3(B, PARAMS_PER_ATOM, 1), tpb, 0, stream>>>(N, d_du_dp_buffer_, d_du_dp);
        gpuErrchk(hipPeekAtLastError());
    }
}

template <typename RealType>
void NonbondedInteractionGroup<RealType>::du_dp_fixed_to_float(
    const int N, const int P, const unsigned long long *du_dp, double *du_dp_float) {

    for (int i = 0; i < N; i++) {
        const int idx = i * PARAMS_PER_ATOM;
        const int idx_charge = idx + PARAM_OFFSET_CHARGE;
        const int idx_sig = idx + PARAM_OFFSET_SIG;
        const int idx_eps = idx + PARAM_OFFSET_EPS;
        const int idx_w = idx + PARAM_OFFSET_W;

        du_dp_float[idx_charge] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DCHARGE>(du_dp[idx_charge]);
        du_dp_float[idx_sig] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DSIG>(du_dp[idx_sig]);
        du_dp_float[idx_eps] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DEPS>(du_dp[idx_eps]);
        du_dp_float[idx_w] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DW>(du_dp[idx_w]);
    }
}

template class NonbondedInteractionGroup<double>;
template class NonbondedInteractionGroup<float>;

} // namespace timemachine
