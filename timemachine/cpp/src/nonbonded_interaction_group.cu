#include "hip/hip_runtime.h"
#include <complex>
#include <hipcub/hipcub.hpp>
#include <string>
#include <vector>

#include "device_buffer.hpp"
#include "fixed_point.hpp"
#include "gpu_utils.cuh"
#include "kernel_utils.cuh"
#include "kernels/k_indices.cuh"
#include "nonbonded_common.hpp"
#include "nonbonded_interaction_group.hpp"
#include "set_utils.hpp"
#include "vendored/hilbert.h"

#include "k_nonbonded.cuh"

namespace timemachine {

template <typename RealType>
NonbondedInteractionGroup<RealType>::NonbondedInteractionGroup(
    const int N, const std::vector<int> &row_atom_idxs, const double beta, const double cutoff)
    : N_(N), NR_(row_atom_idxs.size()), NC_(N_ - NR_),

      kernel_ptrs_({// enumerate over every possible kernel combination
                    // U: Compute U
                    // X: Compute DU_DX
                    // P: Compute DU_DP
                    //                             U  X  P
                    &k_nonbonded_unified<RealType, 0, 0, 0>,
                    &k_nonbonded_unified<RealType, 0, 0, 1>,
                    &k_nonbonded_unified<RealType, 0, 1, 0>,
                    &k_nonbonded_unified<RealType, 0, 1, 1>,
                    &k_nonbonded_unified<RealType, 1, 0, 0>,
                    &k_nonbonded_unified<RealType, 1, 0, 1>,
                    &k_nonbonded_unified<RealType, 1, 1, 0>,
                    &k_nonbonded_unified<RealType, 1, 1, 1>}),

      beta_(beta), cutoff_(cutoff), nblist_(N_), nblist_padding_(0.1), d_sort_storage_(nullptr),
      d_sort_storage_bytes_(0), disable_hilbert_(false) {

    if (NR_ == 0) {
        throw std::runtime_error("row_atom_idxs must be nonempty");
    }
    if (row_atom_idxs.size() == static_cast<long unsigned int>(N)) {
        throw std::runtime_error("must be less then N(" + std::to_string(N) + ") indices");
    }
    verify_atom_idxs(N_, row_atom_idxs);

    cudaSafeMalloc(&d_col_atom_idxs_, N_ * sizeof(*d_col_atom_idxs_));
    cudaSafeMalloc(&d_row_atom_idxs_, N_ * sizeof(*d_row_atom_idxs_));

    cudaSafeMalloc(&d_perm_, N_ * sizeof(*d_perm_));

    cudaSafeMalloc(&d_sorted_x_, N_ * 3 * sizeof(*d_sorted_x_));

    cudaSafeMalloc(&d_sorted_p_, N_ * PARAMS_PER_ATOM * sizeof(*d_sorted_p_));
    cudaSafeMalloc(&d_sorted_du_dx_, N_ * 3 * sizeof(*d_sorted_du_dx_));
    cudaSafeMalloc(&d_sorted_du_dp_, N_ * PARAMS_PER_ATOM * sizeof(*d_sorted_du_dp_));
    cudaSafeMalloc(&d_du_dp_buffer_, N_ * PARAMS_PER_ATOM * sizeof(*d_du_dp_buffer_));

    gpuErrchk(hipHostMalloc(&p_ixn_count_, 1 * sizeof(*p_ixn_count_)));
    gpuErrchk(hipHostMalloc(&p_box_, 3 * 3 * sizeof(*p_box_)));

    cudaSafeMalloc(&d_nblist_x_, N_ * 3 * sizeof(*d_nblist_x_));
    gpuErrchk(hipMemset(d_nblist_x_, 0, N_ * 3 * sizeof(*d_nblist_x_))); // set non-sensical positions
    cudaSafeMalloc(&d_nblist_box_, 3 * 3 * sizeof(*d_nblist_x_));
    gpuErrchk(hipMemset(d_nblist_box_, 0, 3 * 3 * sizeof(*d_nblist_x_)));
    cudaSafeMalloc(&d_rebuild_nblist_, 1 * sizeof(*d_rebuild_nblist_));
    gpuErrchk(hipHostMalloc(&p_rebuild_nblist_, 1 * sizeof(*p_rebuild_nblist_)));

    cudaSafeMalloc(&d_sort_keys_in_, N_ * sizeof(d_sort_keys_in_));
    cudaSafeMalloc(&d_sort_keys_out_, N_ * sizeof(d_sort_keys_out_));
    cudaSafeMalloc(&d_sort_vals_in_, N_ * sizeof(d_sort_vals_in_));

    // initialize hilbert curve
    std::vector<unsigned int> bin_to_idx(HILBERT_GRID_DIM * HILBERT_GRID_DIM * HILBERT_GRID_DIM);
    for (int i = 0; i < HILBERT_GRID_DIM; i++) {
        for (int j = 0; j < HILBERT_GRID_DIM; j++) {
            for (int k = 0; k < HILBERT_GRID_DIM; k++) {

                bitmask_t hilbert_coords[3];
                hilbert_coords[0] = i;
                hilbert_coords[1] = j;
                hilbert_coords[2] = k;

                unsigned int bin = static_cast<unsigned int>(hilbert_c2i(3, HILBERT_N_BITS, hilbert_coords));
                bin_to_idx[i * HILBERT_GRID_DIM * HILBERT_GRID_DIM + j * HILBERT_GRID_DIM + k] = bin;
            }
        }
    }

    cudaSafeMalloc(&d_bin_to_idx_, HILBERT_GRID_DIM * HILBERT_GRID_DIM * HILBERT_GRID_DIM * sizeof(*d_bin_to_idx_));
    gpuErrchk(hipMemcpy(
        d_bin_to_idx_,
        &bin_to_idx[0],
        HILBERT_GRID_DIM * HILBERT_GRID_DIM * HILBERT_GRID_DIM * sizeof(*d_bin_to_idx_),
        hipMemcpyHostToDevice));

    // estimate size needed to do radix sorting, this can use uninitialized data.
    hipcub::DeviceRadixSort::SortPairs(
        d_sort_storage_,
        d_sort_storage_bytes_,
        d_sort_keys_in_,
        d_sort_keys_out_,
        d_sort_vals_in_,
        d_perm_,
        std::max(NC_, NR_));

    gpuErrchk(hipPeekAtLastError());
    cudaSafeMalloc(&d_sort_storage_, d_sort_storage_bytes_);
    this->set_atom_idxs(row_atom_idxs);
};

template <typename RealType> NonbondedInteractionGroup<RealType>::~NonbondedInteractionGroup() {
    gpuErrchk(hipFree(d_col_atom_idxs_));
    gpuErrchk(hipFree(d_row_atom_idxs_));

    gpuErrchk(hipFree(d_du_dp_buffer_));
    gpuErrchk(hipFree(d_perm_));

    gpuErrchk(hipFree(d_bin_to_idx_));
    gpuErrchk(hipFree(d_sorted_x_));

    gpuErrchk(hipFree(d_sorted_p_));
    gpuErrchk(hipFree(d_sorted_du_dx_));
    gpuErrchk(hipFree(d_sorted_du_dp_));

    gpuErrchk(hipFree(d_sort_keys_in_));
    gpuErrchk(hipFree(d_sort_keys_out_));
    gpuErrchk(hipFree(d_sort_vals_in_));
    gpuErrchk(hipFree(d_sort_storage_));

    gpuErrchk(hipHostFree(p_ixn_count_));
    gpuErrchk(hipHostFree(p_box_));

    gpuErrchk(hipFree(d_nblist_x_));
    gpuErrchk(hipFree(d_nblist_box_));
    gpuErrchk(hipFree(d_rebuild_nblist_));
    gpuErrchk(hipHostFree(p_rebuild_nblist_));
};

template <typename RealType> void NonbondedInteractionGroup<RealType>::set_nblist_padding(double val) {
    nblist_padding_ = val;
}

template <typename RealType> void NonbondedInteractionGroup<RealType>::disable_hilbert_sort() {
    disable_hilbert_ = true;
}

template <typename RealType>
void NonbondedInteractionGroup<RealType>::hilbert_sort(
    const int N,
    const unsigned int *d_atom_idxs,
    const double *d_coords,
    const double *d_box,
    unsigned int *d_perm,
    hipStream_t stream) {

    const int tpb = warp_size;
    const int B = ceil_divide(N, tpb);

    k_coords_to_kv_gather<<<B, tpb, 0, stream>>>(
        N, d_atom_idxs, d_coords, d_box, d_bin_to_idx_, d_sort_keys_in_, d_sort_vals_in_);

    gpuErrchk(hipPeekAtLastError());

    hipcub::DeviceRadixSort::SortPairs(
        d_sort_storage_,
        d_sort_storage_bytes_,
        d_sort_keys_in_,
        d_sort_keys_out_,
        d_sort_vals_in_,
        d_perm,
        N,
        0,                            // begin bit
        sizeof(*d_sort_keys_in_) * 8, // end bit
        stream                        // cudaStream
    );

    gpuErrchk(hipPeekAtLastError());
}

template <typename RealType>
void NonbondedInteractionGroup<RealType>::execute_device(
    const int N,
    const int P,
    const double *d_x,
    const double *d_p,   // N * PARAMS_PER_ATOM
    const double *d_box, // 3 * 3
    unsigned long long *d_du_dx,
    unsigned long long *d_du_dp,
    unsigned long long *d_u,
    hipStream_t stream) {

    // (ytz) the nonbonded algorithm proceeds as follows:

    // (done in constructor), construct a hilbert curve mapping each of the HILBERT_GRID_DIM x HILBERT_GRID_DIM x HILBERT_GRID_DIM cells into an index.
    // a. decide if we need to rebuild the neighborlist, if so:
    //     - look up which cell each particle belongs to, and its linear index along the hilbert curve.
    //     - use radix pair sort keyed on the hilbert index with values equal to the atomic index
    //     - resulting sorted values is the permutation array.
    //     - permute coords
    // b. else:
    //     - permute new coords
    // c. permute parameters
    // d. compute the nonbonded interactions using the neighborlist
    // e. inverse permute the forces, du/dps into the original index.
    // f. u is buffered into a per-particle array, and then reduced.

    if (N != N_) {
        throw std::runtime_error(
            "NonbondedInteractionGroup::execute_device(): expected N == N_, got N=" + std::to_string(N) +
            ", N_=" + std::to_string(N_));
    }

    if (P != N_ * PARAMS_PER_ATOM) {
        throw std::runtime_error(
            "NonbondedInteractionGroup::execute_device(): expected P == N_*" + std::to_string(PARAMS_PER_ATOM) +
            ", got P=" + std::to_string(P) + ", N_*" + std::to_string(PARAMS_PER_ATOM) + "=" +
            std::to_string(N_ * PARAMS_PER_ATOM));
    }

    const int tpb = warp_size;
    const int B = ceil_divide(N_, tpb);

    // (ytz) see if we need to rebuild the neighborlist.
    k_check_rebuild_coords_and_box<RealType>
        <<<B, tpb, 0, stream>>>(N_, d_x, d_nblist_x_, d_box, d_nblist_box_, nblist_padding_, d_rebuild_nblist_);

    gpuErrchk(hipPeekAtLastError());

    // we can optimize this away by doing the check on the GPU directly.
    gpuErrchk(hipMemcpyAsync(
        p_rebuild_nblist_, d_rebuild_nblist_, 1 * sizeof(*p_rebuild_nblist_), hipMemcpyDeviceToHost, stream));
    gpuErrchk(hipStreamSynchronize(stream)); // slow!

    if (p_rebuild_nblist_[0] > 0) {

        // (ytz): update the permutation index before building neighborlist, as the neighborlist is tied
        // to a particular sort order
        if (!disable_hilbert_) {
            this->hilbert_sort(NR_, d_row_atom_idxs_, d_x, d_box, d_perm_, stream);
            this->hilbert_sort(NC_, d_col_atom_idxs_, d_x, d_box, d_perm_ + NR_, stream);
        } else {
            gpuErrchk(hipMemcpyAsync(
                d_perm_, d_row_atom_idxs_, NR_ * sizeof(*d_row_atom_idxs_), hipMemcpyDeviceToDevice, stream));
            gpuErrchk(hipMemcpyAsync(
                d_perm_ + NR_, d_col_atom_idxs_, NC_ * sizeof(*d_col_atom_idxs_), hipMemcpyDeviceToDevice, stream));
        }

        // compute new coordinates
        k_gather<<<dim3(B, 3, 1), tpb, 0, stream>>>(N_, d_perm_, d_x, d_sorted_x_);
        gpuErrchk(hipPeekAtLastError());

        nblist_.build_nblist_device(N_, d_sorted_x_, d_box, cutoff_ + nblist_padding_, stream);
        gpuErrchk(hipMemcpyAsync(
            p_ixn_count_, nblist_.get_ixn_count(), 1 * sizeof(*p_ixn_count_), hipMemcpyDeviceToHost, stream));

        gpuErrchk(hipMemcpyAsync(p_box_, d_box, 3 * 3 * sizeof(*d_box), hipMemcpyDeviceToHost, stream));

        // this stream needs to be synchronized so we can be sure that p_ixn_count_ is properly set.
        gpuErrchk(hipStreamSynchronize(stream));

        // Verify that the cutoff and box size are valid together. If cutoff is greater than half the box
        // then a particle can interact with multiple periodic copies.
        const double db_cutoff = (cutoff_ + nblist_padding_) * 2;

        // Verify the width of the box in all dimensions is greater than twice the cutoff
        for (int i = 0; i < 3; i++) {
            if (p_box_[i * 3 + i] < db_cutoff) {
                throw std::runtime_error(
                    "Cutoff with padding is more than half of the box width, neighborlist is no longer reliable");
            }
        }

        gpuErrchk(hipMemsetAsync(d_rebuild_nblist_, 0, sizeof(*d_rebuild_nblist_), stream));
        gpuErrchk(hipMemcpyAsync(d_nblist_x_, d_x, N * 3 * sizeof(*d_x), hipMemcpyDeviceToDevice, stream));
        gpuErrchk(hipMemcpyAsync(d_nblist_box_, d_box, 3 * 3 * sizeof(*d_box), hipMemcpyDeviceToDevice, stream));
    } else {
        k_gather<<<dim3(B, 3, 1), tpb, 0, stream>>>(N, d_perm_, d_x, d_sorted_x_);
        gpuErrchk(hipPeekAtLastError());
    }

    // if the neighborlist is empty, we can return early
    if (p_ixn_count_[0] == 0) {
        return;
    }

    k_gather<<<dim3(B, PARAMS_PER_ATOM, 1), tpb, 0, stream>>>(N, d_perm_, d_p, d_sorted_p_);
    gpuErrchk(hipPeekAtLastError());

    // reset buffers and sorted accumulators
    if (d_du_dx) {
        gpuErrchk(hipMemsetAsync(d_sorted_du_dx_, 0, N * 3 * sizeof(*d_sorted_du_dx_), stream))
    }
    if (d_du_dp) {
        gpuErrchk(hipMemsetAsync(d_sorted_du_dp_, 0, N * PARAMS_PER_ATOM * sizeof(*d_sorted_du_dp_), stream))
    }

    gpuErrchk(hipPeekAtLastError());

    // look up which kernel we need for this computation
    int kernel_idx = 0;
    kernel_idx |= d_du_dp ? 1 << 0 : 0;
    kernel_idx |= d_du_dx ? 1 << 1 : 0;
    kernel_idx |= d_u ? 1 << 2 : 0;

    kernel_ptrs_[kernel_idx]<<<p_ixn_count_[0], tpb, 0, stream>>>(
        N,
        nblist_.get_num_row_idxs(),
        d_sorted_x_,
        d_sorted_p_,
        d_box,
        beta_,
        cutoff_,
        nblist_.get_row_idxs(),
        nblist_.get_ixn_tiles(),
        nblist_.get_ixn_atoms(),
        d_sorted_du_dx_,
        d_sorted_du_dp_,
        d_u // switch to nullptr if we don't request energies
    );

    gpuErrchk(hipPeekAtLastError());

    // coords are N,3
    if (d_du_dx) {
        k_scatter_accum<<<dim3(B, 3, 1), tpb, 0, stream>>>(N, d_perm_, d_sorted_du_dx_, d_du_dx);
        gpuErrchk(hipPeekAtLastError());
    }

    // params are N, PARAMS_PER_ATOM
    // this needs to be an accumulated permute
    if (d_du_dp) {
        k_scatter_assign<<<dim3(B, PARAMS_PER_ATOM, 1), tpb, 0, stream>>>(N, d_perm_, d_sorted_du_dp_, d_du_dp_buffer_);
        gpuErrchk(hipPeekAtLastError());
    }

    if (d_du_dp) {
        k_add_ull_to_ull<<<dim3(B, PARAMS_PER_ATOM, 1), tpb, 0, stream>>>(N, d_du_dp_buffer_, d_du_dp);
        gpuErrchk(hipPeekAtLastError());
    }
}

template <typename RealType>
void NonbondedInteractionGroup<RealType>::set_atom_idxs(const std::vector<int> &atom_idxs) {
    verify_atom_idxs(N_, atom_idxs);
    std::vector<unsigned int> unsigned_idxs = std::vector<unsigned int>(atom_idxs.begin(), atom_idxs.end());

    std::set<unsigned int> unique_row_atom_idxs(unique_idxs(unsigned_idxs));
    // compute set of column atoms as set difference
    std::vector<unsigned int> col_atom_idxs_v = get_indices_difference(N_, unique_row_atom_idxs);
    std::vector<unsigned int> row_atom_idxs_v(set_to_vector(unique_row_atom_idxs));
    DeviceBuffer<unsigned int> d_col(col_atom_idxs_v.size());
    DeviceBuffer<unsigned int> d_row(row_atom_idxs_v.size());
    d_col.copy_from(&col_atom_idxs_v[0]);
    d_row.copy_from(&row_atom_idxs_v[0]);

    hipStream_t stream = static_cast<hipStream_t>(0);
    this->set_atom_idxs_device(col_atom_idxs_v.size(), row_atom_idxs_v.size(), d_col.data, d_row.data, stream);
    gpuErrchk(hipStreamSynchronize(stream));
}

// set_atom_idxs_device is for use when idxs exist on the GPU already and are used as the new idxs to compute the neighborlist on.
template <typename RealType>
void NonbondedInteractionGroup<RealType>::set_atom_idxs_device(
    const int NC,
    const int NR,
    unsigned int *d_in_column_idxs,
    unsigned int *d_in_row_idxs,
    const hipStream_t stream) {
    if (NC + NR != N_) {
        throw std::runtime_error("Total of indices must equal N");
    }
    const size_t tpb = warp_size;

    // The indices must already be on the GPU and are copied into the potential's buffers.
    gpuErrchk(hipMemcpyAsync(
        d_col_atom_idxs_, d_in_column_idxs, NC * sizeof(*d_col_atom_idxs_), hipMemcpyDeviceToDevice, stream));
    gpuErrchk(hipMemcpyAsync(
        d_row_atom_idxs_, d_in_row_idxs, NR * sizeof(*d_row_atom_idxs_), hipMemcpyDeviceToDevice, stream));

    // The neighborlist does not use the indices directly, rather it takes a contiguous set of indices and the ixn group
    // potential will resort the correct particles into the corresponding arrays. We can use the leftover spaces in the
    // two d_*_atom_idxs_ arrays to store these nblist indices.
    // NOTE: The leftover column indices will store the row indices and vice versa.

    k_arange<<<ceil_divide(NR, tpb), tpb, 0, stream>>>(NR, d_col_atom_idxs_ + NC);
    gpuErrchk(hipPeekAtLastError());
    k_arange<<<ceil_divide(NC, tpb), tpb, 0, stream>>>(NC, d_row_atom_idxs_ + NR, NR);
    gpuErrchk(hipPeekAtLastError());

    // Force a NBlist rebuild
    gpuErrchk(hipMemsetAsync(d_rebuild_nblist_, 1, 1 * sizeof(*d_rebuild_nblist_), stream));

    // Offset into the ends of the arrays that now contain the row and column indices for the nblist
    nblist_.set_idxs_device(NC, NR, d_row_atom_idxs_ + NR, d_col_atom_idxs_ + NC, stream);

    // Update the row and column counts
    this->NR_ = NR;
    this->NC_ = NC;
}

template <typename RealType>
void NonbondedInteractionGroup<RealType>::du_dp_fixed_to_float(
    const int N, const int P, const unsigned long long *du_dp, double *du_dp_float) {

    for (int i = 0; i < N; i++) {
        const int idx = i * PARAMS_PER_ATOM;
        const int idx_charge = idx + PARAM_OFFSET_CHARGE;
        const int idx_sig = idx + PARAM_OFFSET_SIG;
        const int idx_eps = idx + PARAM_OFFSET_EPS;
        const int idx_w = idx + PARAM_OFFSET_W;

        du_dp_float[idx_charge] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DCHARGE>(du_dp[idx_charge]);
        du_dp_float[idx_sig] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DSIG>(du_dp[idx_sig]);
        du_dp_float[idx_eps] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DEPS>(du_dp[idx_eps]);
        du_dp_float[idx_w] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DW>(du_dp[idx_w]);
    }
}

template class NonbondedInteractionGroup<double>;
template class NonbondedInteractionGroup<float>;

} // namespace timemachine
