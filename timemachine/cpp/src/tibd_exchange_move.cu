#include "hip/hip_runtime.h"
#include "tibd_exchange_move.hpp"

#include "constants.hpp"
#include "gpu_utils.cuh"
#include "kernels/k_exchange.cuh"
#include "kernels/k_indices.cuh"
#include "kernels/k_nonbonded.cuh"
#include "kernels/k_probability.cuh"
#include "kernels/k_rotations.cuh"
#include "kernels/k_translations.cuh"
#include "math_utils.cuh"
#include "mol_utils.hpp"
#include <hipcub/hipcub.hpp>
#include <math.h>

namespace timemachine {

// Each step will have 6 values for a translation, first 3 is the inner translation and second 3 is outer translation
static const int TIBD_TRANSLATIONS_PER_STEP_XYZXYZ = 6;

template <typename RealType>
TIBDExchangeMove<RealType>::TIBDExchangeMove(
    const int N,
    const std::vector<int> ligand_idxs,
    const std::vector<std::vector<int>> &target_mols,
    const std::vector<double> &params,
    const double temperature,
    const double nb_beta,
    const double cutoff,
    const double radius,
    const int seed,
    const int num_proposals_per_move,
    const int interval,
    const int batch_size)
    : BDExchangeMove<RealType>(
          N,
          target_mols,
          params,
          temperature,
          nb_beta,
          cutoff,
          seed,
          num_proposals_per_move,
          interval,
          batch_size,
          TIBD_TRANSLATIONS_PER_STEP_XYZXYZ * num_proposals_per_move),
      radius_(static_cast<RealType>(radius)), inner_volume_(static_cast<RealType>((4.0 / 3.0) * M_PI * pow(radius, 3))),
      d_rand_states_(DEFAULT_THREADS_PER_BLOCK), d_inner_mols_count_(1), d_identify_indices_(this->num_target_mols_),
      d_partitioned_indices_(this->num_target_mols_), d_temp_storage_buffer_(0), d_center_(3),
      d_uniform_noise_buffer_(num_proposals_per_move), d_targeting_inner_vol_(this->batch_size_),
      d_ligand_idxs_(ligand_idxs), d_src_log_weights_(this->num_target_mols_ * this->batch_size_),
      d_dest_log_weights_(this->num_target_mols_ * this->batch_size_), d_inner_flags_(this->num_target_mols_),
      d_box_volume_(1), d_selected_translation_(this->batch_size_ * 3),
      d_sample_after_segment_offsets_(this->d_sample_segments_offsets_.length),
      d_weights_before_counts_(this->batch_size_), d_weights_after_counts_(this->batch_size_) {

    if (radius <= 0.0) {
        throw std::runtime_error("radius must be greater than 0.0");
    }
    if (d_uniform_noise_buffer_.length != this->d_quaternions_.length / this->QUATERNIONS_PER_STEP) {
        throw std::runtime_error("bug in the code: buffers with random values don't match in batch size");
    }

    // Add 4 to the seed provided to avoid correlating with the four other RNGs
    k_initialize_curand_states<<<
        ceil_divide(d_rand_states_.length, DEFAULT_THREADS_PER_BLOCK),
        DEFAULT_THREADS_PER_BLOCK,
        0>>>(static_cast<int>(d_rand_states_.length), seed + 4, d_rand_states_.data);
    gpuErrchk(hipPeekAtLastError());

    k_arange<<<ceil_divide(this->num_target_mols_, DEFAULT_THREADS_PER_BLOCK), DEFAULT_THREADS_PER_BLOCK, 0>>>(
        this->num_target_mols_, d_identify_indices_.data, 0);
    gpuErrchk(hipPeekAtLastError());

    size_t flagged_bytes = 0;
    // Setup buffer for doing the flagged partition
    gpuErrchk(hipcub::DevicePartition::Flagged(
        nullptr,
        flagged_bytes,
        d_identify_indices_.data,
        d_inner_flags_.data,
        d_partitioned_indices_.data,
        d_inner_mols_count_.data,
        this->num_target_mols_));

    size_t sum_bytes = 0;
    // Will need to compute prefix sums of the count of before and after weights to construct the segment offsets
    gpuErrchk(hipcub::DeviceScan::InclusiveSum(
        nullptr, sum_bytes, d_weights_before_counts_.data, this->d_sample_segments_offsets_.data, this->batch_size_));
    // Take the larger of the two to use as the temp storage data for CUB
    temp_storage_bytes_ = max(flagged_bytes, sum_bytes);

    // Zero out the sample segments offsets, the first index will always be zero and the inclusive sum will be offset by 1
    gpuErrchk(hipMemset(this->d_sample_segments_offsets_.data, 0, this->d_sample_segments_offsets_.size()));
    gpuErrchk(hipMemset(d_sample_after_segment_offsets_.data, 0, d_sample_after_segment_offsets_.size()));
    // Set the inner count to zero and target the inner at the start to ensure that calling `log_probability` produces
    // a zero
    gpuErrchk(hipMemset(d_inner_mols_count_.data, 0, d_inner_mols_count_.size()));
    std::vector<int> h_targeting_inner(d_targeting_inner_vol_.length, 1);
    d_targeting_inner_vol_.copy_from(&h_targeting_inner[0]);

    // Allocate char as temp_storage_bytes_ is in raw bytes and the type doesn't matter in practice.
    // Equivalent to DeviceBuffer<int> buf(temp_storage_bytes_ / sizeof(int))
    d_temp_storage_buffer_.realloc(temp_storage_bytes_);
}

template <typename RealType> TIBDExchangeMove<RealType>::~TIBDExchangeMove() {}

template <typename RealType>
void TIBDExchangeMove<RealType>::move(
    const int N,
    double *d_coords, // [N, 3]
    double *d_box,    // [3, 3]
    hipStream_t stream) {

    if (N != this->N_) {
        throw std::runtime_error("N != N_");
    }
    this->step_++;
    if (this->step_ % this->interval_ != 0) {
        return;
    }

    // Set the stream for the generators
    curandErrchk(hiprandSetStream(this->cr_rng_quat_, stream));
    curandErrchk(hiprandSetStream(this->cr_rng_translations_, stream));
    curandErrchk(hiprandSetStream(this->cr_rng_samples_, stream));
    curandErrchk(hiprandSetStream(this->cr_rng_mh_, stream));

    this->compute_initial_weights(N, d_coords, d_box, stream);

    // Copy the before log weights to the after weights, we will adjust incrementally afterwards
    gpuErrchk(hipMemcpyAsync(
        this->d_log_weights_after_.data,
        this->d_log_weights_before_.data,
        this->d_log_weights_after_.size(),
        hipMemcpyDeviceToDevice,
        stream));

    const int tpb = DEFAULT_THREADS_PER_BLOCK;
    const int mol_blocks = ceil_divide(this->num_target_mols_, tpb);
    const int sample_blocks = ceil_divide(this->batch_size_, tpb);

    dim3 atom_by_atom_grid(ceil_divide(N, tpb), this->mol_size_, 1);

    k_compute_centroid_of_atoms<RealType>
        <<<1, tpb, 0, stream>>>(static_cast<int>(d_ligand_idxs_.length), d_ligand_idxs_.data, d_coords, d_center_.data);
    gpuErrchk(hipPeekAtLastError());

    k_compute_box_volume<<<1, 1, 0, stream>>>(d_box, d_box_volume_.data);
    gpuErrchk(hipPeekAtLastError());

    k_flag_mols_inner_outer<RealType><<<mol_blocks, tpb, 0, stream>>>(
        this->num_target_mols_,
        this->d_atom_idxs_.data,
        this->d_mol_offsets_.data,
        d_center_.data,
        radius_ * radius_,
        d_coords,
        d_box,
        d_inner_flags_.data);
    gpuErrchk(hipPeekAtLastError());

    // Generate all noise upfront for all proposals within a move
    curandErrchk(templateCurandUniform(this->cr_rng_mh_, this->d_mh_noise_.data, this->d_mh_noise_.length));
    // Using the translations RNG from the BDExchangeMove to generate noise for the targeting probability
    curandErrchk(templateCurandUniform(
        this->cr_rng_translations_, this->d_uniform_noise_buffer_.data, this->d_uniform_noise_buffer_.length));
    curandErrchk(
        templateCurandNormal(this->cr_rng_quat_, this->d_quaternions_.data, this->d_quaternions_.length, 0.0, 1.0));
    curandErrchk(
        templateCurandUniform(this->cr_rng_samples_, this->d_sample_noise_.data, this->d_sample_noise_.length));
    k_generate_translations_inside_and_outside_sphere<<<1, d_rand_states_.length, 0, stream>>>(
        this->steps_per_move_, d_box, d_center_.data, radius_, d_rand_states_.data, this->d_translations_.data);
    gpuErrchk(hipPeekAtLastError());

    for (int step = 0; step < this->steps_per_move_; step++) {
        // To ensure determinism between running 1 step per move or K steps per move we have to partition each pass
        // Ordering is consistent, with the tail reversed.
        // https://nvlabs.github.io/cub/structcub_1_1_device_partition.html#a47515ec2a15804719db1b8f3b3124e43
        gpuErrchk(hipcub::DevicePartition::Flagged(
            d_temp_storage_buffer_.data,
            temp_storage_bytes_,
            d_identify_indices_.data,
            d_inner_flags_.data,
            d_partitioned_indices_.data,
            d_inner_mols_count_.data,
            this->num_target_mols_,
            stream));

        k_decide_targeted_moves<<<sample_blocks, tpb, 0, stream>>>(
            this->batch_size_,
            this->num_target_mols_,
            this->d_uniform_noise_buffer_.data + (step * this->batch_size_),
            d_inner_mols_count_.data,
            this->d_translations_.data + (step * TIBD_TRANSLATIONS_PER_STEP_XYZXYZ * this->batch_size_),
            d_targeting_inner_vol_.data,
            d_weights_before_counts_.data,
            d_weights_after_counts_.data,
            d_selected_translation_.data);
        gpuErrchk(hipPeekAtLastError());

        k_separate_weights_for_targeted<RealType><<<mol_blocks, tpb, 0, stream>>>(
            this->num_target_mols_,
            d_targeting_inner_vol_.data,
            d_inner_mols_count_.data,
            d_partitioned_indices_.data,
            this->d_log_weights_before_.data,
            d_src_log_weights_.data);
        gpuErrchk(hipPeekAtLastError());

        gpuErrchk(hipcub::DeviceScan::InclusiveSum(
            d_temp_storage_buffer_.data,
            temp_storage_bytes_,
            d_weights_before_counts_.data,
            this->d_sample_segments_offsets_.data + 1, // Offset by one as the first idx is always 0
            this->batch_size_,
            stream));

        gpuErrchk(hipcub::DeviceScan::InclusiveSum(
            d_temp_storage_buffer_.data,
            temp_storage_bytes_,
            d_weights_after_counts_.data,
            d_sample_after_segment_offsets_.data + 1, // Offset by one as the first idx is always 0
            this->batch_size_,
            stream));

        this->sampler_.sample_given_noise_device(
            this->num_target_mols_ * this->batch_size_,
            this->batch_size_,
            this->d_sample_segments_offsets_.data,
            this->d_log_weights_before_.data,
            this->d_sample_noise_.data + (step * this->num_target_mols_ * this->batch_size_),
            this->d_sampling_intermediate_.data,
            this->d_samples_.data,
            stream);

        this->logsumexp_.sum_device(
            this->num_target_mols_ * this->batch_size_,
            this->batch_size_,
            this->d_sample_segments_offsets_.data,
            d_src_log_weights_.data,
            this->d_lse_max_before_.data,
            this->d_lse_exp_sum_before_.data,
            stream);

        // Selected an index from the src weights, need to remap the samples idx to the mol indices
        k_adjust_sample_idxs<<<sample_blocks, tpb, 0, stream>>>(
            this->batch_size_,
            d_targeting_inner_vol_.data,
            d_inner_mols_count_.data,
            d_partitioned_indices_.data,
            this->d_samples_.data);
        gpuErrchk(hipPeekAtLastError());

        // Don't move translations into computation of the incremental, as different translations can be used
        // by different bias deletion movers (such as targeted insertion)
        // Don't scale the translations as they are computed to be within the region
        this->compute_incremental_weights_device(
            N,
            false,
            d_box,
            d_coords,
            this->d_quaternions_.data + (step * this->QUATERNIONS_PER_STEP),
            this->d_selected_translation_.data,
            stream);

        k_setup_destination_weights_for_targeted<RealType><<<mol_blocks, tpb, 0, stream>>>(
            this->num_target_mols_,
            this->d_samples_.data,
            d_targeting_inner_vol_.data,
            d_inner_mols_count_.data,
            d_partitioned_indices_.data,
            this->d_log_weights_after_.data,
            d_dest_log_weights_.data);
        gpuErrchk(hipPeekAtLastError());

        this->logsumexp_.sum_device(
            this->num_target_mols_ * this->batch_size_,
            this->batch_size_,
            this->d_sample_after_segment_offsets_.data,
            d_dest_log_weights_.data,
            this->d_lse_max_after_.data,
            this->d_lse_exp_sum_after_.data,
            stream);

        k_attempt_exchange_move_targeted<RealType><<<ceil_divide(this->num_target_mols_, tpb), tpb, 0, stream>>>(
            this->num_target_mols_,
            d_targeting_inner_vol_.data,
            d_inner_mols_count_.data,
            d_box_volume_.data,
            inner_volume_,
            this->d_mh_noise_.data + (step * this->batch_size_),
            this->d_samples_.data,
            this->d_lse_max_before_.data,
            this->d_lse_exp_sum_before_.data,
            this->d_lse_max_after_.data,
            this->d_lse_exp_sum_after_.data,
            this->d_target_mol_offsets_.data,
            this->d_intermediate_coords_.data,
            d_coords,
            this->d_log_weights_before_.data,
            this->d_log_weights_after_.data,
            d_inner_flags_.data,
            this->d_num_accepted_.data);
        gpuErrchk(hipPeekAtLastError());
        this->num_attempted_++;
    }
}

template <typename RealType>
std::array<std::vector<double>, 2>
TIBDExchangeMove<RealType>::move_host(const int N, const double *h_coords, const double *h_box) {

    const double box_vol = h_box[0 * 3 + 0] * h_box[1 * 3 + 1] * h_box[2 * 3 + 2];
    if (box_vol <= inner_volume_) {
        throw std::runtime_error("volume of inner radius greater than box volume");
    }

    DeviceBuffer<double> d_coords(N * 3);
    d_coords.copy_from(h_coords);

    DeviceBuffer<double> d_box(3 * 3);
    d_box.copy_from(h_box);

    hipStream_t stream = static_cast<hipStream_t>(0);

    this->move(N, d_coords.data, d_box.data, stream);
    gpuErrchk(hipStreamSynchronize(stream));

    std::vector<double> out_coords(d_coords.length);
    d_coords.copy_to(&out_coords[0]);

    std::vector<double> out_box(d_box.length);
    d_box.copy_to(&out_box[0]);

    return std::array<std::vector<double>, 2>({out_coords, out_box});
}

template <typename RealType> double TIBDExchangeMove<RealType>::raw_log_probability_host() {
    std::vector<RealType> h_log_exp_before(2);
    std::vector<RealType> h_log_exp_after(2);
    this->d_lse_max_before_.copy_to(&h_log_exp_before[0]);
    this->d_lse_exp_sum_before_.copy_to(&h_log_exp_before[1]);
    this->d_lse_max_after_.copy_to(&h_log_exp_after[0]);
    this->d_lse_exp_sum_after_.copy_to(&h_log_exp_after[1]);

    int h_targeting_inner_vol[1];
    d_targeting_inner_vol_.copy_to(h_targeting_inner_vol);

    int local_inner_count[1];
    d_inner_mols_count_.copy_to(local_inner_count);

    RealType h_box_vol;
    d_box_volume_.copy_to(&h_box_vol);

    RealType outer_vol = h_box_vol - inner_volume_;

    const RealType raw_log_acceptance = compute_raw_log_probability_targeted<RealType>(
        h_targeting_inner_vol[0],
        inner_volume_,
        outer_vol,
        local_inner_count[0],
        this->num_target_mols_,
        &h_log_exp_before[0],
        &h_log_exp_before[1],
        &h_log_exp_after[0],
        &h_log_exp_after[1]);

    return static_cast<double>(raw_log_acceptance);
}

template <typename RealType> double TIBDExchangeMove<RealType>::log_probability_host() {
    return min(raw_log_probability_host(), 0.0);
}

template class TIBDExchangeMove<float>;
template class TIBDExchangeMove<double>;

} // namespace timemachine
