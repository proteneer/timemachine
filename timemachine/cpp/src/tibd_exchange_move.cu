#include "hip/hip_runtime.h"
#include "tibd_exchange_move.hpp"

#include "constants.hpp"
#include "gpu_utils.cuh"
#include "kernels/k_exchange.cuh"
#include "kernels/k_indices.cuh"
#include "kernels/k_nonbonded.cuh"
#include "kernels/k_probability.cuh"
#include "kernels/k_rotations.cuh"
#include "kernels/k_translations.cuh"
#include "math_utils.cuh"
#include "mol_utils.hpp"
#include <hipcub/hipcub.hpp>
#include <math.h>

namespace timemachine {

template <typename RealType>
TIBDExchangeMove<RealType>::TIBDExchangeMove(
    const int N,
    const std::vector<int> ligand_idxs,
    const std::vector<std::vector<int>> &target_mols,
    const std::vector<double> &params,
    const double temperature,
    const double nb_beta,
    const double cutoff,
    const double radius,
    const int seed,
    const int proposals_per_move,
    const int interval)
    : BDExchangeMove<RealType>(
          N, target_mols, params, temperature, nb_beta, cutoff, seed, proposals_per_move, interval),
      radius_(static_cast<RealType>(radius)), inner_volume_(static_cast<RealType>((4.0 / 3.0) * M_PI * pow(radius, 3))),
      d_rand_states_(DEFAULT_THREADS_PER_BLOCK), d_inner_mols_count_(1), d_identify_indices_(this->num_target_mols_),
      d_partitioned_indices_(this->num_target_mols_), d_temp_storage_buffer_(0), d_center_(3),
      d_uniform_noise_buffer_(round_up_even(2)), d_targeting_inner_vol_(1), d_ligand_idxs_(ligand_idxs),
      d_src_weights_(this->num_target_mols_), d_dest_weights_(this->num_target_mols_),
      d_inner_flags_(this->num_target_mols_), d_box_volume_(1), p_inner_count_(1), p_targeting_inner_vol_(1) {

    if (radius <= 0.0) {
        throw std::runtime_error("radius must be greater than 0.0");
    }

    // Create event with timings disabled as timings slow down events
    gpuErrchk(hipEventCreateWithFlags(&host_copy_event_, hipEventDisableTiming));

    k_initialize_curand_states<<<1, DEFAULT_THREADS_PER_BLOCK, 0>>>(
        DEFAULT_THREADS_PER_BLOCK, seed, d_rand_states_.data);
    gpuErrchk(hipPeekAtLastError());

    k_arange<<<ceil_divide(this->num_target_mols_, DEFAULT_THREADS_PER_BLOCK), DEFAULT_THREADS_PER_BLOCK, 0>>>(
        this->num_target_mols_, d_identify_indices_.data, 0);
    gpuErrchk(hipPeekAtLastError());

    // Setup buffer for doing the flagged partition
    gpuErrchk(hipcub::DevicePartition::Flagged(
        nullptr,
        temp_storage_bytes_,
        d_identify_indices_.data,
        d_inner_flags_.data,
        d_partitioned_indices_.data,
        d_inner_mols_count_.data,
        this->num_target_mols_));
    // Allocate char as temp_storage_bytes_ is in raw bytes and the type doesn't matter in practice.
    // Equivalent to DeviceBuffer<int> buf(temp_storage_bytes_ / sizeof(int))
    d_temp_storage_buffer_.realloc(temp_storage_bytes_);
}

template <typename RealType> TIBDExchangeMove<RealType>::~TIBDExchangeMove() {
    gpuErrchk(hipEventDestroy(host_copy_event_));
}

template <typename RealType>
void TIBDExchangeMove<RealType>::move(
    const int N,
    double *d_coords, // [N, 3]
    double *d_box,    // [3, 3]
    hipStream_t stream) {

    if (N != this->N_) {
        throw std::runtime_error("N != N_");
    }
    this->step_++;
    if (this->step_ % this->interval_ != 0) {
        return;
    }

    // Set the stream for the generator
    curandErrchk(hiprandSetStream(this->cr_rng_, stream));

    this->compute_initial_weights(N, d_coords, d_box, stream);

    const int tpb = DEFAULT_THREADS_PER_BLOCK;
    const int mol_blocks = ceil_divide(this->num_target_mols_, tpb);

    dim3 atom_by_atom_grid(ceil_divide(N, tpb), this->mol_size_, 1);

    k_compute_centroid_of_atoms<RealType>
        <<<1, tpb, 0, stream>>>(static_cast<int>(d_ligand_idxs_.length), d_ligand_idxs_.data, d_coords, d_center_.data);
    gpuErrchk(hipPeekAtLastError());

    k_compute_box_volume<<<1, 1, 0, stream>>>(d_box, d_box_volume_.data);
    gpuErrchk(hipPeekAtLastError());

    for (int move = 0; move < this->proposals_per_move_; move++) {
        // Run only after the first pass, to maintain meaningful `log_probability_host` values
        if (move > 0) {
            // Run a separate kernel to replace the before log probs and weights with the after if accepted a move
            // Need the weights to sample a value and the log probs are just because they aren't expensive to copy
            k_store_accepted_log_probability_targeted<RealType><<<1, tpb, 0>>>(
                this->num_target_mols_,
                d_targeting_inner_vol_.data,
                d_box_volume_.data,
                inner_volume_,
                this->d_uniform_noise_buffer_.data + 1, // Offset to get the last value for the acceptance criteria
                this->d_log_sum_exp_before_.data,
                this->d_log_sum_exp_after_.data,
                this->d_log_weights_before_.data,
                this->d_log_weights_after_.data);
            gpuErrchk(hipPeekAtLastError());
        }

        k_flag_mols_inner_outer<RealType><<<mol_blocks, tpb, 0, stream>>>(
            this->num_target_mols_,
            this->d_atom_idxs_.data,
            this->d_mol_offsets_.data,
            d_center_.data,
            radius_ * radius_,
            d_coords,
            d_box,
            d_inner_flags_.data);
        gpuErrchk(hipPeekAtLastError());

        gpuErrchk(hipcub::DevicePartition::Flagged(
            d_temp_storage_buffer_.data,
            temp_storage_bytes_,
            d_identify_indices_.data,
            d_inner_flags_.data,
            d_partitioned_indices_.data,
            d_inner_mols_count_.data,
            this->num_target_mols_,
            stream));

        // The this->d_uniform_noise_buffer_ buffer contains the random value for determining where to insert and whether to accept the move
        curandErrchk(templateCurandUniform(
            this->cr_rng_, this->d_uniform_noise_buffer_.data, this->d_uniform_noise_buffer_.length));

        k_decide_targeted_move<<<1, 1, 0, stream>>>(
            this->num_target_mols_,
            this->d_uniform_noise_buffer_.data,
            d_inner_mols_count_.data,
            d_targeting_inner_vol_.data);
        gpuErrchk(hipPeekAtLastError());

        // Copy count and flag to the host, needed to know how many values to look at for
        // sampling and logsumexp
        gpuErrchk(hipMemcpyAsync(
            p_inner_count_.data, d_inner_mols_count_.data, d_inner_mols_count_.size(), hipMemcpyDeviceToHost, stream));
        gpuErrchk(hipMemcpyAsync(
            p_targeting_inner_vol_.data,
            d_targeting_inner_vol_.data,
            d_targeting_inner_vol_.size(),
            hipMemcpyDeviceToHost,
            stream));
        gpuErrchk(hipEventRecord(host_copy_event_, stream));

        // Copy the before log weights to the after weights, we will adjust the after weights incrementally
        gpuErrchk(hipMemcpyAsync(
            this->d_log_weights_after_.data,
            this->d_log_weights_before_.data,
            this->d_log_weights_after_.size(),
            hipMemcpyDeviceToDevice,
            stream));

        k_generate_translations_within_or_outside_a_sphere<<<1, tpb, 0, stream>>>(
            1,
            d_box,
            d_center_.data,
            d_targeting_inner_vol_.data,
            radius_,
            d_rand_states_.data,
            this->d_translations_.data);
        gpuErrchk(hipPeekAtLastError());

        k_separate_weights_for_targeted<RealType><<<mol_blocks, tpb, 0, stream>>>(
            this->num_target_mols_,
            d_targeting_inner_vol_.data,
            d_inner_mols_count_.data,
            d_partitioned_indices_.data,
            this->d_log_weights_before_.data,
            d_src_weights_.data);
        gpuErrchk(hipPeekAtLastError());

        gpuErrchk(hipEventSynchronize(host_copy_event_));
        int inner_count = p_inner_count_.data[0];

        // targeting_inner_vol == 1 indicates that we are targeting the inner volume, starting from the outer mols
        int targeting_inner_vol = p_targeting_inner_vol_.data[0];
        int src_count = targeting_inner_vol == 0 ? inner_count : this->num_target_mols_ - inner_count;
        int dest_count = this->num_target_mols_ - src_count;

        this->logsumexp_.sum_device(src_count, d_src_weights_.data, this->d_log_sum_exp_before_.data, stream);

        // Only sample one mol
        this->sampler_.sample_device(src_count, 1, d_src_weights_.data, this->d_samples_.data, stream);

        // Selected an index from the src weights, need to remap the samples idx to the mol indices
        k_adjust_sample_idx<<<1, 1, 0, stream>>>(
            d_targeting_inner_vol_.data, d_inner_mols_count_.data, d_partitioned_indices_.data, this->d_samples_.data);
        gpuErrchk(hipPeekAtLastError());

        // Don't move translations into computation of the incremental, as different translations can be used
        // by different bias deletion movers (such as targeted insertion)
        // Don't scale the translations as they are computed to be within the region
        this->compute_incremental_weights(N, false, d_coords, d_box, stream);

        k_setup_destination_weights_for_targeted<RealType><<<mol_blocks, tpb, 0, stream>>>(
            this->num_target_mols_,
            this->d_samples_.data,
            d_targeting_inner_vol_.data,
            d_inner_mols_count_.data,
            d_partitioned_indices_.data,
            this->d_log_weights_after_.data,
            d_dest_weights_.data);
        gpuErrchk(hipPeekAtLastError());

        // Add one to the destination count, as we just moved a mol there
        this->logsumexp_.sum_device(dest_count + 1, d_dest_weights_.data, this->d_log_sum_exp_after_.data, stream);

        k_attempt_exchange_move_targeted<RealType><<<ceil_divide(N, tpb), tpb, 0, stream>>>(
            N,
            d_targeting_inner_vol_.data,
            d_box_volume_.data,
            inner_volume_,
            this->d_uniform_noise_buffer_.data + 1, // Offset to get the last value for the acceptance criteria
            this->d_log_sum_exp_before_.data,
            this->d_log_sum_exp_after_.data,
            this->d_intermediate_coords_.data,
            d_coords,
            this->d_num_accepted_.data);
        gpuErrchk(hipPeekAtLastError());
        this->num_attempted_++;
    }
}

template <typename RealType>
std::array<std::vector<double>, 2>
TIBDExchangeMove<RealType>::move_host(const int N, const double *h_coords, const double *h_box) {

    const double box_vol = h_box[0 * 3 + 0] * h_box[1 * 3 + 1] * h_box[2 * 3 + 2];
    if (box_vol <= inner_volume_) {
        throw std::runtime_error("volume of inner radius greater than box volume");
    }

    DeviceBuffer<double> d_coords(N * 3);
    d_coords.copy_from(h_coords);

    DeviceBuffer<double> d_box(3 * 3);
    d_box.copy_from(h_box);

    hipStream_t stream = static_cast<hipStream_t>(0);

    this->move(N, d_coords.data, d_box.data, stream);
    gpuErrchk(hipStreamSynchronize(stream));

    std::vector<double> out_coords(d_coords.length);
    d_coords.copy_to(&out_coords[0]);

    std::vector<double> out_box(d_box.length);
    d_box.copy_to(&out_box[0]);

    return std::array<std::vector<double>, 2>({out_coords, out_box});
}

template <typename RealType> double TIBDExchangeMove<RealType>::log_probability_host() {
    std::vector<RealType> h_log_exp_before(2);
    std::vector<RealType> h_log_exp_after(2);
    this->d_log_sum_exp_before_.copy_to(&h_log_exp_before[0]);
    this->d_log_sum_exp_after_.copy_to(&h_log_exp_after[0]);

    int h_targeting_inner_vol;
    d_targeting_inner_vol_.copy_to(&h_targeting_inner_vol);

    RealType h_box_vol;
    d_box_volume_.copy_to(&h_box_vol);

    RealType before_log_prob = convert_nan_to_inf(compute_logsumexp_final(&h_log_exp_before[0]));
    RealType after_log_prob = convert_nan_to_inf(compute_logsumexp_final(&h_log_exp_after[0]));

    RealType outer_vol = h_box_vol - inner_volume_;

    RealType log_vol_prob = h_targeting_inner_vol == 1 ? log(inner_volume_) - log(h_box_vol - inner_volume_)
                                                       : log(h_box_vol - inner_volume_) - log(inner_volume_);

    double log_prob = min(static_cast<double>(before_log_prob - after_log_prob + log_vol_prob), 0.0);
    return log_prob;
}

template class TIBDExchangeMove<float>;
template class TIBDExchangeMove<double>;

} // namespace timemachine
