#include "hip/hip_runtime.h"
#include "tibd_exchange_move.hpp"

#include "constants.hpp"
#include "gpu_utils.cuh"
#include "kernels/k_exchange.cuh"
#include "kernels/k_indices.cuh"
#include "kernels/k_nonbonded.cuh"
#include "kernels/k_probability.cuh"
#include "kernels/k_rotations.cuh"
#include "kernels/k_translations.cuh"
#include "math_utils.cuh"
#include "mol_utils.hpp"
#include <hipcub/hipcub.hpp>
#include <math.h>

namespace timemachine {

// NOISE_PER_STEP is the uniform generated per step that is used for deciding the targeted move as well as the acceptance
// in the metropolis hasting check.
static const int NOISE_PER_STEP = 2;
// Each step will have 6 values for a translation, first 3 is the inner translation and second 3 is outer translation
static const int TIBD_TRANSLATIONS_PER_STEP_XYZXYZ = 6;

template <typename RealType>
TIBDExchangeMove<RealType>::TIBDExchangeMove(
    const int N,
    const std::vector<int> ligand_idxs,
    const std::vector<std::vector<int>> &target_mols,
    const std::vector<double> &params,
    const double temperature,
    const double nb_beta,
    const double cutoff,
    const double radius,
    const int seed,
    const int proposals_per_move,
    const int interval)
    : BDExchangeMove<RealType>(
          N,
          target_mols,
          params,
          temperature,
          nb_beta,
          cutoff,
          seed,
          proposals_per_move,
          interval,
          round_up_even(TIBD_TRANSLATIONS_PER_STEP_XYZXYZ * proposals_per_move)),
      radius_(static_cast<RealType>(radius)), inner_volume_(static_cast<RealType>((4.0 / 3.0) * M_PI * pow(radius, 3))),
      d_rand_states_(DEFAULT_THREADS_PER_BLOCK), d_inner_mols_count_(1), d_identify_indices_(this->num_target_mols_),
      d_partitioned_indices_(this->num_target_mols_), d_temp_storage_buffer_(0), d_center_(3),
      d_uniform_noise_buffer_(round_up_even(NOISE_PER_STEP * this->proposals_per_move_ * this->samples_per_proposal_)),
      d_targeting_inner_vol_(this->samples_per_proposal_), d_ligand_idxs_(ligand_idxs),
      d_src_weights_(this->num_target_mols_), d_dest_weights_(this->num_target_mols_),
      d_inner_flags_(this->num_target_mols_), d_box_volume_(1), p_inner_count_(1), p_targeting_inner_vol_(1),
      d_selected_translation_(this->samples_per_proposal_ * 3),
      d_sample_after_segments_(this->d_sample_segments_offsets_.length),
      d_weights_before_counts_(this->samples_per_proposal_), d_weights_after_counts_(this->samples_per_proposal_) {

    if (radius <= 0.0) {
        throw std::runtime_error("radius must be greater than 0.0");
    }
    if (d_uniform_noise_buffer_.length / NOISE_PER_STEP != this->d_quaternions_.length / this->QUATERNIONS_PER_STEP) {
        throw std::runtime_error("bug in the code: buffers with random values don't match in batch size");
    }

    // Create event with timings disabled as timings slow down events
    gpuErrchk(hipEventCreateWithFlags(&host_copy_event_, hipEventDisableTiming));

    // Add 3 to the seed provided to avoid correlating with the three other RNGs
    k_initialize_curand_states<<<
        ceil_divide(d_rand_states_.length, DEFAULT_THREADS_PER_BLOCK),
        DEFAULT_THREADS_PER_BLOCK,
        0>>>(static_cast<int>(d_rand_states_.length), seed + 3, d_rand_states_.data);
    gpuErrchk(hipPeekAtLastError());

    k_arange<<<ceil_divide(this->num_target_mols_, DEFAULT_THREADS_PER_BLOCK), DEFAULT_THREADS_PER_BLOCK, 0>>>(
        this->num_target_mols_, d_identify_indices_.data, 0);
    gpuErrchk(hipPeekAtLastError());

    size_t flagged_bytes = 0;
    // Setup buffer for doing the flagged partition
    gpuErrchk(hipcub::DevicePartition::Flagged(
        nullptr,
        flagged_bytes,
        d_identify_indices_.data,
        d_inner_flags_.data,
        d_partitioned_indices_.data,
        d_inner_mols_count_.data,
        this->num_target_mols_));

    size_t sum_bytes = 0;
    // Will need to compute prefix sums of the count of before and after weights to construct the segment offsets
    gpuErrchk(hipcub::DeviceScan::InclusiveSum(
        nullptr,
        sum_bytes,
        d_weights_before_counts_.data,
        this->d_sample_segments_offsets_.data,
        this->samples_per_proposal_));
    // Take the larger of the two to use as the temp storage data for CUB
    temp_storage_bytes_ = max(flagged_bytes, sum_bytes);

    // Zero out the sample segments offsets, the first index will always be zero and the inclusive sum will be offset by 1
    gpuErrchk(hipMemset(this->d_sample_segments_offsets_.data, 0, this->d_sample_segments_offsets_.size()));

    // Allocate char as temp_storage_bytes_ is in raw bytes and the type doesn't matter in practice.
    // Equivalent to DeviceBuffer<int> buf(temp_storage_bytes_ / sizeof(int))
    d_temp_storage_buffer_.realloc(temp_storage_bytes_);

    // Set the inner count to zero and target the inner at the start to ensure that calling `log_probability` produces
    // a zero
    p_inner_count_.data[0] = 0;
    p_targeting_inner_vol_.data[0] = 1;
}

template <typename RealType> TIBDExchangeMove<RealType>::~TIBDExchangeMove() {
    gpuErrchk(hipEventDestroy(host_copy_event_));
}

template <typename RealType>
void TIBDExchangeMove<RealType>::move(
    const int N,
    double *d_coords, // [N, 3]
    double *d_box,    // [3, 3]
    hipStream_t stream) {

    if (N != this->N_) {
        throw std::runtime_error("N != N_");
    }
    this->step_++;
    if (this->step_ % this->interval_ != 0) {
        return;
    }

    // Set the stream for the generators
    curandErrchk(hiprandSetStream(this->cr_rng_quat_, stream));
    curandErrchk(hiprandSetStream(this->cr_rng_translations_, stream));
    curandErrchk(hiprandSetStream(this->cr_rng_samples_, stream));

    this->compute_initial_weights(N, d_coords, d_box, stream);

    // Copy the before log weights to the after weights, we will adjust incrementally afterwards
    gpuErrchk(hipMemcpyAsync(
        this->d_log_weights_after_.data,
        this->d_log_weights_before_.data,
        this->d_log_weights_after_.size(),
        hipMemcpyDeviceToDevice,
        stream));

    const int tpb = DEFAULT_THREADS_PER_BLOCK;
    const int mol_blocks = ceil_divide(this->num_target_mols_, tpb);
    const int sample_blocks = ceil_divide(this->samples_per_proposal_, tpb);

    dim3 atom_by_atom_grid(ceil_divide(N, tpb), this->mol_size_, 1);

    k_compute_centroid_of_atoms<RealType>
        <<<1, tpb, 0, stream>>>(static_cast<int>(d_ligand_idxs_.length), d_ligand_idxs_.data, d_coords, d_center_.data);
    gpuErrchk(hipPeekAtLastError());

    k_compute_box_volume<<<1, 1, 0, stream>>>(d_box, d_box_volume_.data);
    gpuErrchk(hipPeekAtLastError());

    k_flag_mols_inner_outer<RealType><<<mol_blocks, tpb, 0, stream>>>(
        this->num_target_mols_,
        this->d_atom_idxs_.data,
        this->d_mol_offsets_.data,
        d_center_.data,
        radius_ * radius_,
        d_coords,
        d_box,
        d_inner_flags_.data);
    gpuErrchk(hipPeekAtLastError());

    // Generate all noise upfront for all proposals within a move
    // Using the translations RNG from the BDExchangeMove to generate noise for the targeting probability and the acceptance criteria
    curandErrchk(templateCurandUniform(
        this->cr_rng_translations_, this->d_uniform_noise_buffer_.data, this->d_uniform_noise_buffer_.length));
    curandErrchk(
        templateCurandNormal(this->cr_rng_quat_, this->d_quaternions_.data, this->d_quaternions_.length, 0.0, 1.0));
    curandErrchk(
        templateCurandUniform(this->cr_rng_samples_, this->d_sample_noise_.data, this->d_sample_noise_.length));
    k_generate_translations_inside_and_outside_sphere<<<1, d_rand_states_.length, 0, stream>>>(
        this->proposals_per_move_, d_box, d_center_.data, radius_, d_rand_states_.data, this->d_translations_.data);
    gpuErrchk(hipPeekAtLastError());
    for (int step = 0; step < this->proposals_per_move_; step++) {
        // To ensure determinism between running 1 step per move or K steps per move we have to partition each pass
        // Ordering is consistent, with the tail reversed.
        // https://nvlabs.github.io/cub/structcub_1_1_device_partition.html#a47515ec2a15804719db1b8f3b3124e43
        gpuErrchk(hipcub::DevicePartition::Flagged(
            d_temp_storage_buffer_.data,
            temp_storage_bytes_,
            d_identify_indices_.data,
            d_inner_flags_.data,
            d_partitioned_indices_.data,
            d_inner_mols_count_.data,
            this->num_target_mols_,
            stream));

        k_decide_targeted_moves<<<sample_blocks, tpb, 0, stream>>>(
            this->samples_per_proposal_,
            this->num_target_mols_,
            this->d_uniform_noise_buffer_.data + (step * NOISE_PER_STEP * this->samples_per_proposal_),
            d_inner_mols_count_.data,
            this->d_translations_.data + (step * TIBD_TRANSLATIONS_PER_STEP_XYZXYZ * this->samples_per_proposal_),
            d_targeting_inner_vol_.data,
            d_weights_before_counts_.data,
            d_weights_after_counts_.data,
            d_selected_translation_.data);
        gpuErrchk(hipPeekAtLastError());

        // Copy count and flag to the host, needed to know how many values to look at for logsumexp
        gpuErrchk(hipMemcpyAsync(
            p_inner_count_.data, d_inner_mols_count_.data, d_inner_mols_count_.size(), hipMemcpyDeviceToHost, stream));
        gpuErrchk(hipMemcpyAsync(
            p_targeting_inner_vol_.data,
            d_targeting_inner_vol_.data,
            d_targeting_inner_vol_.size(),
            hipMemcpyDeviceToHost,
            stream));
        gpuErrchk(hipEventRecord(host_copy_event_, stream));

        k_separate_weights_for_targeted<RealType><<<mol_blocks, tpb, 0, stream>>>(
            this->num_target_mols_,
            d_targeting_inner_vol_.data,
            d_inner_mols_count_.data,
            d_partitioned_indices_.data,
            this->d_log_weights_before_.data,
            d_src_weights_.data);
        gpuErrchk(hipPeekAtLastError());

        gpuErrchk(hipcub::DeviceScan::InclusiveSum(
            d_temp_storage_buffer_.data,
            temp_storage_bytes_,
            d_weights_before_counts_.data,
            this->d_sample_segments_offsets_.data + 1, // Offset by one as the first idx is always 0
            this->samples_per_proposal_,
            stream));

        // Not yet necessary until SegmentedLogSumExp
        // gpuErrchk(hipcub::DeviceScan::InclusiveSum(
        //     d_temp_storage_buffer_.data,
        //     temp_storage_bytes_,
        //     d_weights_after_counts_.data,
        //     d_sample_after_segments_.data + 1, // Offset by one as the first idx is always 0
        //     this->samples_per_proposal_,
        //     stream));

        this->sampler_.sample_given_noise_device(
            this->num_target_mols_ * this->samples_per_proposal_,
            this->samples_per_proposal_,
            this->d_sample_segments_offsets_.data,
            this->d_log_weights_before_.data,
            this->d_sample_noise_.data + (step * this->num_target_mols_ * this->samples_per_proposal_),
            this->d_sampling_intermediate_.data,
            this->d_samples_.data,
            stream);

        gpuErrchk(hipEventSynchronize(host_copy_event_));
        int inner_count = p_inner_count_.data[0];

        // targeting_inner_vol == 1 indicates that we are targeting the inner volume, starting from the outer mols
        int targeting_inner_vol = p_targeting_inner_vol_.data[0];
        int src_count = targeting_inner_vol == 0 ? inner_count : this->num_target_mols_ - inner_count;
        int dest_count = this->num_target_mols_ - src_count;

        this->logsumexp_.sum_device(src_count, d_src_weights_.data, this->d_log_sum_exp_before_.data, stream);

        // Selected an index from the src weights, need to remap the samples idx to the mol indices
        k_adjust_sample_idxs<<<sample_blocks, tpb, 0, stream>>>(
            this->samples_per_proposal_,
            d_targeting_inner_vol_.data,
            d_inner_mols_count_.data,
            d_partitioned_indices_.data,
            this->d_samples_.data);
        gpuErrchk(hipPeekAtLastError());

        // Don't move translations into computation of the incremental, as different translations can be used
        // by different bias deletion movers (such as targeted insertion)
        // Don't scale the translations as they are computed to be within the region
        this->compute_incremental_weights(
            N,
            false,
            d_box,
            d_coords,
            this->d_quaternions_.data + (step * this->QUATERNIONS_PER_STEP),
            this->d_selected_translation_.data,
            stream);

        k_setup_destination_weights_for_targeted<RealType><<<mol_blocks, tpb, 0, stream>>>(
            this->num_target_mols_,
            this->d_samples_.data,
            d_targeting_inner_vol_.data,
            d_inner_mols_count_.data,
            d_partitioned_indices_.data,
            this->d_log_weights_after_.data,
            d_dest_weights_.data);
        gpuErrchk(hipPeekAtLastError());

        // Add one to the destination count, as we just moved a mol there
        this->logsumexp_.sum_device(dest_count + 1, d_dest_weights_.data, this->d_log_sum_exp_after_.data, stream);

        k_attempt_exchange_move_targeted<RealType><<<ceil_divide(N, tpb), tpb, 0, stream>>>(
            N,
            this->num_target_mols_,
            d_targeting_inner_vol_.data,
            d_inner_mols_count_.data,
            d_box_volume_.data,
            inner_volume_,
            // Offset to get the last value for the acceptance criteria
            this->d_uniform_noise_buffer_.data + (step * NOISE_PER_STEP) + (NOISE_PER_STEP - 1),
            this->d_samples_.data,
            this->d_log_sum_exp_before_.data,
            this->d_log_sum_exp_after_.data,
            this->d_intermediate_coords_.data,
            d_coords,
            this->d_log_weights_before_.data,
            this->d_log_weights_after_.data,
            d_inner_flags_.data,
            this->d_num_accepted_.data);
        gpuErrchk(hipPeekAtLastError());
        this->num_attempted_++;
    }
}

template <typename RealType>
std::array<std::vector<double>, 2>
TIBDExchangeMove<RealType>::move_host(const int N, const double *h_coords, const double *h_box) {

    const double box_vol = h_box[0 * 3 + 0] * h_box[1 * 3 + 1] * h_box[2 * 3 + 2];
    if (box_vol <= inner_volume_) {
        throw std::runtime_error("volume of inner radius greater than box volume");
    }

    DeviceBuffer<double> d_coords(N * 3);
    d_coords.copy_from(h_coords);

    DeviceBuffer<double> d_box(3 * 3);
    d_box.copy_from(h_box);

    hipStream_t stream = static_cast<hipStream_t>(0);

    this->move(N, d_coords.data, d_box.data, stream);
    gpuErrchk(hipStreamSynchronize(stream));

    std::vector<double> out_coords(d_coords.length);
    d_coords.copy_to(&out_coords[0]);

    std::vector<double> out_box(d_box.length);
    d_box.copy_to(&out_box[0]);

    return std::array<std::vector<double>, 2>({out_coords, out_box});
}

template <typename RealType> double TIBDExchangeMove<RealType>::raw_log_probability_host() {
    std::vector<RealType> h_log_exp_before(2);
    std::vector<RealType> h_log_exp_after(2);
    this->d_log_sum_exp_before_.copy_to(&h_log_exp_before[0]);
    this->d_log_sum_exp_after_.copy_to(&h_log_exp_after[0]);

    int h_targeting_inner_vol = p_targeting_inner_vol_.data[0];

    int local_inner_count = p_inner_count_.data[0];

    RealType h_box_vol;
    d_box_volume_.copy_to(&h_box_vol);

    RealType outer_vol = h_box_vol - inner_volume_;

    const RealType raw_log_acceptance = compute_raw_log_probability_targeted<RealType>(
        h_targeting_inner_vol,
        inner_volume_,
        outer_vol,
        local_inner_count,
        this->num_target_mols_,
        &h_log_exp_before[0],
        &h_log_exp_after[0]);

    return static_cast<double>(raw_log_acceptance);
}

template <typename RealType> double TIBDExchangeMove<RealType>::log_probability_host() {
    return min(raw_log_probability_host(), 0.0);
}

template class TIBDExchangeMove<float>;
template class TIBDExchangeMove<double>;

} // namespace timemachine
