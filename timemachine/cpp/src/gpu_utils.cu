#include "gpu_utils.cuh"

hipsparseStatus_t cusparseBsrmm(hipsparseHandle_t         handle,
               hipsparseDirection_t      dirA,
               hipsparseOperation_t      transA,
               hipsparseOperation_t      transB,
               int                      mb,
               int                      n,
               int                      kb,
               int                      nnzb,
               const float*             alpha,
               const hipsparseMatDescr_t descrA,
               const float*             bsrValA,
               const int*               bsrRowPtrA,
               const int*               bsrColIndA,
               int                      blockDim,
               const float*             B,
               int                      ldb,
               const float*             beta,
               float*                   C,
               int                      ldc) {
    return hipsparseSbsrmm(
        handle,
        dirA,
        transA,
        transB,
        mb,
        n,
        kb,
        nnzb,
        alpha,
        descrA,
        bsrValA,
        bsrRowPtrA,
        bsrColIndA,
        blockDim,
        B,
        ldb,
        beta,
        C,
        ldc);

}

hipsparseStatus_t cusparseBsrmm(hipsparseHandle_t         handle,
               hipsparseDirection_t      dirA,
               hipsparseOperation_t      transA,
               hipsparseOperation_t      transB,
               int                      mb,
               int                      n,
               int                      kb,
               int                      nnzb,
               const double*            alpha,
               const hipsparseMatDescr_t descrA,
               const double*            bsrValA,
               const int*               bsrRowPtrA,
               const int*               bsrColIndA,
               int                      blockDim,
               const double*            B,
               int                      ldb,
               const double*            beta,
               double*                  C,
               int                      ldc) {
    return hipsparseDbsrmm(
        handle,
        dirA,
        transA,
        transB,
        mb,
        n,
        kb,
        nnzb,
        alpha,
        descrA,
        bsrValA,
        bsrRowPtrA,
        bsrColIndA,
        blockDim,
        B,
        ldb,
        beta,
        C,
        ldc);

}



hipsparseStatus_t cusparseCsr2bsr(hipsparseHandle_t         handle,
    hipsparseDirection_t      dir,
    int                      m,
    int                      n,
    const hipsparseMatDescr_t descrA,
    const float*             csrValA,
    const int*               csrRowPtrA,
    const int*               csrColIndA,
    int                      blockDim,
    const hipsparseMatDescr_t descrC,
    float*                   bsrValC,
    int*                     bsrRowPtrC,
    int*                     bsrColIndC) {

    return hipsparseScsr2bsr(
        handle,
        dir,
        m,
        n,
        descrA,
        csrValA,
        csrRowPtrA,
        csrColIndA,
        blockDim,
        descrC,
        bsrValC,
        bsrRowPtrC,
        bsrColIndC);

}

hipsparseStatus_t cusparseCsr2bsr(hipsparseHandle_t         handle,
    hipsparseDirection_t      dir,
    int                      m,
    int                      n,
    const hipsparseMatDescr_t descrA,
    const double*             csrValA,
    const int*               csrRowPtrA,
    const int*               csrColIndA,
    int                      blockDim,
    const hipsparseMatDescr_t descrC,
    double*                   bsrValC,
    int*                     bsrRowPtrC,
    int*                     bsrColIndC) {

    return hipsparseDcsr2bsr(
        handle,
        dir,
        m,
        n,
        descrA,
        csrValA,
        csrRowPtrA,
        csrColIndA,
        blockDim,
        descrC,
        bsrValC,
        bsrRowPtrC,
        bsrColIndC);

}


hipsparseStatus_t cusparseCsrgemm2_bufferSizeExt(hipsparseHandle_t         handle,
                                int                      m,
                                int                      n,
                                int                      k,
                                const float*             alpha,
                                const hipsparseMatDescr_t descrA,
                                int                      nnzA,
                                const int*               csrRowPtrA,
                                const int*               csrColIndA,
                                const hipsparseMatDescr_t descrB,
                                int                      nnzB,
                                const int*               csrRowPtrB,
                                const int*               csrColIndB,
                                const float*             beta,
                                const hipsparseMatDescr_t descrD,
                                int                      nnzD,
                                const int*               csrRowPtrD,
                                const int*               csrColIndD,
                                csrgemm2Info_t           info,
                                size_t*                  pBufferSizeInBytes) {
    return hipsparseScsrgemm2_bufferSizeExt(handle,
        m,
        n,
        k,
        alpha,
        descrA,
        nnzA,
        csrRowPtrA,
        csrColIndA,
        descrB,
        nnzB,
        csrRowPtrB,
        csrColIndB,
        beta,
        descrD,
        nnzD,
        csrRowPtrD,
        csrColIndD,
        info,
        pBufferSizeInBytes);

}

hipsparseStatus_t cusparseCsrgemm2_bufferSizeExt(hipsparseHandle_t         handle,
                                int                      m,
                                int                      n,
                                int                      k,
                                const double*            alpha,
                                const hipsparseMatDescr_t descrA,
                                int                      nnzA,
                                const int*               csrRowPtrA,
                                const int*               csrColIndA,
                                const hipsparseMatDescr_t descrB,
                                int                      nnzB,
                                const int*               csrRowPtrB,
                                const int*               csrColIndB,
                                const double*            beta,
                                const hipsparseMatDescr_t descrD,
                                int                      nnzD,
                                const int*               csrRowPtrD,
                                const int*               csrColIndD,
                                csrgemm2Info_t           info,
                                size_t*                  pBufferSizeInBytes) {
    return hipsparseDcsrgemm2_bufferSizeExt(handle,
        m,
        n,
        k,
        alpha,
        descrA,
        nnzA,
        csrRowPtrA,
        csrColIndA,
        descrB,
        nnzB,
        csrRowPtrB,
        csrColIndB,
        beta,
        descrD,
        nnzD,
        csrRowPtrD,
        csrColIndD,
        info,
        pBufferSizeInBytes);

}

hipsparseStatus_t
cusparseCsrgemm(hipsparseHandle_t        handle,
    hipsparseOperation_t      transA,
    hipsparseOperation_t      transB,
    int                      m,
    int                      n,
    int                      k,
    const hipsparseMatDescr_t descrA,
    int                      nnzA,
    const float*             csrValA,
    const int*               csrRowPtrA,
    const int*               csrColIndA,
    const hipsparseMatDescr_t descrB,
    int                      nnzB,
    const float*             csrValB,
    const int*               csrRowPtrB,
    const int*               csrColIndB,
    const hipsparseMatDescr_t descrC,
    float*                   csrValC,
    const int*               csrRowPtrC,
    int*                     csrColIndC) {
    return hipsparseScsrgemm(handle,
         transA,
         transB,
         m,
         n,
         k,
         descrA,
         nnzA,
         csrValA,
         csrRowPtrA,
         csrColIndA,
         descrB,
         nnzB,
         csrValB,
         csrRowPtrB,
         csrColIndB,
         descrC,
         csrValC,
         csrRowPtrC,
         csrColIndC);
}

hipsparseStatus_t
cusparseCsrgemm(hipsparseHandle_t        handle,
    hipsparseOperation_t      transA,
    hipsparseOperation_t      transB,
    int                      m,
    int                      n,
    int                      k,
    const hipsparseMatDescr_t descrA,
    int                      nnzA,
    const double*            csrValA,
    const int*               csrRowPtrA,
    const int*               csrColIndA,
    const hipsparseMatDescr_t descrB,
    int                      nnzB,
    const double*            csrValB,
    const int*               csrRowPtrB,
    const int*               csrColIndB,
    const hipsparseMatDescr_t descrC,
    double*                  csrValC,
    const int*               csrRowPtrC,
    int*                     csrColIndC) {
    return hipsparseDcsrgemm(handle,
         transA,
         transB,
         m,
         n,
         k,
         descrA,
         nnzA,
         csrValA,
         csrRowPtrA,
         csrColIndA,
         descrB,
         nnzB,
         csrValB,
         csrRowPtrB,
         csrColIndB,
         descrC,
         csrValC,
         csrRowPtrC,
         csrColIndC);
}

hipsparseStatus_t cusparseDense2csr(hipsparseHandle_t         handle,
                   int                      m,
                   int                      n,
                   const hipsparseMatDescr_t descrA,
                   const float*             A,
                   int                      lda,
                   const int*               nnzPerRow,
                   float*                   csrValA,
                   int*                     csrRowPtrA,
                   int*                     csrColIndA) {
    return hipsparseSdense2csr(handle,
        m,
        n,
        descrA,
        A,
        lda,
        nnzPerRow,
        csrValA,
        csrRowPtrA,
        csrColIndA);
}

hipsparseStatus_t cusparseDense2csr(hipsparseHandle_t         handle,
                   int                      m,
                   int                      n,
                   const hipsparseMatDescr_t descrA,
                   const double*             A,
                   int                      lda,
                   const int*               nnzPerRow,
                   double*                   csrValA,
                   int*                     csrRowPtrA,
                   int*                     csrColIndA) {
    return hipsparseDdense2csr(handle,
        m,
        n,
        descrA,
        A,
        lda,
        nnzPerRow,
        csrValA,
        csrRowPtrA,
        csrColIndA);
}

hipsparseStatus_t cusparseNnz(hipsparseHandle_t handle,
    hipsparseDirection_t     dirA,
    int                      m,
    int                      n,
    const hipsparseMatDescr_t descrA,
    const double*             A,
    int                      lda,
    int*                     nnzPerRowColumn,
    int*                     nnzTotalDevHostPtr) {

    return hipsparseDnnz(handle,
             dirA,
             m,
             n,
             descrA,
             A,
             lda,
             nnzPerRowColumn,
             nnzTotalDevHostPtr);

}

hipsparseStatus_t cusparseNnz(hipsparseHandle_t handle,
    hipsparseDirection_t     dirA,
    int                      m,
    int                      n,
    const hipsparseMatDescr_t descrA,
    const float*             A,
    int                      lda,
    int*                     nnzPerRowColumn,
    int*                     nnzTotalDevHostPtr) {

    return hipsparseSnnz(handle,
             dirA,
             m,
             n,
             descrA,
             A,
             lda,
             nnzPerRowColumn,
             nnzTotalDevHostPtr);

}

hipsparseStatus_t cusparseNnz(hipsparseHandle_t handle,
    hipsparseDirection_t     dirA,
    int                      m,
    int                      n,
    const hipsparseMatDescr_t descrA,
    const float*             A,
    int                      lda,
    int*                     nnzPerRowColumn,
    int*                     nnzTotalDevHostPtr);


hipblasStatus_t templateSymm(hipblasHandle_t handle,
                           hipblasSideMode_t side, hipblasFillMode_t uplo,
                           int m, int n,
                           const float           *alpha,
                           const float           *A, int lda,
                           const float           *B, int ldb,
                           const float           *beta,
                           float           *C, int ldc) {

  return hipblasSsymm(handle,
    side, uplo,
    m, n,
    alpha,
    A, lda,
    B, ldb, 
    beta,
    C, ldc);
}

hipblasStatus_t templateSymm(hipblasHandle_t handle,
                           hipblasSideMode_t side, hipblasFillMode_t uplo,
                           int m, int n,
                           const double           *alpha,
                           const double           *A, int lda,
                           const double           *B, int ldb,
                           const double           *beta,
                           double           *C, int ldc) {

  return hipblasDsymm(handle,
    side, uplo,
    m, n,
    alpha,
    A, lda,
    B, ldb, 
    beta,
    C, ldc);
}

hipblasStatus_t templateGemm(hipblasHandle_t handle,
   hipblasOperation_t transa, hipblasOperation_t transb,
   int m, int n, int k,
   const float           *alpha,
   const float           *A, int lda,
   const float           *B, int ldb,
   const float           *beta,
   float           *C, int ldc) {
   return hipblasSgemm(handle,
   transa, transb,
   m, n, k,
   alpha,
   A, lda,
   B, ldb,
   beta,
   C, ldc);
}

hipblasStatus_t templateGemm(hipblasHandle_t handle,
   hipblasOperation_t transa, hipblasOperation_t transb,
   int m, int n, int k,
   const double           *alpha,
   const double           *A, int lda,
   const double           *B, int ldb,
   const double           *beta,
   double           *C, int ldc) {
   return hipblasDgemm(handle,
   transa, transb,
   m, n, k,
   alpha,
   A, lda,
   B, ldb,
   beta,
   C, ldc);
}

hiprandStatus_t templateCurandNormal(
    hiprandGenerator_t generator, 
    float *outputPtr, size_t n, 
    float mean, float stddev) {
    return hiprandGenerateNormal(generator, outputPtr, n, mean, stddev);
}

// #include <iostream> 
hiprandStatus_t templateCurandNormal(
    hiprandGenerator_t generator, 
    double *outputPtr, size_t n, 
    double mean, double stddev) {
    // std::cout << "N DOUBLE" << n << std::endl;
    return hiprandGenerateNormalDouble(generator, outputPtr, n, mean, stddev);
}
