#include "hip/hip_runtime.h"
#include "gpu_utils.cuh"

int round_up_even(int count) { return count + (count % 2); }

hiprandStatus_t templateCurandNormal(hiprandGenerator_t generator, float *outputPtr, size_t n, float mean, float stddev) {
    return hiprandGenerateNormal(generator, outputPtr, n, mean, stddev);
}

hiprandStatus_t
templateCurandNormal(hiprandGenerator_t generator, double *outputPtr, size_t n, double mean, double stddev) {
    return hiprandGenerateNormalDouble(generator, outputPtr, n, mean, stddev);
}

void __global__ k_accumulate_energy(
    int N,
    const __int128 *__restrict__ input_buffer, // [N]
    __int128 *__restrict u_buffer              // [1]
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Should only have 1 idx, could have a warp that does a parallel reduce in the future
    if (idx >= 1) {
        return;
    }
    // Read out the value from the u_buffer
    __int128 accum = 0;
    while (idx < N) {
        accum += input_buffer[idx];
        idx += blockDim.x * gridDim.x;
    }
    // This could be a race condition if multiple `k_accumulate_energy` are running
    // on the same u_buffer
    u_buffer[0] += accum;
}
