#include "hip/hip_runtime.h"
#include "gpu_utils.cuh"

namespace timemachine {

int round_up_even(int count) { return count + (count % 2); }

hiprandStatus_t templateCurandNormal(hiprandGenerator_t generator, float *outputPtr, size_t n, float mean, float stddev) {
    return hiprandGenerateNormal(generator, outputPtr, n, mean, stddev);
}

hiprandStatus_t
templateCurandNormal(hiprandGenerator_t generator, double *outputPtr, size_t n, double mean, double stddev) {
    return hiprandGenerateNormalDouble(generator, outputPtr, n, mean, stddev);
}

hiprandStatus_t templateCurandUniform(hiprandGenerator_t generator, float *outputPtr, size_t n) {
    return hiprandGenerateUniform(generator, outputPtr, n);
}

hiprandStatus_t templateCurandUniform(hiprandGenerator_t generator, double *outputPtr, size_t n) {
    return hiprandGenerateUniformDouble(generator, outputPtr, n);
}

void __global__ k_initialize_curand_states(const int count, const int seed, hiprandState_t *states) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    while (idx < count) {
        hiprand_init(seed + idx, 0, 0, &states[idx]);
        idx += gridDim.x * blockDim.x;
    }
}

} // namespace timemachine
