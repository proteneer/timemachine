#include "gpu_utils.cuh"

int round_up_even(int count) {
    if (count % 2 == 1) {
        return count += 1;
    } else {
        return count;
    }
}

hiprandStatus_t templateCurandNormal(hiprandGenerator_t generator, float *outputPtr, size_t n, float mean, float stddev) {
    return hiprandGenerateNormal(generator, outputPtr, n, mean, stddev);
}

hiprandStatus_t
templateCurandNormal(hiprandGenerator_t generator, double *outputPtr, size_t n, double mean, double stddev) {
    return hiprandGenerateNormalDouble(generator, outputPtr, n, mean, stddev);
}
