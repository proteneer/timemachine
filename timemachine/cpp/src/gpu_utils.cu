#include "gpu_utils.cuh"

hiprandStatus_t templateCurandNormal(
    hiprandGenerator_t generator, 
    float *outputPtr, size_t n, 
    float mean, float stddev) {
    return hiprandGenerateNormal(generator, outputPtr, n, mean, stddev);
}

// #include <iostream> 
hiprandStatus_t templateCurandNormal(
    hiprandGenerator_t generator, 
    double *outputPtr, size_t n, 
    double mean, double stddev) {
    // std::cout << "N DOUBLE" << n << std::endl;
    return hiprandGenerateNormalDouble(generator, outputPtr, n, mean, stddev);
}
