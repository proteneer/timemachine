#include "gpu_utils.cuh"

hiprandStatus_t templateCurandNormal(
    hiprandGenerator_t generator,
    float *outputPtr, size_t n,
    float mean, float stddev) {
    return hiprandGenerateNormal(generator, outputPtr, n, mean, stddev);
}

hiprandStatus_t templateCurandNormal(
    hiprandGenerator_t generator,
    double *outputPtr, size_t n,
    double mean, double stddev) {
    return hiprandGenerateNormalDouble(generator, outputPtr, n, mean, stddev);
}
