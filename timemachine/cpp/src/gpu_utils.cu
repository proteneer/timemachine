#include "gpu_utils.cuh"



// hipblasStatus_t hipblasSgemm(hipblasHandle_t handle,
//                            hipblasOperation_t transa, hipblasOperation_t transb,
//                            int m, int n, int k,
//                            const float           *alpha,
//                            const float           *A, int lda,
//                            const float           *B, int ldb,
//                            const float           *beta,
//                            float           *C, int ldc)

// hipblasStatus_t hipblasSsymm(hipblasHandle_t handle,
//                            hipblasSideMode_t side, hipblasFillMode_t uplo,
//                            int m, int n,
//                            const float           *alpha,
//                            const float           *A, int lda,
//                            const float           *B, int ldb,
//                            const float           *beta,
//                            float           *C, int ldc)

hipblasStatus_t templateSymm(hipblasHandle_t handle,
                           hipblasSideMode_t side, hipblasFillMode_t uplo,
                           int m, int n,
                           const float           *alpha,
                           const float           *A, int lda,
                           const float           *B, int ldb,
                           const float           *beta,
                           float           *C, int ldc) {

  return hipblasSsymm(handle,
    side, uplo,
    m, n,
    alpha,
    A, lda,
    B, ldb, 
    beta,
    C, ldc);
}

hipblasStatus_t templateSymm(hipblasHandle_t handle,
                           hipblasSideMode_t side, hipblasFillMode_t uplo,
                           int m, int n,
                           const double           *alpha,
                           const double           *A, int lda,
                           const double           *B, int ldb,
                           const double           *beta,
                           double           *C, int ldc) {

  return hipblasDsymm(handle,
    side, uplo,
    m, n,
    alpha,
    A, lda,
    B, ldb, 
    beta,
    C, ldc);
}

hipblasStatus_t templateGemm(hipblasHandle_t handle,
   hipblasOperation_t transa, hipblasOperation_t transb,
   int m, int n, int k,
   const float           *alpha,
   const float           *A, int lda,
   const float           *B, int ldb,
   const float           *beta,
   float           *C, int ldc) {
   return hipblasSgemm(handle,
   transa, transb,
   m, n, k,
   alpha,
   A, lda,
   B, ldb,
   beta,
   C, ldc);
}

hipblasStatus_t templateGemm(hipblasHandle_t handle,
   hipblasOperation_t transa, hipblasOperation_t transb,
   int m, int n, int k,
   const double           *alpha,
   const double           *A, int lda,
   const double           *B, int ldb,
   const double           *beta,
   double           *C, int ldc) {
   return hipblasDgemm(handle,
   transa, transb,
   m, n, k,
   alpha,
   A, lda,
   B, ldb,
   beta,
   C, ldc);
}

hiprandStatus_t templateCurandNormal(
    hiprandGenerator_t generator, 
    float *outputPtr, size_t n, 
    float mean, float stddev) {
    return hiprandGenerateNormal(generator, outputPtr, n, mean, stddev);
}

// #include <iostream> 
hiprandStatus_t templateCurandNormal(
    hiprandGenerator_t generator, 
    double *outputPtr, size_t n, 
    double mean, double stddev) {
    // std::cout << "N DOUBLE" << n << std::endl;
    return hiprandGenerateNormalDouble(generator, outputPtr, n, mean, stddev);
}
