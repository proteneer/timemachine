#include "gpu_utils.cuh"

namespace timemachine {

int round_up_even(int count) { return count + (count % 2); }

hiprandStatus_t templateCurandNormal(hiprandGenerator_t generator, float *outputPtr, size_t n, float mean, float stddev) {
    return hiprandGenerateNormal(generator, outputPtr, n, mean, stddev);
}

hiprandStatus_t
templateCurandNormal(hiprandGenerator_t generator, double *outputPtr, size_t n, double mean, double stddev) {
    return hiprandGenerateNormalDouble(generator, outputPtr, n, mean, stddev);
}

hiprandStatus_t templateCurandUniform(hiprandGenerator_t generator, float *outputPtr, size_t n) {
    return hiprandGenerateUniform(generator, outputPtr, n);
}

hiprandStatus_t templateCurandUniform(hiprandGenerator_t generator, double *outputPtr, size_t n) {
    return hiprandGenerateUniformDouble(generator, outputPtr, n);
}

} // namespace timemachine
