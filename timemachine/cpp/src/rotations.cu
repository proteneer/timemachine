#include "device_buffer.hpp"
#include "gpu_utils.cuh"
#include "kernels/k_rotations.cuh"
#include "kernels/kernel_utils.cuh"
#include "math_utils.cuh"
#include "rotations.hpp"

namespace timemachine {

// rotate_coordinates_host is used to test the rotation kernel
template <typename RealType>
void rotate_coordinates_host(
    const int N, const int n_rotations, const double *coords, const RealType *quaternions, double *output) {
    DeviceBuffer<double> d_coords(N * 3);
    d_coords.copy_from(coords);

    DeviceBuffer<RealType> d_quats(n_rotations * 4);
    d_quats.copy_from(quaternions);

    DeviceBuffer<double> d_out(n_rotations * N * 3);

    const int tpb = DEFAULT_THREADS_PER_BLOCK;

    dim3 dimGrid(ceil_divide(n_rotations, tpb), N, 1);

    hipStream_t stream = static_cast<hipStream_t>(0);

    k_rotate_coordinates<RealType>
        <<<dimGrid, tpb, 0, stream>>>(N, n_rotations, d_coords.data, d_quats.data, d_out.data);
    gpuErrchk(hipStreamSynchronize(stream));
    d_out.copy_to(output);
}

template <typename RealType>
void rotate_coordinates_and_translate_mol_host(
    const int N,
    const int batch_size,
    const double *mol_coords,
    const double *box,
    const RealType *quaternion,
    const RealType *translation,
    double *output) {
    DeviceBuffer<double> d_coords(N * 3);
    d_coords.copy_from(mol_coords);

    DeviceBuffer<double> d_box(3 * 3);
    d_box.copy_from(box);

    DeviceBuffer<RealType> d_quaternion(batch_size * 4);
    d_quaternion.copy_from(quaternion);

    DeviceBuffer<RealType> d_translation(batch_size * 3);
    d_translation.copy_from(translation);

    int mol_offsets[2] = {0, N};
    DeviceBuffer<int> d_mol_offsets(2);
    d_mol_offsets.copy_from(mol_offsets);

    DeviceBuffer<int> d_offset(1);
    DeviceBuffer<int> d_samples(batch_size);
    // Set the sample to the first (0 index)
    gpuErrchk(hipMemset(d_samples.data, 0, d_samples.size()));
    gpuErrchk(hipMemset(d_offset.data, 0, d_offset.size()));

    DeviceBuffer<double> d_out(d_coords.length * batch_size);

    const int tpb = DEFAULT_THREADS_PER_BLOCK;

    hipStream_t stream = static_cast<hipStream_t>(0);

    k_rotate_and_translate_mols<RealType, true><<<ceil_divide(batch_size, tpb), tpb, 0, stream>>>(
        batch_size,
        batch_size,
        d_offset.data,
        d_coords.data,
        d_box.data,
        d_samples.data,
        d_mol_offsets.data,
        d_quaternion.data,
        d_translation.data,
        d_out.data);
    gpuErrchk(hipStreamSynchronize(stream));
    d_out.copy_to(output);
}

template void rotate_coordinates_host<float>(const int, const int, const double *, const float *, double *);
template void rotate_coordinates_host<double>(const int, const int, const double *, const double *, double *);

template void rotate_coordinates_and_translate_mol_host<float>(
    const int, const int, const double *, const double *, const float *, const float *, double *);
template void rotate_coordinates_and_translate_mol_host<double>(
    const int, const int, const double *, const double *, const double *, const double *, double *);

} // namespace timemachine
