#include "gpu_utils.cuh"
#include "kernel_utils.cuh"
#include "stream_manager.hpp"

namespace timemachine {

StreamManager::StreamManager(){};

StreamManager::~StreamManager() {
    for (const auto &[key, value] : streams_) {
        gpuErrchk(hipStreamDestroy(value));
    }
    for (const auto &[key, value] : events_) {
        gpuErrchk(hipEventDestroy(value));
    }
}

hipStream_t StreamManager::get_stream(int key) {
    if (streams_.count(key) == 1) {
        return streams_[key];
    }
    hipStream_t new_stream;
    // Create stream that doesn't block with the null stream to avoid unintentional blocking.
    gpuErrchk(hipStreamCreateWithFlags(&new_stream, hipStreamNonBlocking));

    streams_[key] = new_stream;
    return new_stream;
};

hipEvent_t StreamManager::get_stream_event(int key) {
    if (events_.count(key) == 1) {
        return events_[key];
    }
    hipEvent_t new_event;
    // Create event with timings disabled as timings slow down events
    gpuErrchk(hipEventCreateWithFlags(&new_event, hipEventDisableTiming));

    events_[key] = new_event;
    return new_event;
};

// sync_from syncs the managed stream with from_stream
void StreamManager::sync_from(int key, hipStream_t from_stream) {
    hipEvent_t event = this->get_stream_event(key);
    gpuErrchk(hipEventRecord(event, from_stream));
    hipStream_t to_stream = this->get_stream(key);
    gpuErrchk(hipStreamWaitEvent(to_stream, event));
};

// sync_to syncs the to_stream from the managed stream
void StreamManager::sync_to(int key, hipStream_t to_stream) {
    hipStream_t from_stream = this->get_stream(key);
    hipEvent_t event = this->get_stream_event(key);
    gpuErrchk(hipEventRecord(event, from_stream));
    gpuErrchk(hipStreamWaitEvent(to_stream, event));
};

} // namespace timemachine
