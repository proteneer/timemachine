#include <chrono>
#include <iostream>
#include <vector>
#include <complex>
#include "centroid_restraint.hpp"
#include "gpu_utils.cuh"
#include "k_centroid_restraint.cuh"

namespace jankmachine {

template <typename RealType>
CentroidRestraint<RealType>::CentroidRestraint(
    const std::vector<int> &group_a_idxs,
    const std::vector<int> &group_b_idxs,
    const std::vector<double> &masses,
    const double kb,
    const double b0
) : N_(masses.size()),
    N_A_(group_a_idxs.size()),
    N_B_(group_b_idxs.size()),
    kb_(kb),
    b0_(b0) {

    for(int i=0; i < group_a_idxs.size(); i++) {
        if(group_a_idxs[i] >= N_ || group_a_idxs[i] < 0) {
            throw std::runtime_error("Invalid group_a_idx!");
        }
    }

    for(int i=0; i < group_b_idxs.size(); i++) {
        if(group_b_idxs[i] >= N_ || group_b_idxs[i] < 0) {
            throw std::runtime_error("Invalid group_a_idx!");
        }
    }

    gpuErrchk(hipMalloc(&d_masses_, N_*sizeof(*d_masses_)));
    gpuErrchk(hipMemcpy(d_masses_, &masses[0], N_*sizeof(*d_masses_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_group_a_idxs_, N_A_*sizeof(*d_group_a_idxs_)));
    gpuErrchk(hipMemcpy(d_group_a_idxs_, &group_a_idxs[0], N_A_*sizeof(*d_group_a_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_group_b_idxs_, N_B_*sizeof(*d_group_b_idxs_)));
    gpuErrchk(hipMemcpy(d_group_b_idxs_, &group_b_idxs[0], N_B_*sizeof(*d_group_b_idxs_), hipMemcpyHostToDevice));

};

template <typename RealType>
CentroidRestraint<RealType>::~CentroidRestraint() {
    gpuErrchk(hipFree(d_masses_));
    gpuErrchk(hipFree(d_group_a_idxs_));
    gpuErrchk(hipFree(d_group_b_idxs_));
};


template <typename RealType>
void CentroidRestraint<RealType>::execute_device(
        const int N,
        const int P,
        const double *d_x,
        const double *d_p,
        const double *d_box,
        const double lambda,
        unsigned long long *d_du_dx,
        double *d_du_dp,
        double *d_du_dl,
        double *d_u,
        hipStream_t stream) {

    int tpb = 32;

    k_centroid_restraint<RealType><<<1, tpb, 0, stream>>>(
        N_,
        d_x,
        d_group_a_idxs_,
        d_group_b_idxs_,
        N_A_,
        N_B_,
        d_masses_,
        kb_,
        b0_,
        d_du_dx,
        d_u
    );
    gpuErrchk(hipPeekAtLastError());

    // auto finish = std::chrono::high_resolution_clock::now();
    // std::chrono::duration<double> elapsed = finish - start;
    // std::cout << "CentroidRestraint Elapsed time: " << elapsed.count() << " s\n";

};

template class CentroidRestraint<double>;
template class CentroidRestraint<float>;

} // namespace jankmachine