#include "centroid_restraint.hpp"
#include "gpu_utils.cuh"
#include "k_centroid_restraint.cuh"
#include <chrono>
#include <complex>
#include <iostream>
#include <vector>

namespace timemachine {

template <typename RealType>
CentroidRestraint<RealType>::CentroidRestraint(
    const std::vector<int> &group_a_idxs, const std::vector<int> &group_b_idxs, const double kb, const double b0)
    : N_A_(group_a_idxs.size()), N_B_(group_b_idxs.size()), kb_(kb), b0_(b0) {

    gpuErrchk(hipMalloc(&d_group_a_idxs_, N_A_ * sizeof(*d_group_a_idxs_)));
    gpuErrchk(hipMemcpy(d_group_a_idxs_, &group_a_idxs[0], N_A_ * sizeof(*d_group_a_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_group_b_idxs_, N_B_ * sizeof(*d_group_b_idxs_)));
    gpuErrchk(hipMemcpy(d_group_b_idxs_, &group_b_idxs[0], N_B_ * sizeof(*d_group_b_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_centroid_a_, 3 * sizeof(*d_centroid_a_)));
    gpuErrchk(hipMalloc(&d_centroid_b_, 3 * sizeof(*d_centroid_b_)));
};

template <typename RealType> CentroidRestraint<RealType>::~CentroidRestraint() {
    gpuErrchk(hipFree(d_group_a_idxs_));
    gpuErrchk(hipFree(d_group_b_idxs_));
    gpuErrchk(hipFree(d_centroid_a_));
    gpuErrchk(hipFree(d_centroid_b_));
};

template <typename RealType>
void CentroidRestraint<RealType>::execute_device(
    const int N,
    const int P,
    const double *d_x,
    const double *d_p,
    const double *d_box,
    const double lambda,
    unsigned long long *d_du_dx,
    unsigned long long *d_du_dp,
    unsigned long long *d_du_dl,
    unsigned long long *d_u,
    hipStream_t stream) {

    int tpb = 32;

    int blocks = (N_B_ + N_A_ + tpb - 1) / tpb;
    gpuErrchk(hipMemsetAsync(d_centroid_a_, 0.0, 3 * sizeof(*d_centroid_a_), stream));
    gpuErrchk(hipMemsetAsync(d_centroid_b_, 0.0, 3 * sizeof(*d_centroid_b_), stream));
    k_calc_centroid<RealType>
        <<<blocks, tpb, 0, stream>>>(d_x, d_group_a_idxs_, d_group_b_idxs_, N_A_, N_B_, d_centroid_a_, d_centroid_b_);

    gpuErrchk(hipPeekAtLastError());

    k_centroid_restraint<RealType><<<blocks, tpb, 0, stream>>>(
        d_x, d_group_a_idxs_, d_group_b_idxs_, N_A_, N_B_, d_centroid_a_, d_centroid_b_, kb_, b0_, d_du_dx, d_u);
    gpuErrchk(hipPeekAtLastError());
};

template class CentroidRestraint<double>;
template class CentroidRestraint<float>;

} // namespace timemachine
