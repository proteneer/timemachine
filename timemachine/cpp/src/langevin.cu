#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hiprand.h"

#include <iostream>
#include <vector>
#include <stdexcept>
#include <cstdio>

#include "langevin.hpp"
#include "gpu_utils.cuh"


template <typename RealType>
__global__ void update_positions(
    const RealType *noise,
    const RealType coeff_a,
    const RealType *coeff_bs, // N x 3, not P x N x 3, but we could just pass in the first index
    const RealType *coeff_cs,
    const RealType *dE_dx,
    const RealType d_t,
    const int N,
    const int D,
    RealType *x_t,
    RealType *v_t) {

    int atom_idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(atom_idx >= N) {
        return;
    }

    int d_idx = blockIdx.y;

    if(d_idx >= 3) {
        return;
    }

    int local_idx = atom_idx*D + d_idx;

    v_t[local_idx] = coeff_a*v_t[local_idx] - coeff_bs[atom_idx]*dE_dx[local_idx] + coeff_cs[atom_idx]*noise[local_idx];
    x_t[local_idx] += v_t[local_idx]*d_t;

}


template<typename RealType>
__global__ void update_derivatives(
    const RealType coeff_a,
    const RealType *coeff_bs, // shape N
    const RealType *d2E_dxdp, 
    const RealType dt,
    const int N,
    const int D,
    RealType *dx_dp_t,
    RealType *dv_dp_t) {

    int atom_idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(atom_idx >= N) {
        return;
    }

    int d_idx = blockIdx.y;
    if(d_idx >= 3) {
        return;
    }
    int p_idx = blockIdx.z;
    int local_idx = p_idx*N*D + atom_idx*D + d_idx;

    // derivative of the above equation
    RealType tmp = coeff_a*dv_dp_t[local_idx] - coeff_bs[atom_idx]*d2E_dxdp[local_idx];
    dv_dp_t[local_idx] = tmp;
    dx_dp_t[local_idx] += dt*tmp;

}


namespace timemachine {


template<typename RealType> 
LangevinOptimizer<RealType>::LangevinOptimizer(
    RealType dt,
    const int num_dims,
    const RealType coeff_a,
    const std::vector<RealType> &coeff_bs,
    const std::vector<RealType> &coeff_cs) :
    dt_(dt),
    coeff_a_(coeff_a),
    d_rng_buffer_(nullptr) {

    gpuErrchk(hipMalloc((void**)&d_coeff_bs_, coeff_bs.size()*sizeof(RealType)));
    gpuErrchk(hipMalloc((void**)&d_coeff_cs_, coeff_cs.size()*sizeof(RealType)));

    gpuErrchk(hipMemcpy(d_coeff_bs_, &coeff_bs[0], coeff_bs.size()*sizeof(RealType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_coeff_cs_, &coeff_cs[0], coeff_cs.size()*sizeof(RealType), hipMemcpyHostToDevice));

    cublasErrchk(hipblasCreate(&cb_handle_));
    curandErrchk(hiprandCreateGenerator(&cr_rng_, HIPRAND_RNG_PSEUDO_PHILOX4_32_10));

    gpuErrchk(hipMalloc((void**)&d_rng_buffer_, coeff_bs.size()*num_dims*sizeof(RealType)));

    hiprandSetPseudoRandomGeneratorSeed(cr_rng_, time(NULL));

}


template<typename RealType> 
LangevinOptimizer<RealType>::~LangevinOptimizer() {
    gpuErrchk(hipFree(d_coeff_bs_));
    gpuErrchk(hipFree(d_coeff_cs_));
    gpuErrchk(hipFree(d_rng_buffer_));

    cublasErrchk(hipblasDestroy(cb_handle_));
    curandErrchk(hiprandDestroyGenerator(cr_rng_));
}

template<typename RealType> 
void LangevinOptimizer<RealType>::step(
    const int N,
    const int D,
    const int DP,
    const RealType *dE_dx,
    const RealType *d2E_dx2,
    RealType *d2E_dxdp, // this is modified in place
    RealType *d_x_t,
    RealType *d_v_t,
    RealType *d_dx_dp_t,
    RealType *d_dv_dp_t,
    const RealType *d_input_noise_buffer) const {

    size_t tpb = 32;
    size_t n_blocks = (N*D + tpb - 1) / tpb;
    if(d2E_dx2 != nullptr && d2E_dxdp != nullptr) {
        hessian_vector_product(N, D, DP, d2E_dx2, d_dx_dp_t, d2E_dxdp);

        dim3 dimGrid_dxdp(n_blocks, D, DP); // x, y, z dims
        update_derivatives<RealType><<<dimGrid_dxdp, tpb>>>(
            coeff_a_,
            d_coeff_bs_,
            d2E_dxdp,
            dt_,
            N,
            D,
            d_dx_dp_t,
            d_dv_dp_t
        );
        gpuErrchk(hipPeekAtLastError());
    }

    const RealType* d_noise_buf = nullptr;

    if(d_input_noise_buffer == nullptr) {
        curandErrchk(templateCurandNormal(cr_rng_, d_rng_buffer_, N*D, 0.0, 1.0));
        d_noise_buf = d_rng_buffer_;
    } else {
        d_noise_buf = d_input_noise_buffer;
    }

    dim3 dimGrid_dx(n_blocks, D);
    update_positions<RealType><<<dimGrid_dx, tpb>>>(
        d_noise_buf,
        coeff_a_,
        d_coeff_bs_,
        d_coeff_cs_,
        dE_dx,
        dt_,
        N,
        D,
        d_x_t,
        d_v_t
    );

    gpuErrchk(hipPeekAtLastError());

}

template<typename RealType> 
void LangevinOptimizer<RealType>::hessian_vector_product(
    const int N,
    const int D,
    const int DP,
    const RealType *d_A,
    RealType *d_B,
    RealType *d_C) const {

    RealType alpha = 1.0;
    RealType beta  = 1.0;
 
    const size_t ND = N*D;

    // this is set to UPPER because of fortran ordering
    cublasErrchk(templateSymm(cb_handle_,
        HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER,
        ND, DP,
        &alpha,
        d_A, ND,
        d_B, ND,
        &beta,
        d_C, ND));

}

template<typename RealType>
void LangevinOptimizer<RealType>::set_coeff_a(RealType a) {
    coeff_a_ = a;
}

template<typename RealType>
void LangevinOptimizer<RealType>::set_coeff_b(int num_atoms, const RealType *cb) {
    gpuErrchk(hipMemcpy(d_coeff_bs_, cb, num_atoms*sizeof(RealType), hipMemcpyHostToDevice));
}

template<typename RealType>
void LangevinOptimizer<RealType>::set_coeff_c(int num_atoms, const RealType *cc) {
    gpuErrchk(hipMemcpy(d_coeff_cs_, cc, num_atoms*sizeof(RealType), hipMemcpyHostToDevice));
}

template<typename RealType>
void LangevinOptimizer<RealType>::set_dt(RealType ndt) {
    dt_ = ndt;
}

}

template class timemachine::LangevinOptimizer<double>;
template class timemachine::LangevinOptimizer<float>;