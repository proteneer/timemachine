#include "device_buffer.hpp"
#include "gpu_utils.cuh"
#include "kernels/k_translations.cuh"
#include "math_utils.cuh"
#include "translations.hpp"

namespace timemachine {

template <typename RealType>
std::vector<RealType> translations_inside_and_outside_sphere_host(
    const int n_translations,
    const std::vector<double> &box,
    const std::vector<RealType> &center,
    const RealType radius,
    const int seed) {

    const int tpb = DEFAULT_THREADS_PER_BLOCK;

    DeviceBuffer<double> d_box(box);
    DeviceBuffer<RealType> d_center(center);
    DeviceBuffer<hiprandState_t> d_states(tpb);
    DeviceBuffer<RealType> d_output(n_translations * 3 * 2);

    hipStream_t stream = static_cast<hipStream_t>(0);

    k_initialize_curand_states<<<1, tpb, 0, stream>>>(static_cast<int>(d_states.length), seed, d_states.data);
    gpuErrchk(hipPeekAtLastError());

    // Generate two translations per translation requested, one inner, one outer
    k_generate_translations_inside_and_outside_sphere<RealType>
        <<<1, tpb, 0, stream>>>(n_translations, d_box.data, d_center.data, radius, d_states.data, d_output.data);
    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipStreamSynchronize(stream));

    std::vector<RealType> h_out(d_output.length);
    d_output.copy_to(&h_out[0]);
    return h_out;
}

template std::vector<float> translations_inside_and_outside_sphere_host<float>(
    const int, const std::vector<double> &box, const std::vector<float> &center, const float, const int);
template std::vector<double> translations_inside_and_outside_sphere_host<double>(
    const int, const std::vector<double> &box, const std::vector<double> &center, const double, const int);

} // namespace timemachine
