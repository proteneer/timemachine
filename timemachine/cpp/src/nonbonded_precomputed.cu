#include "gpu_utils.cuh"
#include "k_nonbonded_precomputed.cuh"
#include "kernel_utils.cuh"
#include "math_utils.cuh"
#include "nonbonded_precomputed.hpp"
#include <vector>

namespace timemachine {

template <typename RealType>
NonbondedPairListPrecomputed<RealType>::NonbondedPairListPrecomputed(
    const std::vector<int> &idxs, const std::vector<double> &w_offsets, const double beta, const double cutoff)
    : B_(idxs.size() / 2), beta_(beta), cutoff_(cutoff) {

    if (idxs.size() % 2 != 0) {
        throw std::runtime_error("idxs.size() must be exactly 2*B!");
    }

    for (int b = 0; b < B_; b++) {
        auto src = idxs[b * 2 + 0];
        auto dst = idxs[b * 2 + 1];
        if (src == dst) {
            throw std::runtime_error(
                "illegal pair with src == dst: " + std::to_string(src) + ", " + std::to_string(dst));
        }
    }

    if (w_offsets.size() != B_) {
        throw std::runtime_error("offset size does not match idxs size");
    }

    gpuErrchk(hipMalloc(&d_idxs_, B_ * 2 * sizeof(*d_idxs_)));
    gpuErrchk(hipMemcpy(d_idxs_, &idxs[0], B_ * 2 * sizeof(*d_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_w_offsets_, B_ * sizeof(*d_w_offsets_)));
    gpuErrchk(hipMemcpy(d_w_offsets_, &w_offsets[0], B_ * sizeof(*d_w_offsets_), hipMemcpyHostToDevice));
};

template <typename RealType> NonbondedPairListPrecomputed<RealType>::~NonbondedPairListPrecomputed() {
    gpuErrchk(hipFree(d_idxs_));
    gpuErrchk(hipFree(d_w_offsets_));
};

template <typename RealType>
void NonbondedPairListPrecomputed<RealType>::execute_device(
    const int N,
    const int P,
    const double *d_x,
    const double *d_p,
    const double *d_box,
    const double lambda,
    unsigned long long *d_du_dx,
    unsigned long long *d_du_dp,
    unsigned long long *d_du_dl,
    unsigned long long *d_u,
    hipStream_t stream) {

    if (P != 3 * B_) {
        throw std::runtime_error(
            "NonbondedPairListPrecomputed::execute_device(): expected P == 3*B, got P=" + std::to_string(P) +
            ", 3*B=" + std::to_string(3 * B_));
    }

    if (B_ > 0) {
        const int tpb = warp_size;
        const int blocks = ceil_divide(B_, tpb);

        k_nonbonded_precomputed<RealType><<<blocks, tpb, 0, stream>>>(
            B_, d_x, d_p, d_box, d_w_offsets_, d_idxs_, beta_, cutoff_, d_du_dx, d_du_dp, d_u);
    }
};

template <typename RealType>
void NonbondedPairListPrecomputed<RealType>::du_dp_fixed_to_float(
    const int N, const int P, const unsigned long long *du_dp, double *du_dp_float) {

    for (int i = 0; i < B_; i++) {
        const int idx_charge = i * 3 + 0;
        const int idx_sig = i * 3 + 1;
        const int idx_eps = i * 3 + 2;

        du_dp_float[idx_charge] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DCHARGE>(du_dp[idx_charge]);
        du_dp_float[idx_sig] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DSIG>(du_dp[idx_sig]);
        du_dp_float[idx_eps] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DEPS>(du_dp[idx_eps]);
    }
};

template class NonbondedPairListPrecomputed<double>;
template class NonbondedPairListPrecomputed<float>;

} // namespace timemachine
