
#include <stdexcept>

#include "custom_bonded_gpu.hpp"
#include "k_harmonic_bond.cuh"
#include "k_harmonic_angle.cuh"
#include "k_periodic_torsion.cuh"
#include "kernel_utils.cuh"

#include <chrono>
#include <iostream>

namespace timemachine {

template <typename RealType>
HarmonicBond<RealType>::HarmonicBond(
    std::vector<int> bond_idxs,
    std::vector<int> param_idxs
) : n_bonds_(bond_idxs.size()/2) {

    gpuErrchk(hipMalloc((void**)&d_param_idxs_, param_idxs.size()*sizeof(*d_param_idxs_)));
    gpuErrchk(hipMalloc((void**)&d_bond_idxs_, bond_idxs.size()*sizeof(*d_bond_idxs_)));
    gpuErrchk(hipMemcpy(d_param_idxs_, &param_idxs[0], param_idxs.size()*sizeof(*d_param_idxs_), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_bond_idxs_, &bond_idxs[0], bond_idxs.size()*sizeof(*d_bond_idxs_), hipMemcpyHostToDevice));

};

template <typename RealType>
HarmonicBond<RealType>::~HarmonicBond() {
    gpuErrchk(hipFree(d_bond_idxs_));
    gpuErrchk(hipFree(d_param_idxs_));
};

template <typename RealType>
void HarmonicBond<RealType>::derivatives_device(
        const int num_confs,
        const int num_atoms,
        const int num_dims,
        const RealType *d_coords,
        const RealType *d_params,
        RealType *d_E,
        RealType *d_dE_dx,
        RealType *d_d2E_dx2,
        // parameter derivatives
        const int num_dp,
        const int *d_param_gather_idxs,
        RealType *d_dE_dp,
        RealType *d_d2E_dxdp) const {

    const auto C = num_confs;
    const auto N = num_atoms;
    const auto B = n_bonds_;

    int tpb = 32;
    int n_blocks = (B + tpb - 1) / tpb;
    int dim_y = 1;

    dim3 dimBlock(tpb);
    dim3 dimGrid(n_blocks, dim_y, C); // x, y, z dims

    // auto start = std::chrono::high_resolution_clock::now();
    if(num_dims == 3) {
        k_harmonic_bond_derivatives<RealType, 3><<<dimGrid, dimBlock>>>(
            N,
            d_coords,
            d_params,
            B,
            d_bond_idxs_,
            d_param_idxs_,
            d_E,
            d_dE_dx,
            d_d2E_dx2,
            // parameter derivatives
            num_dp,
            d_param_gather_idxs,
            d_dE_dp,
            d_d2E_dxdp
        );
    } else if(num_dims == 4) {
        k_harmonic_bond_derivatives<RealType, 4><<<dimGrid, dimBlock>>>(
            N,
            d_coords,
            d_params,
            B,
            d_bond_idxs_,
            d_param_idxs_,
            d_E,
            d_dE_dx,
            d_d2E_dx2,
            // parameter derivatives
            num_dp,
            d_param_gather_idxs,
            d_dE_dp,
            d_d2E_dxdp
        );
    }


    // hipDeviceSynchronize();
    // auto finish = std::chrono::high_resolution_clock::now();
    // std::chrono::duration<double> elapsed = finish - start;
    // std::cout << "Custom harmonic angles elapsed time: " << elapsed.count() << " s\n";

    gpuErrchk(hipPeekAtLastError());

};

template class HarmonicBond<float>;
template class HarmonicBond<double>;

template <typename RealType>
HarmonicAngle<RealType>::HarmonicAngle(
    std::vector<int> angle_idxs,
    std::vector<int> param_idxs
) : n_angles_(angle_idxs.size()/3) {

    gpuErrchk(hipMalloc((void**)&d_param_idxs_, param_idxs.size()*sizeof(*d_param_idxs_)));
    gpuErrchk(hipMalloc((void**)&d_angle_idxs_, angle_idxs.size()*sizeof(*d_angle_idxs_)));
    gpuErrchk(hipMemcpy(d_param_idxs_, &param_idxs[0], param_idxs.size()*sizeof(*d_param_idxs_), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_angle_idxs_, &angle_idxs[0], angle_idxs.size()*sizeof(*d_angle_idxs_), hipMemcpyHostToDevice));

};

template <typename RealType>
HarmonicAngle<RealType>::~HarmonicAngle() {
    gpuErrchk(hipFree(d_angle_idxs_));
    gpuErrchk(hipFree(d_param_idxs_));
};

template <typename RealType>
void HarmonicAngle<RealType>::derivatives_device(
        const int num_confs,
        const int num_atoms,
        const int num_dims,
        const RealType *d_coords,
        const RealType *d_params,
        RealType *d_E,
        RealType *d_dE_dx,
        RealType *d_d2E_dx2,
        // parameter derivatives
        const int num_dp,
        const int *d_param_gather_idxs,
        RealType *d_dE_dp,
        RealType *d_d2E_dxdp) const {

    const auto C = num_confs;
    const auto N = num_atoms;

    int tpb = 32;
    int n_blocks = (n_angles_ + tpb - 1) / tpb;
    int dim_y = 1;

    dim3 dimBlock(tpb);
    dim3 dimGrid(n_blocks, dim_y, C); // x, y, z

    // auto start = std::chrono::high_resolution_clock::now();
    if(num_dims == 3) {
        k_harmonic_angle_derivatives<RealType, 3><<<dimGrid, dimBlock>>>(
            N,
            d_coords,
            d_params,
            n_angles_,
            d_angle_idxs_,
            d_param_idxs_,
            d_E,
            d_dE_dx,
            d_d2E_dx2,
            // parameter derivatives
            num_dp,
            d_param_gather_idxs,
            d_dE_dp,
            d_d2E_dxdp
        );
    } else if(num_dims == 4) {
        k_harmonic_angle_derivatives<RealType, 4><<<dimGrid, dimBlock>>>(
            N,
            d_coords,
            d_params,
            n_angles_,
            d_angle_idxs_,
            d_param_idxs_,
            d_E,
            d_dE_dx,
            d_d2E_dx2,
            // parameter derivatives
            num_dp,
            d_param_gather_idxs,
            d_dE_dp,
            d_d2E_dxdp
        );
    }

    // hipDeviceSynchronize();
    // auto finish = std::chrono::high_resolution_clock::now();
    // std::chrono::duration<double> elapsed = finish - start;
    // std::cout << "Custom harmonic angles elapsed time: " << elapsed.count() << " s\n";

    gpuErrchk(hipPeekAtLastError());

};

template class HarmonicAngle<float>;
template class HarmonicAngle<double>;

template <typename RealType>
PeriodicTorsion<RealType>::PeriodicTorsion(
    std::vector<int> torsion_idxs,
    std::vector<int> param_idxs
) : n_torsions_(torsion_idxs.size()/4),
    d_torsion_idxs_(nullptr),
    d_param_idxs_(nullptr) {

    gpuErrchk(hipMalloc((void**)&d_param_idxs_, param_idxs.size()*sizeof(*d_param_idxs_)));
    gpuErrchk(hipMalloc((void**)&d_torsion_idxs_, torsion_idxs.size()*sizeof(*d_torsion_idxs_)));
    gpuErrchk(hipMemcpy(d_param_idxs_, &param_idxs[0], param_idxs.size()*sizeof(*d_param_idxs_), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_torsion_idxs_, &torsion_idxs[0], torsion_idxs.size()*sizeof(*d_torsion_idxs_), hipMemcpyHostToDevice));


};

template <typename RealType>
PeriodicTorsion<RealType>::~PeriodicTorsion() {
    gpuErrchk(hipFree(d_torsion_idxs_));
    gpuErrchk(hipFree(d_param_idxs_));
};

template <typename RealType>
void PeriodicTorsion<RealType>::derivatives_device(
        const int num_confs,
        const int num_atoms,
        const int num_dims,
        const RealType *d_coords,
        const RealType *d_params,
        RealType *d_E,
        RealType *d_dE_dx,
        RealType *d_d2E_dx2,
        // parameter derivatives
        const int num_dp,
        const int *d_param_gather_idxs,
        RealType *d_dE_dp,
        RealType *d_d2E_dxdp) const {

    const auto C = num_confs;
    const auto N = num_atoms;

    int tpb = 32;
    int n_blocks = (n_torsions_ + tpb - 1) / tpb;
    int dim_y = 1;

    dim3 dimBlock(tpb);
    dim3 dimGrid(n_blocks, dim_y, C); // x, y, z

    if(num_dims == 3) {
        k_periodic_torsion_derivatives<RealType, 3> <<<dimGrid, dimBlock>>>(
            N,
            d_coords,
            d_params,
            n_torsions_,
            d_torsion_idxs_,
            d_param_idxs_,
            d_E,
            d_dE_dx,
            d_d2E_dx2,
            // parameter derivatives
            num_dp,
            d_param_gather_idxs,
            d_dE_dp,
            d_d2E_dxdp
        );
    } else if (num_dims == 4) {
        k_periodic_torsion_derivatives<RealType, 4> <<<dimGrid, dimBlock>>>(
            N,
            d_coords,
            d_params,
            n_torsions_,
            d_torsion_idxs_,
            d_param_idxs_,
            d_E,
            d_dE_dx,
            d_d2E_dx2,
            // parameter derivatives
            num_dp,
            d_param_gather_idxs,
            d_dE_dp,
            d_d2E_dxdp
        );
    }

    // hipDeviceSynchronize();
    // auto finish = std::chrono::high_resolution_clock::now();
    // std::chrono::duration<double> elapsed = finish - start;
    // std::cout << "Custom torsions elapsed time: " << elapsed.count() << " s\n";

    gpuErrchk(hipPeekAtLastError());

};


template class PeriodicTorsion<float>;
template class PeriodicTorsion<double>;


} // namespace timemachine