#include "observable.hpp"
#include "gpu_utils.cuh"
#include <iostream>


namespace timemachine {

AvgPartialUPartialParam::AvgPartialUPartialParam(
    BoundPotential *bp, int freq) : bp_(bp), count_(0), freq_(freq) {
    int P = bp_->size();
    gpuErrchk(hipMalloc(&d_sum_du_dp_, P*sizeof(*d_sum_du_dp_)));
    gpuErrchk(hipMemset(d_sum_du_dp_, 0, P*sizeof(*d_sum_du_dp_)));
}

AvgPartialUPartialParam::~AvgPartialUPartialParam() {
    gpuErrchk(hipFree(d_sum_du_dp_));
}

void AvgPartialUPartialParam::observe(
    int step,
    int N,
    double *d_x_t,
    double *d_box_t,
    double lambda) {

    if(step % freq_ == 0) {
        bp_->execute_device(
            N,
            d_x_t,
            d_box_t,
            lambda,
            nullptr,
            d_sum_du_dp_,
            nullptr,
            nullptr,
            static_cast<hipStream_t>(0) // TBD: parallelize me!
        );
        count_ += 1;
    }

}

void AvgPartialUPartialParam::avg_du_dp(double *h_buf) const {
    gpuErrchk(hipMemcpy(h_buf, d_sum_du_dp_, this->bp_->size()*sizeof(*h_buf), hipMemcpyDeviceToHost));
    for(int i=0; i < this->bp_->size(); i++) {
        h_buf[i] /= count_;
    }
}

}