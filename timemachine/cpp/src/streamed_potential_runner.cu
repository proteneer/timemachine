#include "gpu_utils.cuh"
#include "kernel_utils.cuh"
#include "streamed_potential_runner.hpp"

namespace timemachine {

StreamedPotentialRunner::StreamedPotentialRunner() : streams_(0), events_(0) {
    // Setup the event that is used to sync with the incoming stream
    gpuErrchk(hipEventCreateWithFlags(&sync_event_, hipEventDisableTiming));
};

StreamedPotentialRunner::~StreamedPotentialRunner() {
    for (int i = 0; i < streams_.size(); i++) {
        gpuErrchk(hipStreamDestroy(streams_[i]))
    }
    for (int i = 0; i < events_.size(); i++) {
        gpuErrchk(hipEventDestroy(events_[i]))
    }
    gpuErrchk(hipEventDestroy(sync_event_));
}

hipStream_t StreamedPotentialRunner::_get_potential_stream(int index) {
    auto num_streams = streams_.size();
    if (index < num_streams) {
        return streams_[index];
    }
    // Expect stream to be the next increment
    if (num_streams != index) {
        throw std::runtime_error("Asked for new index out of order");
    }
    hipStream_t new_stream;
    // Create stream that doesn't block with the null stream
    gpuErrchk(hipStreamCreateWithFlags(&new_stream, hipStreamNonBlocking));

    hipEvent_t new_event;
    // Create stream with timings disabled as timings slow down events
    gpuErrchk(hipEventCreateWithFlags(&new_event, hipEventDisableTiming));
    streams_.push_back(new_stream);
    events_.push_back(new_event);
    return new_stream;
}

hipEvent_t StreamedPotentialRunner::_get_potential_event(int index) {
    auto num_events = events_.size();
    if (index >= num_events) {
        throw std::runtime_error("No event with index " + std::to_string(index));
    }
    return events_[index];
}

// wrap execute_device
void StreamedPotentialRunner::execute_potentials(
    std::vector<std::shared_ptr<BoundPotential>> &bps,
    const int N,
    const double *d_x,
    const double *d_box,
    unsigned long long *d_du_dx,
    unsigned long long *d_du_dp,
    unsigned long long *d_u,
    hipStream_t stream) {
    // Always sync the new streams with the incoming stream to ensure that the state
    // of the incoming buffers are valid
    gpuErrchk(hipEventRecord(sync_event_, stream));
    for (int i = 0; i < bps.size(); i++) {
        hipStream_t pot_stream = this->_get_potential_stream(i);

        gpuErrchk(hipStreamWaitEvent(pot_stream, sync_event_));
        bps[i]->execute_device(N, d_x, d_box, d_du_dx, d_du_dp, d_u, pot_stream);
        hipEvent_t event = this->_get_potential_event(i);
        // Tell the main stream to synchronize on all of the potential streams
        gpuErrchk(hipEventRecord(event, pot_stream));
        gpuErrchk(hipStreamWaitEvent(stream, event));
    }
};

} // namespace timemachine
