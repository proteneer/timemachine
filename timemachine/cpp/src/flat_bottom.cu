#include <chrono>
#include <iostream>
#include <vector>
#include <complex>
#include "flat_bottom.hpp"
#include "gpu_utils.cuh"
#include "k_flat_bottom.cuh"

namespace timemachine {

template <typename RealType>
FlatBottom<RealType>::FlatBottom(
    const std::vector<int> &bond_idxs, // [N]
    const std::vector<int> &param_idxs,
    const std::vector<int> &lambda_flags,
    int flat_bottom
) : B_(bond_idxs.size()/2),
    flat_bottom_(flat_bottom) {

    if(flat_bottom_ != 0) {
        throw std::runtime_error("flat bottom must be set to 0 for now");        
    }

    if(bond_idxs.size() % 2 != 0) {
        throw std::runtime_error("bond_idxs.size() must be exactly 2*k");
    }

    for(int b=0; b < B_; b++) {
        auto src = bond_idxs[b*2+0];
        auto dst = bond_idxs[b*2+1];
        if(src == dst) {
            throw std::runtime_error("src == dst");
        }
    }

    gpuErrchk(hipMalloc(&d_lambda_flags_, B_*sizeof(*d_lambda_flags_)));
    gpuErrchk(hipMemcpy(d_lambda_flags_, &lambda_flags[0], B_*sizeof(*d_lambda_flags_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_bond_idxs_, B_*2*sizeof(*d_bond_idxs_)));
    gpuErrchk(hipMemcpy(d_bond_idxs_, &bond_idxs[0], B_*2*sizeof(*d_bond_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_bond_idxs_, B_*2*sizeof(*d_bond_idxs_)));
    gpuErrchk(hipMemcpy(d_bond_idxs_, &bond_idxs[0], B_*2*sizeof(*d_bond_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_param_idxs_, B_*2*sizeof(*d_param_idxs_)));
    gpuErrchk(hipMemcpy(d_param_idxs_, &param_idxs[0], B_*2*sizeof(*d_param_idxs_), hipMemcpyHostToDevice));

};

template <typename RealType>
FlatBottom<RealType>::~FlatBottom() {
    gpuErrchk(hipFree(d_bond_idxs_));
    gpuErrchk(hipFree(d_param_idxs_));
    gpuErrchk(hipFree(d_lambda_flags_));
};

template <typename RealType>
void FlatBottom<RealType>::execute_lambda_inference_device(
    const int N,
    const int P,
    const double *d_coords_primals,
    const double *d_params_primals,
    const double lambda_primal,
    unsigned long long *d_out_coords_primals, // du/dx
    double *d_out_lambda_primals, // du/dl
    double *d_out_energy_primal, // U
    hipStream_t stream) {

    int tpb = 32;
    int blocks = (B_+tpb-1)/tpb;
    k_flat_bottom_inference<RealType><<<blocks, tpb, 0, stream>>>(
        B_,
        d_coords_primals,
        d_params_primals,
        lambda_primal,
        d_bond_idxs_,
        d_param_idxs_,
        d_lambda_flags_,
        d_out_coords_primals,
        d_out_lambda_primals,
        d_out_energy_primal
    );
    gpuErrchk(hipPeekAtLastError());

    // auto finish = std::chrono::high_resolution_clock::now();
    // std::chrono::duration<double> elapsed = finish - start;
    // std::cout << "FlatBottom Elapsed time: " << elapsed.count() << " s\n";

};

template <typename RealType>
void FlatBottom<RealType>::execute_lambda_jvp_device(
    const int N,
    const int P,
    const double *d_coords_primals,
    const double *d_coords_tangents,
    const double *d_params_primals,
    const double lambda_primal, // unused
    const double lambda_tangent, // unused
    double *d_out_coords_primals,
    double *d_out_coords_tangents,
    double *d_out_params_primals,
    double *d_out_params_tangents,
    hipStream_t stream) {

    int tpb = 32;
    int blocks = (B_+tpb-1)/tpb;

    k_flat_bottom_jvp<RealType><<<blocks, tpb, 0, stream>>>(
        B_,
        d_coords_primals,
        d_coords_tangents,
        d_params_primals,
        lambda_primal,
        lambda_tangent,
        d_bond_idxs_,
        d_param_idxs_,
        d_lambda_flags_,
        d_out_coords_primals,
        d_out_coords_tangents,
        d_out_params_primals,
        d_out_params_tangents
    );

    // hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    // auto finish = std::chrono::high_resolution_clock::now();
    // std::chrono::duration<double> elapsed = finish - start;
    // std::cout << "FlatBottom Elapsed time: " << elapsed.count() << " s\n";

}

template class FlatBottom<double>;
template class FlatBottom<float>;

} // namespace timemachine