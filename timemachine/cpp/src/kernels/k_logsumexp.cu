#include "hip/hip_runtime.h"
#include "k_logsumexp.cuh"

namespace timemachine {

// Convert the outputs of LogSumExp kernels into the final logsumexp value

template <typename RealType>
void __global__ k_exp_sub_max(
    const int N,
    const RealType *__restrict__ max,  // [1]
    const RealType *__restrict__ vals, // [N]
    RealType *__restrict__ out         // [N]
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) {
        return;
    }
    out[idx] = exp(vals[idx] - max[0]);
}

template void __global__ k_exp_sub_max<float>(const int, const float *, const float *, float *);
template void __global__ k_exp_sub_max<double>(const int, const double *, const double *, double *);

} // namespace timemachine
