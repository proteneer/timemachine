#include <iostream>

#include "fixed_point.hpp"
#include "gpu_utils.cuh"
#include "potential.hpp"
#include "surreal.cuh"

namespace timemachine {

void Potential::execute_host(
    const int N,
    const int P,
    const double *h_x,           // [N,3]
    const double *h_p,           // [P,]
    const double *h_box,         // [3, 3]
    const double lambda,         // [1]
    unsigned long long *h_du_dx, // [N,3]
    unsigned long long *h_du_dp, // [P]
    unsigned long long *h_du_dl, //
    unsigned long long *h_u) {

    const int &D = Potential::D;

    double *d_x;
    double *d_p;
    double *d_box;

    gpuErrchk(hipMalloc(&d_x, N * D * sizeof(double)));
    gpuErrchk(hipMemcpy(d_x, h_x, N * D * sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_p, P * sizeof(double)));
    gpuErrchk(hipMemcpy(d_p, h_p, P * sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_box, D * D * sizeof(double)));
    gpuErrchk(hipMemcpy(d_box, h_box, D * D * sizeof(double), hipMemcpyHostToDevice));

    unsigned long long *d_du_dx = nullptr;
    unsigned long long *d_du_dp = nullptr;
    unsigned long long *d_du_dl = nullptr;
    unsigned long long *d_u = nullptr;

    // very important that these are initialized to zero since the kernels themselves just accumulate
    if (h_du_dx) {
        gpuErrchk(hipMalloc(&d_du_dx, N * D * sizeof(unsigned long long)));
        gpuErrchk(hipMemset(d_du_dx, 0, N * D * sizeof(unsigned long long)));
    }
    if (h_du_dp) {
        gpuErrchk(hipMalloc(&d_du_dp, P * sizeof(unsigned long long)));
        gpuErrchk(hipMemset(d_du_dp, 0, P * sizeof(unsigned long long)));
    }
    if (h_du_dl) {
        gpuErrchk(hipMalloc(&d_du_dl, N * sizeof(*d_du_dl)));
        gpuErrchk(hipMemset(d_du_dl, 0, N * sizeof(*d_du_dl)));
    }
    if (h_u) {
        gpuErrchk(hipMalloc(&d_u, N * sizeof(*d_u)));
        gpuErrchk(hipMemset(d_u, 0, N * sizeof(*d_u)));
    }

    this->execute_device(N, P, d_x, d_p, d_box, lambda, d_du_dx, d_du_dp, d_du_dl, d_u, static_cast<hipStream_t>(0));

    // outputs
    if (h_du_dx) {
        gpuErrchk(hipMemcpy(h_du_dx, d_du_dx, N * D * sizeof(*h_du_dx), hipMemcpyDeviceToHost));
        gpuErrchk(hipFree(d_du_dx));
    }
    if (h_du_dp) {
        gpuErrchk(hipMemcpy(h_du_dp, d_du_dp, P * sizeof(*h_du_dp), hipMemcpyDeviceToHost));
        gpuErrchk(hipFree(d_du_dp));
    }
    if (h_du_dl) {
        gpuErrchk(hipMemcpy(h_du_dl, d_du_dl, N * sizeof(*h_du_dl), hipMemcpyDeviceToHost));
        gpuErrchk(hipFree(d_du_dl));
    }
    if (h_u) {
        gpuErrchk(hipMemcpy(h_u, d_u, N * sizeof(*h_u), hipMemcpyDeviceToHost));
        gpuErrchk(hipFree(d_u));
    }

    // inputs
    gpuErrchk(hipFree(d_x));
    gpuErrchk(hipFree(d_p));
    gpuErrchk(hipFree(d_box));
};

void Potential::execute_host_du_dx(
    const int N,
    const int P,
    const double *h_x,   // [N,3]
    const double *h_p,   // [P,]
    const double *h_box, // [3, 3]
    const double lambda, // [1]
    unsigned long long *h_du_dx) {

    const int &D = Potential::D;

    double *d_x;
    double *d_p;
    double *d_box;

    gpuErrchk(hipMalloc(&d_x, N * D * sizeof(double)));
    gpuErrchk(hipMemcpy(d_x, h_x, N * D * sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_p, P * sizeof(double)));
    gpuErrchk(hipMemcpy(d_p, h_p, P * sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_box, D * D * sizeof(double)));
    gpuErrchk(hipMemcpy(d_box, h_box, D * D * sizeof(double), hipMemcpyHostToDevice));

    unsigned long long *d_du_dx; // du/dx

    // very important that these are initialized to zero since the kernels themselves just accumulate
    gpuErrchk(hipMalloc(&d_du_dx, N * D * sizeof(unsigned long long)));
    gpuErrchk(hipMemset(d_du_dx, 0, N * D * sizeof(unsigned long long)));

    this->execute_device(
        N, P, d_x, d_p, d_box, lambda, d_du_dx, nullptr, nullptr, nullptr, static_cast<hipStream_t>(0));

    gpuErrchk(hipMemcpy(h_du_dx, d_du_dx, N * D * sizeof(*h_du_dx), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(d_du_dx));
    gpuErrchk(hipFree(d_x));
    gpuErrchk(hipFree(d_p));
    gpuErrchk(hipFree(d_box));
};

void Potential::du_dp_fixed_to_float(const int N, const int P, const unsigned long long *du_dp, double *du_dp_float) {
    for (int i = 0; i < P; i++) {
        du_dp_float[i] = FIXED_TO_FLOAT<double>(du_dp[i]);
    }
}

} // namespace timemachine
