#include <iostream>

#include "fixed_point.hpp"
#include "gpu_utils.cuh"
#include "potential.hpp"
#include "surreal.cuh"

namespace timemachine {

void Potential::execute_host(
    const int N,
    const int P,
    const double *h_x,           // [N,3]
    const double *h_p,           // [P,]
    const double *h_box,         // [3, 3]
    const double lambda,         // [1]
    unsigned long long *h_du_dx, // [N,3]
    double *h_du_dp,             // [P]
    unsigned long long *h_du_dl, //
    unsigned long long *h_u) {

    const int &D = Potential::D;

    double *d_x;
    double *d_p;
    double *d_box;

    gpuErrchk(hipMalloc(&d_x, N * D * sizeof(double)));
    gpuErrchk(hipMemcpy(d_x, h_x, N * D * sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_p, P * sizeof(double)));
    gpuErrchk(hipMemcpy(d_p, h_p, P * sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_box, D * D * sizeof(double)));
    gpuErrchk(hipMemcpy(d_box, h_box, D * D * sizeof(double), hipMemcpyHostToDevice));

    unsigned long long *d_du_dx = nullptr;
    double *d_du_dp = nullptr;
    unsigned long long *d_du_dl = nullptr;
    unsigned long long *d_u = nullptr;

    // very important that these are initialized to zero since the kernels themselves just accumulate
    if (h_du_dx) {
        gpuErrchk(hipMalloc(&d_du_dx, N * D * sizeof(unsigned long long)));
        gpuErrchk(hipMemset(d_du_dx, 0, N * D * sizeof(unsigned long long)));
    }
    if (h_du_dp) {
        gpuErrchk(hipMalloc(&d_du_dp, P * sizeof(unsigned long long)));
        gpuErrchk(hipMemset(d_du_dp, 0, P * sizeof(unsigned long long)));
    }
    if (h_du_dl) {
        gpuErrchk(hipMalloc(&d_du_dl, N * sizeof(*d_du_dl)));
        gpuErrchk(hipMemset(d_du_dl, 0, N * sizeof(*d_du_dl)));
    }
    if (h_u) {
        gpuErrchk(hipMalloc(&d_u, N * sizeof(*d_u)));
        gpuErrchk(hipMemset(d_u, 0, N * sizeof(*d_u)));
    }

    this->execute_device(N, P, d_x, d_p, d_box, lambda, d_du_dx, d_du_dp, d_du_dl, d_u, static_cast<hipStream_t>(0));

    // outputs
    if (h_du_dx) {
        gpuErrchk(hipMemcpy(h_du_dx, d_du_dx, N * D * sizeof(*h_du_dx), hipMemcpyDeviceToHost));
        gpuErrchk(hipFree(d_du_dx));
    }
    if (h_du_dp) {
        gpuErrchk(hipMemcpy(h_du_dp, d_du_dp, P * sizeof(*h_du_dp), hipMemcpyDeviceToHost));
        gpuErrchk(hipFree(d_du_dp));
    }
    if (h_du_dl) {
        gpuErrchk(hipMemcpy(h_du_dl, d_du_dl, N * sizeof(*h_du_dl), hipMemcpyDeviceToHost));
        gpuErrchk(hipFree(d_du_dl));
    }
    if (h_u) {
        gpuErrchk(hipMemcpy(h_u, d_u, N * sizeof(*h_u), hipMemcpyDeviceToHost));
        gpuErrchk(hipFree(d_u));
    }

    // inputs
    gpuErrchk(hipFree(d_x));
    gpuErrchk(hipFree(d_p));
    gpuErrchk(hipFree(d_box));
};

void Potential::execute_host_du_dx(
    const int N,
    const int P,
    const double *h_x,   // [N,3]
    const double *h_p,   // [P,]
    const double *h_box, // [3, 3]
    const double lambda, // [1]
    unsigned long long *h_du_dx) {

    const int &D = Potential::D;

    double *d_x;
    double *d_p;
    double *d_box;

    gpuErrchk(hipMalloc(&d_x, N * D * sizeof(double)));
    gpuErrchk(hipMemcpy(d_x, h_x, N * D * sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_p, P * sizeof(double)));
    gpuErrchk(hipMemcpy(d_p, h_p, P * sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_box, D * D * sizeof(double)));
    gpuErrchk(hipMemcpy(d_box, h_box, D * D * sizeof(double), hipMemcpyHostToDevice));

    unsigned long long *d_du_dx; // du/dx

    // very important that these are initialized to zero since the kernels themselves just accumulate
    gpuErrchk(hipMalloc(&d_du_dx, N * D * sizeof(unsigned long long)));
    gpuErrchk(hipMemset(d_du_dx, 0, N * D * sizeof(unsigned long long)));

    this->execute_device(
        N, P, d_x, d_p, d_box, lambda, d_du_dx, nullptr, nullptr, nullptr, static_cast<hipStream_t>(0));

    gpuErrchk(hipMemcpy(h_du_dx, d_du_dx, N * D * sizeof(*h_du_dx), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(d_du_dx));
    gpuErrchk(hipFree(d_x));
    gpuErrchk(hipFree(d_p));
    gpuErrchk(hipFree(d_box));
};

void Potential::fixed_to_float(
    const int N,
    const int P,
    const unsigned long long *du_dx,
    const double *du_dp,
    const unsigned long long *du_dl,
    const unsigned long long *u,
    double *du_dx_out,
    double *du_dp_out,
    double *du_dl_sum,
    double *u_sum) {

    const int &D = Potential::D;

    for (int i = 0; i < N * D; i++) {
        du_dx_out[i] = FIXED_TO_FLOAT<double>(du_dx[i]);
    }

    for (int i = 0; i < P; i++) {
        du_dp_out[i] = du_dp[i]; // TODO: just a pass-thru for now; update when du_dp is fixed-point
    }

    unsigned long long du_dl_sum_fixed = 0;
    for (int i = 0; i < N; i++) {
        du_dl_sum_fixed += du_dl[i];
    }
    *du_dl_sum = FIXED_TO_FLOAT<double>(du_dl_sum_fixed);

    unsigned long long u_sum_fixed = 0;
    for (int i = 0; i < N; i++) {
        u_sum_fixed += u[i];
    }
    *u_sum = FIXED_TO_FLOAT<double>(u_sum_fixed);
}
} // namespace timemachine
