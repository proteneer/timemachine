#include <stdexcept>

#include "potential.hpp"
#include "kernel_utils.cuh"

namespace timemachine {

template<typename RealType>
void Potential<RealType>::derivatives_host(
    const int num_confs,
    const int num_atoms,
    const int num_dims,
    const int num_params,
    const RealType *h_coords,
    const RealType *h_params,
    RealType *h_E,
    RealType *h_dE_dx,
    RealType *h_d2E_dx2,
    // parameter derivatives
    const int num_dp,
    const int *h_param_gather_idxs,
    RealType *h_dE_dp,
    RealType *h_d2E_dxdp) const {

    const auto C = num_confs;
    const auto N = num_atoms;
    const auto P = num_params;
    const auto DP = num_dp;


    if(num_dims != 3 && num_dims != 4) {
        throw std::runtime_error("Unsupported number of dimensions");
    }

    RealType* d_coords = nullptr;
    RealType* d_params = nullptr;
    int* d_param_gather_idxs = nullptr;

    RealType* d_E = nullptr;
    RealType* d_dE_dx = nullptr;
    RealType* d_d2E_dx2 = nullptr;

    RealType* d_dE_dp = nullptr;
    RealType* d_d2E_dxdp = nullptr;

    gpuErrchk(hipMalloc((void**)&d_coords, C*N*num_dims*sizeof(RealType)));
    gpuErrchk(hipMalloc((void**)&d_params, P*sizeof(RealType)));
    gpuErrchk(hipMalloc((void**)&d_param_gather_idxs, P*sizeof(int)));

    gpuErrchk(hipMemcpy(d_coords, h_coords, C*N*num_dims*sizeof(RealType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_params, h_params, P*sizeof(RealType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_param_gather_idxs, h_param_gather_idxs, P*sizeof(int), hipMemcpyHostToDevice));

    if(h_E != nullptr) {
        gpuErrchk(hipMalloc((void**)&d_E, C*sizeof(RealType)));
        gpuErrchk(hipMemset(d_E, 0, C*sizeof(RealType)));        
    }

    if(h_dE_dx != nullptr) {
        gpuErrchk(hipMalloc((void**)&d_dE_dx, C*N*num_dims*sizeof(RealType)));
        gpuErrchk(hipMemset(d_dE_dx, 0, C*N*num_dims*sizeof(RealType)));
    }

    if(h_d2E_dx2 != nullptr) {
        gpuErrchk(hipMalloc((void**)&d_d2E_dx2, C*N*num_dims*N*num_dims*sizeof(RealType)));
        gpuErrchk(hipMemset(d_d2E_dx2, 0, C*N*num_dims*N*num_dims*sizeof(RealType)));     
    }

    if(h_dE_dp != nullptr) {
        gpuErrchk(hipMalloc((void**)&d_dE_dp, C*DP*sizeof(RealType)));
        gpuErrchk(hipMemset(d_dE_dp, 0, C*DP*sizeof(RealType)));
    }
    if(h_d2E_dxdp != nullptr) {
        gpuErrchk(hipMalloc((void**)&d_d2E_dxdp, C*DP*N*num_dims*sizeof(RealType)));
        gpuErrchk(hipMemset(d_d2E_dxdp, 0, C*DP*N*num_dims*sizeof(RealType)));
    }

    this->derivatives_device(
        C,
        N,
        num_dims,
        d_coords,
        d_params,
        d_E,
        d_dE_dx,
        d_d2E_dx2,

        // parameter derivatives
        num_dp,
        d_param_gather_idxs,
        d_dE_dp,
        d_d2E_dxdp
    );

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMemcpy(h_E, d_E, C*sizeof(RealType), hipMemcpyDeviceToHost));

    if(h_E != nullptr) {
        gpuErrchk(hipMemcpy(h_E, d_E, C*sizeof(RealType), hipMemcpyDeviceToHost));
    }
    if(h_dE_dx != nullptr) {
        gpuErrchk(hipMemcpy(h_dE_dx, d_dE_dx, C*N*num_dims*sizeof(RealType), hipMemcpyDeviceToHost));        
    }
    if(h_d2E_dx2 != nullptr) {
        gpuErrchk(hipMemcpy(h_d2E_dx2, d_d2E_dx2, C*N*num_dims*N*num_dims*sizeof(RealType), hipMemcpyDeviceToHost));        
    }
    if(h_dE_dp != nullptr) {
        gpuErrchk(hipMemcpy(h_dE_dp, d_dE_dp, C*DP*sizeof(RealType), hipMemcpyDeviceToHost));
    }
    if(h_d2E_dxdp != nullptr) {
        gpuErrchk(hipMemcpy(h_d2E_dxdp, d_d2E_dxdp, C*DP*N*num_dims*sizeof(RealType), hipMemcpyDeviceToHost));
    }

    hipFree(d_coords);
    hipFree(d_params);

    hipFree(d_E);
    hipFree(d_dE_dx);
    hipFree(d_d2E_dx2);

    hipFree(d_param_gather_idxs);
    hipFree(d_dE_dp);
    hipFree(d_d2E_dxdp);

}

template class Potential<float>;
template class Potential<double>;

}