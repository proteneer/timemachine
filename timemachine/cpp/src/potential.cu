#include <memory>

#include "device_buffer.hpp"
#include "fixed_point.hpp"
#include "gpu_utils.cuh"
#include "potential.hpp"

#include <chrono>
#include <iostream>

namespace timemachine {

const int Potential::D = 3;

void Potential::execute_batch_host(
    const int coord_batch_size,  // Number of batches of coordinates
    const int N,                 // Number of atoms
    const int param_batch_size,  // Number of batches of parameters
    const int P,                 // Number of parameters
    const int lambda_batch_size, // Number of lambda values
    const double *h_x,           // [coord_batch_size, N, 3]
    const double *h_p,           // [param_batch_size, P]
    const double *h_box,         // [coord_batch_size, 3, 3]
    const double *lambdas,       // [lambda_batch_size]
    unsigned long long *h_du_dx, // [coord_batch_size, param_batch_size, lambda_batch_size, N, 3]
    unsigned long long *h_du_dp, // [coord_batch_size, param_batch_size, lambda_batch_size, P]
    unsigned long long *h_du_dl, // [coord_batch_size, param_batch_size, lambda_batch_size, N]
    unsigned long long *h_u) {   // [coord_batch_size, param_batch_size, lambda_batch_size, N]
    std::unique_ptr<DeviceBuffer<double>> d_p(nullptr);
    if (P > 0) {
        d_p.reset(new DeviceBuffer<double>(param_batch_size * P));
        d_p->copy_from(h_p);
    }

    DeviceBuffer<double> d_box(coord_batch_size * D * D);
    d_box.copy_from(h_box);

    DeviceBuffer<double> d_x_buffer(coord_batch_size * N * D);
    d_x_buffer.copy_from(h_x);

    std::unique_ptr<DeviceBuffer<unsigned long long>> d_du_dx_buffer(nullptr);
    std::unique_ptr<DeviceBuffer<unsigned long long>> d_du_dp_buffer(nullptr);
    std::unique_ptr<DeviceBuffer<unsigned long long>> d_du_dl_buffer(nullptr);
    std::unique_ptr<DeviceBuffer<unsigned long long>> d_u_buffer(nullptr);

    const int total_executions = coord_batch_size * param_batch_size * lambda_batch_size;

    hipStream_t stream;
    gpuErrchk(hipStreamCreate(&stream));

    if (h_du_dx) {
        d_du_dx_buffer.reset(new DeviceBuffer<unsigned long long>(total_executions * N * D));
        gpuErrchk(hipMemsetAsync(d_du_dx_buffer->data, 0, d_du_dx_buffer->size, stream));
    }

    if (h_du_dp) {
        d_du_dp_buffer.reset(new DeviceBuffer<unsigned long long>(total_executions * P));
        gpuErrchk(hipMemsetAsync(d_du_dp_buffer->data, 0, d_du_dp_buffer->size, stream));
    }

    if (h_du_dl) {
        d_du_dl_buffer.reset(new DeviceBuffer<unsigned long long>(total_executions * N));
        gpuErrchk(hipMemsetAsync(d_du_dl_buffer->data, 0, d_du_dl_buffer->size, stream));
    }

    if (h_u) {
        d_u_buffer.reset(new DeviceBuffer<unsigned long long>(total_executions * N));
        gpuErrchk(hipMemsetAsync(d_u_buffer->data, 0, d_u_buffer->size, stream));
    }

    for (unsigned int i = 0; i < coord_batch_size; i++) {
        for (unsigned int j = 0; j < param_batch_size; j++) {
            for (unsigned int k = 0; k < lambda_batch_size; k++) {
                unsigned int offset_factor = (i * param_batch_size * lambda_batch_size) + (j * lambda_batch_size) + k;
                this->execute_device(
                    N,
                    P,
                    d_x_buffer.data + (i * N * D),
                    P > 0 ? d_p->data + (j * P) : nullptr,
                    d_box.data + (i * D * D),
                    lambdas[k],
                    d_du_dx_buffer ? d_du_dx_buffer->data + (offset_factor * N * D) : nullptr,
                    d_du_dp_buffer ? d_du_dp_buffer->data + (offset_factor * P) : nullptr,
                    d_du_dl_buffer ? d_du_dl_buffer->data + (offset_factor * N) : nullptr,
                    d_u_buffer ? d_u_buffer->data + (offset_factor * N) : nullptr,
                    stream);
            }
        }
    }
    gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk(hipStreamDestroy(stream));

    if (h_du_dx) {
        d_du_dx_buffer->copy_to(h_du_dx);
    }

    if (h_du_dp) {
        d_du_dp_buffer->copy_to(h_du_dp);
    }

    if (h_du_dl) {
        d_du_dl_buffer->copy_to(h_du_dl);
    }

    if (h_u) {
        d_u_buffer->copy_to(h_u);
    }
}

void Potential::execute_host(
    const int N,
    const int P,
    const double *h_x,           // [N,3]
    const double *h_p,           // [P,]
    const double *h_box,         // [3, 3]
    const double lambda,         // [1]
    unsigned long long *h_du_dx, // [N,3]
    unsigned long long *h_du_dp, // [P]
    unsigned long long *h_du_dl, //
    unsigned long long *h_u) {

    const int &D = Potential::D;

    DeviceBuffer<double> d_x(N * D);
    DeviceBuffer<double> d_box(D * D);

    d_x.copy_from(h_x);
    d_box.copy_from(h_box);

    std::unique_ptr<DeviceBuffer<double>> d_p;
    std::unique_ptr<DeviceBuffer<unsigned long long>> d_du_dx;
    std::unique_ptr<DeviceBuffer<unsigned long long>> d_du_dp;
    std::unique_ptr<DeviceBuffer<unsigned long long>> d_du_dl;
    std::unique_ptr<DeviceBuffer<unsigned long long>> d_u;

    if (P > 0) {
        d_p.reset(new DeviceBuffer<double>(P));
        d_p->copy_from(h_p);
    }

    // very important that these are initialized to zero since the kernels themselves just accumulate
    if (h_du_dx) {
        d_du_dx.reset(new DeviceBuffer<unsigned long long>(N * D));
        gpuErrchk(hipMemset(d_du_dx->data, 0, d_du_dx->size));
    }
    if (h_du_dp) {
        d_du_dp.reset(new DeviceBuffer<unsigned long long>(P));
        gpuErrchk(hipMemset(d_du_dp->data, 0, d_du_dp->size));
    }
    if (h_du_dl) {
        d_du_dl.reset(new DeviceBuffer<unsigned long long>(N));
        gpuErrchk(hipMemset(d_du_dl->data, 0, d_du_dl->size));
    }
    if (h_u) {
        d_u.reset(new DeviceBuffer<unsigned long long>(N));
        gpuErrchk(hipMemset(d_u->data, 0, d_u->size));
    }

    this->execute_device(
        N,
        P,
        d_x.data,
        P > 0 ? d_p->data : nullptr,
        d_box.data,
        lambda,
        d_du_dx ? d_du_dx->data : nullptr,
        d_du_dp ? d_du_dp->data : nullptr,
        d_du_dl ? d_du_dl->data : nullptr,
        d_u ? d_u->data : nullptr,
        static_cast<hipStream_t>(0));

    // outputs
    if (h_du_dx) {
        d_du_dx->copy_to(h_du_dx);
    }
    if (h_du_dp) {
        d_du_dp->copy_to(h_du_dp);
    }
    if (h_du_dl) {
        d_du_dl->copy_to(h_du_dl);
    }
    if (h_u) {
        d_u->copy_to(h_u);
    }
};

void Potential::execute_host_du_dx(
    const int N,
    const int P,
    const double *h_x,   // [N,3]
    const double *h_p,   // [P,]
    const double *h_box, // [3, 3]
    const double lambda, // [1]
    unsigned long long *h_du_dx) {

    const int &D = Potential::D;

    double *d_x;
    double *d_p;
    double *d_box;

    gpuErrchk(hipMalloc(&d_x, N * D * sizeof(double)));
    gpuErrchk(hipMemcpy(d_x, h_x, N * D * sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_p, P * sizeof(double)));
    gpuErrchk(hipMemcpy(d_p, h_p, P * sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_box, D * D * sizeof(double)));
    gpuErrchk(hipMemcpy(d_box, h_box, D * D * sizeof(double), hipMemcpyHostToDevice));

    unsigned long long *d_du_dx; // du/dx

    // very important that these are initialized to zero since the kernels themselves just accumulate
    gpuErrchk(hipMalloc(&d_du_dx, N * D * sizeof(unsigned long long)));
    gpuErrchk(hipMemset(d_du_dx, 0, N * D * sizeof(unsigned long long)));

    this->execute_device(
        N, P, d_x, d_p, d_box, lambda, d_du_dx, nullptr, nullptr, nullptr, static_cast<hipStream_t>(0));

    gpuErrchk(hipMemcpy(h_du_dx, d_du_dx, N * D * sizeof(*h_du_dx), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(d_du_dx));
    gpuErrchk(hipFree(d_x));
    gpuErrchk(hipFree(d_p));
    gpuErrchk(hipFree(d_box));
};

void Potential::du_dp_fixed_to_float(const int N, const int P, const unsigned long long *du_dp, double *du_dp_float) {
    for (int i = 0; i < P; i++) {
        du_dp_float[i] = FIXED_TO_FLOAT<double>(du_dp[i]);
    }
}

} // namespace timemachine
