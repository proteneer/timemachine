#include "gpu_utils.cuh"
#include "k_lambda_transformer.cuh"
#include "k_nonbonded_pair_list.cuh"
#include "math_utils.cuh"
#include "nonbonded_pair_list.hpp"
#include <stdexcept>
#include <vector>

namespace timemachine {

template <typename RealType, bool Negated>
NonbondedPairList<RealType, Negated>::NonbondedPairList(
    const std::vector<int> &pair_idxs,          // [M, 2]
    const std::vector<double> &scales,          // [M, 2]
    const std::vector<int> &lambda_plane_idxs,  // [N]
    const std::vector<int> &lambda_offset_idxs, // [N]
    const double beta,
    const double cutoff)
    : N_(lambda_offset_idxs.size()), M_(pair_idxs.size() / 2), beta_(beta), cutoff_(cutoff) {

    if (pair_idxs.size() % 2 != 0) {
        throw std::runtime_error("pair_idxs.size() must be even, but got " + std::to_string(pair_idxs.size()));
    }

    for (int i = 0; i < M_; i++) {
        auto src = pair_idxs[i * 2 + 0];
        auto dst = pair_idxs[i * 2 + 1];
        if (src == dst) {
            throw std::runtime_error(
                "illegal pair with src == dst: " + std::to_string(src) + ", " + std::to_string(dst));
        }
    }

    if (scales.size() / 2 != M_) {
        throw std::runtime_error(
            "expected same number of pairs and scale tuples, but got " + std::to_string(M_) +
            " != " + std::to_string(scales.size() / 2));
    }

    gpuErrchk(hipMalloc(&d_pair_idxs_, M_ * 2 * sizeof(*d_pair_idxs_)));
    gpuErrchk(hipMemcpy(d_pair_idxs_, &pair_idxs[0], M_ * 2 * sizeof(*d_pair_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_lambda_plane_idxs_, N_ * sizeof(*d_lambda_plane_idxs_)));
    gpuErrchk(hipMemcpy(
        d_lambda_plane_idxs_, &lambda_plane_idxs[0], N_ * sizeof(*d_lambda_plane_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_lambda_offset_idxs_, N_ * sizeof(*d_lambda_offset_idxs_)));
    gpuErrchk(hipMemcpy(
        d_lambda_offset_idxs_, &lambda_offset_idxs[0], N_ * sizeof(*d_lambda_offset_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_w_, N_ * sizeof(*d_w_)));

    gpuErrchk(hipMalloc(&d_scales_, M_ * 2 * sizeof(*d_scales_)));
    gpuErrchk(hipMemcpy(d_scales_, &scales[0], M_ * 2 * sizeof(*d_scales_), hipMemcpyHostToDevice));
};

template <typename RealType, bool Negated> NonbondedPairList<RealType, Negated>::~NonbondedPairList() {
    gpuErrchk(hipFree(d_pair_idxs_));
    gpuErrchk(hipFree(d_scales_));
    gpuErrchk(hipFree(d_lambda_plane_idxs_));
    gpuErrchk(hipFree(d_lambda_offset_idxs_));
    gpuErrchk(hipFree(d_w_));
};

template <typename RealType, bool Negated>
void NonbondedPairList<RealType, Negated>::execute_device(
    const int N,
    const int P,
    const double *d_x,
    const double *d_p,
    const double *d_box,
    const double lambda,
    unsigned long long *d_du_dx,
    unsigned long long *d_du_dp,
    unsigned long long *d_du_dl,
    unsigned long long *d_u,
    hipStream_t stream) {

    const int tpb = 32;

    int num_blocks = ceil_divide(N, tpb);
    dim3 dimGrid(num_blocks, 3, 1);

    k_compute_w_coords<<<dimGrid, tpb, 0, stream>>>(
        N, lambda, cutoff_, d_lambda_plane_idxs_, d_lambda_offset_idxs_, d_w_);
    gpuErrchk(hipPeekAtLastError());

    int num_blocks_pairs = ceil_divide(M_, tpb);

    k_nonbonded_pair_list<RealType, Negated><<<num_blocks_pairs, tpb, 0, stream>>>(
        M_, d_x, d_p, d_box, d_w_, d_pair_idxs_, d_scales_, beta_, cutoff_, d_du_dx, d_du_dp, d_u);

    gpuErrchk(hipPeekAtLastError());
}

// TODO: this implementation is duplicated from NonbondedAllPairs
template <typename RealType, bool Negated>
void NonbondedPairList<RealType, Negated>::du_dp_fixed_to_float(
    const int N, const int P, const unsigned long long *du_dp, double *du_dp_float) {

    for (int i = 0; i < N; i++) {
        const int idx_charge = i * 3 + 0;
        const int idx_sig = i * 3 + 1;
        const int idx_eps = i * 3 + 2;
        du_dp_float[idx_charge] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DCHARGE>(du_dp[idx_charge]);
        du_dp_float[idx_sig] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DSIG>(du_dp[idx_sig]);
        du_dp_float[idx_eps] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DEPS>(du_dp[idx_eps]);
    }
}

template class NonbondedPairList<double, true>;
template class NonbondedPairList<float, true>;

template class NonbondedPairList<double, false>;
template class NonbondedPairList<float, false>;

} // namespace timemachine
