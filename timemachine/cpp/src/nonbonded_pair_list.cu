#include "energy_accumulation.hpp"
#include "gpu_utils.cuh"
#include "k_nonbonded_pair_list.cuh"
#include "kernels/kernel_utils.cuh"
#include "math_utils.cuh"
#include "nonbonded_pair_list.hpp"
#include <stdexcept>
#include <vector>

namespace timemachine {

template <typename RealType, bool Negated>
NonbondedPairList<RealType, Negated>::NonbondedPairList(
    const std::vector<int> &pair_idxs, // [M, 2]
    const std::vector<double> &scales, // [M, 2]
    const double beta,
    const double cutoff)
    : M_(pair_idxs.size() / 2), beta_(beta), cutoff_(cutoff) {

    if (pair_idxs.size() % 2 != 0) {
        throw std::runtime_error("pair_idxs.size() must be even, but got " + std::to_string(pair_idxs.size()));
    }

    for (int i = 0; i < M_; i++) {
        auto src = pair_idxs[i * 2 + 0];
        auto dst = pair_idxs[i * 2 + 1];
        if (src == dst) {
            throw std::runtime_error(
                "illegal pair with src == dst: " + std::to_string(src) + ", " + std::to_string(dst));
        }
    }

    if (scales.size() / 2 != M_) {
        throw std::runtime_error(
            "expected same number of pairs and scale tuples, but got " + std::to_string(M_) +
            " != " + std::to_string(scales.size() / 2));
    }

    cudaSafeMalloc(&d_u_buffer_, M_ * sizeof(*d_u_buffer_));

    cudaSafeMalloc(&d_pair_idxs_, M_ * 2 * sizeof(*d_pair_idxs_));
    gpuErrchk(hipMemcpy(d_pair_idxs_, &pair_idxs[0], M_ * 2 * sizeof(*d_pair_idxs_), hipMemcpyHostToDevice));

    cudaSafeMalloc(&d_scales_, M_ * 2 * sizeof(*d_scales_));
    gpuErrchk(hipMemcpy(d_scales_, &scales[0], M_ * 2 * sizeof(*d_scales_), hipMemcpyHostToDevice));
};

template <typename RealType, bool Negated> NonbondedPairList<RealType, Negated>::~NonbondedPairList() {
    gpuErrchk(hipFree(d_pair_idxs_));
    gpuErrchk(hipFree(d_scales_));
    gpuErrchk(hipFree(d_u_buffer_));
};

template <typename RealType, bool Negated>
void NonbondedPairList<RealType, Negated>::execute_device(
    const int N,
    const int P,
    const double *d_x,
    const double *d_p,
    const double *d_box,
    unsigned long long *d_du_dx,
    unsigned long long *d_du_dp,
    __int128 *d_u,
    hipStream_t stream) {

    if (M_ > 0) {
        const int tpb = DEFAULT_THREADS_PER_BLOCK;
        const int num_blocks_pairs = ceil_divide(M_, tpb);

        k_nonbonded_pair_list<RealType, Negated><<<num_blocks_pairs, tpb, 0, stream>>>(
            M_,
            d_x,
            d_p,
            d_box,
            d_pair_idxs_,
            d_scales_,
            beta_,
            cutoff_,
            d_du_dx,
            d_du_dp,
            d_u == nullptr ? nullptr : d_u_buffer_);

        gpuErrchk(hipPeekAtLastError());

        if (d_u) {
            accumulate_energy(M_, d_u_buffer_, d_u, stream);
        }
    }
}

// TODO: this implementation is duplicated from NonbondedAllPairs
template <typename RealType, bool Negated>
void NonbondedPairList<RealType, Negated>::du_dp_fixed_to_float(
    const int N, const int P, const unsigned long long *du_dp, double *du_dp_float) {

    for (int i = 0; i < N; i++) {
        const int idx = i * PARAMS_PER_ATOM;
        const int idx_charge = idx + PARAM_OFFSET_CHARGE;
        const int idx_sig = idx + PARAM_OFFSET_SIG;
        const int idx_eps = idx + PARAM_OFFSET_EPS;
        const int idx_w = idx + PARAM_OFFSET_W;

        du_dp_float[idx_charge] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DCHARGE>(du_dp[idx_charge]);
        du_dp_float[idx_sig] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DSIG>(du_dp[idx_sig]);
        du_dp_float[idx_eps] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DEPS>(du_dp[idx_eps]);
        du_dp_float[idx_w] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DW>(du_dp[idx_w]);
    }
}

template class NonbondedPairList<double, true>;
template class NonbondedPairList<float, true>;

template class NonbondedPairList<double, false>;
template class NonbondedPairList<float, false>;

} // namespace timemachine
