#include "hip/hip_runtime.h"
#include "constants.hpp"
#include "gpu_utils.cuh"
#include "kernel_utils.cuh"
#include "langevin_integrator.hpp"
#include "math_utils.cuh"

#include "kernels/k_integrator.cuh"

namespace timemachine {

LangevinIntegrator::LangevinIntegrator(
    int N, const double *masses, double temperature, double dt, double friction, int seed)
    : N_(N), temperature_(temperature), dt_(dt), friction_(friction) {

    ca_ = exp(-friction * dt);

    const double kT = BOLTZ * temperature;
    const double ccs_adjustment = sqrt(1 - exp(-2 * friction * dt));

    std::vector<double> h_ccs(N_);
    std::vector<double> h_cbs(N_);
    for (int i = 0; i < N_; i++) {
        h_cbs[i] = dt_ / masses[i];
        h_ccs[i] = ccs_adjustment * sqrt(kT / masses[i]);
    }

    d_cbs_ = gpuErrchkCudaMallocAndCopy(h_cbs.data(), N_);
    d_ccs_ = gpuErrchkCudaMallocAndCopy(h_ccs.data(), N_);

    curandErrchk(hiprandCreateGenerator(&cr_rng_, HIPRAND_RNG_PSEUDO_DEFAULT));
    gpuErrchk(hipMalloc(&d_noise_, round_up_even(N_ * 3) * sizeof(double)));
    curandErrchk(hiprandSetPseudoRandomGeneratorSeed(cr_rng_, seed));

    gpuErrchk(hipMalloc(&d_du_dx_, N_ * 3 * sizeof(*d_du_dx_)));
}

LangevinIntegrator::~LangevinIntegrator() {
    gpuErrchk(hipFree(d_cbs_));
    gpuErrchk(hipFree(d_ccs_));
    gpuErrchk(hipFree(d_noise_));
    gpuErrchk(hipFree(d_du_dx_));
    curandErrchk(hiprandDestroyGenerator(cr_rng_));
}

void LangevinIntegrator::step_fwd(
    std::vector<BoundPotential *> &bps, double *d_x_t, double *d_v_t, double *d_box_t, hipStream_t stream) {

    gpuErrchk(hipMemsetAsync(d_du_dx_, 0, N_ * 3 * sizeof(*d_du_dx_), stream));

    for (int i = 0; i < bps.size(); i++) {
        bps[i]->execute_device(
            N_,
            d_x_t,
            d_box_t,
            d_du_dx_, // we only need the forces
            nullptr,
            nullptr,
            stream);
    }

    const int D = 3;
    size_t tpb = warp_size;
    size_t n_blocks = ceil_divide(N_, tpb);
    dim3 dimGrid_dx(n_blocks, D);

    curandErrchk(hiprandSetStream(cr_rng_, stream));
    curandErrchk(templateCurandNormal(cr_rng_, d_noise_, round_up_even(N_ * D), 0.0, 1.0));

    update_forward_baoab<double>
        <<<dimGrid_dx, tpb, 0, stream>>>(N_, D, ca_, d_cbs_, d_ccs_, d_noise_, d_x_t, d_v_t, d_du_dx_, dt_);

    gpuErrchk(hipPeekAtLastError());
}

void LangevinIntegrator::initialize(
    std::vector<BoundPotential *> &bps, double *d_x_t, double *d_v_t, double *d_box_t, hipStream_t stream){};

void LangevinIntegrator::finalize(
    std::vector<BoundPotential *> &bps, double *d_x_t, double *d_v_t, double *d_box_t, hipStream_t stream){};

} // end namespace timemachine
