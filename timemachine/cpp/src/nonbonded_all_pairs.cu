#include "vendored/jitify.hpp"
#include <algorithm>
#include <cassert>
#include <chrono>
#include <complex>
#include <cstdlib>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <numeric>
#include <optional>
#include <vector>

#include "fixed_point.hpp"
#include "gpu_utils.cuh"
#include "nonbonded_all_pairs.hpp"
#include "vendored/hilbert.h"

#include "k_nonbonded.cuh"

#include <fstream>
#include <streambuf>
#include <string>

namespace timemachine {

template <typename RealType, bool Interpolated>
NonbondedAllPairs<RealType, Interpolated>::NonbondedAllPairs(
    const std::vector<int> &lambda_plane_idxs,  // [N]
    const std::vector<int> &lambda_offset_idxs, // [N]
    const double beta,
    const double cutoff,
    const std::optional<std::set<int>> &atom_idxs,
    const std::string &kernel_src
    // const std::string &transform_lambda_charge,
    // const std::string &transform_lambda_sigma,
    // const std::string &transform_lambda_epsilon,
    // const std::string &transform_lambda_w
    )
    : N_(lambda_offset_idxs.size()), K_(atom_idxs ? atom_idxs->size() : N_), beta_(beta), cutoff_(cutoff),
      d_atom_idxs_(nullptr), nblist_(K_), nblist_padding_(0.1), d_sort_storage_(nullptr), d_sort_storage_bytes_(0),
      disable_hilbert_(false),

      kernel_ptrs_({// enumerate over every possible kernel combination
                    // U: Compute U
                    // X: Compute DU_DL
                    // L: Compute DU_DX
                    // P: Compute DU_DP
                    //                             U  X  L  P
                    &k_nonbonded_unified<RealType, 0, 0, 0, 0>,
                    &k_nonbonded_unified<RealType, 0, 0, 0, 1>,
                    &k_nonbonded_unified<RealType, 0, 0, 1, 0>,
                    &k_nonbonded_unified<RealType, 0, 0, 1, 1>,
                    &k_nonbonded_unified<RealType, 0, 1, 0, 0>,
                    &k_nonbonded_unified<RealType, 0, 1, 0, 1>,
                    &k_nonbonded_unified<RealType, 0, 1, 1, 0>,
                    &k_nonbonded_unified<RealType, 0, 1, 1, 1>,
                    &k_nonbonded_unified<RealType, 1, 0, 0, 0>,
                    &k_nonbonded_unified<RealType, 1, 0, 0, 1>,
                    &k_nonbonded_unified<RealType, 1, 0, 1, 0>,
                    &k_nonbonded_unified<RealType, 1, 0, 1, 1>,
                    &k_nonbonded_unified<RealType, 1, 1, 0, 0>,
                    &k_nonbonded_unified<RealType, 1, 1, 0, 1>,
                    &k_nonbonded_unified<RealType, 1, 1, 1, 0>,
                    &k_nonbonded_unified<RealType, 1, 1, 1, 1>}),

      compute_w_coords_instance_(kernel_cache_.program(kernel_src.c_str()).kernel("k_compute_w_coords").instantiate()),
      compute_gather_interpolated_(
          kernel_cache_.program(kernel_src.c_str()).kernel("k_gather_interpolated").instantiate()),
      compute_add_du_dp_interpolated_(
          kernel_cache_.program(kernel_src.c_str()).kernel("k_add_du_dp_interpolated").instantiate()) {

    if (lambda_offset_idxs.size() != lambda_plane_idxs.size()) {
        throw std::runtime_error("lambda offset idxs and plane idxs need to be equivalent");
    }

    gpuErrchk(hipMalloc(&d_lambda_plane_idxs_, N_ * sizeof(*d_lambda_plane_idxs_)));
    gpuErrchk(hipMemcpy(
        d_lambda_plane_idxs_, &lambda_plane_idxs[0], N_ * sizeof(*d_lambda_plane_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_lambda_offset_idxs_, N_ * sizeof(*d_lambda_offset_idxs_)));
    gpuErrchk(hipMemcpy(
        d_lambda_offset_idxs_, &lambda_offset_idxs[0], N_ * sizeof(*d_lambda_offset_idxs_), hipMemcpyHostToDevice));

    if (atom_idxs) {
        gpuErrchk(hipMalloc(&d_atom_idxs_, K_ * sizeof(*d_atom_idxs_)));
        std::vector<int> atom_idxs_v(atom_idxs->begin(), atom_idxs->end());
        gpuErrchk(hipMemcpy(d_atom_idxs_, &atom_idxs_v[0], K_ * sizeof(*d_atom_idxs_), hipMemcpyHostToDevice));
    }

    gpuErrchk(hipMalloc(&d_sorted_atom_idxs_, K_ * sizeof(*d_sorted_atom_idxs_)));

    gpuErrchk(hipMalloc(&d_gathered_x_, K_ * 3 * sizeof(*d_gathered_x_)));

    gpuErrchk(hipMalloc(&d_w_, N_ * sizeof(*d_w_)));
    gpuErrchk(hipMalloc(&d_dw_dl_, N_ * sizeof(*d_dw_dl_)));

    gpuErrchk(hipMalloc(&d_gathered_w_, K_ * sizeof(*d_gathered_w_)));
    gpuErrchk(hipMalloc(&d_gathered_dw_dl_, K_ * sizeof(*d_gathered_dw_dl_)));

    gpuErrchk(hipMalloc(&d_gathered_p_, K_ * 3 * sizeof(*d_gathered_p_)));         // interpolated
    gpuErrchk(hipMalloc(&d_gathered_dp_dl_, K_ * 3 * sizeof(*d_gathered_dp_dl_))); // interpolated
    gpuErrchk(hipMalloc(&d_gathered_du_dx_, K_ * 3 * sizeof(*d_gathered_du_dx_)));
    gpuErrchk(hipMalloc(&d_gathered_du_dp_, K_ * 3 * sizeof(*d_gathered_du_dp_)));

    gpuErrchk(hipMalloc(&d_du_dp_buffer_, N_ * 3 * sizeof(*d_du_dp_buffer_)));

    gpuErrchk(hipHostMalloc(&p_ixn_count_, 1 * sizeof(*p_ixn_count_)));

    gpuErrchk(hipMalloc(&d_nblist_x_, N_ * 3 * sizeof(*d_nblist_x_)));
    gpuErrchk(hipMemset(d_nblist_x_, 0, N_ * 3 * sizeof(*d_nblist_x_))); // set non-sensical positions
    gpuErrchk(hipMalloc(&d_nblist_box_, 3 * 3 * sizeof(*d_nblist_x_)));
    gpuErrchk(hipMemset(d_nblist_box_, 0, 3 * 3 * sizeof(*d_nblist_x_)));
    gpuErrchk(hipMalloc(&d_rebuild_nblist_, 1 * sizeof(*d_rebuild_nblist_)));
    gpuErrchk(hipHostMalloc(&p_rebuild_nblist_, 1 * sizeof(*p_rebuild_nblist_)));

    gpuErrchk(hipMalloc(&d_sort_keys_in_, K_ * sizeof(d_sort_keys_in_)));
    gpuErrchk(hipMalloc(&d_sort_keys_out_, K_ * sizeof(d_sort_keys_out_)));
    gpuErrchk(hipMalloc(&d_sort_vals_in_, K_ * sizeof(d_sort_vals_in_)));

    // initialize hilbert curve
    std::vector<unsigned int> bin_to_idx(HILBERT_GRID_DIM * HILBERT_GRID_DIM * HILBERT_GRID_DIM);
    for (int i = 0; i < HILBERT_GRID_DIM; i++) {
        for (int j = 0; j < HILBERT_GRID_DIM; j++) {
            for (int k = 0; k < HILBERT_GRID_DIM; k++) {

                bitmask_t hilbert_coords[3];
                hilbert_coords[0] = i;
                hilbert_coords[1] = j;
                hilbert_coords[2] = k;

                unsigned int bin = static_cast<unsigned int>(hilbert_c2i(3, HILBERT_N_BITS, hilbert_coords));
                bin_to_idx[i * HILBERT_GRID_DIM * HILBERT_GRID_DIM + j * HILBERT_GRID_DIM + k] = bin;
            }
        }
    }

    gpuErrchk(
        hipMalloc(&d_bin_to_idx_, HILBERT_GRID_DIM * HILBERT_GRID_DIM * HILBERT_GRID_DIM * sizeof(*d_bin_to_idx_)));
    gpuErrchk(hipMemcpy(
        d_bin_to_idx_,
        &bin_to_idx[0],
        HILBERT_GRID_DIM * HILBERT_GRID_DIM * HILBERT_GRID_DIM * sizeof(*d_bin_to_idx_),
        hipMemcpyHostToDevice));

    // estimate size needed to do radix sorting, this can use uninitialized data.
    hipcub::DeviceRadixSort::SortPairs(
        nullptr, d_sort_storage_bytes_, d_sort_keys_in_, d_sort_keys_out_, d_sort_vals_in_, d_sorted_atom_idxs_, K_);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMalloc(&d_sort_storage_, d_sort_storage_bytes_));
};

template <typename RealType, bool Interpolated> NonbondedAllPairs<RealType, Interpolated>::~NonbondedAllPairs() {

    gpuErrchk(hipFree(d_lambda_plane_idxs_));
    gpuErrchk(hipFree(d_lambda_offset_idxs_));

    if (d_atom_idxs_) {
        gpuErrchk(hipFree(d_atom_idxs_));
    }

    gpuErrchk(hipFree(d_du_dp_buffer_));
    gpuErrchk(hipFree(d_sorted_atom_idxs_));

    gpuErrchk(hipFree(d_bin_to_idx_));
    gpuErrchk(hipFree(d_gathered_x_));

    gpuErrchk(hipFree(d_w_));
    gpuErrchk(hipFree(d_dw_dl_));
    gpuErrchk(hipFree(d_gathered_w_));
    gpuErrchk(hipFree(d_gathered_dw_dl_));
    gpuErrchk(hipFree(d_gathered_p_));
    gpuErrchk(hipFree(d_gathered_dp_dl_));
    gpuErrchk(hipFree(d_gathered_du_dx_));
    gpuErrchk(hipFree(d_gathered_du_dp_));

    gpuErrchk(hipFree(d_sort_keys_in_));
    gpuErrchk(hipFree(d_sort_keys_out_));
    gpuErrchk(hipFree(d_sort_vals_in_));
    gpuErrchk(hipFree(d_sort_storage_));

    gpuErrchk(hipHostFree(p_ixn_count_));

    gpuErrchk(hipFree(d_nblist_x_));
    gpuErrchk(hipFree(d_nblist_box_));
    gpuErrchk(hipFree(d_rebuild_nblist_));
    gpuErrchk(hipHostFree(p_rebuild_nblist_));
};

template <typename RealType, bool Interpolated>
void NonbondedAllPairs<RealType, Interpolated>::set_nblist_padding(double val) {
    nblist_padding_ = val;
}

template <typename RealType, bool Interpolated> void NonbondedAllPairs<RealType, Interpolated>::disable_hilbert_sort() {
    disable_hilbert_ = true;
}

template <typename RealType, bool Interpolated>
void NonbondedAllPairs<RealType, Interpolated>::hilbert_sort(
    const double *d_coords, const double *d_box, hipStream_t stream) {

    const int tpb = 32;
    const int B = ceil_divide(K_, tpb);

    if (d_atom_idxs_) {
        k_coords_to_kv_gather<<<B, tpb, 0, stream>>>(
            K_, d_atom_idxs_, d_coords, d_box, d_bin_to_idx_, d_sort_keys_in_, d_sort_vals_in_);
    } else {
        // N_ == K_
        k_coords_to_kv<<<B, tpb, 0, stream>>>(K_, d_coords, d_box, d_bin_to_idx_, d_sort_keys_in_, d_sort_vals_in_);
    }

    gpuErrchk(hipPeekAtLastError());

    hipcub::DeviceRadixSort::SortPairs(
        d_sort_storage_,
        d_sort_storage_bytes_,
        d_sort_keys_in_,
        d_sort_keys_out_,
        d_sort_vals_in_,
        d_sorted_atom_idxs_,
        K_,
        0,                            // begin bit
        sizeof(*d_sort_keys_in_) * 8, // end bit
        stream                        // cudaStream
    );

    gpuErrchk(hipPeekAtLastError());
}

template <typename RealType, bool Interpolated>
void NonbondedAllPairs<RealType, Interpolated>::execute_device(
    const int N,
    const int P,
    const double *d_x,
    const double *d_p,   // 2 * N * 3
    const double *d_box, // 3 * 3
    const double lambda,
    unsigned long long *d_du_dx,
    unsigned long long *d_du_dp,
    unsigned long long *d_du_dl,
    unsigned long long *d_u,
    hipStream_t stream) {

    // (ytz) the nonbonded algorithm proceeds as follows:

    // (done in constructor), construct a hilbert curve mapping each of the HILBERT_GRID_DIM x HILBERT_GRID_DIM x HILBERT_GRID_DIM cells into an index.
    // a. decide if we need to rebuild the neighborlist, if so:
    //     - look up which cell each particle belongs to, and its linear index along the hilbert curve.
    //     - use radix pair sort keyed on the hilbert index with values equal to the atomic index
    //     - resulting sorted values is the permutation array.
    //     - permute lambda plane/offsets, coords
    // b. else:
    //     - permute new coords
    // c. permute parameters
    // d. compute the nonbonded interactions using the neighborlist
    // e. inverse permute the forces, du/dps into the original index.
    // f. u and du/dl is buffered into a per-particle array, and then reduced.
    // g. note that du/dl is not an exact per-particle du/dl - it is only used for reduction purposes.

    if (N != N_) {
        throw std::runtime_error(
            "NonbondedAllPairs::execute_device(): expected N == N_, got N=" + std::to_string(N) +
            ", N_=" + std::to_string(N_));
    }

    const int M = Interpolated ? 2 : 1;

    if (P != M * N_ * 3) {
        throw std::runtime_error(
            "NonbondedAllPairs::execute_device(): expected P == M*N_*3, got P=" + std::to_string(P) +
            ", M*N_*3=" + std::to_string(M * N_ * 3));
    }

    // identify which tiles contain interpolated parameters

    const int tpb = 32;

    // (ytz) see if we need to rebuild the neighborlist.
    if (d_atom_idxs_) {
        k_check_rebuild_coords_and_box_gather<RealType><<<ceil_divide(K_, tpb), tpb, 0, stream>>>(
            K_, d_atom_idxs_, d_x, d_nblist_x_, d_box, d_nblist_box_, nblist_padding_, d_rebuild_nblist_);
    } else {
        k_check_rebuild_coords_and_box<RealType><<<ceil_divide(K_, tpb), tpb, 0, stream>>>(
            K_, d_x, d_nblist_x_, d_box, d_nblist_box_, nblist_padding_, d_rebuild_nblist_);
    }
    gpuErrchk(hipPeekAtLastError());

    // we can optimize this away by doing the check on the GPU directly.
    gpuErrchk(hipMemcpyAsync(
        p_rebuild_nblist_, d_rebuild_nblist_, 1 * sizeof(*p_rebuild_nblist_), hipMemcpyDeviceToHost, stream));
    gpuErrchk(hipStreamSynchronize(stream)); // slow!

    if (p_rebuild_nblist_[0] > 0) {

        // (ytz): update the permutation index before building neighborlist, as the neighborlist is tied
        // to a particular sort order
        if (!disable_hilbert_) {
            this->hilbert_sort(d_x, d_box, stream);
        } else {
            if (d_atom_idxs_) {
                gpuErrchk(hipMemcpyAsync(
                    d_sorted_atom_idxs_, d_atom_idxs_, K_ * sizeof(*d_atom_idxs_), hipMemcpyDeviceToDevice, stream));
                gpuErrchk(hipPeekAtLastError());
            } else {
                // N_ == K_
                k_arange<<<ceil_divide(K_, tpb), tpb, 0, stream>>>(K_, d_sorted_atom_idxs_);
            }
        }

        // compute new coordinates, new lambda_idxs, new_plane_idxs
        k_gather<<<dim3(ceil_divide(K_, tpb), 3, 1), tpb, 0, stream>>>(K_, d_sorted_atom_idxs_, d_x, d_gathered_x_);
        gpuErrchk(hipPeekAtLastError());
        nblist_.build_nblist_device(K_, d_gathered_x_, d_box, cutoff_ + nblist_padding_, stream);
        gpuErrchk(hipMemcpyAsync(
            p_ixn_count_, nblist_.get_ixn_count(), 1 * sizeof(*p_ixn_count_), hipMemcpyDeviceToHost, stream));

        std::vector<double> h_box(9);
        gpuErrchk(hipMemcpyAsync(&h_box[0], d_box, 3 * 3 * sizeof(*d_box), hipMemcpyDeviceToHost, stream));

        // this stream needs to be synchronized so we can be sure that p_ixn_count_ is properly set.
        hipStreamSynchronize(stream);

        // Verify that the cutoff and box size are valid together. If cutoff is greater than half the box
        // then a particle can interact with multiple periodic copies.
        const double db_cutoff = (cutoff_ + nblist_padding_) * 2;

        // Verify that box is orthogonal and the width of the box in all dimensions is greater than twice the cutoff
        for (int i = 0; i < 9; i++) {
            if (i == 0 || i == 4 || i == 8) {
                if (h_box[i] < db_cutoff) {
                    throw std::runtime_error(
                        "Cutoff with padding is more than half of the box width, neighborlist is no longer reliable");
                }
            } else if (h_box[i] != 0.0) {
                throw std::runtime_error("Provided non-ortholinear box, unable to compute nonbonded energy");
            }
        }

        gpuErrchk(hipMemsetAsync(d_rebuild_nblist_, 0, sizeof(*d_rebuild_nblist_), stream));
        gpuErrchk(hipMemcpyAsync(d_nblist_x_, d_x, N * 3 * sizeof(*d_x), hipMemcpyDeviceToDevice, stream));
        gpuErrchk(hipMemcpyAsync(d_nblist_box_, d_box, 3 * 3 * sizeof(*d_box), hipMemcpyDeviceToDevice, stream));
    } else {
        k_gather<<<dim3(ceil_divide(K_, tpb), 3, 1), tpb, 0, stream>>>(K_, d_sorted_atom_idxs_, d_x, d_gathered_x_);
        gpuErrchk(hipPeekAtLastError());
    }

    // do parameter interpolation here
    if (Interpolated) {
        hipError_t result =
            compute_gather_interpolated_.configure(dim3(ceil_divide(K_, tpb), 3, 1), tpb, 0, stream)
                .launch(lambda, K_, d_sorted_atom_idxs_, d_p, d_p + N * 3, d_gathered_p_, d_gathered_dp_dl_);
        if (result != 0) {
            throw std::runtime_error("Driver call to k_gather_interpolated failed");
        }
    } else {
        k_gather<<<dim3(ceil_divide(K_, tpb), 3, 1), tpb, 0, stream>>>(K_, d_sorted_atom_idxs_, d_p, d_gathered_p_);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipMemsetAsync(d_gathered_dp_dl_, 0, K_ * 3 * sizeof(*d_gathered_dp_dl_), stream))
    }

    // reset buffers and sorted accumulators
    if (d_du_dx) {
        gpuErrchk(hipMemsetAsync(d_gathered_du_dx_, 0, K_ * 3 * sizeof(*d_gathered_du_dx_), stream))
    }
    if (d_du_dp) {
        gpuErrchk(hipMemsetAsync(d_gathered_du_dp_, 0, K_ * 3 * sizeof(*d_gathered_du_dp_), stream))
    }

    // update new w coordinates
    // (tbd): cache lambda value for equilibrium calculations
    hipError_t result = compute_w_coords_instance_.configure(ceil_divide(N_, tpb), tpb, 0, stream)
                          .launch(N, lambda, cutoff_, d_lambda_plane_idxs_, d_lambda_offset_idxs_, d_w_, d_dw_dl_);
    if (result != 0) {
        throw std::runtime_error("Driver call to k_compute_w_coords");
    }

    gpuErrchk(hipPeekAtLastError());
    k_gather_2x<<<ceil_divide(K_, tpb), tpb, 0, stream>>>(
        K_, d_sorted_atom_idxs_, d_w_, d_dw_dl_, d_gathered_w_, d_gathered_dw_dl_);
    gpuErrchk(hipPeekAtLastError());

    // look up which kernel we need for this computation
    int kernel_idx = 0;
    kernel_idx |= d_du_dp ? 1 << 0 : 0;
    kernel_idx |= d_du_dl ? 1 << 1 : 0;
    kernel_idx |= d_du_dx ? 1 << 2 : 0;
    kernel_idx |= d_u ? 1 << 3 : 0;

    kernel_ptrs_[kernel_idx]<<<p_ixn_count_[0], tpb, 0, stream>>>(
        K_,
        0,
        d_gathered_x_,
        d_gathered_p_,
        d_box,
        d_gathered_dp_dl_,
        d_gathered_w_,
        d_gathered_dw_dl_,
        beta_,
        cutoff_,
        nblist_.get_ixn_tiles(),
        nblist_.get_ixn_atoms(),
        d_gathered_du_dx_,
        d_gathered_du_dp_,
        d_du_dl, // switch to nullptr if we don't request du_dl
        d_u      // switch to nullptr if we don't request energies
    );

    gpuErrchk(hipPeekAtLastError());

    // coords are N,3
    if (d_du_dx) {
        k_scatter_accum<<<dim3(ceil_divide(K_, tpb), 3, 1), tpb, 0, stream>>>(
            K_, d_sorted_atom_idxs_, d_gathered_du_dx_, d_du_dx);
        gpuErrchk(hipPeekAtLastError());
    }

    // params are N,3
    // this needs to be an accumulated permute
    if (d_du_dp) {
        // scattered assignment updates K_ <= N_ elements; the rest should be 0
        gpuErrchk(hipMemset(d_du_dp_buffer_, 0, N_ * 3 * sizeof(*d_du_dp_buffer_)));
        k_scatter_assign<<<dim3(ceil_divide(K_, tpb), 3, 1), tpb, 0, stream>>>(
            K_, d_sorted_atom_idxs_, d_gathered_du_dp_, d_du_dp_buffer_);
        gpuErrchk(hipPeekAtLastError());
    }

    if (d_du_dp) {
        if (Interpolated) {
            hipError_t result =
                compute_add_du_dp_interpolated_.configure(dim3(ceil_divide(N_, tpb), 3, 1), tpb, 0, stream)
                    .launch(lambda, N, d_du_dp_buffer_, d_du_dp);
            if (result != 0) {
                throw std::runtime_error("Driver call to k_add_du_dp_interpolated failed");
            }
        } else {
            k_add_ull_to_ull<<<dim3(ceil_divide(N_, tpb), 3, 1), tpb, 0, stream>>>(N, d_du_dp_buffer_, d_du_dp);
        }
        gpuErrchk(hipPeekAtLastError());
    }
}

template <typename RealType, bool Interpolated>
void NonbondedAllPairs<RealType, Interpolated>::du_dp_fixed_to_float(
    const int N, const int P, const unsigned long long *du_dp, double *du_dp_float) {

    // In the interpolated case we have derivatives for the initial and final parameters
    const int num_tuples = Interpolated ? N * 2 : N;

    for (int i = 0; i < num_tuples; i++) {
        const int idx_charge = i * 3 + 0;
        const int idx_sig = i * 3 + 1;
        const int idx_eps = i * 3 + 2;
        du_dp_float[idx_charge] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DCHARGE>(du_dp[idx_charge]);
        du_dp_float[idx_sig] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DSIG>(du_dp[idx_sig]);
        du_dp_float[idx_eps] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DEPS>(du_dp[idx_eps]);
    }
}

template class NonbondedAllPairs<double, true>;
template class NonbondedAllPairs<float, true>;
template class NonbondedAllPairs<double, false>;
template class NonbondedAllPairs<float, false>;

} // namespace timemachine
