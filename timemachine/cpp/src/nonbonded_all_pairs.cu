#include <hipcub/hipcub.hpp>
#include <string>

#include "device_buffer.hpp"
#include "energy_accumulation.hpp"
#include "fixed_point.hpp"
#include "gpu_utils.cuh"
#include "kernels/k_indices.cuh"
#include "kernels/k_nonbonded.cuh"
#include "kernels/k_nonbonded_common.cuh"
#include "kernels/kernel_utils.cuh"
#include "nonbonded_all_pairs.hpp"
#include "nonbonded_common.hpp"
#include "vendored/hilbert.h"

#include <numeric>

static const int STEPS_PER_SORT = 100;

namespace timemachine {

template <typename RealType>
NonbondedAllPairs<RealType>::NonbondedAllPairs(
    const int N,
    const double beta,
    const double cutoff,
    const std::optional<std::set<int>> &atom_idxs,
    const bool disable_hilbert_sort,
    const double nblist_padding)
    : N_(N), K_(atom_idxs ? atom_idxs->size() : N_), beta_(beta), cutoff_(cutoff), steps_since_last_sort_(0),
      d_atom_idxs_(nullptr), nblist_(N_), nblist_padding_(nblist_padding), d_sort_storage_(nullptr),
      d_sort_storage_bytes_(0), disable_hilbert_(disable_hilbert_sort),

      kernel_ptrs_({// enumerate over every possible kernel combination
                    // Set threads to 1 if not computing energy to reduced unused shared memory
                    // U: Compute U
                    // X: Compute DU_DX
                    // P: Compute DU_DP
                    //                                                                 U  X  P
                    &k_nonbonded_unified<RealType, NONBONDED_KERNEL_THREADS_PER_BLOCK, 0, 0, 0>,
                    &k_nonbonded_unified<RealType, NONBONDED_KERNEL_THREADS_PER_BLOCK, 0, 0, 1>,
                    &k_nonbonded_unified<RealType, NONBONDED_KERNEL_THREADS_PER_BLOCK, 0, 1, 0>,
                    &k_nonbonded_unified<RealType, NONBONDED_KERNEL_THREADS_PER_BLOCK, 0, 1, 1>,
                    &k_nonbonded_unified<RealType, NONBONDED_KERNEL_THREADS_PER_BLOCK, 1, 0, 0>,
                    &k_nonbonded_unified<RealType, NONBONDED_KERNEL_THREADS_PER_BLOCK, 1, 0, 1>,
                    &k_nonbonded_unified<RealType, NONBONDED_KERNEL_THREADS_PER_BLOCK, 1, 1, 0>,
                    &k_nonbonded_unified<RealType, NONBONDED_KERNEL_THREADS_PER_BLOCK, 1, 1, 1>}) {

    std::vector<int> atom_idxs_h;
    if (atom_idxs) {
        atom_idxs_h = std::vector<int>(atom_idxs->begin(), atom_idxs->end());
    } else {
        atom_idxs_h = std::vector<int>(N_);
        std::iota(atom_idxs_h.begin(), atom_idxs_h.end(), 0);
    }
    verify_atom_idxs(N_, atom_idxs_h);

    cudaSafeMalloc(&d_atom_idxs_, N_ * sizeof(*d_atom_idxs_));

    cudaSafeMalloc(&d_sorted_atom_idxs_, N_ * sizeof(*d_sorted_atom_idxs_));

    cudaSafeMalloc(&d_gathered_x_, N_ * 3 * sizeof(*d_gathered_x_));
    cudaSafeMalloc(&d_u_buffer_, NONBONDED_KERNEL_BLOCKS * sizeof(*d_u_buffer_));

    cudaSafeMalloc(&d_gathered_p_, N_ * PARAMS_PER_ATOM * sizeof(*d_gathered_p_));
    cudaSafeMalloc(&d_gathered_du_dx_, N_ * 3 * sizeof(*d_gathered_du_dx_));
    cudaSafeMalloc(&d_gathered_du_dp_, N_ * PARAMS_PER_ATOM * sizeof(*d_gathered_du_dp_));

    cudaSafeMalloc(&d_nblist_x_, N_ * 3 * sizeof(*d_nblist_x_));
    gpuErrchk(hipMemset(d_nblist_x_, 0, N_ * 3 * sizeof(*d_nblist_x_))); // set non-sensical positions
    cudaSafeMalloc(&d_nblist_box_, 3 * 3 * sizeof(*d_nblist_box_));
    gpuErrchk(hipMemset(d_nblist_box_, 0, 3 * 3 * sizeof(*d_nblist_box_)));
    cudaSafeMalloc(&d_rebuild_nblist_, 1 * sizeof(*d_rebuild_nblist_));
    gpuErrchk(hipHostMalloc(&p_rebuild_nblist_, 1 * sizeof(*p_rebuild_nblist_)));

    cudaSafeMalloc(&d_sort_keys_in_, N_ * sizeof(*d_sort_keys_in_));
    cudaSafeMalloc(&d_sort_keys_out_, N_ * sizeof(*d_sort_keys_out_));
    cudaSafeMalloc(&d_sort_vals_in_, N_ * sizeof(*d_sort_vals_in_));

    // initialize hilbert curve
    std::vector<unsigned int> bin_to_idx(HILBERT_GRID_DIM * HILBERT_GRID_DIM * HILBERT_GRID_DIM);
    for (int i = 0; i < HILBERT_GRID_DIM; i++) {
        for (int j = 0; j < HILBERT_GRID_DIM; j++) {
            for (int k = 0; k < HILBERT_GRID_DIM; k++) {

                bitmask_t hilbert_coords[3];
                hilbert_coords[0] = i;
                hilbert_coords[1] = j;
                hilbert_coords[2] = k;

                unsigned int bin = static_cast<unsigned int>(hilbert_c2i(3, HILBERT_N_BITS, hilbert_coords));
                bin_to_idx[i * HILBERT_GRID_DIM * HILBERT_GRID_DIM + j * HILBERT_GRID_DIM + k] = bin;
            }
        }
    }

    cudaSafeMalloc(&d_bin_to_idx_, HILBERT_GRID_DIM * HILBERT_GRID_DIM * HILBERT_GRID_DIM * sizeof(*d_bin_to_idx_));
    gpuErrchk(hipMemcpy(
        d_bin_to_idx_,
        &bin_to_idx[0],
        HILBERT_GRID_DIM * HILBERT_GRID_DIM * HILBERT_GRID_DIM * sizeof(*d_bin_to_idx_),
        hipMemcpyHostToDevice));

    // estimate size needed to do radix sorting, this can use uninitialized data.
    hipcub::DeviceRadixSort::SortPairs(
        nullptr, d_sort_storage_bytes_, d_sort_keys_in_, d_sort_keys_out_, d_sort_vals_in_, d_sorted_atom_idxs_, K_);

    gpuErrchk(hipPeekAtLastError());

    cudaSafeMalloc(&d_sort_storage_, d_sort_storage_bytes_);

    this->set_atom_idxs(atom_idxs_h);

    // Create event with timings disabled as timings slow down events
    gpuErrchk(hipEventCreateWithFlags(&nblist_flag_sync_event_, hipEventDisableTiming));
};

template <typename RealType> NonbondedAllPairs<RealType>::~NonbondedAllPairs() {

    gpuErrchk(hipFree(d_atom_idxs_));

    gpuErrchk(hipFree(d_sorted_atom_idxs_));

    gpuErrchk(hipFree(d_bin_to_idx_));
    gpuErrchk(hipFree(d_gathered_x_));
    gpuErrchk(hipFree(d_u_buffer_));

    gpuErrchk(hipFree(d_gathered_p_));
    gpuErrchk(hipFree(d_gathered_du_dx_));
    gpuErrchk(hipFree(d_gathered_du_dp_));

    gpuErrchk(hipFree(d_sort_keys_in_));
    gpuErrchk(hipFree(d_sort_keys_out_));
    gpuErrchk(hipFree(d_sort_vals_in_));
    gpuErrchk(hipFree(d_sort_storage_));

    gpuErrchk(hipFree(d_nblist_x_));
    gpuErrchk(hipFree(d_nblist_box_));
    gpuErrchk(hipFree(d_rebuild_nblist_));
    gpuErrchk(hipHostFree(p_rebuild_nblist_));

    gpuErrchk(hipEventDestroy(nblist_flag_sync_event_));
};

// Set atom idxs upon which to compute the non-bonded potential. This will trigger a neighborlist rebuild.
template <typename RealType> void NonbondedAllPairs<RealType>::set_atom_idxs(const std::vector<int> &atom_idxs) {
    verify_atom_idxs(N_, atom_idxs);
    const hipStream_t stream = static_cast<hipStream_t>(0);
    std::vector<unsigned int> unsigned_idxs = std::vector<unsigned int>(atom_idxs.begin(), atom_idxs.end());
    DeviceBuffer<unsigned int> atom_idxs_buffer(atom_idxs.size());
    atom_idxs_buffer.copy_from(&unsigned_idxs[0]);
    this->set_atom_idxs_device(atom_idxs.size(), atom_idxs_buffer.data, stream);
    gpuErrchk(hipStreamSynchronize(stream));
}

template <typename RealType> std::vector<int> NonbondedAllPairs<RealType>::get_atom_idxs() {
    std::vector<unsigned int> atom_idxs_buffer(K_);
    gpuErrchk(hipMemcpy(&atom_idxs_buffer[0], d_atom_idxs_, K_ * sizeof(*d_atom_idxs_), hipMemcpyDeviceToHost));
    std::vector<int> atom_idxs = std::vector<int>(atom_idxs_buffer.begin(), atom_idxs_buffer.end());
    return atom_idxs;
}

template <typename RealType>
void NonbondedAllPairs<RealType>::set_atom_idxs_device(
    const int K, const unsigned int *d_in_atom_idxs, const hipStream_t stream) {
    if (K < 1) {
        throw std::runtime_error("K must be at least 1");
    }
    if (K > N_) {
        throw std::runtime_error("number of idxs must be less than or equal to N");
    }
    gpuErrchk(
        hipMemcpyAsync(d_atom_idxs_, d_in_atom_idxs, K * sizeof(*d_atom_idxs_), hipMemcpyDeviceToDevice, stream));
    nblist_.resize_device(K, stream);
    // Force the rebuild of the nblist
    gpuErrchk(hipMemsetAsync(d_rebuild_nblist_, 1, 1 * sizeof(*d_rebuild_nblist_), stream));
    this->K_ = K;
    // Reset the steps so that we do a new sort
    this->steps_since_last_sort_ = 0;
}

template <typename RealType> bool NonbondedAllPairs<RealType>::needs_sort() {
    return steps_since_last_sort_ % STEPS_PER_SORT == 0;
}

template <typename RealType>
void NonbondedAllPairs<RealType>::sort(const double *d_coords, const double *d_box, hipStream_t stream) {
    // We must rebuild the neighborlist after sorting, as the neighborlist is tied to a particular sort order
    if (!disable_hilbert_) {
        this->hilbert_sort(d_coords, d_box, stream);
    } else {
        gpuErrchk(hipMemcpyAsync(
            d_sorted_atom_idxs_, d_atom_idxs_, K_ * sizeof(*d_atom_idxs_), hipMemcpyDeviceToDevice, stream));
    }
    gpuErrchk(hipMemsetAsync(d_rebuild_nblist_, 1, sizeof(*d_rebuild_nblist_), stream));
    // Set the pinned memory to indicate that we need to rebuild
    p_rebuild_nblist_[0] = 1;
}

template <typename RealType>
void NonbondedAllPairs<RealType>::hilbert_sort(const double *d_coords, const double *d_box, hipStream_t stream) {

    const int tpb = DEFAULT_THREADS_PER_BLOCK;
    const int B = ceil_divide(K_, tpb);

    k_coords_to_kv_gather<<<B, tpb, 0, stream>>>(
        K_, d_atom_idxs_, d_coords, d_box, d_bin_to_idx_, d_sort_keys_in_, d_sort_vals_in_);

    gpuErrchk(hipPeekAtLastError());

    hipcub::DeviceRadixSort::SortPairs(
        d_sort_storage_,
        d_sort_storage_bytes_,
        d_sort_keys_in_,
        d_sort_keys_out_,
        d_sort_vals_in_,
        d_sorted_atom_idxs_,
        K_,
        0,                            // begin bit
        sizeof(*d_sort_keys_in_) * 8, // end bit
        stream                        // cudaStream
    );

    gpuErrchk(hipPeekAtLastError());
}

template <typename RealType>
void NonbondedAllPairs<RealType>::execute_device(
    const int N,
    const int P,
    const double *d_x,
    const double *d_p,   // N * PARAMS_PER_ATOM
    const double *d_box, // 3 * 3
    unsigned long long *d_du_dx,
    unsigned long long *d_du_dp,
    __int128 *d_u,
    hipStream_t stream) {

    // (ytz) the nonbonded algorithm proceeds as follows:

    // (done in constructor), construct a hilbert curve mapping each of the HILBERT_GRID_DIM x HILBERT_GRID_DIM x HILBERT_GRID_DIM cells into an index.
    // a. decide if we need to rebuild the neighborlist, if so:
    //     - look up which cell each particle belongs to, and its linear index along the hilbert curve.
    //     - use radix pair sort keyed on the hilbert index with values equal to the atomic index
    //     - resulting sorted values is the permutation array.
    //     - permute coords
    // b. else:
    //     - permute new coords
    // c. permute parameters
    // d. compute the nonbonded interactions using the neighborlist
    // e. inverse permute the forces, du/dps into the original index.
    // f. u is buffered into a per-particle array, and then reduced.

    if (N != N_) {
        throw std::runtime_error(
            "NonbondedAllPairs::execute_device(): expected N == N_, got N=" + std::to_string(N) +
            ", N_=" + std::to_string(N_));
    }

    if (P != N_ * PARAMS_PER_ATOM) {
        throw std::runtime_error(
            "NonbondedAllPairs::execute_device(): expected P == N_*" + std::to_string(PARAMS_PER_ATOM) + ", got P=" +
            std::to_string(P) + ", N_*" + std::to_string(PARAMS_PER_ATOM) + "=" + std::to_string(N_ * PARAMS_PER_ATOM));
    }

    const int tpb = DEFAULT_THREADS_PER_BLOCK;

    if (this->needs_sort()) {
        // Sorting always triggers a neighborlist rebuild
        this->sort(d_x, d_box, stream);
    } else {
        // (ytz) see if we need to rebuild the neighborlist.
        k_check_rebuild_coords_and_box_gather<RealType><<<ceil_divide(K_, tpb), tpb, 0, stream>>>(
            K_, d_atom_idxs_, d_x, d_nblist_x_, d_box, d_nblist_box_, nblist_padding_, d_rebuild_nblist_);
        gpuErrchk(hipPeekAtLastError());

        // we can optimize this away by doing the check on the GPU directly.
        gpuErrchk(hipMemcpyAsync(
            p_rebuild_nblist_, d_rebuild_nblist_, 1 * sizeof(*p_rebuild_nblist_), hipMemcpyDeviceToHost, stream));
        gpuErrchk(hipEventRecord(nblist_flag_sync_event_, stream));
    }
    // compute new coordinates/params
    k_gather_coords_and_params<<<dim3(ceil_divide(K_, tpb), PARAMS_PER_ATOM, 1), tpb, 0, stream>>>(
        K_, d_sorted_atom_idxs_, d_x, d_p, d_gathered_x_, d_gathered_p_);
    gpuErrchk(hipPeekAtLastError());

    // reset buffers and sorted accumulators
    if (d_du_dx) {
        gpuErrchk(hipMemsetAsync(d_gathered_du_dx_, 0, K_ * 3 * sizeof(*d_gathered_du_dx_), stream));
    }
    if (d_du_dp) {
        gpuErrchk(hipMemsetAsync(d_gathered_du_dp_, 0, K_ * PARAMS_PER_ATOM * sizeof(*d_gathered_du_dp_), stream));
    }
    // Syncing to an event allows having additional kernels run while we synchronize
    // Note that if no event is recorded, this is effectively a no-op, such as in the case of sorting.
    gpuErrchk(hipEventSynchronize(nblist_flag_sync_event_));
    if (p_rebuild_nblist_[0] > 0) {

        nblist_.build_nblist_device(K_, d_gathered_x_, d_box, cutoff_ + nblist_padding_, stream);

        gpuErrchk(hipMemsetAsync(d_rebuild_nblist_, 0, sizeof(*d_rebuild_nblist_), stream));
        gpuErrchk(hipMemcpyAsync(d_nblist_x_, d_x, N * 3 * sizeof(*d_x), hipMemcpyDeviceToDevice, stream));
        gpuErrchk(hipMemcpyAsync(d_nblist_box_, d_box, 3 * 3 * sizeof(*d_box), hipMemcpyDeviceToDevice, stream));
    }

    // look up which kernel we need for this computation
    int kernel_idx = 0;
    kernel_idx |= d_du_dp ? 1 << 0 : 0;
    kernel_idx |= d_du_dx ? 1 << 1 : 0;
    kernel_idx |= d_u ? 1 << 2 : 0;

    kernel_ptrs_[kernel_idx]<<<NONBONDED_KERNEL_BLOCKS, NONBONDED_KERNEL_THREADS_PER_BLOCK, 0, stream>>>(
        K_,
        nblist_.get_num_row_idxs(),
        nblist_.get_ixn_count(),
        d_gathered_x_,
        d_gathered_p_,
        d_box,
        beta_,
        cutoff_,
        nblist_.get_row_idxs(),
        nblist_.get_ixn_tiles(),
        nblist_.get_ixn_atoms(),
        d_gathered_du_dx_,
        d_gathered_du_dp_,
        d_u == nullptr ? nullptr : d_u_buffer_ // switch to nullptr if we don't request energies,
    );
    gpuErrchk(hipPeekAtLastError());

    // coords are N,3
    if (d_du_dx) {
        k_scatter_accum<<<dim3(ceil_divide(K_, tpb), 3, 1), tpb, 0, stream>>>(
            K_, d_sorted_atom_idxs_, d_gathered_du_dx_, d_du_dx);
        gpuErrchk(hipPeekAtLastError());
    }

    // params are N, PARAMS_PER_ATOM
    // this needs to be an accumulated permute
    if (d_du_dp) {
        k_scatter_accum<<<dim3(ceil_divide(K_, tpb), PARAMS_PER_ATOM, 1), tpb, 0, stream>>>(
            K_, d_sorted_atom_idxs_, d_gathered_du_dp_, d_du_dp);
        gpuErrchk(hipPeekAtLastError());
    }

    if (d_u) {
        accumulate_energy(NONBONDED_KERNEL_BLOCKS, d_u_buffer_, d_u, stream);
    }
    // Increment steps
    steps_since_last_sort_++;
}

template <typename RealType>
void NonbondedAllPairs<RealType>::du_dp_fixed_to_float(
    const int N, const int P, const unsigned long long *du_dp, double *du_dp_float) {

    for (int i = 0; i < N; i++) {
        const int idx = i * PARAMS_PER_ATOM;
        const int idx_charge = idx + PARAM_OFFSET_CHARGE;
        const int idx_sig = idx + PARAM_OFFSET_SIG;
        const int idx_eps = idx + PARAM_OFFSET_EPS;
        const int idx_w = idx + PARAM_OFFSET_W;

        du_dp_float[idx_charge] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DCHARGE>(du_dp[idx_charge]);
        du_dp_float[idx_sig] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DSIG>(du_dp[idx_sig]);
        du_dp_float[idx_eps] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DEPS>(du_dp[idx_eps]);
        du_dp_float[idx_w] = FIXED_TO_FLOAT_DU_DP<double, FIXED_EXPONENT_DU_DW>(du_dp[idx_w]);
    }
}

template class NonbondedAllPairs<double>;
template class NonbondedAllPairs<float>;

} // namespace timemachine
