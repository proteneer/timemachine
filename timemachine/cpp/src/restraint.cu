#include <chrono>
#include <iostream>
#include <vector>
#include <complex>
#include "restraint.hpp"
#include "gpu_utils.cuh"
#include "k_restraint.cuh"

namespace timemachine {

template <typename RealType>
Restraint<RealType>::Restraint(
    const std::vector<int> &bond_idxs, // [N]
    const std::vector<double> &params,
    const std::vector<int> &lambda_flags
) : B_(bond_idxs.size()/2) {


    if(bond_idxs.size() % 2 != 0) {
        throw std::runtime_error("bond_idxs.size() must be exactly 2*B");
    }

    if(params.size() % 3 != 0) {
        throw std::runtime_error("params.size() must be exactly 3*B");
    }
    
    if(params.size()/3 != B_) {
        throw std::runtime_error("params.size() must be equal to B*3");
    }

    for(int b=0; b < B_; b++) {
        auto src = bond_idxs[b*2+0];
        auto dst = bond_idxs[b*2+1];
        if(src == dst) {
            throw std::runtime_error("src == dst");
        }
    }

    gpuErrchk(hipMalloc(&d_lambda_flags_, B_*sizeof(*d_lambda_flags_)));
    gpuErrchk(hipMemcpy(d_lambda_flags_, &lambda_flags[0], B_*sizeof(*d_lambda_flags_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_bond_idxs_, B_*2*sizeof(*d_bond_idxs_)));
    gpuErrchk(hipMemcpy(d_bond_idxs_, &bond_idxs[0], B_*2*sizeof(*d_bond_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_bond_idxs_, B_*2*sizeof(*d_bond_idxs_)));
    gpuErrchk(hipMemcpy(d_bond_idxs_, &bond_idxs[0], B_*2*sizeof(*d_bond_idxs_), hipMemcpyHostToDevice));


    gpuErrchk(hipMalloc(&d_params_, B_*3*sizeof(*d_params_)));
    gpuErrchk(hipMemcpy(d_params_, &params[0], B_*3*sizeof(*d_params_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_du_dp_primals_, B_*3*sizeof(*d_du_dp_primals_)));
    gpuErrchk(hipMemset(d_du_dp_primals_, 0, B_*3*sizeof(*d_du_dp_primals_)));

    gpuErrchk(hipMalloc(&d_du_dp_tangents_, B_*3*sizeof(*d_du_dp_tangents_)));
    gpuErrchk(hipMemset(d_du_dp_tangents_, 0, B_*3*sizeof(*d_du_dp_tangents_)));

};

template <typename RealType>
Restraint<RealType>::~Restraint() {
    gpuErrchk(hipFree(d_bond_idxs_));
    gpuErrchk(hipFree(d_lambda_flags_));

    gpuErrchk(hipFree(d_params_));
    gpuErrchk(hipFree(d_du_dp_primals_));
    gpuErrchk(hipFree(d_du_dp_tangents_));
};



template <typename RealType>
void Restraint<RealType>::get_du_dp_primals(double *buf) {
    gpuErrchk(hipMemcpy(buf, d_du_dp_primals_, B_*3*sizeof(*d_params_), hipMemcpyDeviceToHost));
}

template <typename RealType>
void Restraint<RealType>::get_du_dp_tangents(double *buf) {
    gpuErrchk(hipMemcpy(buf, d_du_dp_tangents_, B_*3*sizeof(*d_params_), hipMemcpyDeviceToHost));
}


template <typename RealType>
void Restraint<RealType>::execute_lambda_inference_device(
    const int N,
    // const int P,
    const double *d_coords_primals,
    // const double *d_params_primals,
    const double lambda_primal,
    unsigned long long *d_out_coords_primals, // du/dx
    double *d_out_lambda_primals, // du/dl
    double *d_out_energy_primal, // U
    hipStream_t stream) {

    int tpb = 32;
    int blocks = (B_+tpb-1)/tpb;
    k_restraint_inference<RealType><<<blocks, tpb, 0, stream>>>(
        B_,
        d_coords_primals,
        d_params_,
        lambda_primal,
        d_bond_idxs_,
        d_lambda_flags_,
        d_out_coords_primals,
        d_out_lambda_primals,
        d_out_energy_primal
    );
    gpuErrchk(hipPeekAtLastError());

    // auto finish = std::chrono::high_resolution_clock::now();
    // std::chrono::duration<double> elapsed = finish - start;
    // std::cout << "Restraint Elapsed time: " << elapsed.count() << " s\n";

};

template <typename RealType>
void Restraint<RealType>::execute_lambda_jvp_device(
    const int N,
    // const int P,
    const double *d_coords_primals,
    const double *d_coords_tangents,
    // const double *d_params_primals,
    const double lambda_primal, // unused
    const double lambda_tangent, // unused
    double *d_out_coords_primals,
    double *d_out_coords_tangents,
    // double *d_out_params_primals,
    // double *d_out_params_tangents,
    hipStream_t stream) {

    int tpb = 32;
    int blocks = (B_+tpb-1)/tpb;

    k_restraint_jvp<RealType><<<blocks, tpb, 0, stream>>>(
        B_,
        d_coords_primals,
        d_coords_tangents,
        d_params_,
        lambda_primal,
        lambda_tangent,
        d_bond_idxs_,
        d_lambda_flags_,
        d_out_coords_primals,
        d_out_coords_tangents,
        d_du_dp_primals_,
        d_du_dp_tangents_
    );

    // hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    // auto finish = std::chrono::high_resolution_clock::now();
    // std::chrono::duration<double> elapsed = finish - start;
    // std::cout << "Restraint Elapsed time: " << elapsed.count() << " s\n";

}

template class Restraint<double>;
template class Restraint<float>;

} // namespace timemachine