#include <cassert>
#include <chrono>
#include <iostream>
#include <vector>
#include <algorithm>
#include <complex>
#include <cstdlib>
#include <hipcub/hipcub.hpp>

#include "nonbonded.hpp"
#include "hilbert.h"
#include "gpu_utils.cuh"

#include "k_nonbonded.cuh"

namespace timemachine {

template <typename RealType>
Nonbonded<RealType>::Nonbonded(
    const std::vector<int> &exclusion_idxs, // [E,2]
    const std::vector<double> &scales, // [E, 2]
    const std::vector<int> &lambda_offset_idxs, // [N]
    double beta,
    double cutoff
) :  N_(lambda_offset_idxs.size()),
    cutoff_(cutoff),
    E_(exclusion_idxs.size()/2),
    nblist_(lambda_offset_idxs.size()),
    beta_(beta),
    d_sort_storage_(nullptr),
    d_sort_storage_bytes_(0) {

    if(lambda_offset_idxs.size() != N_) {
        throw std::runtime_error("lambda offset idxs need to have size N");
    }

    if(scales.size()/2 != E_) {
        throw std::runtime_error("bad scales size!");
    }

    gpuErrchk(hipMalloc(&d_lambda_offset_idxs_, N_*sizeof(*d_lambda_offset_idxs_)));
    gpuErrchk(hipMemcpy(d_lambda_offset_idxs_, &lambda_offset_idxs[0], N_*sizeof(*d_lambda_offset_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_du_dl_buffer_, N_*sizeof(*d_du_dl_buffer_)));
    gpuErrchk(hipMalloc(&d_u_buffer_, N_*sizeof(*d_u_buffer_)));

    gpuErrchk(hipMalloc(&d_du_dl_reduce_sum_, 1*sizeof(*d_du_dl_reduce_sum_)));
    gpuErrchk(hipMalloc(&d_u_reduce_sum_, 1*sizeof(*d_u_reduce_sum_)));

    gpuErrchk(hipMalloc(&d_perm_, N_*sizeof(*d_perm_)));

    gpuErrchk(hipMalloc(&d_sorted_lambda_offset_idxs_, N_*sizeof(*d_sorted_lambda_offset_idxs_)));
    gpuErrchk(hipMalloc(&d_sorted_x_, N_*3*sizeof(*d_sorted_x_)));
    gpuErrchk(hipMalloc(&d_sorted_p_, N_*3*sizeof(*d_sorted_p_)));
    gpuErrchk(hipMalloc(&d_sorted_du_dx_, N_*3*sizeof(*d_sorted_du_dx_)));
    gpuErrchk(hipMalloc(&d_sorted_du_dp_, N_*3*sizeof(*d_sorted_du_dp_)));
    gpuErrchk(hipMalloc(&d_du_dp_buffer_, N_*3*sizeof(*d_du_dp_buffer_)));

    gpuErrchk(hipMalloc(&d_exclusion_idxs_, E_*2*sizeof(*d_exclusion_idxs_)));
    gpuErrchk(hipMemcpy(d_exclusion_idxs_, &exclusion_idxs[0], E_*2*sizeof(*d_exclusion_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_scales_, E_*2*sizeof(*d_scales_)));
    gpuErrchk(hipMemcpy(d_scales_, &scales[0], E_*2*sizeof(*d_scales_), hipMemcpyHostToDevice));
    
    gpuErrchk(hipHostMalloc(&p_ixn_count_, 1*sizeof(*p_ixn_count_)));

    gpuErrchk(hipMalloc(&d_sort_keys_in_, N_*sizeof(d_sort_keys_in_)));
    gpuErrchk(hipMalloc(&d_sort_keys_out_, N_*sizeof(d_sort_keys_out_)));
    gpuErrchk(hipMalloc(&d_sort_vals_in_, N_*sizeof(d_sort_vals_in_)));

    // initialize hilbert curve
    std::vector<unsigned int> bin_to_idx(256*256*256);
    for(int i=0; i < 256; i++) {
        for(int j=0; j < 256; j++) {
            for(int k=0; k < 256; k++) {

                bitmask_t hilbert_coords[3];
                hilbert_coords[0] = i;
                hilbert_coords[1] = j;
                hilbert_coords[2] = k;

                unsigned int bin = static_cast<unsigned int>(hilbert_c2i(3, 8, hilbert_coords));
                bin_to_idx[i*256*256 + j*256 + k] = bin;

            }
        }
    }

    gpuErrchk(hipMalloc(&d_bin_to_idx_, 256*256*256*sizeof(*d_bin_to_idx_)));
    gpuErrchk(hipMemcpy(d_bin_to_idx_, &bin_to_idx[0], 256*256*256*sizeof(*d_bin_to_idx_), hipMemcpyHostToDevice));   

    // estimate size needed to do radix sorting, this can use uninitialized data.
    hipcub::DeviceRadixSort::SortPairs(
        d_sort_storage_,
        d_sort_storage_bytes_,
        d_sort_keys_in_,
        d_sort_keys_out_,
        d_sort_vals_in_,
        d_perm_,
        N_
    );

    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipMalloc(&d_sort_storage_, d_sort_storage_bytes_));

};


template <typename RealType>
Nonbonded<RealType>::~Nonbonded() {

    gpuErrchk(hipFree(d_exclusion_idxs_));
    gpuErrchk(hipFree(d_scales_));
    gpuErrchk(hipFree(d_lambda_offset_idxs_));

    gpuErrchk(hipFree(d_du_dl_reduce_sum_));
    gpuErrchk(hipFree(d_u_reduce_sum_));

    gpuErrchk(hipFree(d_du_dp_buffer_));
    gpuErrchk(hipFree(d_du_dl_buffer_));
    gpuErrchk(hipFree(d_u_buffer_));
    gpuErrchk(hipFree(d_perm_)); // nullptr if we never built nblist
    
    gpuErrchk(hipFree(d_bin_to_idx_));
    gpuErrchk(hipFree(d_sorted_x_));
    gpuErrchk(hipFree(d_sorted_p_));
    gpuErrchk(hipFree(d_sorted_du_dx_));
    gpuErrchk(hipFree(d_sorted_du_dp_));
    gpuErrchk(hipFree(d_sorted_lambda_offset_idxs_));

    gpuErrchk(hipFree(d_sort_keys_in_));
    gpuErrchk(hipFree(d_sort_keys_out_));
    gpuErrchk(hipFree(d_sort_vals_in_));
    gpuErrchk(hipFree(d_sort_storage_));

    gpuErrchk(hipHostFree(p_ixn_count_));
};


template <typename RealType>
void Nonbonded<RealType>::hilbert_sort(
    const double *d_coords,
    const double *d_box,
    hipStream_t stream) {

    const int B = (N_+32-1)/32;
    const int tpb = 32;

    k_coords_to_kv<<<B, tpb, 0, stream>>>(N_, d_coords, d_box, d_bin_to_idx_, d_sort_keys_in_, d_sort_vals_in_);

    gpuErrchk(hipPeekAtLastError());

    hipcub::DeviceRadixSort::SortPairs(
        d_sort_storage_,
        d_sort_storage_bytes_,
        d_sort_keys_in_,
        d_sort_keys_out_,
        d_sort_vals_in_,
        d_perm_,
        N_,
        0, // begin bit
        sizeof(*d_sort_keys_in_)*8, // end bit
        stream // cudaStream
    );

    gpuErrchk(hipPeekAtLastError());

}

template <typename RealType>
void Nonbonded<RealType>::execute_device(
        const int N,
        const int P,
        const double *d_x,
        const double *d_p, // N*3
        const double *d_box,
        const double lambda,
        unsigned long long *d_du_dx,
        double *d_du_dp,
        double *d_du_dl,
        double *d_u,
        hipStream_t stream) {

    // (ytz) the nonbonded algorithm proceeds as follows:

    // 0. (done in constructor), construct a hilbert curve mapping each of the 256x256x256 cells into an index.
    // 1. look up which cell each particle belongs to, and its linear index along the hilbert curve.
    // 2. use radix pair sort keyed on the hilbert index with values equal to the atomic index
    // 3. resulting sorted values is the permutation array.
    // 4. permute coords, params, lambda_offsets
    // 5. compute the neighborlist into tiles
    // 6. compute the nonbonded interactions using the neighborlist
    // 7. inverse permute the forces, du/dps into the original index.
    // 8. u and du/dl is buffered into a per-particle array, and then reduced.
    // 9. note that du/dl is not an exact per-particle du/dl - it is only used for reduction purposes.

    // assert(N == N_);
    // assert(P == N_*3);



    if(N != N_) {
        throw std::runtime_error("N != N_");
    }

    const int B = (N+32-1)/32;
    const int tpb = 32;

    // randomly re-hilbert sort
    this->hilbert_sort(d_x, d_box, stream);

	dim3 dimGrid(B, 3, 1);

    k_permute<<<dimGrid, tpb, 0, stream>>>(N, d_perm_, d_p, d_sorted_p_);
    gpuErrchk(hipPeekAtLastError());

    k_permute<<<B, tpb, 0, stream>>>(N, d_perm_, d_lambda_offset_idxs_, d_sorted_lambda_offset_idxs_);
    gpuErrchk(hipPeekAtLastError());

    k_permute<<<dimGrid, tpb, 0, stream>>>(N, d_perm_, d_x, d_sorted_x_);
    gpuErrchk(hipPeekAtLastError());

    // hipDeviceSynchronize();
    // auto start = std::chrono::high_resolution_clock::now();

    nblist_.build_nblist_device(
        N,
        d_sorted_x_,
        d_box,
        cutoff_,
        stream
    );

    gpuErrchk(hipMemcpyAsync(p_ixn_count_, nblist_.get_ixn_count(), 1*sizeof(*p_ixn_count_), hipMemcpyDeviceToHost, stream));
    // this stream needs to be synchronized so we can be sure that p_ixn_count_ is properly set.
    // reset buffers and sorted accumulators
    if(d_du_dx) {
	   gpuErrchk(hipMemsetAsync(d_sorted_du_dx_, 0, N*3*sizeof(*d_sorted_du_dx_), stream))
    }
    if(d_du_dp) {
	   gpuErrchk(hipMemsetAsync(d_sorted_du_dp_, 0, N*3*sizeof(*d_sorted_du_dp_), stream))
    }
    if(d_du_dl) {
        gpuErrchk(hipMemsetAsync(d_du_dl_buffer_, 0, N*sizeof(*d_du_dl_buffer_), stream));
        gpuErrchk(hipMemsetAsync(d_du_dl_reduce_sum_, 0, 1*sizeof(*d_du_dl_reduce_sum_), stream)); 
    }
    if(d_u) {
        gpuErrchk(hipMemsetAsync(d_u_buffer_, 0, N*sizeof(*d_u_buffer_), stream));
        gpuErrchk(hipMemsetAsync(d_u_reduce_sum_, 0, 1*sizeof(*d_u_reduce_sum_), stream));
    }

    gpuErrchk(hipStreamSynchronize(stream));


    k_nonbonded<RealType><<<p_ixn_count_[0], 32, 0, stream>>>(
        N,
        d_sorted_x_,
        d_sorted_p_,
        d_box,
        lambda,
        d_sorted_lambda_offset_idxs_,
        beta_,
        cutoff_,
        nblist_.get_ixn_tiles(),
        nblist_.get_ixn_atoms(),
        d_du_dx ? d_sorted_du_dx_ : nullptr,
        d_du_dp ? d_sorted_du_dp_ : nullptr,
        d_du_dl ? d_du_dl_buffer_ : nullptr, // switch to nullptr if we don't request du_dl
        d_u ? d_u_buffer_ : nullptr // switch to nullptr if we don't request energies
    );

    // hipDeviceSynchronize();

    // auto end = std::chrono::high_resolution_clock::now();

    // std::chrono::duration<double> elapsed = end - start;
    // std::cout << "NB Forces time: " << elapsed.count() << "ms\n";

    // hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    // coords are N,3
    if(d_du_dx) {
        k_inv_permute_accum<<<dimGrid, tpb, 0, stream>>>(N, d_perm_, d_sorted_du_dx_, d_du_dx);
        gpuErrchk(hipPeekAtLastError());
    }

    // params are N,3
    // this needs to be an accumlated permute
    if(d_du_dp) {
        k_inv_permute_assign<<<dimGrid, tpb, 0, stream>>>(N, d_perm_, d_sorted_du_dp_, d_du_dp_buffer_);
        gpuErrchk(hipPeekAtLastError());
    }

    // exclusions use the non-sorted version
    if(E_ > 0) {

        const int tpb = 32;
        dim3 dimGridExclusions((E_+tpb-1)/tpb, 1, 1);

        k_nonbonded_exclusions<RealType><<<dimGridExclusions, tpb, 0, stream>>>(
            E_,
            d_x,
            d_p,
            d_box,
            lambda,
            d_lambda_offset_idxs_,
            d_exclusion_idxs_,
            d_scales_,
            beta_,
            cutoff_,
            d_du_dx,
            d_du_dp_buffer_,
            d_du_dl ? d_du_dl_buffer_ : nullptr, // switch to nullptr if we don't request du_dl
            d_u ? d_u_buffer_ : nullptr // switch to nullptr if we don't request energies
        );
        gpuErrchk(hipPeekAtLastError());
    }

    if(d_du_dp) {
        k_add_ull_to_real<<<dimGrid, tpb, 0, stream>>>(N, d_du_dp_buffer_, d_du_dp);
        gpuErrchk(hipPeekAtLastError());
    }

    // (ytz): we must accumulate in fixed point to get the cancellation of nans
    // otherwise if we convert prematurely floating points become messed up

    if(d_du_dl) {
        k_reduce_buffer<<<B, 32, 0, stream>>>(N, d_du_dl_buffer_, d_du_dl_reduce_sum_);
        gpuErrchk(hipPeekAtLastError());
        k_final_add<<<1, 32, 0, stream>>>(d_du_dl_reduce_sum_, d_du_dl);
        gpuErrchk(hipPeekAtLastError());
    }

    if(d_u) {
        k_reduce_buffer<<<B, 32, 0, stream>>>(N, d_u_buffer_, d_u_reduce_sum_);
        gpuErrchk(hipPeekAtLastError());
        k_final_add<<<1, 32, 0, stream>>>(d_u_reduce_sum_, d_u);
        gpuErrchk(hipPeekAtLastError());
    }
    
}

template class Nonbonded<double>;
template class Nonbonded<float>;

} // namespace timemachine
