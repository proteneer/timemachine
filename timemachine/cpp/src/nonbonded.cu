#include <chrono>
#include <iostream>
#include <vector>
#include <complex>
#include "nonbonded.hpp"
#include "kernel_utils.cuh"
#include "k_nonbonded_deterministic.cuh"

namespace timemachine {

template <typename RealType, int D>
Nonbonded<RealType, D>::Nonbonded(
    const std::vector<int> &charge_param_idxs,
    const std::vector<int> &lj_param_idxs,
    const std::vector<int> &exclusion_idxs, // [E,2]
    const std::vector<int> &charge_scale_idxs, // [E]
    const std::vector<int> &lj_scale_idxs, // [E]
    double cutoff
) :  N_(charge_param_idxs.size()), cutoff_(cutoff), E_(charge_scale_idxs.size()) {

    if(charge_scale_idxs.size()*2 != exclusion_idxs.size()) {
        throw std::runtime_error("charge scale idxs size not half of exclusion size!");
    }

    if(charge_scale_idxs.size() != lj_scale_idxs.size()) {
        throw std::runtime_error("Charge scale idxs does not match LJ scale idxs!");
    }

    if(charge_param_idxs.size()*2 != lj_param_idxs.size()) {
        throw std::runtime_error("Charge param idxs not half of lj param idxs!");
    }

    int tpb = 32;
    int B = (N_+tpb-1)/tpb;

    gpuErrchk(hipMalloc(&d_exclusion_idxs_, E_*2*sizeof(*d_exclusion_idxs_)));
    gpuErrchk(hipMemcpy(d_exclusion_idxs_, &exclusion_idxs[0], E_*2*sizeof(*d_exclusion_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_charge_scale_idxs_, E_*sizeof(*d_charge_scale_idxs_)));
    gpuErrchk(hipMemcpy(d_charge_scale_idxs_, &charge_scale_idxs[0], E_*sizeof(*d_charge_scale_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_lj_scale_idxs_, E_*sizeof(*d_lj_scale_idxs_)));
    gpuErrchk(hipMemcpy(d_lj_scale_idxs_, &lj_scale_idxs[0], E_*sizeof(*d_lj_scale_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_charge_param_idxs_, N_*sizeof(*d_charge_param_idxs_)));
    gpuErrchk(hipMemcpy(d_charge_param_idxs_, &charge_param_idxs[0], N_*sizeof(*d_charge_param_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_lj_param_idxs_, N_*2*sizeof(*d_lj_param_idxs_)));
    gpuErrchk(hipMemcpy(d_lj_param_idxs_, &lj_param_idxs[0], N_*2*sizeof(*d_lj_param_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_block_bounds_ctr_, B*D*sizeof(*d_block_bounds_ctr_)));
    gpuErrchk(hipMalloc(&d_block_bounds_ext_, B*D*sizeof(*d_block_bounds_ext_)));

};

template <typename RealType, int D>
Nonbonded<RealType, D>::~Nonbonded() {
    gpuErrchk(hipFree(d_charge_param_idxs_));
    gpuErrchk(hipFree(d_lj_param_idxs_));
    gpuErrchk(hipFree(d_exclusion_idxs_));
    gpuErrchk(hipFree(d_charge_scale_idxs_));
    gpuErrchk(hipFree(d_lj_scale_idxs_));
    gpuErrchk(hipFree(d_block_bounds_ctr_));
    gpuErrchk(hipFree(d_block_bounds_ext_));
};

template <typename RealType, int D>
void Nonbonded<RealType, D>::execute_device(
    const int N,
    const int P,
    const double *d_coords,
    const double *d_coords_tangents,
    const double *d_params,
    unsigned long long *d_out_coords,
    double *d_out_coords_tangents,
    double *d_out_params_tangents
) {

    if(N != N_) {
        throw std::runtime_error("N != N_");
    }

    int tpb = 32;
    int B = (N_+tpb-1)/tpb;

    gpuErrchk(hipMemset(d_block_bounds_ctr_, 0, B*D*sizeof(*d_block_bounds_ctr_)));
    gpuErrchk(hipMemset(d_block_bounds_ext_, 0, B*D*sizeof(*d_block_bounds_ext_)));

    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    k_find_block_bounds<<<1, B>>>(
        N,
        D,
        B,
        d_coords,
        d_block_bounds_ctr_,
        d_block_bounds_ext_
    );
    hipDeviceSynchronize();

    gpuErrchk(hipPeekAtLastError());

    dim3 dimGrid(B, B, 1); // x, y, z dims
    dim3 dimGridExclusions((E_+tpb-1)/tpb, 1, 1);

    auto start = std::chrono::high_resolution_clock::now();
    if(d_coords_tangents == nullptr) {

        // these can be ran in two streams
        // gpuErrchk(hipMemset(d_out_coords, 0, N*D*sizeof(*d_out_coords)));

        // tbd run in two streams?

        k_nonbonded_inference<RealType, D><<<dimGrid, tpb>>>(
            N,
            d_coords,
            d_params,
            d_charge_param_idxs_,
            d_lj_param_idxs_,
            cutoff_,
            d_block_bounds_ctr_,
            d_block_bounds_ext_,
            d_out_coords
        );

        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        if(E_ > 0) {
            k_nonbonded_exclusion_inference<RealType, D><<<dimGridExclusions, tpb>>>(
                E_,
                d_coords,
                d_params,
                d_exclusion_idxs_,
                d_charge_scale_idxs_,
                d_lj_scale_idxs_,
                d_charge_param_idxs_,
                d_lj_param_idxs_,
                cutoff_,
                d_out_coords
            );
            hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());
        }


        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        // auto finish = std::chrono::high_resolution_clock::now();
        // std::chrono::duration<double> elapsed = finish - start;
        // std::cout << "Nonbonded Elapsed time: " << elapsed.count() << " s\n";

    } else {

        // do *not* accumulate tangents here
        // gpuErrchk(hipMemset(d_out_coords_tangents, 0, N*D*sizeof(RealType)));
        // gpuErrchk(hipMemset(d_out_params_tangents, 0, P*sizeof(RealType)));

        k_nonbonded_jvp<RealType, D><<<dimGrid, tpb>>>(
            N,
            d_coords,
            d_coords_tangents,
            d_params,
            d_charge_param_idxs_,
            d_lj_param_idxs_,
            cutoff_,
            d_block_bounds_ctr_,
            d_block_bounds_ext_,
            d_out_coords_tangents,
            d_out_params_tangents
        );

        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        if(E_ > 0) {
            k_nonbonded_exclusion_jvp<RealType, D><<<dimGridExclusions, tpb>>>(
                E_,
                d_coords,
                d_coords_tangents,
                d_params,
                d_exclusion_idxs_,
                d_charge_scale_idxs_,
                d_lj_scale_idxs_,
                d_charge_param_idxs_,
                d_lj_param_idxs_,
                cutoff_,
                d_out_coords_tangents,
                d_out_params_tangents
            );            

            hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());
        }


        // auto finish = std::chrono::high_resolution_clock::now();
        // std::chrono::duration<double> elapsed = finish - start;
        // std::cout << "Nonbonded JVP Elapsed time: " << elapsed.count() << " s\n";

    }


};

template class Nonbonded<double, 4>;
template class Nonbonded<double, 3>;

template class Nonbonded<float, 4>;
template class Nonbonded<float, 3>;

} // namespace timemachine