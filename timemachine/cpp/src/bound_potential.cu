#include "bound_potential.hpp"
#include "gpu_utils.cuh"

namespace timemachine {

BoundPotential::BoundPotential(std::shared_ptr<Potential> potential, const std::vector<double> &params)
    : size(params.size()), buffer_size_(size), d_p(buffer_size_), potential(potential) {
    set_params(params);
}

void BoundPotential::execute_device(
    const int N,
    const double *d_x,
    const double *d_box,
    unsigned long long *d_du_dx,
    unsigned long long *d_du_dp,
    __int128 *d_u,
    hipStream_t stream) {
    this->potential->execute_device(
        N, this->size, d_x, this->size > 0 ? this->d_p.data : nullptr, d_box, d_du_dx, d_du_dp, d_u, stream);
}

void BoundPotential::execute_host(
    const int N,
    const double *h_x,           // [N,3]
    const double *h_box,         // [3, 3]
    unsigned long long *h_du_dx, // [N, 3]
    __int128 *h_u                // [1]
) {

    const int D = 3;

    DeviceBuffer<double> d_x(N * D);
    DeviceBuffer<double> d_box(D * D);

    d_x.copy_from(h_x);
    d_box.copy_from(h_box);

    std::unique_ptr<DeviceBuffer<unsigned long long>> d_du_dx(nullptr);
    std::unique_ptr<DeviceBuffer<__int128>> d_u(nullptr);

    hipStream_t stream = static_cast<hipStream_t>(0);
    // very important that these are initialized to zero since the kernels themselves just accumulate
    if (h_du_dx != nullptr) {
        d_du_dx.reset(new DeviceBuffer<unsigned long long>(N * D));
        gpuErrchk(hipMemsetAsync(d_du_dx->data, 0, d_du_dx->size, stream));
    }
    if (h_u != nullptr) {
        d_u.reset(new DeviceBuffer<__int128>(1));
        gpuErrchk(hipMemsetAsync(d_u->data, 0, d_u->size, stream));
    }

    this->execute_device(
        N,
        d_x.data,
        d_box.data,
        h_du_dx != nullptr ? d_du_dx->data : nullptr,
        nullptr,
        h_u != nullptr ? d_u->data : nullptr,
        stream);
    gpuErrchk(hipStreamSynchronize(stream));

    if (h_du_dx) {
        d_du_dx->copy_to(h_du_dx);
    }
    if (h_u) {
        d_u->copy_to(h_u);
    }
};

void BoundPotential::set_params(const std::vector<double> &params) {
    if (params.size() != buffer_size_) {
        throw std::runtime_error(
            "parameter size is not equal to device buffer size: " + std::to_string(params.size()) +
            " != " + std::to_string(buffer_size_));
    }
    d_p.copy_from(params.data());
    this->size = params.size();
}

void BoundPotential::set_params_device(const int new_size, const double *d_new_params, const hipStream_t stream) {
    if (static_cast<size_t>(new_size) > buffer_size_) {
        throw std::runtime_error(
            "parameter size is greater than device buffer size: " + std::to_string(new_size) + " > " +
            std::to_string(buffer_size_));
    }
    gpuErrchk(hipMemcpyAsync(d_p.data, d_new_params, new_size * sizeof(*d_p.data), hipMemcpyDeviceToDevice, stream));
    this->size = new_size;
}
} // namespace timemachine
