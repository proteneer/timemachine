#include "bound_potential.hpp"
#include "gpu_utils.cuh"

namespace timemachine {

BoundPotential::BoundPotential(std::shared_ptr<Potential> potential, std::vector<int> shape, const double *h_p)
    : shape(shape), d_p(nullptr), potential(potential) {
    if (this->size() > 0) {
        d_p.reset(new DeviceBuffer<double>(this->size()));
        d_p->copy_from(h_p);
    }
}

void BoundPotential::execute_host(
    const int N,
    const double *h_x,           // [N,3]
    const double *h_box,         // [3, 3]
    unsigned long long *h_du_dx, // [N,3]
    unsigned long long *h_u) {

    const int D = 3;

    DeviceBuffer<double> d_x(N * D);
    DeviceBuffer<double> d_box(D * D);

    d_x.copy_from(h_x);
    d_box.copy_from(h_box);

    DeviceBuffer<unsigned long long> d_du_dx(N * D);
    DeviceBuffer<unsigned long long> d_u(N);

    // very important that these are initialized to zero since the kernels themselves just accumulate
    gpuErrchk(hipMemset(d_du_dx.data, 0, d_du_dx.size));
    gpuErrchk(hipMemset(d_u.data, 0, d_u.size));

    hipStream_t stream = static_cast<hipStream_t>(0);
    this->execute_device(N, d_x.data, d_box.data, d_du_dx.data, nullptr, d_u.data, stream);
    gpuErrchk(hipStreamSynchronize(stream));
    d_du_dx.copy_to(h_du_dx);
    d_u.copy_to(h_u);
};

int BoundPotential::size() const {
    if (shape.size() == 0) {
        return 0;
    }
    int total = 1;
    for (auto s : shape) {
        total *= s;
    }
    return total;
}

} // namespace timemachine
