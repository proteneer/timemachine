#include "chiral_bond_restraint.hpp"
#include "gpu_utils.cuh"
#include "k_chiral_restraint.cuh"
#include "kernel_utils.cuh"
#include "math_utils.cuh"
#include <vector>

namespace timemachine {

template <typename RealType>
ChiralBondRestraint<RealType>::ChiralBondRestraint(const std::vector<int> &idxs, const std::vector<int> &signs)
    : R_(idxs.size() / 4) {

    if (idxs.size() % 4 != 0) {
        throw std::runtime_error("idxs.size() must be exactly 4*R!");
    }

    if (R_ != signs.size()) {
        throw std::runtime_error("signs.size() must be exactly R!");
    }

    for (auto s : signs) {
        if (s != -1 && s != 1) {
            throw std::runtime_error("signs must be comprised exclusively of 1 or -1");
        }
    }

    cudaSafeMalloc(&d_idxs_, R_ * 4 * sizeof(*d_idxs_));
    gpuErrchk(hipMemcpy(d_idxs_, &idxs[0], R_ * 4 * sizeof(*d_idxs_), hipMemcpyHostToDevice));

    cudaSafeMalloc(&d_signs_, R_ * sizeof(*d_signs_));
    gpuErrchk(hipMemcpy(d_signs_, &signs[0], R_ * sizeof(*d_signs_), hipMemcpyHostToDevice));
};

template <typename RealType> ChiralBondRestraint<RealType>::~ChiralBondRestraint() {
    gpuErrchk(hipFree(d_idxs_));
    gpuErrchk(hipFree(d_signs_));
};

template <typename RealType>
void ChiralBondRestraint<RealType>::execute_device(
    const int N,
    const int P,
    const double *d_x,
    const double *d_p,
    const double *d_box,
    unsigned long long *d_du_dx,
    unsigned long long *d_du_dp,
    unsigned long long *d_u,
    hipStream_t stream) {

    if (P != R_) {
        throw std::runtime_error(
            "ChiralBondRestraint::execute_device(): expected P == R, got P=" + std::to_string(P) +
            ", R=" + std::to_string(R_));
    }

    if (R_ > 0) {
        const int tpb = DEFAULT_THREADS_PER_BLOCK;
        const int blocks = ceil_divide(R_, tpb);

        k_chiral_bond_restraint<RealType>
            <<<blocks, tpb, 0, stream>>>(R_, d_x, d_p, d_idxs_, d_signs_, d_du_dx, d_du_dp, d_u);
        gpuErrchk(hipPeekAtLastError());
    }
};

template class ChiralBondRestraint<double>;
template class ChiralBondRestraint<float>;

} // namespace timemachine
