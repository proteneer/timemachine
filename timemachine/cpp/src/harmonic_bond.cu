#include "gpu_utils.cuh"
#include "harmonic_bond.hpp"
#include "k_harmonic_bond.cuh"
#include "kernel_utils.cuh"
#include "math_utils.cuh"
#include <vector>

namespace timemachine {

template <typename RealType>
HarmonicBond<RealType>::HarmonicBond(
    const std::vector<int> &bond_idxs, const std::vector<int> &lambda_mult, const std::vector<int> &lambda_offset)
    : B_(bond_idxs.size() / 2) {

    if (bond_idxs.size() % 2 != 0) {
        throw std::runtime_error("bond_idxs.size() must be exactly 2*k!");
    }

    if (lambda_mult.size() > 0 && lambda_mult.size() != B_) {
        throw std::runtime_error("bad lambda_mult size()");
    }

    if (lambda_offset.size() > 0 && lambda_offset.size() != B_) {
        throw std::runtime_error("bad lambda_offset size()");
    }

    for (int b = 0; b < B_; b++) {
        auto src = bond_idxs[b * 2 + 0];
        auto dst = bond_idxs[b * 2 + 1];
        if (src == dst) {
            throw std::runtime_error("src == dst");
        }
    }

    gpuErrchk(hipMalloc(&d_bond_idxs_, B_ * 2 * sizeof(*d_bond_idxs_)));
    gpuErrchk(hipMemcpy(d_bond_idxs_, &bond_idxs[0], B_ * 2 * sizeof(*d_bond_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_lambda_mult_, B_ * sizeof(*d_lambda_mult_)));
    gpuErrchk(hipMalloc(&d_lambda_offset_, B_ * sizeof(*d_lambda_offset_)));

    if (lambda_mult.size() > 0) {
        gpuErrchk(hipMemcpy(d_lambda_mult_, &lambda_mult[0], B_ * sizeof(*d_lambda_mult_), hipMemcpyHostToDevice));
    } else {
        initializeArray(B_, d_lambda_mult_, 0);
    }

    if (lambda_offset.size() > 0) {
        gpuErrchk(
            hipMemcpy(d_lambda_offset_, &lambda_offset[0], B_ * sizeof(*d_lambda_offset_), hipMemcpyHostToDevice));
    } else {
        // can't memset this
        initializeArray(B_, d_lambda_offset_, 1);
    }
};

template <typename RealType> HarmonicBond<RealType>::~HarmonicBond() {
    gpuErrchk(hipFree(d_bond_idxs_));
    gpuErrchk(hipFree(d_lambda_mult_));
    gpuErrchk(hipFree(d_lambda_offset_));
};

template <typename RealType>
void HarmonicBond<RealType>::execute_device(
    const int N,
    const int P,
    const double *d_x,
    const double *d_p,
    const double *d_box,
    const double lambda,
    unsigned long long *d_du_dx,
    unsigned long long *d_du_dp,
    unsigned long long *d_du_dl,
    unsigned long long *d_u,
    hipStream_t stream) {

    if (P != 2 * B_) {
        throw std::runtime_error(
            "HarmonicBond::execute_device(): expected P == 2*B, got P=" + std::to_string(P) +
            ", 2*B=" + std::to_string(2 * B_));
    }

    if (B_ > 0) {
        const int tpb = warp_size;
        const int blocks = ceil_divide(B_, tpb);

        k_harmonic_bond<RealType><<<blocks, tpb, 0, stream>>>(
            B_, d_x, d_p, lambda, d_lambda_mult_, d_lambda_offset_, d_bond_idxs_, d_du_dx, d_du_dp, d_du_dl, d_u);
        gpuErrchk(hipPeekAtLastError());
    }
};

template class HarmonicBond<double>;
template class HarmonicBond<float>;

} // namespace timemachine
