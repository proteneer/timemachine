#include "gpu_utils.cuh"
#include "harmonic_bond.hpp"
#include "k_harmonic_bond.cuh"
#include "kernel_utils.cuh"
#include "math_utils.cuh"
#include <vector>

namespace timemachine {

template <typename RealType>
HarmonicBond<RealType>::HarmonicBond(const std::vector<int> &bond_idxs) : B_(bond_idxs.size() / 2) {

    if (bond_idxs.size() % 2 != 0) {
        throw std::runtime_error("bond_idxs.size() must be exactly 2*k!");
    }

    for (int b = 0; b < B_; b++) {
        auto src = bond_idxs[b * 2 + 0];
        auto dst = bond_idxs[b * 2 + 1];
        if (src == dst) {
            throw std::runtime_error("src == dst");
        }
    }

    gpuErrchk(hipMalloc(&d_bond_idxs_, B_ * 2 * sizeof(*d_bond_idxs_)));
    gpuErrchk(hipMemcpy(d_bond_idxs_, &bond_idxs[0], B_ * 2 * sizeof(*d_bond_idxs_), hipMemcpyHostToDevice));
};

template <typename RealType> HarmonicBond<RealType>::~HarmonicBond() { gpuErrchk(hipFree(d_bond_idxs_)); };

template <typename RealType>
void HarmonicBond<RealType>::execute_device(
    const int N,
    const int P,
    const double *d_x,
    const double *d_p,
    const double *d_box,
    const double lambda,
    unsigned long long *d_du_dx,
    unsigned long long *d_du_dp,
    unsigned long long *d_du_dl,
    unsigned long long *d_u,
    hipStream_t stream) {

    if (P != 2 * B_) {
        throw std::runtime_error(
            "HarmonicBond::execute_device(): expected P == 2*B, got P=" + std::to_string(P) +
            ", 2*B=" + std::to_string(2 * B_));
    }

    if (B_ > 0) {
        const int tpb = warp_size;
        const int blocks = ceil_divide(B_, tpb);

        k_harmonic_bond<RealType>
            <<<blocks, tpb, 0, stream>>>(B_, d_x, d_p, lambda, d_bond_idxs_, d_du_dx, d_du_dp, d_du_dl, d_u);
        gpuErrchk(hipPeekAtLastError());
    }
};

template class HarmonicBond<double>;
template class HarmonicBond<float>;

} // namespace timemachine
